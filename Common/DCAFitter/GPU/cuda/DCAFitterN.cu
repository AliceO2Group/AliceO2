// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

#ifdef __HIPCC__
#include "hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include "GPUCommonDef.h"
#include "DCAFitter/DCAFitterN.h"
// #include "MathUtils/SMatrixGPU.h"

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}
namespace o2::vertexing::device
{
namespace kernel
{
GPUg() void printKernel(o2::vertexing::DCAFitterN<2>* ft)
{
  if (threadIdx.x == 0) {
    printf(" =============== GPU DCA Fitter ================\n");
    ft->print();
    printf(" ===============================================\n");
  }
}

GPUg() void processKernel(o2::vertexing::DCAFitterN<2>* ft, o2::track::TrackParCov* t1, o2::track::TrackParCov* t2, int* res)
{
  *res = ft->process(*t1, *t2);
}
} // namespace kernel

void print(o2::vertexing::DCAFitterN<2>* ft,
           const int nBlocks,
           const int nThreads)
{
  DCAFitterN<2>* ft_device;
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&ft_device), sizeof(o2::vertexing::DCAFitterN<2>)));
  gpuCheckError(hipMemcpy(ft_device, ft, sizeof(o2::vertexing::DCAFitterN<2>), hipMemcpyHostToDevice));

  kernel::printKernel<<<nBlocks, nThreads>>>(ft_device);

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

int process(o2::vertexing::DCAFitterN<2>* fitter,
            o2::track::TrackParCov& track1,
            o2::track::TrackParCov& track2,
            const int nBlocks,
            const int nThreads)
{
  DCAFitterN<2>* ft_device;
  o2::track::TrackParCov* t1_device;
  o2::track::TrackParCov* t2_device;
  int result, *result_device;

  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&ft_device), sizeof(o2::vertexing::DCAFitterN<2>)));
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&t1_device), sizeof(o2::track::TrackParCov)));
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&t2_device), sizeof(o2::track::TrackParCov)));
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&result_device), sizeof(int)));

  gpuCheckError(hipMemcpy(ft_device, fitter, sizeof(o2::vertexing::DCAFitterN<2>), hipMemcpyHostToDevice));
  gpuCheckError(hipMemcpy(t1_device, &track1, sizeof(o2::track::TrackParCov), hipMemcpyHostToDevice));
  gpuCheckError(hipMemcpy(t2_device, &track2, sizeof(o2::track::TrackParCov), hipMemcpyHostToDevice));

  kernel::processKernel<<<nBlocks, nThreads>>>(ft_device, t1_device, t2_device, result_device);

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());

  gpuCheckError(hipMemcpy(&result, result_device, sizeof(int), hipMemcpyDeviceToHost));
  gpuCheckError(hipMemcpy(fitter, ft_device, sizeof(o2::vertexing::DCAFitterN<2>), hipMemcpyDeviceToHost));
  gpuCheckError(hipMemcpy(&track1, t1_device, sizeof(o2::track::TrackParCov), hipMemcpyDeviceToHost));
  gpuCheckError(hipMemcpy(&track2, t2_device, sizeof(o2::track::TrackParCov), hipMemcpyDeviceToHost));
  gpuCheckError(hipFree(ft_device));
  gpuCheckError(hipFree(t1_device));
  gpuCheckError(hipFree(t2_device));

  gpuCheckError(hipFree(result_device));

  return result;
}

} // namespace o2::vertexing::device