// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

#ifdef __HIPCC__
#include "hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include "GPUCommonDef.h"
#include "DCAFitter/DCAFitterN.h"
// #include "MathUtils/SMatrixGPU.h"

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}
namespace o2::vertexing::device
{
namespace kernel
{
template <typename Fitter>
GPUg() void printKernel(Fitter* ft)
{
  if (threadIdx.x == 0) {
    printf(" =============== GPU DCA Fitter %d prongs ================\n", Fitter::getNProngs());
    ft->print();
    printf(" =========================================================\n");
  }
}

template <typename Fitter, typename... Tr>
GPUg() void processKernel(Fitter* ft, int* res, Tr*... tracks)
{
  *res = ft->process(*tracks...);
}
} // namespace kernel

/// CPU handlers
template <typename Fitter>
void print(const int nBlocks,
           const int nThreads,
           Fitter& ft)
{
  Fitter* ft_device;
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&ft_device), sizeof(Fitter)));
  gpuCheckError(hipMemcpy(ft_device, &ft, sizeof(Fitter), hipMemcpyHostToDevice));

  kernel::printKernel<<<nBlocks, nThreads>>>(ft_device);

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

template <typename Fitter, class... Tr>
int process(const int nBlocks,
            const int nThreads,
            Fitter& fitter,
            Tr&... args)
{
  Fitter* ft_device;
  std::array<o2::track::TrackParCov*, Fitter::getNProngs()> tracks_device;
  int result, *result_device;

  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&ft_device), sizeof(Fitter)));
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&result_device), sizeof(int)));

  int iArg{0};
  ([&] {
    gpuCheckError(hipMalloc(reinterpret_cast<void**>(&(tracks_device[iArg])), sizeof(o2::track::TrackParCov)));
    gpuCheckError(hipMemcpy(tracks_device[iArg], &args, sizeof(o2::track::TrackParCov), hipMemcpyHostToDevice));
    ++iArg;
  }(),
   ...);

  gpuCheckError(hipMemcpy(ft_device, &fitter, sizeof(Fitter), hipMemcpyHostToDevice));

  std::apply([&](auto&&... args) { kernel::processKernel<<<nBlocks, nThreads>>>(ft_device, result_device, args...); }, tracks_device);

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());

  gpuCheckError(hipMemcpy(&result, result_device, sizeof(int), hipMemcpyDeviceToHost));
  gpuCheckError(hipMemcpy(&fitter, ft_device, sizeof(Fitter), hipMemcpyDeviceToHost));
  iArg = 0;
  ([&] {
    gpuCheckError(hipMemcpy(&args, tracks_device[iArg], sizeof(o2::track::TrackParCov), hipMemcpyDeviceToHost));
    gpuCheckError(hipFree(tracks_device[iArg]));
    ++iArg;
  }(),
   ...);

  gpuCheckError(hipFree(result_device));

  return result;
}

template int process(const int, const int, o2::vertexing::DCAFitterN<2>&, o2::track::TrackParCov&, o2::track::TrackParCov&);
template int process(const int, const int, o2::vertexing::DCAFitterN<3>&, o2::track::TrackParCov&, o2::track::TrackParCov&, o2::track::TrackParCov&);
template void print(const int, const int, o2::vertexing::DCAFitterN<2>&);
template void print(const int, const int, o2::vertexing::DCAFitterN<3>&);
} // namespace o2::vertexing::device