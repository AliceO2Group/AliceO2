#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

#ifdef __HIPCC__
#include "hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include "GPUCommonDef.h"
#include "DCAFitter/DCAFitterN.h"
#include "DeviceInterface/GPUInterface.h"

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}
namespace o2::vertexing::device
{
namespace kernel
{
GPUg() void warmUpGpuKernel()
{
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

template <typename Fitter>
GPUg() void printKernel(Fitter* fitter)
{
  if (threadIdx.x == 0) {
    printf(" =============== GPU DCA Fitter %d prongs =================\n", Fitter::getNProngs());
    fitter->print();
    printf(" =========================================================\n");
  }
}

template <typename Fitter, typename... Tr>
GPUg() void processKernel(Fitter* fitter, int* res, Tr*... tracks)
{
  *res = fitter->process(*tracks...);
}

template <typename Fitter, typename... Tr>
GPUg() void processBatchKernel(Fitter* fitters, int* results, size_t off, size_t N, Tr*... tracks)
{
  for (auto iThread{blockIdx.x * blockDim.x + threadIdx.x}; iThread < N; iThread += blockDim.x * gridDim.x) {
    results[iThread + off] = fitters[iThread + off].process(tracks[iThread + off]...);
  }
}

} // namespace kernel

/// CPU handlers
template <typename Fitter>
void print(const int nBlocks,
           const int nThreads,
           Fitter& fitter)
{
  Fitter* fitter_device;
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&fitter_device), sizeof(Fitter)));
  gpuCheckError(hipMemcpy(fitter_device, &fitter, sizeof(Fitter), hipMemcpyHostToDevice));

  kernel::printKernel<<<nBlocks, nThreads>>>(fitter_device);

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

template <typename Fitter, class... Tr>
int process(const int nBlocks,
            const int nThreads,
            Fitter& fitter,
            Tr&... args)
{
  Fitter* fitter_device;
  std::array<o2::track::TrackParCov*, Fitter::getNProngs()> tracks_device;
  int result, *result_device;

  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&fitter_device), sizeof(Fitter)));
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&result_device), sizeof(int)));

  int iArg{0};
  ([&] {
    gpuCheckError(hipMalloc(reinterpret_cast<void**>(&(tracks_device[iArg])), sizeof(o2::track::TrackParCov)));
    gpuCheckError(hipMemcpy(tracks_device[iArg], &args, sizeof(o2::track::TrackParCov), hipMemcpyHostToDevice));
    ++iArg;
  }(),
   ...);

  gpuCheckError(hipMemcpy(fitter_device, &fitter, sizeof(Fitter), hipMemcpyHostToDevice));

  std::apply([&](auto&&... args) { kernel::processKernel<<<nBlocks, nThreads>>>(fitter_device, result_device, args...); }, tracks_device);

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());

  gpuCheckError(hipMemcpy(&result, result_device, sizeof(int), hipMemcpyDeviceToHost));
  gpuCheckError(hipMemcpy(&fitter, fitter_device, sizeof(Fitter), hipMemcpyDeviceToHost));
  iArg = 0;
  ([&] {
    gpuCheckError(hipMemcpy(&args, tracks_device[iArg], sizeof(o2::track::TrackParCov), hipMemcpyDeviceToHost));
    gpuCheckError(hipFree(tracks_device[iArg]));
    ++iArg;
  }(),
   ...);

  gpuCheckError(hipFree(fitter_device));
  gpuCheckError(hipFree(result_device));

  return result;
}

template <typename Fitter, class... Tr>
void processBulk(const int nBlocks,
                 const int nThreads,
                 const int nStreams,
                 std::vector<Fitter>& fitters,
                 std::vector<int>& results,
                 std::vector<Tr>&... args)
{
  auto* gpuInterface = GPUInterface::Instance();
  kernel::warmUpGpuKernel<<<1, 1>>>();

  // Benchmarking events
  // std::vector<hipEvent_t> start(nStreams), stop(nStreams);
  // hipEvent_t totalStart, totalStop;
  // gpuCheckError(hipEventCreate(&totalStart));
  // gpuCheckError(hipEventCreate(&totalStop));
  // for (int iBatch{0}; iBatch < nStreams; ++iBatch) {
  //   gpuCheckError(hipEventCreate(&start[iBatch]));
  //   gpuCheckError(hipEventCreate(&stop[iBatch]));
  // }

  // Tracks
  std::array<o2::track::TrackParCov*, Fitter::getNProngs()> tracks_device;
  int iArg{0};
  ([&] {
    gpuInterface->registerBuffer(reinterpret_cast<void*>(args.data()), sizeof(Tr) * args.size());
    gpuInterface->allocDevice(reinterpret_cast<void**>(&(tracks_device[iArg])), sizeof(Tr) * args.size());
    ++iArg;
  }(),
   ...);

  // Fitters
  gpuInterface->registerBuffer(reinterpret_cast<void*>(fitters.data()), sizeof(Fitter) * fitters.size());
  Fitter* fitters_device;
  gpuInterface->allocDevice(reinterpret_cast<void**>(&fitters_device), sizeof(Fitter) * fitters.size());

  // Results
  gpuInterface->registerBuffer(reinterpret_cast<void*>(results.data()), sizeof(int) * fitters.size());
  int* results_device;
  gpuInterface->allocDevice(reinterpret_cast<void**>(&results_device), sizeof(int) * fitters.size());

  // gpuCheckError(hipEventRecord(totalStart));
  int totalSize = fitters.size();
  int batchSize = totalSize / nStreams;
  int remainder = totalSize % nStreams;

  for (int iBatch{0}; iBatch < nStreams; ++iBatch) {
    auto& stream = gpuInterface->getNextStream();
    auto offset = iBatch * batchSize + std::min(iBatch, remainder);
    auto nFits = batchSize + (iBatch < remainder ? 1 : 0);

    gpuCheckError(hipMemcpyAsync(fitters_device + offset, fitters.data() + offset, sizeof(Fitter) * nFits, hipMemcpyHostToDevice, stream));
    iArg = 0;
    ([&] {
      gpuCheckError(hipMemcpyAsync(tracks_device[iArg] + offset, args.data() + offset, sizeof(Tr) * nFits, hipMemcpyHostToDevice, stream));
      ++iArg;
    }(),
     ...);
    // gpuCheckError(hipEventRecord(start[iBatch]));
    std::apply([&](auto&&... args) { kernel::processBatchKernel<<<nBlocks, nThreads, 0, stream>>>(fitters_device, results_device, offset, nFits, args...); }, tracks_device);
    // gpuCheckError(hipEventRecord(stop[iBatch]));

    gpuCheckError(hipPeekAtLastError());
    gpuCheckError(hipStreamSynchronize(stream));
    iArg = 0;
    ([&] {
      gpuCheckError(hipMemcpyAsync(args.data() + offset, tracks_device[iArg] + offset, sizeof(Tr) * nFits, hipMemcpyDeviceToHost, stream));
      ++iArg;
    }(),
     ...);
    gpuCheckError(hipMemcpyAsync(fitters.data() + offset, fitters_device + offset, sizeof(Fitter) * nFits, hipMemcpyDeviceToHost, stream));
    gpuCheckError(hipMemcpyAsync(results.data() + offset, results_device + offset, sizeof(int) * nFits, hipMemcpyDeviceToHost, stream));
  }
  ([&] { gpuInterface->unregisterBuffer(args.data()); }(), ...);
  // gpuCheckError(hipEventRecord(totalStop));

  for (auto* tracksD : tracks_device) {
    gpuInterface->freeDevice(tracksD);
  }

  gpuInterface->freeDevice(fitters_device);
  gpuInterface->freeDevice(results_device);
  gpuInterface->unregisterBuffer(fitters.data());
  gpuInterface->unregisterBuffer(results.data());
  // float milliseconds = 0;
  // gpuCheckError(hipEventElapsedTime(&milliseconds, start, stop));

  // LOGP(info, "Kernel run in: {} ms using {} blocks and {} threads.", milliseconds, nBlocks, nThreads);
  // return results;
}

template void processBulk(const int,
                          const int,
                          const int,
                          std::vector<o2::vertexing::DCAFitterN<2>>&,
                          std::vector<int>&,
                          std::vector<o2::track::TrackParCov>&,
                          std::vector<o2::track::TrackParCov>&);
template void processBulk(const int,
                          const int,
                          const int,
                          std::vector<o2::vertexing::DCAFitterN<3>>&,
                          std::vector<int>&,
                          std::vector<o2::track::TrackParCov>&,
                          std::vector<o2::track::TrackParCov>&,
                          std::vector<o2::track::TrackParCov>&);
template int process(const int, const int, o2::vertexing::DCAFitterN<2>&, o2::track::TrackParCov&, o2::track::TrackParCov&);
template int process(const int, const int, o2::vertexing::DCAFitterN<3>&, o2::track::TrackParCov&, o2::track::TrackParCov&, o2::track::TrackParCov&);
template void print(const int, const int, o2::vertexing::DCAFitterN<2>&);
template void print(const int, const int, o2::vertexing::DCAFitterN<3>&);
} // namespace o2::vertexing::device