// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

#ifdef __HIPCC__
#include "hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include "GPUCommonDef.h"
#include "DCAFitter/DCAFitterN.h"
// #include "MathUtils/SMatrixGPU.h"

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}
namespace o2::vertexing::gpu
{
namespace kernel
{
GPUg() void printKernel(o2::vertexing::DCAFitterN<2>* ft)
{
  if (threadIdx.x == 0) {
    printf(" =============== GPU DCA Fitter ================\n");
    ft->print();
    printf(" ===============================================\n");
  }
}

GPUg() void processKernel(o2::vertexing::DCAFitterN<2>* ft, o2::track::TrackParCov* t1, o2::track::TrackParCov* t2, int* res)
{
  *res = ft->process(*t1, *t2);
}

void printKHost(o2::vertexing::DCAFitterN<2>* ft, int th, int bl)
{
  DCAFitterN<2>* ft_device;
  gpuCheckError(hipMalloc(reinterpret_cast<void**>(&ft_device), sizeof(o2::vertexing::DCAFitterN<2>)));
  gpuCheckError(hipMemcpy(ft_device, ft, sizeof(o2::vertexing::DCAFitterN<2>), hipMemcpyHostToDevice));
  LOGP(info, "ft: {} ft_device: {} size: {}", (void*)ft, (void*)ft_device, sizeof(o2::vertexing::DCAFitterN<2>));
  printKernel<<<bl, th>>>(ft);
  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
  // static_assert(false);
}
} // namespace kernel

} // namespace o2::vertexing::gpu
