// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

#ifdef __HIPCC__
#include "hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include "DCAFitter/DCAFitterN.h"
#include "DCAFitterNKernels.h"
#include "ReconstructionDataFormats/Track.h"

#include <iostream>
#include <cstdint>

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    std::cout<< "GPUassert: " << hipGetErrorString(code) <<" "<< file <<" "<< line <<std::endl;
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}

namespace o2::vertexing
{
void doProcessingOnGPU(o2::vertexing::DCAFitterN<2>* ft, o2::track::TrackParCov* t1, o2::track::TrackParCov* t2)
{
  o2::vertexing::DCAFitterN<2>* ft_device;
  o2::track::TrackParCov* t1_device;
  o2::track::TrackParCov* t2_device;

  gpuCheckError(hipMalloc(&ft_device, sizeof(o2::vertexing::DCAFitterN<2>)));
  gpuCheckError(hipMalloc(&t1_device, sizeof(o2::track::TrackParCov)));
  gpuCheckError(hipMalloc(&t2_device, sizeof(o2::track::TrackParCov)));

  gpuCheckError(hipMemcpy(ft_device, ft, sizeof(o2::vertexing::DCAFitterN<2>), hipMemcpyHostToDevice));
  gpuCheckError(hipMemcpy(t1_device, t1, sizeof(o2::track::TrackParCov), hipMemcpyHostToDevice));
  gpuCheckError(hipMemcpy(t2_device, t2, sizeof(o2::track::TrackParCov), hipMemcpyHostToDevice));

  gpu::processKernel<<<1, 1>>>(ft_device, t1_device, t2_device);
}
} // namespace o2::vertexing