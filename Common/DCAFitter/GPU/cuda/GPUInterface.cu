// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

#ifdef __HIPCC__
#include "hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include <iostream>

#include "DeviceInterface/GPUInterface.h"

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}

namespace o2::vertexing::device
{

GPUInterface::GPUInterface(size_t N)
{
  resize(N);
  for (auto& st : mStreams) {
    gpuCheckError(hipStreamCreate(&st));
  }
}

GPUInterface::~GPUInterface()
{
  for (auto& st : mStreams) {
    gpuCheckError(hipStreamDestroy(st));
  }
}

void GPUInterface::resize(size_t N)
{
  mPool.resize(N);
  mStreams.resize(N);
}

void GPUInterface::registerBuffer(void* addr, size_t bufferSize)
{
  gpuCheckError(hipHostRegister(addr, bufferSize, hipHostRegisterDefault));
}

void GPUInterface::unregisterBuffer(void* addr)
{
  gpuCheckError(hipHostUnregister(addr));
}

GPUInterface* GPUInterface::sGPUInterface = nullptr;
GPUInterface* GPUInterface::Instance()
{
  if (sGPUInterface == nullptr) {
    sGPUInterface = new GPUInterface(8); // FIXME: get some configurable param to do so.
  }
  return sGPUInterface;
}

void GPUInterface::allocDevice(void** addrPtr, size_t bufferSize)
{
  gpuCheckError(hipMalloc(addrPtr, bufferSize));
}

void GPUInterface::freeDevice(void* addr)
{
  gpuCheckError(hipFree(addr));
}

Stream& GPUInterface::getStream(short N)
{
  return mStreams[N % mStreams.size()];
}
} // namespace o2::vertexing::device