// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \brief Helper interface to the GPU device, meant to be compatible with manual allocation/streams and GPUReconstruction ones.
/// \author matteo.concas@cern.ch

#ifdef __HIPCC__
#include "hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include <iostream>
#include <cstdlib>

#include "DeviceInterface/GPUInterface.h"

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
#define gpuCheckErrorSoft(x)                   \
  {                                            \
    gpuAssert((x), __FILE__, __LINE__, false); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    std::cerr << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}

namespace o2::vertexing::device
{

GPUInterface::GPUInterface(size_t N)
{
  resize(N);
  for (auto& st : mStreams) {
    gpuCheckError(hipStreamCreate(&st));
  }
}

GPUInterface::~GPUInterface()
{
  for (auto& st : mStreams) {
    gpuCheckError(hipStreamDestroy(st));
  }
}

void GPUInterface::resize(size_t N)
{
  mPool.resize(N);
  mStreams.resize(N);
}

void GPUInterface::registerBuffer(void* addr, size_t bufferSize)
{
  gpuCheckError(hipHostRegister(addr, bufferSize, hipHostRegisterDefault));
}

void GPUInterface::unregisterBuffer(void* addr)
{
  gpuCheckError(hipHostUnregister(addr));
}

GPUInterface* GPUInterface::sGPUInterface = nullptr;
GPUInterface* GPUInterface::Instance()
{
  if (sGPUInterface == nullptr) {
    const auto* envValue = std::getenv("GPUINTERFACE_NSTREAMS");
    sGPUInterface = new GPUInterface(envValue == nullptr ? 8 : std::stoi(envValue));
  }
  return sGPUInterface;
}

void GPUInterface::allocDevice(void** addrPtr, size_t bufferSize)
{
  gpuCheckError(hipMalloc(addrPtr, bufferSize));
}

void GPUInterface::freeDevice(void* addr)
{
  gpuCheckError(hipFree(addr));
}

Stream& GPUInterface::getStream(unsigned short N)
{
  return mStreams[N % mStreams.size()];
}

Stream& GPUInterface::getNextStream()
{
  unsigned short next = mLastUsedStream.fetch_add(1) % mStreams.size(); // wrap-around + automatic wrap-around beyond 65535
  return mStreams[next];
}
} // namespace o2::vertexing::device