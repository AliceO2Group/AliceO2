#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>
#include <iostream>

#define NVRTC_SAFE_CALL(x)                              \
  do {                                                  \
    hiprtcResult result = x;                             \
    if (result != HIPRTC_SUCCESS) {                      \
      std::cerr << "\nerror: " #x " failed with error " \
                << hiprtcGetErrorString(result) << '\n'; \
      exit(1);                                          \
    }                                                   \
  } while (0)

#define CUDA_SAFE_CALL(x)                               \
  do {                                                  \
    hipError_t result = x;                                \
    if (result != hipSuccess) {                       \
      const char* msg;                                  \
      hipDrvGetErrorName(result, &msg);                     \
      std::cerr << "\nerror: " #x " failed with error " \
                << msg << '\n';                         \
      exit(1);                                          \
    }                                                   \
  } while (0)

int main(int argc, char** argv)
{
  //Read Sourcecode from file
  unsigned int filesize;
  FILE* pFile;
  //Open file
  if ((pFile = fopen("source.cu", "rb")) == NULL)
    exit(1);
  //Optain File Size
  fseek(pFile, 0, SEEK_END);
  filesize = ftell(pFile);
  rewind(pFile);
  //Read file
  char* sourceCode = new char[filesize + 1];
  if (fread(sourceCode, 1, filesize, pFile) != filesize)
    exit(1);
  //Make sourceCode 0-terminated
  sourceCode[filesize] = 0;
  fclose(pFile);

  hiprtcProgram prog;
  NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog,      // prog
                                     sourceCode, // buffer
                                     "saxpy.cu", // name
                                     0,          // numHeaders
                                     NULL,       // headers
                                     NULL));     // includeNames
  delete[] sourceCode;
  //const char *opts[] = {"-default-device -std=c++17  --extended-lambda -Xptxas -O4 -Xcompiler -O4 -use_fast_math --ftz=true"};
  const char* opts[] = {"-default-device", "--std=c++17", "-use_fast_math", "-ftz=true"};
  hiprtcResult compileResult = hiprtcCompileProgram(prog,                           // prog
                                                  sizeof(opts) / sizeof(opts[0]), // numOptions
                                                  opts);                          // options
  size_t logSize;
  NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
  char* log = new char[logSize];
  NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, log));
  std::cout << log << '\n';
  delete[] log;
  if (compileResult != HIPRTC_SUCCESS) {
    exit(1);
  }
  size_t ptxSize;
  NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
  char* ptx = new char[ptxSize];
  NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));
  NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
  hipModule_t module;
  hipFunction_t kernel;
  CUDA_SAFE_CALL(hipModuleLoadDataEx(&module, ptx, 0, 0, 0));
  CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, "foo"));
  void* args[] = {};
  CUDA_SAFE_CALL(
    hipModuleLaunchKernel(kernel,
                   1, 1, 1,   // grid dim
                   32, 1, 1,  // block dim
                   0, NULL,   // shared mem and stream
                   args, 0)); // arguments
  return 0;
}
