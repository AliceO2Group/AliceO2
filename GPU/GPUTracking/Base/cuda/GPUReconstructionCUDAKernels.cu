#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDAKernels.cu
/// \author David Rohr

#include "GPUReconstructionCUDADef.h"
#include "GPUReconstructionCUDAIncludes.h"

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"
#include "CUDAThrustHelpers.h"

using namespace GPUCA_NAMESPACE::gpu;

#ifdef GPUCA_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

#include "GPUReconstructionIncludesDeviceAll.h"

template <>
void GPUReconstructionCUDABackend::runKernelBackendInternal<GPUMemClean16, 0>(krnlSetup& _xyz, void* const& ptr, unsigned long const& size)
{
  GPUDebugTiming timer(mProcessingSettings.debugLevel, nullptr, mInternals->Streams, _xyz, this);
  GPUFailedMsg(hipMemsetAsync(ptr, 0, size, mInternals->Streams[_xyz.x.stream]));
}

template <class T, int I, typename... Args>
void GPUReconstructionCUDABackend::runKernelBackendInternal(krnlSetup& _xyz, const Args&... args)
{
  GPUDebugTiming timer(mProcessingSettings.deviceTimers && mProcessingSettings.debugLevel > 0, (void**)mDebugEvents, mInternals->Streams, _xyz, this);
#if !defined(GPUCA_KERNEL_COMPILE_MODE) || GPUCA_KERNEL_COMPILE_MODE != 1
  if (!mProcessingSettings.rtc.enable) {
    backendInternal<T, I>::runKernelBackendMacro(_xyz, this, args...);
  } else
#endif
  {
    auto& x = _xyz.x;
    auto& y = _xyz.y;
    const void* pArgs[sizeof...(Args) + 3]; // 3 is max: cons mem + y.start + y.num
    int arg_offset = 0;
#ifdef GPUCA_NO_CONSTANT_MEMORY
    arg_offset = 1;
    pArgs[0] = &mDeviceConstantMem;
#endif
    pArgs[arg_offset] = &y.start;
    GPUReconstructionCUDAInternals::getArgPtrs(&pArgs[arg_offset + 1 + (y.num > 1)], args...);
    if (y.num <= 1) {
      GPUFailedMsg(hipModuleLaunchKernel(*mInternals->kernelFunctions[mInternals->getRTCkernelNum<false, T, I>()], x.nBlocks, 1, 1, x.nThreads, 1, 1, 0, mInternals->Streams[x.stream], (void**)pArgs, nullptr));
    } else {
      pArgs[arg_offset + 1] = &y.num;
      GPUFailedMsg(hipModuleLaunchKernel(*mInternals->kernelFunctions[mInternals->getRTCkernelNum<true, T, I>()], x.nBlocks, 1, 1, x.nThreads, 1, 1, 0, mInternals->Streams[x.stream], (void**)pArgs, nullptr));
    }
  }
  if (mProcessingSettings.checkKernelFailures) {
    if (GPUDebug(GetKernelName<T, I>(), _xyz.x.stream, true)) {
      throw std::runtime_error("Kernel Failure");
    }
  }
}

template <class T, int I, typename... Args>
int GPUReconstructionCUDABackend::runKernelBackend(krnlSetup& _xyz, Args... args)
{
  auto& x = _xyz.x;
  auto& z = _xyz.z;
  if (z.evList) {
    for (int k = 0; k < z.nEvents; k++) {
      GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[x.stream], ((hipEvent_t*)z.evList)[k], 0));
    }
  }
  runKernelBackendInternal<T, I>(_xyz, args...);
  GPUFailedMsg(hipGetLastError());
  if (z.ev) {
    GPUFailedMsg(hipEventRecord(*(hipEvent_t*)z.ev, mInternals->Streams[x.stream]));
  }
  return 0;
}

#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward) \
  GPUCA_KRNL_PROP(x_class, x_attributes)                          \
  template int GPUReconstructionCUDABackend::runKernelBackend<GPUCA_M_KRNL_TEMPLATE(x_class)>(krnlSetup & _xyz GPUCA_M_STRIP(x_arguments));
#else
#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 2
#define GPUCA_KRNL_DEFONLY
#endif
#undef GPUCA_KRNL_REG
#define GPUCA_KRNL_REG(args) __launch_bounds__(GPUCA_M_MAX2_3(GPUCA_M_STRIP(args)))
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward)             \
  GPUCA_KRNL_PROP(x_class, x_attributes)                                      \
  GPUCA_KRNL_WRAP(GPUCA_KRNL_, x_class, x_attributes, x_arguments, x_forward) \
  template int GPUReconstructionCUDABackend::runKernelBackend<GPUCA_M_KRNL_TEMPLATE(x_class)>(krnlSetup & _xyz GPUCA_M_STRIP(x_arguments));
#define GPUCA_KRNL_CALL_single(x_class, x_attributes, x_arguments, x_forward) \
  GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))<<<x.nBlocks, x.nThreads, 0, me->mInternals->Streams[x.stream]>>>(GPUCA_CONSMEM_CALL y.start, args...);
#define GPUCA_KRNL_CALL_multi(x_class, x_attributes, x_arguments, x_forward) \
  GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)<<<x.nBlocks, x.nThreads, 0, me->mInternals->Streams[x.stream]>>>(GPUCA_CONSMEM_CALL y.start, y.num, args...);
#endif
#include "GPUReconstructionKernels.h"
#undef GPUCA_KRNL

#ifndef GPUCA_NO_CONSTANT_MEMORY
static GPUReconstructionDeviceBase::deviceConstantMemRegistration registerConstSymbol([]() {
  void* retVal = nullptr;
  GPUReconstructionCUDA::GPUFailedMsgI(hipGetSymbolAddress(&retVal, HIP_SYMBOL(gGPUConstantMemBuffer)));
  return retVal;
});
#endif
