#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDAKernels.cu
/// \author David Rohr

#include "GPUReconstructionCUDADef.h"
#include "GPUReconstructionCUDAIncludes.h"

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"
#include "CUDAThrustHelpers.h"

using namespace GPUCA_NAMESPACE::gpu;

#ifdef GPUCA_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

#include "GPUReconstructionIncludesDeviceAll.h"

#if defined(__HIPCC__) && defined(GPUCA_HAS_GLOBAL_SYMBOL_CONSTANT_MEM)
__global__ void gGPUConstantMemBuffer_dummy(int* p) { *p = *(int*)&gGPUConstantMemBuffer; }
#endif

template <>
inline void GPUReconstructionCUDABackend::runKernelBackendInternal<GPUMemClean16, 0>(const krnlSetupTime& _xyz, void* const& ptr, unsigned long const& size)
{
  GPUFailedMsg(hipMemsetAsync(ptr, 0, size, mInternals->Streams[_xyz.x.stream]));
}

template <class T, int I, typename... Args>
inline void GPUReconstructionCUDABackend::runKernelBackendInternal(const krnlSetupTime& _xyz, const Args&... args)
{
#if !defined(GPUCA_KERNEL_COMPILE_MODE) || GPUCA_KERNEL_COMPILE_MODE != 1
  if (!mProcessingSettings.rtc.enable) {
    backendInternal<T, I>::runKernelBackendMacro(_xyz, this, args...);
  } else
#endif
  {
    auto& x = _xyz.x;
    auto& y = _xyz.y;
    const void* pArgs[sizeof...(Args) + 3]; // 3 is max: cons mem + y.start + y.num
    int arg_offset = 0;
#ifdef GPUCA_NO_CONSTANT_MEMORY
    arg_offset = 1;
    pArgs[0] = &mDeviceConstantMem;
#endif
    pArgs[arg_offset] = &y.start;
    GPUReconstructionCUDAInternals::getArgPtrs(&pArgs[arg_offset + 1 + (y.num > 1)], args...);
    if (y.num <= 1) {
      GPUFailedMsg(hipModuleLaunchKernel(*mInternals->kernelFunctions[getRTCkernelNum<false, T, I>()], x.nBlocks, 1, 1, x.nThreads, 1, 1, 0, mInternals->Streams[x.stream], (void**)pArgs, nullptr));
    } else {
      pArgs[arg_offset + 1] = &y.num;
      GPUFailedMsg(hipModuleLaunchKernel(*mInternals->kernelFunctions[getRTCkernelNum<true, T, I>()], x.nBlocks, 1, 1, x.nThreads, 1, 1, 0, mInternals->Streams[x.stream], (void**)pArgs, nullptr));
    }
  }
}

template <class T, int I, typename... Args>
int GPUReconstructionCUDABackend::runKernelBackend(const krnlSetupArgs<T, I, Args...>& args)
{
  auto& x = args.s.x;
  auto& z = args.s.z;
  if (z.evList) {
    for (int k = 0; k < z.nEvents; k++) {
      GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[x.stream], ((hipEvent_t*)z.evList)[k], 0));
    }
  }
  {
    GPUDebugTiming timer(mProcessingSettings.deviceTimers && mProcessingSettings.debugLevel > 0, (deviceEvent*)mDebugEvents, mInternals->Streams, args.s, this);
    std::apply([this, &args](auto&... vals) { this->runKernelBackendInternal<T, I, Args...>(args.s, vals...); }, args.v);
  }
  GPUFailedMsg(hipGetLastError());
  if (mProcessingSettings.checkKernelFailures) {
    if (GPUDebug(GetKernelName<T, I>(), args.s.x.stream, true)) {
      throw std::runtime_error("Kernel Failure");
    }
  }
  if (z.ev) {
    GPUFailedMsg(hipEventRecord(*(hipEvent_t*)z.ev, mInternals->Streams[x.stream]));
  }
  return 0;
}

#undef GPUCA_KRNL_REG
#define GPUCA_KRNL_REG(args) __launch_bounds__(GPUCA_M_MAX2_3(GPUCA_M_STRIP(args)))

#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward, x_types) \
  GPUCA_KRNL_PROP(x_class, x_attributes)                                   \
  template int GPUReconstructionCUDABackend::runKernelBackend<GPUCA_M_KRNL_TEMPLATE(x_class)>(const krnlSetupArgs<GPUCA_M_KRNL_TEMPLATE(x_class) GPUCA_M_STRIP(x_types)>& args);
#else
#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 2
#define GPUCA_KRNL_DEFONLY
#endif

#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward, x_types)             \
  GPUCA_KRNL_PROP(x_class, x_attributes)                                               \
  GPUCA_KRNL_WRAP(GPUCA_KRNL_, x_class, x_attributes, x_arguments, x_forward, x_types) \
  template int GPUReconstructionCUDABackend::runKernelBackend<GPUCA_M_KRNL_TEMPLATE(x_class)>(const krnlSetupArgs<GPUCA_M_KRNL_TEMPLATE(x_class) GPUCA_M_STRIP(x_types)>& args);
#ifndef __HIPCC__ // CUDA version
#define GPUCA_KRNL_CALL_single(x_class, ...) \
  GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))<<<x.nBlocks, x.nThreads, 0, me->mInternals->Streams[x.stream]>>>(GPUCA_CONSMEM_CALL y.start, args...);
#define GPUCA_KRNL_CALL_multi(x_class, ...) \
  GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)<<<x.nBlocks, x.nThreads, 0, me->mInternals->Streams[x.stream]>>>(GPUCA_CONSMEM_CALL y.start, y.num, args...);
#else // HIP version
#undef GPUCA_KRNL_CUSTOM
#define GPUCA_KRNL_CUSTOM(args) GPUCA_M_STRIP(args)
#define GPUCA_KRNL_CALL_single(x_class, ...) \
  hipLaunchKernelGGL(HIP_KERNEL_NAME(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))), dim3(x.nBlocks), dim3(x.nThreads), 0, me->mInternals->Streams[x.stream], GPUCA_CONSMEM_CALL y.start, args...);
#define GPUCA_KRNL_CALL_multi(x_class, ...) \
  hipLaunchKernelGGL(HIP_KERNEL_NAME(GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)), dim3(x.nBlocks), dim3(x.nThreads), 0, me->mInternals->Streams[x.stream], GPUCA_CONSMEM_CALL y.start, y.num, args...);
#endif // __HIPCC__
#endif

#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL

template <bool multi, class T, int I>
int GPUReconstructionCUDABackend::getRTCkernelNum(int k)
{
  static int num = k;
  if (num < 0) {
    throw std::runtime_error("Invalid kernel");
  }
  return num;
}

#define GPUCA_KRNL(x_class, ...)                                                                            \
  template int GPUReconstructionCUDABackend::getRTCkernelNum<false, GPUCA_M_KRNL_TEMPLATE(x_class)>(int k); \
  template int GPUReconstructionCUDABackend::getRTCkernelNum<true, GPUCA_M_KRNL_TEMPLATE(x_class)>(int k);
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL

void GPUReconstructionCUDABackend::getRTCKernelCalls(std::vector<std::string>& kernels)
{
#define GPUCA_KRNL(...) GPUCA_KRNL_WRAP(GPUCA_KRNL_LOAD_, __VA_ARGS__)
#define GPUCA_KRNL_LOAD_single(...) kernels.emplace_back(GPUCA_M_STR(GPUCA_KRNLGPU_SINGLE(__VA_ARGS__)));
#define GPUCA_KRNL_LOAD_multi(...) kernels.emplace_back(GPUCA_M_STR(GPUCA_KRNLGPU_MULTI(__VA_ARGS__)));
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
#undef GPUCA_KRNL_LOAD_single
#undef GPUCA_KRNL_LOAD_multi
}

#ifndef GPUCA_NO_CONSTANT_MEMORY
static GPUReconstructionDeviceBase::deviceConstantMemRegistration registerConstSymbol([]() {
  void* retVal = nullptr;
  GPUReconstructionCUDA::GPUFailedMsgI(hipGetSymbolAddress(&retVal, HIP_SYMBOL(gGPUConstantMemBuffer)));
  return retVal;
});
#endif
