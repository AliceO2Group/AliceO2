#include "hip/hip_runtime.h"
// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDA.cu
/// \author David Rohr

#define GPUCA_GPUTYPE_TURING
#define GPUCA_UNROLL(CUDA, HIP) GPUCA_M_UNROLL_##CUDA
#define GPUdic(CUDA, HIP) GPUCA_GPUdic_select_##CUDA()

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wshadow"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#pragma GCC diagnostic pop

#ifdef __clang__
#define assert(...)
#endif

#include "GPUDef.h"

#ifndef GPUCA_NO_CONSTANT_MEMORY
#define GPUCA_CONSMEM_PTR
#define GPUCA_CONSMEM_CALL
#define GPUCA_CONSMEM (gGPUConstantMemBuffer.v)
#else
#define GPUCA_CONSMEM_PTR const GPUConstantMem *gGPUConstantMemBuffer,
#define GPUCA_CONSMEM_CALL me->mDeviceConstantMem,
#define GPUCA_CONSMEM ((GPUConstantMem&)(*gGPUConstantMemBuffer))
#endif
#define GPUCA_KRNL_BACKEND_CLASS GPUReconstructionCUDABackend

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"
#include "GPUReconstructionIncludes.h"

static constexpr size_t REQUIRE_MIN_MEMORY = 1024u * 1024 * 1024;
static constexpr size_t REQUIRE_MEMORY_RESERVED = 512u * 1024 * 1024;
static constexpr size_t REQUIRE_FREE_MEMORY_RESERVED = 2048u * 1024 * 1024;

using namespace GPUCA_NAMESPACE::gpu;

#ifdef GPUCA_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

__global__ void dummyInitKernel(void* foo)
{
}

#if defined(HAVE_O2HEADERS) && !defined(GPUCA_NO_ITS_TRAITS)
#include "ITStrackingCUDA/TrackerTraitsNV.h"
#include "ITStrackingCUDA/VertexerTraitsGPU.h"
#else
namespace o2
{
namespace its
{
class TrackerTraitsNV : public TrackerTraits
{
};
class VertexerTraitsGPU : public VertexerTraits
{
};
} // namespace its
} // namespace o2
#endif

class GPUDebugTiming
{
 public:
  GPUDebugTiming(bool d, void** t, hipStream_t* s, GPUReconstruction::krnlSetup& x, GPUReconstructionCUDABackend* r = nullptr) : mDeviceTimers(t), mStreams(s), mXYZ(x), mRec(r), mDo(d)
  {
    if (mDo) {
      if (mDeviceTimers) {
        GPUFailedMsg(hipEventRecord((hipEvent_t)mDeviceTimers[0], mStreams[mXYZ.x.stream]));
      } else {
        mTimer.ResetStart();
      }
    }
  }
  ~GPUDebugTiming()
  {
    if (mDo) {
      if (mDeviceTimers) {
        GPUFailedMsg(hipEventRecord((hipEvent_t)mDeviceTimers[1], mStreams[mXYZ.x.stream]));
        GPUFailedMsg(hipEventSynchronize((hipEvent_t)mDeviceTimers[1]));
        float v;
        GPUFailedMsg(hipEventElapsedTime(&v, (hipEvent_t)mDeviceTimers[0], (hipEvent_t)mDeviceTimers[1]));
        mXYZ.t = v * 1.e-3;
      } else {
        GPUFailedMsg(hipStreamSynchronize(mStreams[mXYZ.x.stream]));
        mXYZ.t = mTimer.GetCurrentElapsedTime();
      }
    }
  }

 private:
  void** mDeviceTimers;
  hipStream_t* mStreams;
  GPUReconstruction::krnlSetup& mXYZ;
  GPUReconstructionCUDABackend* mRec;
  HighResTimer mTimer;
  bool mDo;
};

#include "GPUReconstructionIncludesDevice.h"

/*
// Not using templated kernel any more, since nvidia profiler does not resolve template names
template <class T, int I, typename... Args>
GPUg() void runKernelCUDA(GPUCA_CONSMEM_PTR int iSlice_internal, Args... args)
{
  GPUshared() typename T::GPUSharedMemory smem;
  T::template Thread<I>(get_num_groups(0), get_local_size(0), get_group_id(0), get_local_id(0), smem, T::Processor(GPUCA_CONSMEM)[iSlice_internal], args...);
}
*/

#undef GPUCA_KRNL_REG
#define GPUCA_KRNL_REG(args) __launch_bounds__(GPUCA_M_MAX2_3(GPUCA_M_STRIP(args)))
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward) \
  GPUCA_KRNL_PROP(x_class, x_attributes)                          \
  GPUCA_KRNL_WRAP(GPUCA_KRNL_, x_class, x_attributes, x_arguments, x_forward)
#define GPUCA_KRNL_CALL_single(x_class, x_attributes, x_arguments, x_forward) \
  GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))<<<x.nBlocks, x.nThreads, 0, me->mInternals->Streams[x.stream]>>>(GPUCA_CONSMEM_CALL y.start, args...);
#define GPUCA_KRNL_CALL_multi(x_class, x_attributes, x_arguments, x_forward) \
  GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)<<<x.nBlocks, x.nThreads, 0, me->mInternals->Streams[x.stream]>>>(GPUCA_CONSMEM_CALL y.start, y.num, args...);

#include "GPUReconstructionKernels.h"
#undef GPUCA_KRNL

template <>
void GPUReconstructionCUDABackend::runKernelBackendInternal<GPUMemClean16, 0>(krnlSetup& _xyz, void* const& ptr, unsigned long const& size)
{
  GPUDebugTiming timer(mProcessingSettings.debugLevel, nullptr, mInternals->Streams, _xyz, this);
  GPUFailedMsg(hipMemsetAsync(ptr, 0, size, mInternals->Streams[_xyz.x.stream]));
}

template <class T, int I, typename... Args>
void GPUReconstructionCUDABackend::runKernelBackendInternal(krnlSetup& _xyz, const Args&... args)
{
  GPUDebugTiming timer(mProcessingSettings.deviceTimers, (void**)mDebugEvents, mInternals->Streams, _xyz);
  backendInternal<T, I>::runKernelBackendMacro(_xyz, this, args...);
}

template <class T, int I, typename... Args>
int GPUReconstructionCUDABackend::runKernelBackend(krnlSetup& _xyz, const Args&... args)
{
  auto& x = _xyz.x;
  auto& z = _xyz.z;
  if (z.evList) {
    for (int k = 0; k < z.nEvents; k++) {
      GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[x.stream], ((hipEvent_t*)z.evList)[k], 0));
    }
  }
  runKernelBackendInternal<T, I>(_xyz, args...);
  GPUFailedMsg(hipGetLastError());
  if (z.ev) {
    GPUFailedMsg(hipEventRecord(*(hipEvent_t*)z.ev, mInternals->Streams[x.stream]));
  }
  return 0;
}

GPUReconstructionCUDABackend::GPUReconstructionCUDABackend(const GPUSettingsDeviceBackend& cfg) : GPUReconstructionDeviceBase(cfg, sizeof(GPUReconstructionDeviceBase))
{
  if (mMaster == nullptr) {
    mInternals = new GPUReconstructionCUDAInternals;
  }
  mDeviceBackendSettings.deviceType = DeviceType::CUDA;
}

GPUReconstructionCUDABackend::~GPUReconstructionCUDABackend()
{
  Exit(); // Make sure we destroy everything (in particular the ITS tracker) before we exit CUDA
  if (mMaster == nullptr) {
    delete mInternals;
  }
}

GPUReconstruction* GPUReconstruction_Create_CUDA(const GPUSettingsDeviceBackend& cfg) { return new GPUReconstructionCUDA(cfg); }

void GPUReconstructionCUDABackend::GetITSTraits(std::unique_ptr<o2::its::TrackerTraits>* trackerTraits, std::unique_ptr<o2::its::VertexerTraits>* vertexerTraits)
{
  if (trackerTraits) {
    trackerTraits->reset(new o2::its::TrackerTraitsNV);
  }
  if (vertexerTraits) {
    vertexerTraits->reset(new o2::its::VertexerTraitsGPU);
  }
}

void GPUReconstructionCUDABackend::UpdateSettings()
{
  GPUCA_GPUReconstructionUpdateDefailts();
}

int GPUReconstructionCUDABackend::InitDevice_Runtime()
{
  if (mMaster == nullptr) {
    hipDeviceProp_t hipDeviceProp_t;
    int count, bestDevice = -1;
    double bestDeviceSpeed = -1, deviceSpeed;
    if (GPUFailedMsgI(hipInit(0))) {
      GPUError("Error initializing CUDA!");
      return (1);
    }
    if (GPUFailedMsgI(hipGetDeviceCount(&count))) {
      GPUError("Error getting CUDA Device Count");
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Available CUDA devices:");
    }
    const int reqVerMaj = 2;
    const int reqVerMin = 0;
    std::vector<bool> devicesOK(count, false);
    std::vector<size_t> devMemory(count, 0);
    bool contextCreated = false;
    for (int i = 0; i < count; i++) {
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Examining device %d", i);
      }
      size_t free, total;
      hipDevice_t tmpDevice;
      if (GPUFailedMsgI(hipDeviceGet(&tmpDevice, i))) {
        GPUError("Could not set CUDA device!");
        return (1);
      }
      if (GPUFailedMsgI(hipCtxCreate(&mInternals->CudaContext, 0, tmpDevice))) {
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Couldn't create context for device %d. Skipping it.", i);
        }
        continue;
      }
      contextCreated = true;
      if (GPUFailedMsgI(hipMemGetInfo(&free, &total))) {
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Error obtaining CUDA memory info about device %d! Skipping it.", i);
        }
        GPUFailedMsg(hipCtxDestroy(mInternals->CudaContext));
        continue;
      }
      if (count > 1) {
        GPUFailedMsg(hipCtxDestroy(mInternals->CudaContext));
        contextCreated = false;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained current memory usage for device %d", i);
      }
      if (GPUFailedMsgI(hipGetDeviceProperties(&hipDeviceProp_t, i))) {
        continue;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained device properties for device %d", i);
      }
      int deviceOK = true;
      const char* deviceFailure = "";
      if (hipDeviceProp_t.major < reqVerMaj || (hipDeviceProp_t.major == reqVerMaj && hipDeviceProp_t.minor < reqVerMin)) {
        deviceOK = false;
        deviceFailure = "Too low device revision";
      } else if (free < std::max(mDeviceMemorySize, REQUIRE_MIN_MEMORY)) {
        deviceOK = false;
        deviceFailure = "Insufficient GPU memory";
      }

      deviceSpeed = (double)hipDeviceProp_t.multiProcessorCount * (double)hipDeviceProp_t.clockRate * (double)hipDeviceProp_t.warpSize * (double)free * (double)hipDeviceProp_t.major * (double)hipDeviceProp_t.major;
      if (mProcessingSettings.debugLevel >= 2) {
        GPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lld / %lld)%s %s", deviceOK ? " " : "[", i, hipDeviceProp_t.name, hipDeviceProp_t.major, hipDeviceProp_t.minor, (long long int)free, (long long int)hipDeviceProp_t.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
      }
      if (!deviceOK) {
        continue;
      }
      devicesOK[i] = true;
      devMemory[i] = std::min(free, total - REQUIRE_MEMORY_RESERVED);
      if (deviceSpeed > bestDeviceSpeed) {
        bestDevice = i;
        bestDeviceSpeed = deviceSpeed;
      } else {
        if (mProcessingSettings.debugLevel >= 2 && mProcessingSettings.deviceNum < 0) {
          GPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
        }
      }
    }

    bool noDevice = false;
    if (bestDevice == -1) {
      GPUWarning("No %sCUDA Device available, aborting CUDA Initialisation", count ? "appropriate " : "");
      GPUImportant("Requiring Revision %d.%d, Mem: %lld", reqVerMaj, reqVerMin, (long long int)std::max(mDeviceMemorySize, REQUIRE_MIN_MEMORY));
      noDevice = true;
    } else if (mProcessingSettings.deviceNum > -1) {
      if (mProcessingSettings.deviceNum >= (signed)count) {
        GPUError("Requested device ID %d does not exist", mProcessingSettings.deviceNum);
        noDevice = true;
      } else if (!devicesOK[mProcessingSettings.deviceNum]) {
        GPUError("Unsupported device requested (%d)", mProcessingSettings.deviceNum);
        noDevice = true;
      } else {
        bestDevice = mProcessingSettings.deviceNum;
      }
    }
    if (noDevice) {
      if (contextCreated) {
        GPUFailedMsgI(hipCtxDestroy(mInternals->CudaContext));
      }
      return (1);
    }
    mDeviceId = bestDevice;

    GPUFailedMsgI(hipGetDeviceProperties(&hipDeviceProp_t, mDeviceId));

    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Using CUDA Device %s with Properties:", hipDeviceProp_t.name);
      GPUInfo("\ttotalGlobalMem = %lld", (unsigned long long int)hipDeviceProp_t.totalGlobalMem);
      GPUInfo("\tsharedMemPerBlock = %lld", (unsigned long long int)hipDeviceProp_t.sharedMemPerBlock);
      GPUInfo("\tregsPerBlock = %d", hipDeviceProp_t.regsPerBlock);
      GPUInfo("\twarpSize = %d", hipDeviceProp_t.warpSize);
      GPUInfo("\tmemPitch = %lld", (unsigned long long int)hipDeviceProp_t.memPitch);
      GPUInfo("\tmaxThreadsPerBlock = %d", hipDeviceProp_t.maxThreadsPerBlock);
      GPUInfo("\tmaxThreadsDim = %d %d %d", hipDeviceProp_t.maxThreadsDim[0], hipDeviceProp_t.maxThreadsDim[1], hipDeviceProp_t.maxThreadsDim[2]);
      GPUInfo("\tmaxGridSize = %d %d %d", hipDeviceProp_t.maxGridSize[0], hipDeviceProp_t.maxGridSize[1], hipDeviceProp_t.maxGridSize[2]);
      GPUInfo("\ttotalConstMem = %lld", (unsigned long long int)hipDeviceProp_t.totalConstMem);
      GPUInfo("\tmajor = %d", hipDeviceProp_t.major);
      GPUInfo("\tminor = %d", hipDeviceProp_t.minor);
      GPUInfo("\tclockRate = %d", hipDeviceProp_t.clockRate);
      GPUInfo("\tmemoryClockRate = %d", hipDeviceProp_t.memoryClockRate);
      GPUInfo("\tmultiProcessorCount = %d", hipDeviceProp_t.multiProcessorCount);
      GPUInfo("\ttextureAlignment = %lld", (unsigned long long int)hipDeviceProp_t.textureAlignment);
      GPUInfo(" ");
    }
    mBlockCount = hipDeviceProp_t.multiProcessorCount;
    mWarpSize = 32;
    mMaxThreads = std::max<int>(mMaxThreads, hipDeviceProp_t.maxThreadsPerBlock * mBlockCount);
    mDeviceName = hipDeviceProp_t.name;
    mDeviceName += " (CUDA GPU)";

    if (hipDeviceProp_t.major < 3) {
      GPUError("Unsupported CUDA Device");
      return (1);
    }

#ifdef GPUCA_USE_TEXTURES
    if (GPUCA_SLICE_DATA_MEMORY * NSLICES > (size_t)hipDeviceProp_t.maxTexture1DLinear) {
      GPUError("Invalid maximum texture size of device: %lld < %lld\n", (long long int)hipDeviceProp_t.maxTexture1DLinear, (long long int)(GPUCA_SLICE_DATA_MEMORY * NSLICES));
      return (1);
    }
#endif
#ifndef GPUCA_NO_CONSTANT_MEMORY
    if (gGPUConstantMemBufferSize > hipDeviceProp_t.totalConstMem) {
      GPUError("Insufficient constant memory available on GPU %d < %d!", (int)hipDeviceProp_t.totalConstMem, (int)gGPUConstantMemBufferSize);
      return (1);
    }
#endif

    if (contextCreated == 0 && GPUFailedMsgI(hipCtxCreate(&mInternals->CudaContext, hipDeviceScheduleAuto, mDeviceId))) {
      GPUError("Could not set CUDA Device!");
      return (1);
    }

    if (GPUFailedMsgI(hipDeviceSetLimit(hipLimitStackSize, GPUCA_GPU_STACK_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (GPUFailedMsgI(hipDeviceSetLimit(hipLimitMallocHeapSize, GPUCA_GPU_HEAP_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }

    if (mDeviceMemorySize == 2) {
      mDeviceMemorySize = devMemory[mDeviceId] * 2 / 3; // Leave 1/3 of GPU memory for event display
    } else if (mDeviceMemorySize == 1) {
      mDeviceMemorySize = devMemory[mDeviceId] - REQUIRE_FREE_MEMORY_RESERVED; // Take all GPU memory but 1/2 GB
    }

    if (mDeviceMemorySize > hipDeviceProp_t.totalGlobalMem || GPUFailedMsgI(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize))) {
      GPUError("CUDA Memory Allocation Error (trying %lld bytes, %lld available)", (long long int)mDeviceMemorySize, (long long int)hipDeviceProp_t.totalGlobalMem);
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (GPUFailedMsgI(hipHostMalloc(&mHostMemoryBase, mHostMemorySize))) {
      GPUError("Error allocating Page Locked Host Memory (trying %lld bytes)", (long long int)mHostMemorySize);
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 1) {
      GPUInfo("Memory ptrs: GPU (%lld bytes): %p - Host (%lld bytes): %p", (long long int)mDeviceMemorySize, mDeviceMemoryBase, (long long int)mHostMemorySize, mHostMemoryBase);
      memset(mHostMemoryBase, 0xDD, mHostMemorySize);
      if (GPUFailedMsgI(hipMemset(mDeviceMemoryBase, 0xDD, mDeviceMemorySize))) {
        GPUError("Error during CUDA memset");
        GPUFailedMsgI(hipDeviceReset());
        return (1);
      }
    }

    for (int i = 0; i < mNStreams; i++) {
      if (GPUFailedMsgI(hipStreamCreateWithFlags(&mInternals->Streams[i], hipStreamNonBlocking))) {
        GPUError("Error creating CUDA Stream");
        GPUFailedMsgI(hipDeviceReset());
        return (1);
      }
    }

    void* devPtrConstantMem;
#ifndef GPUCA_NO_CONSTANT_MEMORY
    if (GPUFailedMsgI(hipGetSymbolAddress(&devPtrConstantMem, HIP_SYMBOL(gGPUConstantMemBuffer)))) {
      GPUError("Error getting ptr to constant memory");
      GPUFailedMsgI(hipDeviceReset());
      return 1;
    }
#else
    if (GPUFailedMsgI(hipMalloc(&devPtrConstantMem, gGPUConstantMemBufferSize))) {
      GPUError("CUDA Memory Allocation Error");
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
#endif
    mDeviceConstantMem = (GPUConstantMem*)devPtrConstantMem;

    dummyInitKernel<<<mBlockCount, 256>>>(mDeviceMemoryBase);
    GPUInfo("CUDA Initialisation successfull (Device %d: %s (Frequency %d, Cores %d), %lld / %lld bytes host / global memory, Stack frame %d, Constant memory %lld)", mDeviceId, hipDeviceProp_t.name, hipDeviceProp_t.clockRate, hipDeviceProp_t.multiProcessorCount, (long long int)mHostMemorySize,
            (long long int)mDeviceMemorySize, (int)GPUCA_GPU_STACK_SIZE, (long long int)gGPUConstantMemBufferSize);
  } else {
    GPUReconstructionCUDABackend* master = dynamic_cast<GPUReconstructionCUDABackend*>(mMaster);
    mDeviceId = master->mDeviceId;
    mBlockCount = master->mBlockCount;
    mWarpSize = master->mWarpSize;
    mMaxThreads = master->mMaxThreads;
    mDeviceName = master->mDeviceName;
    mDeviceConstantMem = master->mDeviceConstantMem;
    mInternals = master->mInternals;
    GPUFailedMsgI(hipCtxPushCurrent(mInternals->CudaContext));
  }

  if (mProcessingSettings.debugLevel >= 1) {
  }
  for (unsigned int i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (unsigned int j = 0; j < mEvents[i].size(); j++) {
      if (GPUFailedMsgI(hipEventCreate(&events[j]))) {
        GPUError("Error creating event");
        GPUFailedMsgI(hipDeviceReset());
        return 1;
      }
    }
  }

  if (GPUFailedMsgI(hipCtxPopCurrent(&mInternals->CudaContext))) {
    GPUError("Error popping CUDA context!");
    return (1);
  }

  return (0);
}

int GPUReconstructionCUDABackend::ExitDevice_Runtime()
{
  // Uninitialize CUDA
  GPUFailedMsgI(hipCtxPushCurrent(mInternals->CudaContext));

  SynchronizeGPU();
  for (unsigned int i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (unsigned int j = 0; j < mEvents[i].size(); j++) {
      GPUFailedMsgI(hipEventDestroy(events[j]));
    }
  }

  if (mMaster == nullptr) {
    GPUFailedMsgI(hipFree(mDeviceMemoryBase));
#ifdef GPUCA_NO_CONSTANT_MEMORY
    GPUFailedMsgI(hipFree(mDeviceConstantMem));
#endif

    for (int i = 0; i < mNStreams; i++) {
      GPUFailedMsgI(hipStreamDestroy(mInternals->Streams[i]));
    }

    GPUFailedMsgI(hipHostFree(mHostMemoryBase));
    GPUFailedMsgI(hipCtxDestroy(mInternals->CudaContext));
    GPUInfo("CUDA Uninitialized");
  } else {
    GPUFailedMsgI(hipCtxPopCurrent(&mInternals->CudaContext));
  }
  mDeviceMemoryBase = nullptr;
  mHostMemoryBase = nullptr;

  /*if (GPUFailedMsgI(hipDeviceReset())) { // No longer doing this, another thread might have used the GPU
    GPUError("Could not uninitialize GPU");
    return (1);
  }*/

  return (0);
}

size_t GPUReconstructionCUDABackend::GPUMemCpy(void* dst, const void* src, size_t size, int stream, int toGPU, deviceEvent* ev, deviceEvent* evList, int nEvents)
{
  if (mProcessingSettings.debugLevel >= 3) {
    stream = -1;
  }
  if (stream == -1) {
    SynchronizeGPU();
    GPUFailedMsg(hipMemcpy(dst, src, size, toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost));
  } else {
    if (evList == nullptr) {
      nEvents = 0;
    }
    for (int k = 0; k < nEvents; k++) {
      GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[stream], ((hipEvent_t*)evList)[k], 0));
    }
    GPUFailedMsg(hipMemcpyAsync(dst, src, size, toGPU == -2 ? hipMemcpyDeviceToDevice : toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost, mInternals->Streams[stream]));
  }
  if (ev) {
    GPUFailedMsg(hipEventRecord(*(hipEvent_t*)ev, mInternals->Streams[stream == -1 ? 0 : stream]));
  }
  return size;
}

size_t GPUReconstructionCUDABackend::TransferMemoryInternal(GPUMemoryResource* res, int stream, deviceEvent* ev, deviceEvent* evList, int nEvents, bool toGPU, const void* src, void* dst)
{
  if (!(res->Type() & GPUMemoryResource::MEMORY_GPU)) {
    if (mProcessingSettings.debugLevel >= 4) {
      GPUInfo("Skipped transfer of non-GPU memory resource: %s", res->Name());
    }
    return 0;
  }
  if (mProcessingSettings.debugLevel >= 3) {
    GPUInfo("Copying to %s: %s - %lld bytes", toGPU ? "GPU" : "Host", res->Name(), (long long int)res->Size());
  }
  return GPUMemCpy(dst, src, res->Size(), stream, toGPU, ev, evList, nEvents);
}

size_t GPUReconstructionCUDABackend::WriteToConstantMemory(size_t offset, const void* src, size_t size, int stream, deviceEvent* ev)
{
#ifndef GPUCA_NO_CONSTANT_MEMORY
  if (stream == -1) {
    GPUFailedMsg(hipMemcpyToSymbol(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice));
  } else {
    GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice, mInternals->Streams[stream]));
  }

#else
  if (stream == -1) {
    GPUFailedMsg(hipMemcpy(((char*)mDeviceConstantMem) + offset, src, size, hipMemcpyHostToDevice));
  } else {
    GPUFailedMsg(hipMemcpyAsync(((char*)mDeviceConstantMem) + offset, src, size, hipMemcpyHostToDevice, mInternals->Streams[stream]));
  }

#endif
  if (ev && stream != -1) {
    GPUFailedMsg(hipEventRecord(*(hipEvent_t*)ev, mInternals->Streams[stream]));
  }
  return size;
}

void GPUReconstructionCUDABackend::ReleaseEvent(deviceEvent* ev) {}
void GPUReconstructionCUDABackend::RecordMarker(deviceEvent* ev, int stream) { GPUFailedMsg(hipEventRecord(*(hipEvent_t*)ev, mInternals->Streams[stream])); }

GPUReconstructionCUDABackend::GPUThreadContextCUDA::GPUThreadContextCUDA(GPUReconstructionCUDAInternals* context) : GPUThreadContext(), mContext(context)
{
  if (mContext->cudaContextObtained++ == 0) {
    hipCtxPushCurrent(mContext->CudaContext);
  }
}
GPUReconstructionCUDABackend::GPUThreadContextCUDA::~GPUThreadContextCUDA()
{
  if (--mContext->cudaContextObtained == 0) {
    hipCtxPopCurrent(&mContext->CudaContext);
  }
}
std::unique_ptr<GPUReconstruction::GPUThreadContext> GPUReconstructionCUDABackend::GetThreadContext() { return std::unique_ptr<GPUThreadContext>(new GPUThreadContextCUDA(mInternals)); }

void GPUReconstructionCUDABackend::SynchronizeGPU() { GPUFailedMsg(hipDeviceSynchronize()); }
void GPUReconstructionCUDABackend::SynchronizeStream(int stream) { GPUFailedMsg(hipStreamSynchronize(mInternals->Streams[stream])); }

void GPUReconstructionCUDABackend::SynchronizeEvents(deviceEvent* evList, int nEvents)
{
  for (int i = 0; i < nEvents; i++) {
    GPUFailedMsg(hipEventSynchronize(((hipEvent_t*)evList)[i]));
  }
}

void GPUReconstructionCUDABackend::StreamWaitForEvents(int stream, deviceEvent* evList, int nEvents)
{
  for (int i = 0; i < nEvents; i++) {
    GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[stream], ((hipEvent_t*)evList)[i], 0));
  }
}

bool GPUReconstructionCUDABackend::IsEventDone(deviceEvent* evList, int nEvents)
{
  for (int i = 0; i < nEvents; i++) {
    hipError_t retVal = hipEventSynchronize(((hipEvent_t*)evList)[i]);
    if (retVal == hipErrorNotReady) {
      return false;
    }
    GPUFailedMsg(retVal);
  }
  return (true);
}

int GPUReconstructionCUDABackend::GPUDebug(const char* state, int stream)
{
  // Wait for CUDA-Kernel to finish and check for CUDA errors afterwards, in case of debugmode
  hipError_t cuErr;
  cuErr = hipGetLastError();
  if (cuErr != hipSuccess) {
    GPUError("Cuda Error %s while running kernel (%s) (Stream %d)", hipGetErrorString(cuErr), state, stream);
    return (1);
  }
  if (mProcessingSettings.debugLevel == 0) {
    return (0);
  }
  if (GPUFailedMsgI(hipDeviceSynchronize())) {
    GPUError("CUDA Error while synchronizing (%s) (Stream %d)", state, stream);
    return (1);
  }
  if (mProcessingSettings.debugLevel >= 3) {
    GPUInfo("GPU Sync Done");
  }
  return (0);
}

int GPUReconstructionCUDABackend::PrepareTextures()
{
#ifdef GPUCA_USE_TEXTURES
  hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
  size_t offset;
  GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSLICES * GPUCA_SLICE_DATA_MEMORY));
  hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
  GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSLICES * GPUCA_SLICE_DATA_MEMORY));
#endif
  return (0);
}

int GPUReconstructionCUDABackend::registerMemoryForGPU(const void* ptr, size_t size)
{
  return GPUFailedMsgI(hipHostRegister((void*)ptr, size, hipHostRegisterDefault));
}

int GPUReconstructionCUDABackend::unregisterMemoryForGPU(const void* ptr)
{
  return GPUFailedMsgI(hipHostUnregister((void*)ptr));
}

void GPUReconstructionCUDABackend::PrintKernelOccupancies()
{
  int maxBlocks, threads, suggestedBlocks;
  hipFuncAttributes attr;
  GPUFailedMsg(hipCtxPushCurrent(mInternals->CudaContext));
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward) GPUCA_KRNL_WRAP(GPUCA_KRNL_LOAD_, x_class, x_attributes, x_arguments, x_forward)
#define GPUCA_KRNL_LOAD_single(x_class, x_attributes, x_arguments, x_forward)                                                          \
  GPUFailedMsg(hipOccupancyMaxPotentialBlockSize(&suggestedBlocks, &threads, GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))));        \
  GPUFailedMsg(hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocks, GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class)), threads, 0)); \
  GPUFailedMsg(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(GPUCA_M_CAT(krnl_), GPUCA_M_KRNL_NAME(x_class))));                                          \
  GPUInfo("Kernel: %50s Block size: %4d, Maximum active blocks: %3d, Suggested blocks: %3d, Regs: %3d, smem: %3d", GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))), threads, maxBlocks, suggestedBlocks, attr.numRegs, (int)attr.sharedSizeBytes);
#define GPUCA_KRNL_LOAD_multi(x_class, x_attributes, x_arguments, x_forward)                                                                    \
  GPUFailedMsg(hipOccupancyMaxPotentialBlockSize(&suggestedBlocks, &threads, GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)));        \
  GPUFailedMsg(hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocks, GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi), threads, 0)); \
  GPUFailedMsg(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(GPUCA_M_CAT3(krnl_), GPUCA_M_KRNL_NAME(x_class), _multi)));                                          \
  GPUInfo("Kernel: %50s Block size: %4d, Maximum active blocks: %3d, Suggested blocks: %3d, Regs: %3d, smem: %3d", GPUCA_M_STR(GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)), threads, maxBlocks, suggestedBlocks, attr.numRegs, (int)attr.sharedSizeBytes);
#include "GPUReconstructionKernels.h"
#undef GPUCA_KRNL
#undef GPUCA_KRNL_LOAD_single
#undef GPUCA_KRNL_LOAD_multi
  GPUFailedMsg(hipCtxPopCurrent(&mInternals->CudaContext));
}

void GPUReconstructionCUDABackend::startGPUProfiling()
{
  GPUFailedMsg(hipProfilerStart());
}

void GPUReconstructionCUDABackend::endGPUProfiling()
{
  GPUFailedMsg(hipProfilerStop());
}
