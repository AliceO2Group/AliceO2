#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDA.cu
/// \author David Rohr

#define GPUCA_GPUCODE_HOSTONLY
#include "GPUReconstructionCUDADef.h"
#include "GPUReconstructionCUDAIncludes.h"

#include <hip/hip_runtime_api.h>

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"
#include "CUDAThrustHelpers.h"
#include "GPUReconstructionIncludes.h"
#include "GPUParamRTC.h"

#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
#include "utils/qGetLdBinarySymbols.h"
#ifndef __HIPCC__ // CUDA
#define PER_KERNEL_OBJECT_EXT _fatbin
#else // HIP
#define PER_KERNEL_OBJECT_EXT _hip_cxx_o
#endif
#define GPUCA_KRNL(x_class, ...) QGET_LD_BINARY_SYMBOLS(GPUCA_M_CAT3(cuda_kernel_module_fatbin_krnl_, GPUCA_M_KRNL_NAME(x_class), PER_KERNEL_OBJECT_EXT))
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
#endif

static constexpr size_t REQUIRE_MIN_MEMORY = 1024L * 1024 * 1024;
static constexpr size_t REQUIRE_MEMORY_RESERVED = 512L * 1024 * 1024;
static constexpr size_t REQUIRE_FREE_MEMORY_RESERVED_PER_SM = 40L * 1024 * 1024;
static constexpr size_t RESERVE_EXTRA_MEM_THRESHOLD = 10L * 1024 * 1024 * 1024;
static constexpr size_t RESERVE_EXTRA_MEM_OFFSET = 1L * 512 * 1024 * 1024;

using namespace GPUCA_NAMESPACE::gpu;

__global__ void dummyInitKernel(void*) {}

#include "GPUReconstructionIncludesITS.h"

GPUReconstructionCUDABackend::GPUReconstructionCUDABackend(const GPUSettingsDeviceBackend& cfg) : GPUReconstructionDeviceBase(cfg, sizeof(GPUReconstructionDeviceBase))
{
  if (mMaster == nullptr) {
    mInternals = new GPUReconstructionCUDAInternals;
  }
}

GPUReconstructionCUDABackend::~GPUReconstructionCUDABackend()
{
  if (mMaster == nullptr) {
    delete mInternals;
  }
}

int32_t GPUReconstructionCUDABackend::GPUFailedMsgAI(const int64_t error, const char* file, int32_t line)
{
  // Check for CUDA Error and in the case of an error display the corresponding error string
  if (error == hipSuccess) {
    return (0);
  }
  GPUError("CUDA Error: %ld / %s (%s:%d)", error, hipGetErrorString((hipError_t)error), file, line);
  return 1;
}

void GPUReconstructionCUDABackend::GPUFailedMsgA(const int64_t error, const char* file, int32_t line)
{
  if (GPUFailedMsgAI(error, file, line)) {
    static bool runningCallbacks = false;
    if (IsInitialized() && runningCallbacks == false) {
      runningCallbacks = true;
      CheckErrorCodes(false, true);
    }
    throw std::runtime_error("CUDA Failure");
  }
}

GPUReconstructionCUDA::GPUReconstructionCUDA(const GPUSettingsDeviceBackend& cfg) : GPUReconstructionKernels(cfg)
{
  mDeviceBackendSettings.deviceType = DeviceType::CUDA;
#ifndef __HIPCC__ // CUDA
  mRtcSrcExtension = ".cu";
  mRtcBinExtension = ".fatbin";
#else // HIP
  mRtcSrcExtension = ".hip";
  mRtcBinExtension = ".o";
#endif
}

GPUReconstructionCUDA::~GPUReconstructionCUDA()
{
  Exit(); // Make sure we destroy everything (in particular the ITS tracker) before we exit CUDA
}

GPUReconstruction* GPUReconstruction_Create_CUDA(const GPUSettingsDeviceBackend& cfg) { return new GPUReconstructionCUDA(cfg); }

void GPUReconstructionCUDA::GetITSTraits(std::unique_ptr<o2::its::TrackerTraits>* trackerTraits, std::unique_ptr<o2::its::VertexerTraits>* vertexerTraits, std::unique_ptr<o2::its::TimeFrame>* timeFrame)
{
  if (trackerTraits) {
    trackerTraits->reset(new o2::its::TrackerTraitsGPU);
  }
  if (vertexerTraits) {
    vertexerTraits->reset(new o2::its::VertexerTraitsGPU);
  }
  if (timeFrame) {
    timeFrame->reset(new o2::its::gpu::TimeFrameGPU);
  }
}

void GPUReconstructionCUDA::UpdateAutomaticProcessingSettings()
{
  GPUCA_GPUReconstructionUpdateDefaults();
}

int32_t GPUReconstructionCUDA::InitDevice_Runtime()
{
#ifndef __HIPCC__ // CUDA
  constexpr int32_t reqVerMaj = 2;
  constexpr int32_t reqVerMin = 0;
#endif
  if (mProcessingSettings.rtc.enable && mProcessingSettings.rtc.runTest == 2) {
    genAndLoadRTC();
    exit(0);
  }

  if (mMaster == nullptr) {
    hipDeviceProp_t deviceProp;
    int32_t count, bestDevice = -1;
    double bestDeviceSpeed = -1, deviceSpeed;
    if (GPUFailedMsgI(hipGetDeviceCount(&count))) {
      GPUError("Error getting CUDA Device Count");
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Available CUDA devices:");
    }
    std::vector<bool> devicesOK(count, false);
    std::vector<size_t> devMemory(count, 0);
    bool contextCreated = false;
    for (int32_t i = 0; i < count; i++) {
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Examining device %d", i);
      }
      size_t free, total;
#ifndef __HIPCC__ // CUDA
      if (GPUFailedMsgI(cudaInitDevice(i, 0, 0))) {
#else // HIP
      if (GPUFailedMsgI(hipSetDevice(i))) {
#endif
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Couldn't create context for device %d. Skipping it.", i);
        }
        continue;
      }
      contextCreated = true;
      if (GPUFailedMsgI(hipMemGetInfo(&free, &total))) {
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Error obtaining CUDA memory info about device %d! Skipping it.", i);
        }
        GPUFailedMsg(hipDeviceReset());
        continue;
      }
      if (count > 1) {
        GPUFailedMsg(hipDeviceReset());
        contextCreated = false;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained current memory usage for device %d", i);
      }
      if (GPUFailedMsgI(hipGetDeviceProperties(&deviceProp, i))) {
        continue;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained device properties for device %d", i);
      }
      int32_t deviceOK = true;
      [[maybe_unused]] const char* deviceFailure = "";
#ifndef __HIPCC__
      if (deviceProp.major < reqVerMaj || (deviceProp.major == reqVerMaj && deviceProp.minor < reqVerMin)) {
        deviceOK = false;
        deviceFailure = "Too low device revision";
      }
#endif
      if (free < std::max<size_t>(mDeviceMemorySize, REQUIRE_MIN_MEMORY)) {
        deviceOK = false;
        deviceFailure = "Insufficient GPU memory";
      }

      deviceSpeed = (double)deviceProp.multiProcessorCount * (double)deviceProp.clockRate * (double)deviceProp.warpSize * (double)free * (double)deviceProp.major * (double)deviceProp.major;
      if (mProcessingSettings.debugLevel >= 2) {
        GPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lu / %lu)%s %s", deviceOK ? " " : "[", i, deviceProp.name, deviceProp.major, deviceProp.minor, free, (size_t)deviceProp.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
      }
      if (!deviceOK) {
        continue;
      }
      devicesOK[i] = true;
      devMemory[i] = std::min<size_t>(free, std::max<int64_t>(0, total - REQUIRE_MEMORY_RESERVED));
      if (deviceSpeed > bestDeviceSpeed) {
        bestDevice = i;
        bestDeviceSpeed = deviceSpeed;
      } else {
        if (mProcessingSettings.debugLevel >= 2 && mProcessingSettings.deviceNum < 0) {
          GPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
        }
      }
    }

    bool noDevice = false;
    if (bestDevice == -1) {
      GPUWarning("No %sCUDA Device available, aborting CUDA Initialisation (Required mem: %ld)", count ? "appropriate " : "", (int64_t)mDeviceMemorySize);
#ifndef __HIPCC__
      GPUImportant("Requiring Revision %d.%d, Mem: %lu", reqVerMaj, reqVerMin, std::max<size_t>(mDeviceMemorySize, REQUIRE_MIN_MEMORY));
#endif
      noDevice = true;
    } else if (mProcessingSettings.deviceNum > -1) {
      if (mProcessingSettings.deviceNum >= (signed)count) {
        GPUError("Requested device ID %d does not exist", mProcessingSettings.deviceNum);
        noDevice = true;
      } else if (!devicesOK[mProcessingSettings.deviceNum]) {
        GPUError("Unsupported device requested (%d)", mProcessingSettings.deviceNum);
        noDevice = true;
      } else {
        bestDevice = mProcessingSettings.deviceNum;
      }
    }
    if (noDevice) {
      if (contextCreated) {
        GPUFailedMsgI(hipDeviceReset());
      }
      return (1);
    }
    mDeviceId = bestDevice;

    GPUFailedMsgI(hipGetDeviceProperties(&deviceProp, mDeviceId));

    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Using CUDA Device %s with Properties:", deviceProp.name);
      GPUInfo("\ttotalGlobalMem = %ld", (uint64_t)deviceProp.totalGlobalMem);
      GPUInfo("\tsharedMemPerBlock = %ld", (uint64_t)deviceProp.sharedMemPerBlock);
      GPUInfo("\tregsPerBlock = %d", deviceProp.regsPerBlock);
      GPUInfo("\twarpSize = %d", deviceProp.warpSize);
      GPUInfo("\tmemPitch = %ld", (uint64_t)deviceProp.memPitch);
      GPUInfo("\tmaxThreadsPerBlock = %d", deviceProp.maxThreadsPerBlock);
      GPUInfo("\tmaxThreadsDim = %d %d %d", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
      GPUInfo("\tmaxGridSize = %d %d %d", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
      GPUInfo("\ttotalConstMem = %ld", (uint64_t)deviceProp.totalConstMem);
      GPUInfo("\tmajor = %d", deviceProp.major);
      GPUInfo("\tminor = %d", deviceProp.minor);
      GPUInfo("\tclockRate = %d", deviceProp.clockRate);
      GPUInfo("\tmemoryClockRate = %d", deviceProp.memoryClockRate);
      GPUInfo("\tmultiProcessorCount = %d", deviceProp.multiProcessorCount);
      GPUInfo("\ttextureAlignment = %ld", (uint64_t)deviceProp.textureAlignment);
      GPUInfo(" ");
    }
    if (deviceProp.warpSize != GPUCA_WARP_SIZE) {
      throw std::runtime_error("Invalid warp size on GPU");
    }
    mBlockCount = deviceProp.multiProcessorCount;
    mMaxThreads = std::max<int32_t>(mMaxThreads, deviceProp.maxThreadsPerBlock * mBlockCount);
#ifndef __HIPCC__ // CUDA
    mWarpSize = 32;
#else // HIP
    mWarpSize = 64;
#endif
    mDeviceName = deviceProp.name;
    mDeviceName += " (CUDA GPU)";

    if (deviceProp.major < 3) {
      GPUError("Unsupported CUDA Device");
      return (1);
    }

#ifdef GPUCA_USE_TEXTURES
    if (GPUCA_SLICE_DATA_MEMORY * NSLICES > (size_t)deviceProp.maxTexture1DLinear) {
      GPUError("Invalid maximum texture size of device: %ld < %ld\n", (int64_t)deviceProp.maxTexture1DLinear, (int64_t)(GPUCA_SLICE_DATA_MEMORY * NSLICES));
      return (1);
    }
#endif
#ifndef GPUCA_NO_CONSTANT_MEMORY
    if (gGPUConstantMemBufferSize > deviceProp.totalConstMem) {
      GPUError("Insufficient constant memory available on GPU %d < %d!", (int32_t)deviceProp.totalConstMem, (int32_t)gGPUConstantMemBufferSize);
      return (1);
    }
#endif

#ifndef __HIPCC__ // CUDA
    if (contextCreated == 0 && GPUFailedMsgI(cudaInitDevice(mDeviceId, 0, 0))) {
#else // HIP
    if (contextCreated == 0 && GPUFailedMsgI(hipSetDevice(mDeviceId))) {
#endif
      GPUError("Could not set CUDA Device!");
      return (1);
    }

#ifndef __HIPCC__ // CUDA
    if (GPUFailedMsgI(hipDeviceSetLimit(hipLimitStackSize, GPUCA_GPU_STACK_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (GPUFailedMsgI(hipDeviceSetLimit(hipLimitMallocHeapSize, mProcessingSettings.deterministicGPUReconstruction ? std::max<size_t>(1024 * 1024 * 1024, GPUCA_GPU_HEAP_SIZE) : GPUCA_GPU_HEAP_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
#else // HIP
    if (GPUFailedMsgI(hipSetDeviceFlags(hipDeviceScheduleBlockingSync))) {
      GPUError("Could not set HIP Device flags!");
      return (1);
    }
#endif

    if (mDeviceMemorySize == 1 || mDeviceMemorySize == 2) {
      mDeviceMemorySize = std::max<int64_t>(0, devMemory[mDeviceId] - REQUIRE_FREE_MEMORY_RESERVED_PER_SM * deviceProp.multiProcessorCount); // Take all GPU memory but some reserve
      if (mDeviceMemorySize >= RESERVE_EXTRA_MEM_THRESHOLD) {
        mDeviceMemorySize -= RESERVE_EXTRA_MEM_OFFSET;
      }
    }
    if (mDeviceMemorySize == 2) {
      mDeviceMemorySize = mDeviceMemorySize * 2 / 3; // Leave 1/3 of GPU memory for event display
    }

    if (mProcessingSettings.debugLevel >= 3) {
      GPUInfo("Allocating memory on GPU");
    }
    if (mDeviceMemorySize > deviceProp.totalGlobalMem || GPUFailedMsgI(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize))) {
      size_t free, total;
      GPUFailedMsg(hipMemGetInfo(&free, &total));
      GPUError("CUDA Memory Allocation Error (trying %ld bytes, %ld available on GPU, %ld free)", (int64_t)mDeviceMemorySize, (int64_t)deviceProp.totalGlobalMem, (int64_t)free);
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 3) {
      GPUInfo("Allocating memory on Host");
    }
    if (GPUFailedMsgI(hipHostMalloc(&mHostMemoryBase, mHostMemorySize))) {
      GPUError("Error allocating Page Locked Host Memory (trying %ld bytes)", (int64_t)mHostMemorySize);
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 1) {
      GPUInfo("Memory ptrs: GPU (%ld bytes): %p - Host (%ld bytes): %p", (int64_t)mDeviceMemorySize, mDeviceMemoryBase, (int64_t)mHostMemorySize, mHostMemoryBase);
      memset(mHostMemoryBase, 0xDD, mHostMemorySize);
      if (GPUFailedMsgI(hipMemset(mDeviceMemoryBase, 0xDD, mDeviceMemorySize))) {
        GPUError("Error during CUDA memset");
        GPUFailedMsgI(hipDeviceReset());
        return (1);
      }
    }

    for (int32_t i = 0; i < mNStreams; i++) {
      if (GPUFailedMsgI(hipStreamCreateWithFlags(&mInternals->Streams[i], hipStreamNonBlocking))) {
        GPUError("Error creating CUDA Stream");
        GPUFailedMsgI(hipDeviceReset());
        return (1);
      }
    }

#ifndef __HIPCC__ // CUDA
    dummyInitKernel<<<mBlockCount, 256>>>(mDeviceMemoryBase);
#else // HIP
    hipLaunchKernelGGL(HIP_KERNEL_NAME(dummyInitKernel), dim3(mBlockCount), dim3(256), 0, 0, mDeviceMemoryBase);
#endif

#ifndef GPUCA_ALIROOT_LIB
    if (mProcessingSettings.rtc.enable) {
      genAndLoadRTC();
    }
#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
    else {
#define GPUCA_KRNL(x_class, ...)                                        \
  mInternals->kernelModules.emplace_back(std::make_unique<hipModule_t>()); \
  GPUFailedMsg(hipModuleLoadData(mInternals->kernelModules.back().get(), GPUCA_M_CAT3(_binary_cuda_kernel_module_fatbin_krnl_, GPUCA_M_KRNL_NAME(x_class), GPUCA_M_CAT(PER_KERNEL_OBJECT_EXT, _start))));
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
      loadKernelModules(true, false);
    }
#endif
#endif
    void* devPtrConstantMem = nullptr;
#ifndef GPUCA_NO_CONSTANT_MEMORY
    runConstantRegistrators();
    devPtrConstantMem = mDeviceConstantMemList[0];
    for (uint32_t i = 0; i < mInternals->kernelModules.size(); i++) {
#ifndef __HIPCC__
      hipDeviceptr_t tmp; // CUDA has a custom type, that initializes to zero and cannot be initialized with nullptr
#else
      hipDeviceptr_t tmp = nullptr; // HIP just uses void*
#endif
      size_t tmpSize = 0;
      GPUFailedMsg(hipModuleGetGlobal(&tmp, &tmpSize, *mInternals->kernelModules[i], "gGPUConstantMemBuffer"));
      mDeviceConstantMemList.emplace_back((void*)tmp);
    }
#else
    GPUFailedMsg(hipMalloc(&devPtrConstantMem, gGPUConstantMemBufferSize));
#endif
    mDeviceConstantMem = (GPUConstantMem*)devPtrConstantMem;

    GPUInfo("CUDA Initialisation successfull (Device %d: %s (Frequency %d, Cores %d), %ld / %ld bytes host / global memory, Stack frame %d, Constant memory %ld)", mDeviceId, deviceProp.name, deviceProp.clockRate, deviceProp.multiProcessorCount, (int64_t)mHostMemorySize, (int64_t)mDeviceMemorySize, (int32_t)GPUCA_GPU_STACK_SIZE, (int64_t)gGPUConstantMemBufferSize);
  } else {
    GPUReconstructionCUDA* master = dynamic_cast<GPUReconstructionCUDA*>(mMaster);
    mDeviceId = master->mDeviceId;
    mBlockCount = master->mBlockCount;
    mWarpSize = master->mWarpSize;
    mMaxThreads = master->mMaxThreads;
    mDeviceName = master->mDeviceName;
    mDeviceConstantMem = master->mDeviceConstantMem;
    mDeviceConstantMemList.resize(master->mDeviceConstantMemList.size());
    std::copy(master->mDeviceConstantMemList.begin(), master->mDeviceConstantMemList.end(), mDeviceConstantMemList.begin());
    mInternals = master->mInternals;
    GPUFailedMsg(hipSetDevice(mDeviceId));

    GPUInfo("CUDA Initialized from master");
  }

  for (uint32_t i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (uint32_t j = 0; j < mEvents[i].size(); j++) {
#ifndef __HIPCC__ // CUDA
      if (GPUFailedMsgI(hipEventCreate(&events[j]))) {
#else
      if (GPUFailedMsgI(hipEventCreateWithFlags(&events[j], hipEventBlockingSync))) {
#endif
        GPUError("Error creating event");
        GPUFailedMsgI(hipDeviceReset());
        return 1;
      }
    }
  }

  return (0);
}

void GPUReconstructionCUDA::genAndLoadRTC()
{
  std::string filename = "";
  uint32_t nCompile = 0;
  if (genRTC(filename, nCompile)) {
    throw std::runtime_error("Runtime compilation failed");
  }
  for (uint32_t i = 0; i < nCompile; i++) {
    if (mProcessingSettings.rtc.runTest != 2) {
      mInternals->kernelModules.emplace_back(std::make_unique<hipModule_t>());
      GPUFailedMsg(hipModuleLoad(mInternals->kernelModules.back().get(), (filename + "_" + std::to_string(i) + mRtcBinExtension).c_str()));
    }
    remove((filename + "_" + std::to_string(i) + mRtcSrcExtension).c_str());
    remove((filename + "_" + std::to_string(i) + mRtcBinExtension).c_str());
  }
  if (mProcessingSettings.rtc.runTest == 2) {
    return;
  }
  loadKernelModules(mProcessingSettings.rtc.compilePerKernel);
}

int32_t GPUReconstructionCUDA::ExitDevice_Runtime()
{
  // Uninitialize CUDA
  GPUFailedMsg(hipSetDevice(mDeviceId));
  SynchronizeGPU();
  unregisterRemainingRegisteredMemory();

  for (uint32_t i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (uint32_t j = 0; j < mEvents[i].size(); j++) {
      GPUFailedMsgI(hipEventDestroy(events[j]));
    }
  }

  if (mMaster == nullptr) {
    GPUFailedMsgI(hipFree(mDeviceMemoryBase));
#ifdef GPUCA_NO_CONSTANT_MEMORY
    GPUFailedMsgI(hipFree(mDeviceConstantMem));
#endif

    for (int32_t i = 0; i < mNStreams; i++) {
      GPUFailedMsgI(hipStreamDestroy(mInternals->Streams[i]));
    }

    GPUFailedMsgI(hipHostFree(mHostMemoryBase));
    for (uint32_t i = 0; i < mInternals->kernelModules.size(); i++) {
      GPUFailedMsg(hipModuleUnload(*mInternals->kernelModules[i]));
    }

    GPUFailedMsgI(hipDeviceReset());
    GPUInfo("CUDA Uninitialized");
  }
  mDeviceMemoryBase = nullptr;
  mHostMemoryBase = nullptr;

  return (0);
}

size_t GPUReconstructionCUDA::GPUMemCpy(void* dst, const void* src, size_t size, int32_t stream, int32_t toGPU, deviceEvent* ev, deviceEvent* evList, int32_t nEvents)
{
  if (mProcessingSettings.debugLevel >= 3) {
    stream = -1;
  }
  if (stream == -1) {
    SynchronizeGPU();
    GPUFailedMsg(hipMemcpy(dst, src, size, toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost));
  } else {
    if (evList == nullptr) {
      nEvents = 0;
    }
    for (int32_t k = 0; k < nEvents; k++) {
      GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[stream], evList[k].get<hipEvent_t>(), 0));
    }
    GPUFailedMsg(hipMemcpyAsync(dst, src, size, toGPU == -2 ? hipMemcpyDeviceToDevice : toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost, mInternals->Streams[stream]));
  }
  if (ev) {
    GPUFailedMsg(hipEventRecord(ev->get<hipEvent_t>(), mInternals->Streams[stream == -1 ? 0 : stream]));
  }
  if (mProcessingSettings.serializeGPU & 2) {
    GPUDebug(("GPUMemCpy " + std::to_string(toGPU)).c_str(), stream, true);
  }
  return size;
}

size_t GPUReconstructionCUDA::WriteToConstantMemory(size_t offset, const void* src, size_t size, int32_t stream, deviceEvent* ev)
{
  for (uint32_t i = 0; i < 1 + mDeviceConstantMemList.size(); i++) {
    void* basePtr = i ? mDeviceConstantMemList[i - 1] : mDeviceConstantMem;
    if (basePtr == nullptr || (i && basePtr == (void*)mDeviceConstantMem)) {
      continue;
    }
    if (stream == -1) {
      GPUFailedMsg(hipMemcpy(((char*)basePtr) + offset, src, size, hipMemcpyHostToDevice));
    } else {
      GPUFailedMsg(hipMemcpyAsync(((char*)basePtr) + offset, src, size, hipMemcpyHostToDevice, mInternals->Streams[stream]));
    }
  }
  if (ev && stream != -1) {
    GPUFailedMsg(hipEventRecord(ev->get<hipEvent_t>(), mInternals->Streams[stream]));
  }
  if (mProcessingSettings.serializeGPU & 2) {
    GPUDebug("WriteToConstantMemory", stream, true);
  }
  return size;
}

void GPUReconstructionCUDA::ReleaseEvent(deviceEvent ev) {}
void GPUReconstructionCUDA::RecordMarker(deviceEvent ev, int32_t stream) { GPUFailedMsg(hipEventRecord(ev.get<hipEvent_t>(), mInternals->Streams[stream])); }

std::unique_ptr<GPUReconstruction::GPUThreadContext> GPUReconstructionCUDA::GetThreadContext()
{
  GPUFailedMsg(hipSetDevice(mDeviceId));
  return std::unique_ptr<GPUThreadContext>(new GPUThreadContext);
}

void GPUReconstructionCUDA::SynchronizeGPU() { GPUFailedMsg(hipDeviceSynchronize()); }
void GPUReconstructionCUDA::SynchronizeStream(int32_t stream) { GPUFailedMsg(hipStreamSynchronize(mInternals->Streams[stream])); }

void GPUReconstructionCUDA::SynchronizeEvents(deviceEvent* evList, int32_t nEvents)
{
  for (int32_t i = 0; i < nEvents; i++) {
    GPUFailedMsg(hipEventSynchronize(evList[i].get<hipEvent_t>()));
  }
}

void GPUReconstructionCUDA::StreamWaitForEvents(int32_t stream, deviceEvent* evList, int32_t nEvents)
{
  for (int32_t i = 0; i < nEvents; i++) {
    GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[stream], evList[i].get<hipEvent_t>(), 0));
  }
}

bool GPUReconstructionCUDA::IsEventDone(deviceEvent* evList, int32_t nEvents)
{
  for (int32_t i = 0; i < nEvents; i++) {
    hipError_t retVal = hipEventSynchronize(evList[i].get<hipEvent_t>());
    if (retVal == hipErrorNotReady) {
      return false;
    }
    GPUFailedMsg(retVal);
  }
  return (true);
}

int32_t GPUReconstructionCUDA::GPUDebug(const char* state, int32_t stream, bool force)
{
  // Wait for CUDA-Kernel to finish and check for CUDA errors afterwards, in case of debugmode
  hipError_t cuErr;
  cuErr = hipGetLastError();
  if (cuErr != hipSuccess) {
    GPUError("CUDA Error %s while running (%s) (Stream %d)", hipGetErrorString(cuErr), state, stream);
    return (1);
  }
  if (!force && mProcessingSettings.debugLevel <= 0) {
    return (0);
  }
  if (GPUFailedMsgI(stream == -1 ? hipDeviceSynchronize() : hipStreamSynchronize(mInternals->Streams[stream]))) {
    GPUError("CUDA Error while synchronizing (%s) (Stream %d)", state, stream);
    return (1);
  }
  if (mProcessingSettings.debugLevel >= 3) {
    GPUInfo("GPU Sync Done");
  }
  return (0);
}

int32_t GPUReconstructionCUDA::registerMemoryForGPU_internal(const void* ptr, size_t size)
{
  if (mProcessingSettings.debugLevel >= 3) {
    GPUInfo("Registering %zu bytes of memory for GPU", size);
  }
  return GPUFailedMsgI(hipHostRegister((void*)ptr, size, hipHostRegisterDefault));
}

int32_t GPUReconstructionCUDA::unregisterMemoryForGPU_internal(const void* ptr)
{
  return GPUFailedMsgI(hipHostUnregister((void*)ptr));
}

void GPUReconstructionCUDABackend::PrintKernelOccupancies()
{
  int32_t maxBlocks = 0, threads = 0, suggestedBlocks = 0, nRegs = 0, sMem = 0;
  GPUFailedMsg(hipSetDevice(mDeviceId));
  for (uint32_t i = 0; i < mInternals->kernelFunctions.size(); i++) {
    GPUFailedMsg(hipModuleOccupancyMaxPotentialBlockSize(&suggestedBlocks, &threads, *mInternals->kernelFunctions[i], 0, 0));
    GPUFailedMsg(hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocks, *mInternals->kernelFunctions[i], threads, 0));
    GPUFailedMsg(hipFuncGetAttribute(&nRegs, HIP_FUNC_ATTRIBUTE_NUM_REGS, *mInternals->kernelFunctions[i]));
    GPUFailedMsg(hipFuncGetAttribute(&sMem, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, *mInternals->kernelFunctions[i]));
    GPUInfo("Kernel: %50s Block size: %4d, Maximum active blocks: %3d, Suggested blocks: %3d, Regs: %3d, smem: %3d", mInternals->kernelNames[i].c_str(), threads, maxBlocks, suggestedBlocks, nRegs, sMem);
  }
}

void GPUReconstructionCUDA::loadKernelModules(bool perKernel, bool perSingleMulti)
{
  uint32_t j = 0;
#define GPUCA_KRNL(...)                          \
  GPUCA_KRNL_WRAP(GPUCA_KRNL_LOAD_, __VA_ARGS__) \
  j += !perSingleMulti;
#define GPUCA_KRNL_LOAD_single(x_class, ...)                                                                                                                                               \
  getRTCkernelNum<false, GPUCA_M_KRNL_TEMPLATE(x_class)>(mInternals->kernelFunctions.size());                                                                                              \
  mInternals->kernelFunctions.emplace_back(new hipFunction_t);                                                                                                                                \
  mInternals->kernelNames.emplace_back(GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))));                                                                                       \
  if (mProcessingSettings.debugLevel >= 3) {                                                                                                                                               \
    GPUInfo("Loading kernel %s (j = %u)", GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))), j);                                                                                 \
  }                                                                                                                                                                                        \
  GPUFailedMsg(hipModuleGetFunction(mInternals->kernelFunctions.back().get(), *mInternals->kernelModules[perKernel ? j : 0], GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class))))); \
  j += perSingleMulti;
#define GPUCA_KRNL_LOAD_multi(x_class, ...)                                                                                                                                                         \
  getRTCkernelNum<true, GPUCA_M_KRNL_TEMPLATE(x_class)>(mInternals->kernelFunctions.size());                                                                                                        \
  mInternals->kernelFunctions.emplace_back(new hipFunction_t);                                                                                                                                         \
  mInternals->kernelNames.emplace_back(GPUCA_M_STR(GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)));                                                                                       \
  if (mProcessingSettings.debugLevel >= 3) {                                                                                                                                                        \
    GPUInfo("Loading kernel %s (j = %u)", GPUCA_M_STR(GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)), j);                                                                                 \
  }                                                                                                                                                                                                 \
  GPUFailedMsg(hipModuleGetFunction(mInternals->kernelFunctions.back().get(), *mInternals->kernelModules[perKernel ? j : 0], GPUCA_M_STR(GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi)))); \
  j += perSingleMulti;
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
#undef GPUCA_KRNL_LOAD_single
#undef GPUCA_KRNL_LOAD_multi

  if (j != mInternals->kernelModules.size()) {
    GPUFatal("Did not load all kernels (%u < %u)", j, (uint32_t)mInternals->kernelModules.size());
  }
}

#ifndef __HIPCC__ // CUDA
int32_t GPUReconstructionCUDA::PrepareTextures()
{
#ifdef GPUCA_USE_TEXTURES
  hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
  size_t offset;
  GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSLICES * GPUCA_SLICE_DATA_MEMORY));
  hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
  GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSLICES * GPUCA_SLICE_DATA_MEMORY));
#endif
  return (0);
}

void GPUReconstructionCUDA::startGPUProfiling()
{
  GPUFailedMsg(hipProfilerStart());
}

void GPUReconstructionCUDA::endGPUProfiling()
{
  GPUFailedMsg(hipProfilerStop());
}
#else  // HIP
void* GPUReconstructionHIP::getGPUPointer(void* ptr)
{
  void* retVal = nullptr;
  GPUFailedMsg(hipHostGetDevicePointer(&retVal, ptr, 0));
  return retVal;
}
#endif // __HIPCC__

namespace GPUCA_NAMESPACE::gpu
{
template class GPUReconstructionKernels<GPUReconstructionCUDABackend>;
}
