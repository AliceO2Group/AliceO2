#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDA.cu
/// \author David Rohr

#define GPUCA_GPUCODE_HOSTONLY
#include "GPUReconstructionCUDADef.h"
#include "GPUReconstructionCUDAIncludes.h"

#include <hip/hip_runtime_api.h>

#include "GPUReconstructionCUDA.h"
#include "GPUReconstructionCUDAInternals.h"
#include "CUDAThrustHelpers.h"
#include "GPUReconstructionIncludes.h"
#include "GPUParamRTC.h"

#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
#include "utils/qGetLdBinarySymbols.h"
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward) QGET_LD_BINARY_SYMBOLS(GPUCA_M_CAT3(cuda_kernel_module_fatbin_krnl_, GPUCA_M_KRNL_NAME(x_class), _fatbin))
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
#endif

static constexpr size_t REQUIRE_MIN_MEMORY = 1024L * 1024 * 1024;
static constexpr size_t REQUIRE_MEMORY_RESERVED = 512L * 1024 * 1024;
static constexpr size_t REQUIRE_FREE_MEMORY_RESERVED_PER_SM = 40L * 1024 * 1024;
static constexpr size_t RESERVE_EXTRA_MEM_THRESHOLD = 10L * 1024 * 1024 * 1024;
static constexpr size_t RESERVE_EXTRA_MEM_OFFSET = 1L * 512 * 1024 * 1024;

using namespace GPUCA_NAMESPACE::gpu;

__global__ void dummyInitKernel(void*) {}

#include "GPUReconstructionIncludesITS.h"

GPUReconstructionCUDABackend::GPUReconstructionCUDABackend(const GPUSettingsDeviceBackend& cfg) : GPUReconstructionDeviceBase(cfg, sizeof(GPUReconstructionDeviceBase))
{
  if (mMaster == nullptr) {
    mInternals = new GPUReconstructionCUDAInternals;
  }
}

GPUReconstructionCUDABackend::~GPUReconstructionCUDABackend()
{
  if (mMaster == nullptr) {
    for (unsigned int i = 0; i < mInternals->kernelModules.size(); i++) {
      GPUFailedMsg(hipModuleUnload(*mInternals->kernelModules[i]));
    }
    delete mInternals;
  }
}

int GPUReconstructionCUDABackend::GPUFailedMsgAI(const long long int error, const char* file, int line)
{
  // Check for CUDA Error and in the case of an error display the corresponding error string
  if (error == hipSuccess) {
    return (0);
  }
  GPUError("CUDA Error: %lld / %s (%s:%d)", error, hipGetErrorString((hipError_t)error), file, line);
  return 1;
}

void GPUReconstructionCUDABackend::GPUFailedMsgA(const long long int error, const char* file, int line)
{
  if (GPUFailedMsgAI(error, file, line)) {
    static bool runningCallbacks = false;
    if (IsInitialized() && runningCallbacks == false) {
      runningCallbacks = true;
      CheckErrorCodes(false, true);
    }
    throw std::runtime_error("CUDA Failure");
  }
}

GPUReconstructionCUDA::GPUReconstructionCUDA(const GPUSettingsDeviceBackend& cfg) : GPUReconstructionKernels(cfg)
{
  mDeviceBackendSettings.deviceType = DeviceType::CUDA;
}

GPUReconstructionCUDA::~GPUReconstructionCUDA()
{
  Exit(); // Make sure we destroy everything (in particular the ITS tracker) before we exit CUDA
}

GPUReconstruction* GPUReconstruction_Create_CUDA(const GPUSettingsDeviceBackend& cfg) { return new GPUReconstructionCUDA(cfg); }

void GPUReconstructionCUDA::GetITSTraits(std::unique_ptr<o2::its::TrackerTraits>* trackerTraits, std::unique_ptr<o2::its::VertexerTraits>* vertexerTraits, std::unique_ptr<o2::its::TimeFrame>* timeFrame)
{
  if (trackerTraits) {
    trackerTraits->reset(new o2::its::TrackerTraitsGPU);
  }
  if (vertexerTraits) {
    vertexerTraits->reset(new o2::its::VertexerTraitsGPU);
  }
  if (timeFrame) {
    timeFrame->reset(new o2::its::gpu::TimeFrameGPU);
  }
}

void GPUReconstructionCUDA::UpdateAutomaticProcessingSettings()
{
  GPUCA_GPUReconstructionUpdateDefaults();
}

int GPUReconstructionCUDA::InitDevice_Runtime()
{
#ifndef __HIPCC__ // CUDA
  constexpr int reqVerMaj = 2;
  constexpr int reqVerMin = 0;
#endif
  if (mMaster == nullptr) {
    hipDeviceProp_t deviceProp;
    int count, bestDevice = -1;
    double bestDeviceSpeed = -1, deviceSpeed;
    if (GPUFailedMsgI(hipGetDeviceCount(&count))) {
      GPUError("Error getting CUDA Device Count");
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Available CUDA devices:");
    }
    std::vector<bool> devicesOK(count, false);
    std::vector<size_t> devMemory(count, 0);
    bool contextCreated = false;
    for (int i = 0; i < count; i++) {
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Examining device %d", i);
      }
      size_t free, total;
#ifndef __HIPCC__ // CUDA
      if (GPUFailedMsgI(cudaInitDevice(i, 0, 0))) {
#else // HIP
      if (GPUFailedMsgI(hipSetDevice(i))) {
#endif
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Couldn't create context for device %d. Skipping it.", i);
        }
        continue;
      }
      contextCreated = true;
      if (GPUFailedMsgI(hipMemGetInfo(&free, &total))) {
        if (mProcessingSettings.debugLevel >= 4) {
          GPUWarning("Error obtaining CUDA memory info about device %d! Skipping it.", i);
        }
        GPUFailedMsg(hipDeviceReset());
        continue;
      }
      if (count > 1) {
        GPUFailedMsg(hipDeviceReset());
        contextCreated = false;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained current memory usage for device %d", i);
      }
      if (GPUFailedMsgI(hipGetDeviceProperties(&deviceProp, i))) {
        continue;
      }
      if (mProcessingSettings.debugLevel >= 4) {
        GPUInfo("Obtained device properties for device %d", i);
      }
      int deviceOK = true;
      const char* deviceFailure = "";
#ifndef __HIPCC__
      if (deviceProp.major < reqVerMaj || (deviceProp.major == reqVerMaj && deviceProp.minor < reqVerMin)) {
        deviceOK = false;
        deviceFailure = "Too low device revision";
      }
#endif
      if (free < std::max<size_t>(mDeviceMemorySize, REQUIRE_MIN_MEMORY)) {
        deviceOK = false;
        deviceFailure = "Insufficient GPU memory";
      }

      deviceSpeed = (double)deviceProp.multiProcessorCount * (double)deviceProp.clockRate * (double)deviceProp.warpSize * (double)free * (double)deviceProp.major * (double)deviceProp.major;
      if (mProcessingSettings.debugLevel >= 2) {
        GPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lu / %lu)%s %s", deviceOK ? " " : "[", i, deviceProp.name, deviceProp.major, deviceProp.minor, free, (size_t)deviceProp.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
      }
      if (!deviceOK) {
        continue;
      }
      devicesOK[i] = true;
      devMemory[i] = std::min<size_t>(free, std::max<long int>(0, total - REQUIRE_MEMORY_RESERVED));
      if (deviceSpeed > bestDeviceSpeed) {
        bestDevice = i;
        bestDeviceSpeed = deviceSpeed;
      } else {
        if (mProcessingSettings.debugLevel >= 2 && mProcessingSettings.deviceNum < 0) {
          GPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
        }
      }
    }

    bool noDevice = false;
    if (bestDevice == -1) {
      GPUWarning("No %sCUDA Device available, aborting CUDA Initialisation (Required mem: %lld)", count ? "appropriate " : "", (long long int)mDeviceMemorySize);
#ifndef __HIPCC__
      GPUImportant("Requiring Revision %d.%d, Mem: %lu", reqVerMaj, reqVerMin, std::max<size_t>(mDeviceMemorySize, REQUIRE_MIN_MEMORY));
#endif
      noDevice = true;
    } else if (mProcessingSettings.deviceNum > -1) {
      if (mProcessingSettings.deviceNum >= (signed)count) {
        GPUError("Requested device ID %d does not exist", mProcessingSettings.deviceNum);
        noDevice = true;
      } else if (!devicesOK[mProcessingSettings.deviceNum]) {
        GPUError("Unsupported device requested (%d)", mProcessingSettings.deviceNum);
        noDevice = true;
      } else {
        bestDevice = mProcessingSettings.deviceNum;
      }
    }
    if (noDevice) {
      if (contextCreated) {
        GPUFailedMsgI(hipDeviceReset());
      }
      return (1);
    }
    mDeviceId = bestDevice;

    GPUFailedMsgI(hipGetDeviceProperties(&deviceProp, mDeviceId));

    if (mProcessingSettings.debugLevel >= 2) {
      GPUInfo("Using CUDA Device %s with Properties:", deviceProp.name);
      GPUInfo("\ttotalGlobalMem = %lld", (unsigned long long int)deviceProp.totalGlobalMem);
      GPUInfo("\tsharedMemPerBlock = %lld", (unsigned long long int)deviceProp.sharedMemPerBlock);
      GPUInfo("\tregsPerBlock = %d", deviceProp.regsPerBlock);
      GPUInfo("\twarpSize = %d", deviceProp.warpSize);
      GPUInfo("\tmemPitch = %lld", (unsigned long long int)deviceProp.memPitch);
      GPUInfo("\tmaxThreadsPerBlock = %d", deviceProp.maxThreadsPerBlock);
      GPUInfo("\tmaxThreadsDim = %d %d %d", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
      GPUInfo("\tmaxGridSize = %d %d %d", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
      GPUInfo("\ttotalConstMem = %lld", (unsigned long long int)deviceProp.totalConstMem);
      GPUInfo("\tmajor = %d", deviceProp.major);
      GPUInfo("\tminor = %d", deviceProp.minor);
      GPUInfo("\tclockRate = %d", deviceProp.clockRate);
      GPUInfo("\tmemoryClockRate = %d", deviceProp.memoryClockRate);
      GPUInfo("\tmultiProcessorCount = %d", deviceProp.multiProcessorCount);
      GPUInfo("\ttextureAlignment = %lld", (unsigned long long int)deviceProp.textureAlignment);
      GPUInfo(" ");
    }
    if (deviceProp.warpSize != GPUCA_WARP_SIZE) {
      throw std::runtime_error("Invalid warp size on GPU");
    }
    mBlockCount = deviceProp.multiProcessorCount;
    mMaxThreads = std::max<int>(mMaxThreads, deviceProp.maxThreadsPerBlock * mBlockCount);
#ifndef __HIPCC__ // CUDA
    mWarpSize = 32;
#else // HIP
    mWarpSize = 64;
#endif
    mDeviceName = deviceProp.name;
    mDeviceName += " (CUDA GPU)";

    if (deviceProp.major < 3) {
      GPUError("Unsupported CUDA Device");
      return (1);
    }

#ifdef GPUCA_USE_TEXTURES
    if (GPUCA_SLICE_DATA_MEMORY * NSLICES > (size_t)deviceProp.maxTexture1DLinear) {
      GPUError("Invalid maximum texture size of device: %lld < %lld\n", (long long int)deviceProp.maxTexture1DLinear, (long long int)(GPUCA_SLICE_DATA_MEMORY * NSLICES));
      return (1);
    }
#endif
#ifndef GPUCA_NO_CONSTANT_MEMORY
    if (gGPUConstantMemBufferSize > deviceProp.totalConstMem) {
      GPUError("Insufficient constant memory available on GPU %d < %d!", (int)deviceProp.totalConstMem, (int)gGPUConstantMemBufferSize);
      return (1);
    }
#endif

#ifndef __HIPCC__ // CUDA
    if (contextCreated == 0 && GPUFailedMsgI(cudaInitDevice(mDeviceId, 0, 0))) {
#else // HIP
    if (contextCreated == 0 && GPUFailedMsgI(hipSetDevice(mDeviceId))) {
#endif
      GPUError("Could not set CUDA Device!");
      return (1);
    }

#ifndef __HIPCC__ // CUDA
    if (GPUFailedMsgI(hipDeviceSetLimit(hipLimitStackSize, GPUCA_GPU_STACK_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (GPUFailedMsgI(hipDeviceSetLimit(hipLimitMallocHeapSize, mProcessingSettings.deterministicGPUReconstruction ? std::max<size_t>(1024 * 1024 * 1024, GPUCA_GPU_HEAP_SIZE) : GPUCA_GPU_HEAP_SIZE))) {
      GPUError("Error setting CUDA stack size");
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
#else // HIP
    if (GPUFailedMsgI(hipSetDeviceFlags(hipDeviceScheduleBlockingSync))) {
      GPUError("Could not set HIP Device!");
      return (1);
    }
#endif

    if (mDeviceMemorySize == 1 || mDeviceMemorySize == 2) {
      mDeviceMemorySize = std::max<long int>(0, devMemory[mDeviceId] - REQUIRE_FREE_MEMORY_RESERVED_PER_SM * deviceProp.multiProcessorCount); // Take all GPU memory but some reserve
      if (mDeviceMemorySize >= RESERVE_EXTRA_MEM_THRESHOLD) {
        mDeviceMemorySize -= RESERVE_EXTRA_MEM_OFFSET;
      }
    }
    if (mDeviceMemorySize == 2) {
      mDeviceMemorySize = mDeviceMemorySize * 2 / 3; // Leave 1/3 of GPU memory for event display
    }

    if (mDeviceMemorySize > deviceProp.totalGlobalMem || GPUFailedMsgI(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize))) {
      size_t free, total;
      GPUFailedMsg(hipMemGetInfo(&free, &total));
      GPUError("CUDA Memory Allocation Error (trying %lld bytes, %lld available on GPU, %lld free)", (long long int)mDeviceMemorySize, (long long int)deviceProp.totalGlobalMem, (long long int)free);
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (GPUFailedMsgI(hipHostMalloc(&mHostMemoryBase, mHostMemorySize))) {
      GPUError("Error allocating Page Locked Host Memory (trying %lld bytes)", (long long int)mHostMemorySize);
      GPUFailedMsgI(hipDeviceReset());
      return (1);
    }
    if (mProcessingSettings.debugLevel >= 1) {
      GPUInfo("Memory ptrs: GPU (%lld bytes): %p - Host (%lld bytes): %p", (long long int)mDeviceMemorySize, mDeviceMemoryBase, (long long int)mHostMemorySize, mHostMemoryBase);
      memset(mHostMemoryBase, 0xDD, mHostMemorySize);
      if (GPUFailedMsgI(hipMemset(mDeviceMemoryBase, 0xDD, mDeviceMemorySize))) {
        GPUError("Error during CUDA memset");
        GPUFailedMsgI(hipDeviceReset());
        return (1);
      }
    }

    for (int i = 0; i < mNStreams; i++) {
      if (GPUFailedMsgI(hipStreamCreateWithFlags(&mInternals->Streams[i], hipStreamNonBlocking))) {
        GPUError("Error creating CUDA Stream");
        GPUFailedMsgI(hipDeviceReset());
        return (1);
      }
    }

#ifndef __HIPCC__ // CUDA
    dummyInitKernel<<<mBlockCount, 256>>>(mDeviceMemoryBase);
#else // HIP
    hipLaunchKernelGGL(HIP_KERNEL_NAME(dummyInitKernel), dim3(mBlockCount), dim3(256), 0, 0, mDeviceMemoryBase);
#endif

#ifndef GPUCA_ALIROOT_LIB
    if (mProcessingSettings.rtc.enable) {
      std::string filename = "";
      unsigned int nCompile = 0;
      if (genRTC(filename, nCompile)) {
        throw std::runtime_error("Runtime compilation failed");
      }
      for (unsigned int i = 0; i < nCompile; i++) {
        mInternals->kernelModules.emplace_back(std::make_unique<hipModule_t>());
        GPUFailedMsg(hipModuleLoad(mInternals->kernelModules.back().get(), (filename + "_" + std::to_string(i) + ".cubin").c_str()));
        remove((filename + "_" + std::to_string(i) + ".cu").c_str());
        remove((filename + "_" + std::to_string(i) + ".cubin").c_str());
      }
      loadKernelModules(mProcessingSettings.rtc.compilePerKernel);
    }
#if defined(GPUCA_KERNEL_COMPILE_MODE) && GPUCA_KERNEL_COMPILE_MODE == 1
    else {
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward)       \
  mInternals->kernelModules.emplace_back(std::make_unique<hipModule_t>()); \
  GPUFailedMsg(hipModuleLoadData(mInternals->kernelModules.back().get(), GPUCA_M_CAT3(_binary_cuda_kernel_module_fatbin_krnl_, GPUCA_M_KRNL_NAME(x_class), _fatbin_start)));
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
      loadKernelModules(true, false);
    }
#endif
#endif
    void* devPtrConstantMem = nullptr;
#ifndef GPUCA_NO_CONSTANT_MEMORY
    runConstantRegistrators();
    devPtrConstantMem = mDeviceConstantMemList[0];
    for (unsigned int i = 0; i < mInternals->kernelModules.size(); i++) {
      hipDeviceptr_t tmp;
      GPUFailedMsg(hipModuleGetGlobal(&tmp, nullptr, *mInternals->kernelModules[i], "gGPUConstantMemBuffer"));
      mDeviceConstantMemList.emplace_back((void*)tmp);
    }
#else
    GPUFailedMsg(hipMalloc(&devPtrConstantMem, gGPUConstantMemBufferSize));
#endif
    mDeviceConstantMem = (GPUConstantMem*)devPtrConstantMem;

    GPUInfo("CUDA Initialisation successfull (Device %d: %s (Frequency %d, Cores %d), %lld / %lld bytes host / global memory, Stack frame %d, Constant memory %lld)", mDeviceId, deviceProp.name, deviceProp.clockRate, deviceProp.multiProcessorCount, (long long int)mHostMemorySize, (long long int)mDeviceMemorySize, (int)GPUCA_GPU_STACK_SIZE, (long long int)gGPUConstantMemBufferSize);
  } else {
    GPUReconstructionCUDA* master = dynamic_cast<GPUReconstructionCUDA*>(mMaster);
    mDeviceId = master->mDeviceId;
    mBlockCount = master->mBlockCount;
    mWarpSize = master->mWarpSize;
    mMaxThreads = master->mMaxThreads;
    mDeviceName = master->mDeviceName;
    mDeviceConstantMem = master->mDeviceConstantMem;
    mDeviceConstantMemList.resize(master->mDeviceConstantMemList.size());
    std::copy(master->mDeviceConstantMemList.begin(), master->mDeviceConstantMemList.end(), mDeviceConstantMemList.begin());
    mInternals = master->mInternals;
    GPUFailedMsg(hipSetDevice(mDeviceId));

    GPUInfo("CUDA Initialized from master");
  }

  for (unsigned int i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (unsigned int j = 0; j < mEvents[i].size(); j++) {
#ifndef __HIPCC__ // CUDA
      if (GPUFailedMsgI(hipEventCreate(&events[j]))) {
#else
      if (GPUFailedMsgI(hipEventCreateWithFlags(&events[j], hipEventBlockingSync))) {
#endif
        GPUError("Error creating event");
        GPUFailedMsgI(hipDeviceReset());
        return 1;
      }
    }
  }

  return (0);
}

int GPUReconstructionCUDA::ExitDevice_Runtime()
{
  // Uninitialize CUDA
  GPUFailedMsg(hipSetDevice(mDeviceId));
  SynchronizeGPU();
  unregisterRemainingRegisteredMemory();

  for (unsigned int i = 0; i < mEvents.size(); i++) {
    hipEvent_t* events = (hipEvent_t*)mEvents[i].data();
    for (unsigned int j = 0; j < mEvents[i].size(); j++) {
      GPUFailedMsgI(hipEventDestroy(events[j]));
    }
  }

  if (mMaster == nullptr) {
    GPUFailedMsgI(hipFree(mDeviceMemoryBase));
#ifdef GPUCA_NO_CONSTANT_MEMORY
    GPUFailedMsgI(hipFree(mDeviceConstantMem));
#endif

    for (int i = 0; i < mNStreams; i++) {
      GPUFailedMsgI(hipStreamDestroy(mInternals->Streams[i]));
    }

    GPUFailedMsgI(hipHostFree(mHostMemoryBase));
    GPUFailedMsgI(hipDeviceReset());
    GPUInfo("CUDA Uninitialized");
  }
  mDeviceMemoryBase = nullptr;
  mHostMemoryBase = nullptr;

  return (0);
}

size_t GPUReconstructionCUDA::GPUMemCpy(void* dst, const void* src, size_t size, int stream, int toGPU, deviceEvent ev, deviceEvent* evList, int nEvents)
{
  if (mProcessingSettings.debugLevel >= 3) {
    stream = -1;
  }
  if (stream == -1) {
    SynchronizeGPU();
    GPUFailedMsg(hipMemcpy(dst, src, size, toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost));
  } else {
    if (evList == nullptr) {
      nEvents = 0;
    }
    for (int k = 0; k < nEvents; k++) {
      GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[stream], ((hipEvent_t*)evList)[k], 0));
    }
    GPUFailedMsg(hipMemcpyAsync(dst, src, size, toGPU == -2 ? hipMemcpyDeviceToDevice : toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost, mInternals->Streams[stream]));
  }
  if (ev) {
    GPUFailedMsg(hipEventRecord(*(hipEvent_t*)ev, mInternals->Streams[stream == -1 ? 0 : stream]));
  }
  return size;
}

size_t GPUReconstructionCUDA::TransferMemoryInternal(GPUMemoryResource* res, int stream, deviceEvent ev, deviceEvent* evList, int nEvents, bool toGPU, const void* src, void* dst)
{
  if (!(res->Type() & GPUMemoryResource::MEMORY_GPU)) {
    if (mProcessingSettings.debugLevel >= 4) {
      GPUInfo("Skipped transfer of non-GPU memory resource: %s", res->Name());
    }
    return 0;
  }
  if (mProcessingSettings.debugLevel >= 3 && (strcmp(res->Name(), "ErrorCodes") || mProcessingSettings.debugLevel >= 4)) {
    GPUInfo("Copying to %s: %s - %lld bytes", toGPU ? "GPU" : "Host", res->Name(), (long long int)res->Size());
  }
  return GPUMemCpy(dst, src, res->Size(), stream, toGPU, ev, evList, nEvents);
}

size_t GPUReconstructionCUDA::WriteToConstantMemory(size_t offset, const void* src, size_t size, int stream, deviceEvent ev)
{
  for (unsigned int i = 0; i < 1 + mDeviceConstantMemList.size(); i++) {
    void* basePtr = i ? mDeviceConstantMemList[i - 1] : mDeviceConstantMem;
    if (basePtr == nullptr || (i && basePtr == (void*)mDeviceConstantMem)) {
      continue;
    }
    if (stream == -1) {
      GPUFailedMsg(hipMemcpy(((char*)basePtr) + offset, src, size, hipMemcpyHostToDevice));
    } else {
      GPUFailedMsg(hipMemcpyAsync(((char*)basePtr) + offset, src, size, hipMemcpyHostToDevice, mInternals->Streams[stream]));
    }
  }
  if (ev && stream != -1) {
    GPUFailedMsg(hipEventRecord(*(hipEvent_t*)ev, mInternals->Streams[stream]));
  }
  return size;
}

void GPUReconstructionCUDA::ReleaseEvent(deviceEvent ev) {}
void GPUReconstructionCUDA::RecordMarker(deviceEvent ev, int stream) { GPUFailedMsg(hipEventRecord(*(hipEvent_t*)ev, mInternals->Streams[stream])); }

std::unique_ptr<GPUReconstruction::GPUThreadContext> GPUReconstructionCUDA::GetThreadContext()
{
  GPUFailedMsg(hipSetDevice(mDeviceId));
  return std::unique_ptr<GPUThreadContext>(new GPUThreadContext);
}

void GPUReconstructionCUDA::SynchronizeGPU() { GPUFailedMsg(hipDeviceSynchronize()); }
void GPUReconstructionCUDA::SynchronizeStream(int stream) { GPUFailedMsg(hipStreamSynchronize(mInternals->Streams[stream])); }

void GPUReconstructionCUDA::SynchronizeEvents(deviceEvent* evList, int nEvents)
{
  for (int i = 0; i < nEvents; i++) {
    GPUFailedMsg(hipEventSynchronize(((hipEvent_t*)evList)[i]));
  }
}

void GPUReconstructionCUDA::StreamWaitForEvents(int stream, deviceEvent* evList, int nEvents)
{
  for (int i = 0; i < nEvents; i++) {
    GPUFailedMsg(hipStreamWaitEvent(mInternals->Streams[stream], ((hipEvent_t*)evList)[i], 0));
  }
}

bool GPUReconstructionCUDA::IsEventDone(deviceEvent* evList, int nEvents)
{
  for (int i = 0; i < nEvents; i++) {
    hipError_t retVal = hipEventSynchronize(((hipEvent_t*)evList)[i]);
    if (retVal == hipErrorNotReady) {
      return false;
    }
    GPUFailedMsg(retVal);
  }
  return (true);
}

int GPUReconstructionCUDA::GPUDebug(const char* state, int stream, bool force)
{
  // Wait for CUDA-Kernel to finish and check for CUDA errors afterwards, in case of debugmode
  hipError_t cuErr;
  cuErr = hipGetLastError();
  if (cuErr != hipSuccess) {
    GPUError("CUDA Error %s while running kernel (%s) (Stream %d)", hipGetErrorString(cuErr), state, stream);
    return (1);
  }
  if (!force && mProcessingSettings.debugLevel <= 0) {
    return (0);
  }
  if (GPUFailedMsgI(stream == -1 ? hipDeviceSynchronize() : hipStreamSynchronize(mInternals->Streams[stream]))) {
    GPUError("CUDA Error while synchronizing (%s) (Stream %d)", state, stream);
    return (1);
  }
  if (mProcessingSettings.debugLevel >= 3) {
    GPUInfo("GPU Sync Done");
  }
  return (0);
}

int GPUReconstructionCUDA::registerMemoryForGPU_internal(const void* ptr, size_t size)
{
  return GPUFailedMsgI(hipHostRegister((void*)ptr, size, hipHostRegisterDefault));
}

int GPUReconstructionCUDA::unregisterMemoryForGPU_internal(const void* ptr)
{
  return GPUFailedMsgI(hipHostUnregister((void*)ptr));
}

void GPUReconstructionCUDABackend::PrintKernelOccupancies()
{
  int maxBlocks = 0, threads = 0, suggestedBlocks = 0, nRegs = 0, sMem = 0;
  GPUFailedMsg(hipSetDevice(mDeviceId));
  for (unsigned int i = 0; i < mInternals->kernelFunctions.size(); i++) {
    GPUFailedMsg(hipModuleOccupancyMaxPotentialBlockSize(&suggestedBlocks, &threads, *mInternals->kernelFunctions[i], 0, 0));
    GPUFailedMsg(hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocks, *mInternals->kernelFunctions[i], threads, 0));
    GPUFailedMsg(hipFuncGetAttribute(&nRegs, HIP_FUNC_ATTRIBUTE_NUM_REGS, *mInternals->kernelFunctions[i]));
    GPUFailedMsg(hipFuncGetAttribute(&sMem, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, *mInternals->kernelFunctions[i]));
    GPUInfo("Kernel: %50s Block size: %4d, Maximum active blocks: %3d, Suggested blocks: %3d, Regs: %3d, smem: %3d", mInternals->kernelNames[i].c_str(), threads, maxBlocks, suggestedBlocks, nRegs, sMem);
  }
}

int GPUReconstructionCUDA::loadKernelModules(bool perKernel, bool perSingleMulti)
{
  int j = 0;
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward)                  \
  GPUCA_KRNL_WRAP(GPUCA_KRNL_LOAD_, x_class, x_attributes, x_arguments, x_forward) \
  j += !perSingleMulti;
#define GPUCA_KRNL_LOAD_single(x_class, x_attributes, x_arguments, x_forward)                        \
  getRTCkernelNum<false, GPUCA_M_KRNL_TEMPLATE(x_class)>(mInternals->kernelFunctions.size());        \
  mInternals->kernelFunctions.emplace_back(new hipFunction_t);                                          \
  mInternals->kernelNames.emplace_back(GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class)))); \
  GPUFailedMsg(hipModuleGetFunction(mInternals->kernelFunctions.back().get(), *mInternals->kernelModules[perKernel ? (j += perSingleMulti) : 0], GPUCA_M_STR(GPUCA_M_CAT(krnl_, GPUCA_M_KRNL_NAME(x_class)))));
#define GPUCA_KRNL_LOAD_multi(x_class, x_attributes, x_arguments, x_forward)                                  \
  getRTCkernelNum<true, GPUCA_M_KRNL_TEMPLATE(x_class)>(mInternals->kernelFunctions.size());                  \
  mInternals->kernelFunctions.emplace_back(new hipFunction_t);                                                   \
  mInternals->kernelNames.emplace_back(GPUCA_M_STR(GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi))); \
  GPUFailedMsg(hipModuleGetFunction(mInternals->kernelFunctions.back().get(), *mInternals->kernelModules[perKernel ? (j += perSingleMulti) : 0], GPUCA_M_STR(GPUCA_M_CAT3(krnl_, GPUCA_M_KRNL_NAME(x_class), _multi))));
#include "GPUReconstructionKernelList.h"
#undef GPUCA_KRNL
#undef GPUCA_KRNL_LOAD_single
#undef GPUCA_KRNL_LOAD_multi

  for (unsigned int i = 0; i < mInternals->kernelNames.size(); i++) {
    if (mProcessingSettings.debugLevel >= 3) {
      GPUInfo("Loaded module for kernel %s", mInternals->kernelNames[i].c_str());
    }
  }
  return 0;
}

#ifndef __HIPCC__ // CUDA
int GPUReconstructionCUDA::PrepareTextures()
{
#ifdef GPUCA_USE_TEXTURES
  hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
  size_t offset;
  GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSLICES * GPUCA_SLICE_DATA_MEMORY));
  hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
  GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, mProcessorsShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSLICES * GPUCA_SLICE_DATA_MEMORY));
#endif
  return (0);
}

void GPUReconstructionCUDA::startGPUProfiling()
{
  GPUFailedMsg(hipProfilerStart());
}

void GPUReconstructionCUDA::endGPUProfiling()
{
  GPUFailedMsg(hipProfilerStop());
}
#else  // HIP
void* GPUReconstructionHIP::getGPUPointer(void* ptr)
{
  void* retVal = nullptr;
  GPUFailedMsg(hipHostGetDevicePointer(&retVal, ptr, 0));
  return retVal;
}
#endif // __HIPCC__

namespace GPUCA_NAMESPACE::gpu
{
template class GPUReconstructionKernels<GPUReconstructionCUDABackend>;
}
