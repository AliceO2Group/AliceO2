// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file GPUReconstructionCUDArtc.cu
/// \author David Rohr

#include "GPUReconstructionCUDArtcPre.h"

#define GPUCA_GPUTYPE_TURING
#include "GPUReconstructionIncludesDevice.h"

#ifndef GPUCA_GPUCODE_DEVICE
#error RTC Preprocessing must run on device code
#endif
#ifdef GPUCA_NO_CONSTANT_MEMORY
#error CUDA RTC does not support processing without constant memory
#endif

extern "C" {
#undef GPUCA_KRNL_REG
#define GPUCA_KRNL_REG(args) __launch_bounds__(GPUCA_M_MAX2_3(GPUCA_M_STRIP(args)))
#define GPUCA_KRNL(x_class, x_attributes, x_arguments, x_forward) GPUCA_KRNL_WRAP(GPUCA_KRNL_LOAD_, x_class, x_attributes, x_arguments, x_forward)
#define GPUCA_KRNL_LOAD_single(x_class, x_attributes, x_arguments, x_forward) GPUCA_KRNLGPU_SINGLE(x_class, x_attributes, x_arguments, x_forward)
#define GPUCA_KRNL_LOAD_multi(x_class, x_attributes, x_arguments, x_forward) GPUCA_KRNLGPU_MULTI(x_class, x_attributes, x_arguments, x_forward)
#include "GPUReconstructionKernels.h"
#undef GPUCA_KRNL
}
