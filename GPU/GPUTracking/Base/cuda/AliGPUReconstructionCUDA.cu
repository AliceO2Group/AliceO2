#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>
#define GPUCA_GPUTYPE_PASCAL

#include "AliGPUReconstructionCUDA.h"
#include "AliGPUReconstructionCUDAInternals.h"
#include "AliGPUReconstructionIncludes.h"

__constant__ uint4 gGPUConstantMemBuffer[(sizeof(AliGPUConstantMem) + sizeof(uint4) - 1) / sizeof(uint4)];
__constant__ char& gGPUConstantMemBufferChar = (char&) gGPUConstantMemBuffer;
__constant__ AliGPUConstantMem& gGPUConstantMem = (AliGPUConstantMem&) gGPUConstantMemBufferChar;

#ifdef GPUCA_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

#ifdef HAVE_O2HEADERS
#include "ITStrackingCUDA/TrackerTraitsNV.h"
#else
namespace o2 { namespace ITS { class TrackerTraitsNV : public TrackerTraits {}; }}
#endif

#include "AliGPUReconstructionIncludesDevice.h"

template <class T, int I, typename... Args> GPUg() void runKernelCUDA(int iSlice, Args... args)
{
	GPUshared() typename T::AliGPUTPCSharedMemory smem;
	T::template Thread<I>(get_num_groups(0), get_local_size(0), get_group_id(0), get_local_id(0), smem, T::Worker(gGPUConstantMem)[iSlice], args...);
}

template <class T, int I, typename... Args> GPUg() void runKernelCUDAMulti(int firstSlice, int nSliceCount, Args... args)
{
	const int iSlice = nSliceCount * (get_group_id(0) + (get_num_groups(0) % nSliceCount != 0 && nSliceCount * (get_group_id(0) + 1) % get_num_groups(0) != 0)) / get_num_groups(0);
	const int nSliceBlockOffset = get_num_groups(0) * iSlice / nSliceCount;
	const int sliceBlockId = get_group_id(0) - nSliceBlockOffset;
	const int sliceGridDim = get_num_groups(0) * (iSlice + 1) / nSliceCount - get_num_groups(0) * (iSlice) / nSliceCount;
	GPUshared() typename T::AliGPUTPCSharedMemory smem;
	T::template Thread<I>(sliceGridDim, get_local_size(0), sliceBlockId, get_local_id(0), smem, T::Worker(gGPUConstantMem)[firstSlice + iSlice], args...);
}

template <class T, int I, typename... Args> int AliGPUReconstructionCUDABackend::runKernelBackend(const krnlExec& x, const krnlRunRange& y, const krnlEvent& z, const Args&... args)
{
	if (x.device == krnlDeviceType::CPU) return AliGPUReconstructionCPU::runKernelImpl(classArgument<T, I>(), x, y, z, args...);
	if (z.evList) for (int k = 0;k < z.nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[x.stream], ((hipEvent_t*) z.evList)[k], 0));
	if (y.num <= 1)
	{
		runKernelCUDA<T, I> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>>(y.start, args...);
	}
	else
	{
		runKernelCUDAMulti<T, I> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>> (y.start, y.num, args...);
	}
	if (z.ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) z.ev, mInternals->CudaStreams[x.stream]));
	return 0;
}

AliGPUReconstructionCUDABackend::AliGPUReconstructionCUDABackend(const AliGPUSettingsProcessing& cfg) : AliGPUReconstructionDeviceBase(cfg)
{
	mInternals = new AliGPUReconstructionCUDAInternals;
	mProcessingSettings.deviceType = DeviceType::CUDA;
}

AliGPUReconstructionCUDABackend::~AliGPUReconstructionCUDABackend()
{
	mChains.clear(); //Make sure we destroy the ITS tracker before we exit CUDA
	GPUFailedMsgI(hipDeviceReset());
	delete mInternals;
}

AliGPUReconstruction* AliGPUReconstruction_Create_CUDA(const AliGPUSettingsProcessing& cfg)
{
	return new AliGPUReconstructionCUDA(cfg);
}

void AliGPUReconstructionCUDABackend::GetITSTraits(std::unique_ptr<o2::ITS::TrackerTraits>& trackerTraits, std::unique_ptr<o2::ITS::VertexerTraits>& vertexerTraits)
{
	trackerTraits.reset(new o2::ITS::TrackerTraitsNV);
	vertexerTraits.reset(new o2::ITS::VertexerTraits);
}

int AliGPUReconstructionCUDABackend::InitDevice_Runtime()
{
	//Find best CUDA device, initialize and allocate memory

	hipDeviceProp_t hipDeviceProp_t;

	int count, bestDevice = -1;
	double bestDeviceSpeed = -1, deviceSpeed;
	if (GPUFailedMsgI(hipGetDeviceCount(&count)))
	{
		GPUError("Error getting CUDA Device Count");
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 2) GPUInfo("Available CUDA devices:");
	const int reqVerMaj = 2;
	const int reqVerMin = 0;
	for (int i = 0;i < count;i++)
	{
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Examining device %d\n", i);
		size_t free, total;
		hipInit(0);
		hipDevice_t tmpDevice;
		hipDeviceGet(&tmpDevice, i);
		hipCtx_t tmpContext;
		hipCtxCreate(&tmpContext, 0, tmpDevice);
		if(hipMemGetInfo(&free, &total)) std::cout << "Error\n";
		hipCtxDestroy(tmpContext);
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained current memory usage for device %d\n", i);
		if (GPUFailedMsgI(hipGetDeviceProperties(&hipDeviceProp_t, i))) continue;
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained device properties for device %d\n", i);
		int deviceOK = true;
		const char* deviceFailure = "";
		if (hipDeviceProp_t.major >= 9) {deviceOK = false; deviceFailure = "Invalid Revision";}
		else if (hipDeviceProp_t.major < reqVerMaj || (hipDeviceProp_t.major == reqVerMaj && hipDeviceProp_t.minor < reqVerMin)) {deviceOK = false; deviceFailure = "Too low device revision";}
		else if (free < mDeviceMemorySize) {deviceOK = false; deviceFailure = "Insufficient GPU memory";}

		deviceSpeed = (double) hipDeviceProp_t.multiProcessorCount * (double) hipDeviceProp_t.clockRate * (double) hipDeviceProp_t.warpSize * (double) free * (double) hipDeviceProp_t.major * (double) hipDeviceProp_t.major;
		if (mDeviceProcessingSettings.debugLevel >= 2) GPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lld / %lld)%s %s", deviceOK ? " " : "[", i, hipDeviceProp_t.name, hipDeviceProp_t.major, hipDeviceProp_t.minor, (long long int) free, (long long int) hipDeviceProp_t.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
		if (!deviceOK) continue;
		if (deviceSpeed > bestDeviceSpeed)
		{
			bestDevice = i;
			bestDeviceSpeed = deviceSpeed;
		}
		else
		{
			if (mDeviceProcessingSettings.debugLevel >= 0) GPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
		}
	}
	if (bestDevice == -1)
	{
		GPUWarning("No %sCUDA Device available, aborting CUDA Initialisation", count ? "appropriate " : "");
		GPUImportant("Requiring Revision %d.%d, Mem: %lld", reqVerMaj, reqVerMin, (long long int) mDeviceMemorySize);
		return(1);
	}

	if (mDeviceProcessingSettings.deviceNum > -1)
	{
		if (mDeviceProcessingSettings.deviceNum < (signed) count)
		{
			bestDevice = mDeviceProcessingSettings.deviceNum;
		}
		else
		{
			GPUWarning("Requested device ID %d non existend, falling back to default device id %d", mDeviceProcessingSettings.deviceNum, bestDevice);
		}
	}
	fDeviceId = bestDevice;

	GPUFailedMsgI(hipGetDeviceProperties(&hipDeviceProp_t ,fDeviceId));

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		GPUInfo("Using CUDA Device %s with Properties:", hipDeviceProp_t.name);
		GPUInfo("totalGlobalMem = %lld", (unsigned long long int) hipDeviceProp_t.totalGlobalMem);
		GPUInfo("sharedMemPerBlock = %lld", (unsigned long long int) hipDeviceProp_t.sharedMemPerBlock);
		GPUInfo("regsPerBlock = %d", hipDeviceProp_t.regsPerBlock);
		GPUInfo("warpSize = %d", hipDeviceProp_t.warpSize);
		GPUInfo("memPitch = %lld", (unsigned long long int) hipDeviceProp_t.memPitch);
		GPUInfo("maxThreadsPerBlock = %d", hipDeviceProp_t.maxThreadsPerBlock);
		GPUInfo("maxThreadsDim = %d %d %d", hipDeviceProp_t.maxThreadsDim[0], hipDeviceProp_t.maxThreadsDim[1], hipDeviceProp_t.maxThreadsDim[2]);
		GPUInfo("maxGridSize = %d %d %d", hipDeviceProp_t.maxGridSize[0], hipDeviceProp_t.maxGridSize[1], hipDeviceProp_t.maxGridSize[2]);
		GPUInfo("totalConstMem = %lld", (unsigned long long int) hipDeviceProp_t.totalConstMem);
		GPUInfo("major = %d", hipDeviceProp_t.major);
		GPUInfo("minor = %d", hipDeviceProp_t.minor);
		GPUInfo("clockRate = %d", hipDeviceProp_t.clockRate);
		GPUInfo("memoryClockRate = %d", hipDeviceProp_t.memoryClockRate);
		GPUInfo("multiProcessorCount = %d", hipDeviceProp_t.multiProcessorCount);
		GPUInfo("textureAlignment = %lld", (unsigned long long int) hipDeviceProp_t.textureAlignment);
	}
	mCoreCount = hipDeviceProp_t.multiProcessorCount;

	if (hipDeviceProp_t.major < 1 || (hipDeviceProp_t.major == 1 && hipDeviceProp_t.minor < 2))
	{
		GPUError( "Unsupported CUDA Device" );
		return(1);
	}

#ifdef GPUCA_USE_TEXTURES
	if (GPUCA_SLICE_DATA_MEMORY * NSLICES > (size_t) hipDeviceProp_t.maxTexture1DLinear)
	{
		GPUError("Invalid maximum texture size of device: %lld < %lld\n", (long long int) hipDeviceProp_t.maxTexture1DLinear, (long long int) (GPUCA_SLICE_DATA_MEMORY * NSLICES));
		return(1);
	}
#endif

	mNStreams = std::max(mDeviceProcessingSettings.nStreams, 3);

	if (hipCtxCreate(&mInternals->CudaContext, hipDeviceScheduleAuto, fDeviceId) != hipSuccess)
	{
		GPUError("Could not set CUDA Device!");
		return(1);
	}

	if (mDeviceMemorySize > hipDeviceProp_t.totalGlobalMem || GPUFailedMsgI(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize)))
	{
		GPUError("CUDA Memory Allocation Error");
		GPUFailedMsgI(hipDeviceReset());
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) GPUInfo("GPU Memory used: %lld", (long long int) mDeviceMemorySize);
	if (GPUFailedMsgI(hipHostMalloc(&mHostMemoryBase, mHostMemorySize)))
	{
		GPUError("Error allocating Page Locked Host Memory");
		GPUFailedMsgI(hipDeviceReset());
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) GPUInfo("Host Memory used: %lld", (long long int) mHostMemorySize);

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		memset(mHostMemoryBase, 0xDD, mHostMemorySize);
		if (GPUFailedMsgI(hipMemset(mDeviceMemoryBase, 0xDD, mDeviceMemorySize)))
		{
			GPUError("Error during CUDA memset");
			GPUFailedMsgI(hipDeviceReset());
			return(1);
		}
	}

	for (int i = 0;i < mNStreams;i++)
	{
		if (GPUFailedMsgI(hipStreamCreate(&mInternals->CudaStreams[i])))
		{
			GPUError("Error creating CUDA Stream");
			GPUFailedMsgI(hipDeviceReset());
			return(1);
		}
	}
	
	void* devPtrConstantMem;
	if (GPUFailedMsgI(hipGetSymbolAddress(&devPtrConstantMem, HIP_SYMBOL(gGPUConstantMemBuffer))))
	{
		GPUError("Error getting ptr to constant memory");
		GPUFailedMsgI(hipDeviceReset());
		return 1;
	}
	mDeviceConstantMem = (AliGPUConstantMem*) devPtrConstantMem;
	
	for (unsigned int i = 0;i < mEvents.size();i++)
	{
		hipEvent_t *events = (hipEvent_t*) mEvents[i].first;
		for (unsigned int j = 0;j < mEvents[i].second;j++)
		{
			if (GPUFailedMsgI(hipEventCreate(&events[j])))
			{
				GPUError("Error creating event");
				GPUFailedMsgI(hipDeviceReset());
				return 1;
			}
		}
	}

	ReleaseThreadContext();
	GPUInfo("CUDA Initialisation successfull (Device %d: %s, Thread %d, %lld/%lld bytes used)", fDeviceId, hipDeviceProp_t.name, mThreadId, (long long int) mHostMemorySize, (long long int) mDeviceMemorySize);

	return(0);
}

int AliGPUReconstructionCUDABackend::ExitDevice_Runtime()
{
	//Uninitialize CUDA
	ActivateThreadContext();

	SynchronizeGPU();

	GPUFailedMsgI(hipFree(mDeviceMemoryBase));
	mDeviceMemoryBase = nullptr;

	for (int i = 0;i < mNStreams;i++)
	{
		GPUFailedMsgI(hipStreamDestroy(mInternals->CudaStreams[i]));
	}

	GPUFailedMsgI(hipHostFree(mHostMemoryBase));
	mHostMemoryBase = nullptr;
	
	for (unsigned int i = 0;i < mEvents.size();i++)
	{
		hipEvent_t *events = (hipEvent_t*) mEvents[i].first;
		for (unsigned int j = 0;j < mEvents[i].second;j++)
		{
			GPUFailedMsgI(hipEventDestroy(events[j]));
		}
	}

	if (GPUFailedMsgI(hipDeviceReset()))
	{
		GPUError("Could not uninitialize GPU");
		return(1);
	}

	hipCtxDestroy(mInternals->CudaContext);

	GPUInfo("CUDA Uninitialized");
	return(0);
}

void AliGPUReconstructionCUDABackend::TransferMemoryInternal(AliGPUMemoryResource* res, int stream, deviceEvent* ev, deviceEvent* evList, int nEvents, bool toGPU, void* src, void* dst)
{
	if (!(res->Type() & AliGPUMemoryResource::MEMORY_GPU))
	{
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Skipped transfer of non-GPU memory resource: %s\n", res->Name());
		return;
	}
	if (mDeviceProcessingSettings.debugLevel >= 3) stream = -1;
	if (mDeviceProcessingSettings.debugLevel >= 3) printf(toGPU ? "Copying to GPU: %s\n" : "Copying to Host: %s\n", res->Name());
	if (stream == -1)
	{
		if (stream == -1) SynchronizeGPU();
		GPUFailedMsg(hipMemcpy(dst, src, res->Size(), toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost));
	}
	else
	{
		if (evList == nullptr) nEvents = 0;
		for (int k = 0;k < nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[stream], ((hipEvent_t*) evList)[k], 0));
		GPUFailedMsg(hipMemcpyAsync(dst, src, res->Size(), toGPU ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost, mInternals->CudaStreams[stream]));
	}
	if (ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream == -1 ? 0 : stream]));
}

void AliGPUReconstructionCUDABackend::WriteToConstantMemory(size_t offset, const void* src, size_t size, int stream, deviceEvent* ev)
{
	if (stream == -1) GPUFailedMsg(hipMemcpyToSymbol(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice));
	else GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice, mInternals->CudaStreams[stream]));
	if (ev && stream != -1) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
}

void AliGPUReconstructionCUDABackend::ReleaseEvent(deviceEvent* ev) {}

void AliGPUReconstructionCUDABackend::RecordMarker(deviceEvent* ev, int stream)
{
	GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
}

void AliGPUReconstructionCUDABackend::ActivateThreadContext()
{
	hipCtxPushCurrent(mInternals->CudaContext);
}
void AliGPUReconstructionCUDABackend::ReleaseThreadContext()
{
	hipCtxPopCurrent(&mInternals->CudaContext);
}

void AliGPUReconstructionCUDABackend::SynchronizeGPU()
{
	GPUFailedMsg(hipDeviceSynchronize());
}

void AliGPUReconstructionCUDABackend::SynchronizeStream(int stream)
{
	GPUFailedMsg(hipStreamSynchronize(mInternals->CudaStreams[stream]));
}

void AliGPUReconstructionCUDABackend::SynchronizeEvents(deviceEvent* evList, int nEvents)
{
	for (int i = 0;i < nEvents;i++)
	{
		GPUFailedMsg(hipEventSynchronize(((hipEvent_t*) evList)[i]));
	}
}

bool AliGPUReconstructionCUDABackend::IsEventDone(deviceEvent* evList, int nEvents)
{
	for (int i = 0;i < nEvents;i++)
	{
		hipError_t retVal = hipEventSynchronize(((hipEvent_t*) evList)[i]);
		if (retVal == hipErrorNotReady) return false;
		GPUFailedMsg(retVal);
	}
	return(true);
}

int AliGPUReconstructionCUDABackend::GPUDebug(const char* state, int stream)
{
	//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards, in case of debugmode
	if (mDeviceProcessingSettings.debugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		GPUError("Cuda Error %s while running kernel (%s) (Stream %d)", hipGetErrorString(cuErr), state, stream);
		return(1);
	}
	if (GPUFailedMsgI(hipDeviceSynchronize()))
	{
		GPUError("CUDA Error while synchronizing (%s) (Stream %d)", state, stream);
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 3) GPUInfo("GPU Sync Done");
	return(0);
}

int AliGPUReconstructionCUDABackend::PrepareTextures()
{
#ifdef GPUCA_USE_TEXTURES
	hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
	size_t offset;
	GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSLICES * GPUCA_SLICE_DATA_MEMORY));
	hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
	GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSLICES * GPUCA_SLICE_DATA_MEMORY));
#endif
	return(0);
}

void AliGPUReconstructionCUDABackend::SetThreadCounts()
{
	fThreadCount = GPUCA_THREAD_COUNT;
	fBlockCount = mCoreCount;
	fConstructorBlockCount = fBlockCount * (mDeviceProcessingSettings.trackletConstructorInPipeline ? 1 : GPUCA_BLOCK_COUNT_CONSTRUCTOR_MULTIPLIER);
	fSelectorBlockCount = fBlockCount * GPUCA_BLOCK_COUNT_SELECTOR_MULTIPLIER;
	fConstructorThreadCount = GPUCA_THREAD_COUNT_CONSTRUCTOR;
	fSelectorThreadCount = GPUCA_THREAD_COUNT_SELECTOR;
	fFinderThreadCount = GPUCA_THREAD_COUNT_FINDER;
	fTRDThreadCount = GPUCA_THREAD_COUNT_TRD;
}
