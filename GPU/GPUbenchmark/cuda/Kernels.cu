#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Kernels.{cu, hip.cxx}
/// \author: mconcas@cern.ch

#include "../Shared/Kernels.h"
#if defined(__HIPCC__)
#include "hip/hip_runtime.h"
#endif
#include <cstdio>

// Memory partitioning legend
//
// |----------------------region 0-----------------|----------------------region 1-----------------| regions -> deafult: 2, to test lower and upper RAM
// |--chunk 0--|--chunk 1--|--chunk 2--|                  ***                          |--chunk n--| chunks  -> default size: 1GB (sing block pins)
// |__________________________________________scratch______________________________________________| scratch -> default size: 95% free GPU RAM

#define GPUCHECK(error)                                                                        \
  if (error != hipSuccess) {                                                                  \
    printf("%serror: '%s'(%d) at %s:%d%s\n", KRED, hipGetErrorString(error), error, __FILE__, \
           __LINE__, KNRM);                                                                    \
    failed("API returned error code.");                                                        \
  }

double bytesToKB(size_t s) { return (double)s / (1024.0); }
double bytesToGB(size_t s) { return (double)s / GB; }

template <class T>
std::string getType()
{
  if (typeid(T).name() == typeid(char).name()) {
    return std::string{"char"};
  }
  if (typeid(T).name() == typeid(size_t).name()) {
    return std::string{"unsigned_long"};
  }
  if (typeid(T).name() == typeid(int).name()) {
    return std::string{"int"};
  }
  if (typeid(T).name() == typeid(int4).name()) {
    return std::string{"int4"};
  }
  return std::string{"unknown"};
}

namespace o2
{
namespace benchmark
{

template <class chunk_t>
inline chunk_t* getPartPtr(chunk_t* scratchPtr, float chunkReservedGB, int partNumber)
{
  return reinterpret_cast<chunk_t*>(reinterpret_cast<char*>(scratchPtr) + static_cast<size_t>(GB * chunkReservedGB) * partNumber);
}

namespace gpu
{
//////////////////
// Kernels go here
// Read
template <class chunk_t>
__global__ void readChunkSBKernel(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  chunk_t sink{0}; // local memory -> excluded from bandwidth accounting
  size_t last{0};
  for (last = threadIdx.x; last < chunkSize; last += blockDim.x) {
    sink += chunkPtr[last]; // 1 read operation, performed "chunkSize" times
  }
  chunkPtr[last] = sink; // writing done once
}

template <class chunk_t>
__global__ void readChunkMBKernel(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  chunk_t sink{0}; // local memory -> excluded from bandwidth accounting
  size_t last{0};
  for (last = blockIdx.x * blockDim.x + threadIdx.x; last < chunkSize; last += blockDim.x * gridDim.x) {
    sink += chunkPtr[last]; // 1 read operation, performed "chunkSize" times
  }
  chunkPtr[last] = sink;
}

// Write
template <class chunk_t>
__global__ void writeChunkSBKernel(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  for (size_t i = threadIdx.x; i < chunkSize; i += blockDim.x) {
    chunkPtr[i] = 0;
  }
}

template <class chunk_t>
__global__ void writeChunkMBKernel(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    chunkPtr[i] = 0;
  }
}

// Copy
template <class chunk_t>
__global__ void copyChunkSBKernel(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  for (size_t i = threadIdx.x; i < chunkSize; i += blockDim.x) {
    chunkPtr[chunkSize - i - 1] = chunkPtr[i];
  }
}

template <class chunk_t>
__global__ void copyChunkMBKernel(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    chunkPtr[chunkSize - i - 1] = chunkPtr[i];
  }
}
} // namespace gpu

void printDeviceProp(int deviceId)
{
  const int w1 = 34;
  std::cout << std::left;
  std::cout << std::setw(w1)
            << "--------------------------------------------------------------------------------"
            << std::endl;
  std::cout << std::setw(w1) << "device#" << deviceId << std::endl;

  hipDeviceProp_t props;
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));

  std::cout << std::setw(w1) << "Name: " << props.name << std::endl;
  std::cout << std::setw(w1) << "pciBusID: " << props.pciBusID << std::endl;
  std::cout << std::setw(w1) << "pciDeviceID: " << props.pciDeviceID << std::endl;
  std::cout << std::setw(w1) << "pciDomainID: " << props.pciDomainID << std::endl;
  std::cout << std::setw(w1) << "multiProcessorCount: " << props.multiProcessorCount << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << std::setw(w1) << "isMultiGpuBoard: " << props.isMultiGpuBoard << std::endl;
  std::cout << std::setw(w1) << "clockRate: " << (float)props.clockRate / 1000.0 << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "memoryClockRate: " << (float)props.memoryClockRate / 1000.0 << " Mhz"
            << std::endl;
  std::cout << std::setw(w1) << "memoryBusWidth: " << props.memoryBusWidth << std::endl;
  std::cout << std::setw(w1) << "clockInstructionRate: " << (float)props.clockRate / 1000.0
            << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "totalGlobalMem: " << std::fixed << std::setprecision(2)
            << bytesToGB(props.totalGlobalMem) << " GB" << std::endl;
#if !defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.sharedMemPerMultiprocessor) << " KB" << std::endl;
#endif
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.maxSharedMemoryPerMultiProcessor) << " KB" << std::endl;
#endif
  std::cout << std::setw(w1) << "totalConstMem: " << props.totalConstMem << std::endl;
  std::cout << std::setw(w1) << "sharedMemPerBlock: " << (float)props.sharedMemPerBlock / 1024.0 << " KB"
            << std::endl;
  std::cout << std::setw(w1) << "canMapHostMemory: " << props.canMapHostMemory << std::endl;
  std::cout << std::setw(w1) << "regsPerBlock: " << props.regsPerBlock << std::endl;
  std::cout << std::setw(w1) << "warpSize: " << props.warpSize << std::endl;
  std::cout << std::setw(w1) << "l2CacheSize: " << props.l2CacheSize << std::endl;
  std::cout << std::setw(w1) << "computeMode: " << props.computeMode << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.x: " << props.maxThreadsDim[0] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.y: " << props.maxThreadsDim[1] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.z: " << props.maxThreadsDim[2] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.x: " << props.maxGridSize[0] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.y: " << props.maxGridSize[1] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.z: " << props.maxGridSize[2] << std::endl;
  std::cout << std::setw(w1) << "major: " << props.major << std::endl;
  std::cout << std::setw(w1) << "minor: " << props.minor << std::endl;
  std::cout << std::setw(w1) << "concurrentKernels: " << props.concurrentKernels << std::endl;
  std::cout << std::setw(w1) << "cooperativeLaunch: " << props.cooperativeLaunch << std::endl;
  std::cout << std::setw(w1) << "cooperativeMultiDeviceLaunch: " << props.cooperativeMultiDeviceLaunch << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "arch.hasGlobalInt32Atomics: " << props.arch.hasGlobalInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalFloatAtomicExch: " << props.arch.hasGlobalFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt32Atomics: " << props.arch.hasSharedInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedFloatAtomicExch: " << props.arch.hasSharedFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasFloatAtomicAdd: " << props.arch.hasFloatAtomicAdd << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalInt64Atomics: " << props.arch.hasGlobalInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt64Atomics: " << props.arch.hasSharedInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasDoubles: " << props.arch.hasDoubles << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpVote: " << props.arch.hasWarpVote << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpBallot: " << props.arch.hasWarpBallot << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpShuffle: " << props.arch.hasWarpShuffle << std::endl;
  std::cout << std::setw(w1) << "arch.hasFunnelShift: " << props.arch.hasFunnelShift << std::endl;
  std::cout << std::setw(w1) << "arch.hasThreadFenceSystem: " << props.arch.hasThreadFenceSystem << std::endl;
  std::cout << std::setw(w1) << "arch.hasSyncThreadsExt: " << props.arch.hasSyncThreadsExt << std::endl;
  std::cout << std::setw(w1) << "arch.hasSurfaceFuncs: " << props.arch.hasSurfaceFuncs << std::endl;
  std::cout << std::setw(w1) << "arch.has3dGrid: " << props.arch.has3dGrid << std::endl;
  std::cout << std::setw(w1) << "arch.hasDynamicParallelism: " << props.arch.hasDynamicParallelism << std::endl;
  std::cout << std::setw(w1) << "gcnArchName: " << props.gcnArchName << std::endl;
#endif
  std::cout << std::setw(w1) << "isIntegrated: " << props.integrated << std::endl;
  std::cout << std::setw(w1) << "maxTexture1D: " << props.maxTexture1D << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.width: " << props.maxTexture2D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.height: " << props.maxTexture2D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.width: " << props.maxTexture3D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.height: " << props.maxTexture3D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.depth: " << props.maxTexture3D[2] << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "isLargeBar: " << props.isLargeBar << std::endl;
  std::cout << std::setw(w1) << "asicRevision: " << props.asicRevision << std::endl;
#endif

  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));
  std::cout << std::setw(w1) << "peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;
  std::cout << std::setw(w1) << "non-peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (!isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;

  size_t free, total;
  GPUCHECK(hipMemGetInfo(&free, &total));

  std::cout << std::fixed << std::setprecision(2);
  std::cout << std::setw(w1) << "memInfo.total: " << bytesToGB(total) << " GB" << std::endl;
  std::cout << std::setw(w1) << "memInfo.free:  " << bytesToGB(free) << " GB (" << std::setprecision(0)
            << (float)free / total * 100.0 << "%)" << std::endl;
}

template <class chunk_t>
template <typename... T>
float GPUbenchmark<chunk_t>::runSequential(void (*kernel)(chunk_t*, T...),
                                           int nLaunches,
                                           int chunkId,
                                           int nBlocks,
                                           int nThreads,
                                           T&... args) // run for each chunk
{
  hipEvent_t start, stop;
  hipStream_t stream;
  GPUCHECK(hipStreamCreate(&stream));

  GPUCHECK(hipSetDevice(mOptions.deviceId));
  chunk_t* chunkPtr = getPartPtr<chunk_t>(mState.scratchPtr, mState.chunkReservedGB, chunkId);

  // Warm up
  (*kernel)<<<nBlocks, nThreads, 0, stream>>>(chunkPtr, args...);

  GPUCHECK(hipEventCreate(&start));
  GPUCHECK(hipEventCreate(&stop));

  GPUCHECK(hipEventRecord(start));
  for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) {           // Schedule all the requested kernel launches
    (*kernel)<<<nBlocks, nThreads, 0, stream>>>(chunkPtr, args...); // NOLINT: clang-tidy false-positive
  }
  GPUCHECK(hipEventRecord(stop)); // record checkpoint

  GPUCHECK(hipEventSynchronize(stop)); // synchronize executions
  float milliseconds{0.f};
  GPUCHECK(hipEventElapsedTime(&milliseconds, start, stop));
  GPUCHECK(hipEventDestroy(start));
  GPUCHECK(hipEventDestroy(stop));

  GPUCHECK(hipStreamDestroy(stream));
  return milliseconds;
}

template <class chunk_t>
template <typename... T>
std::vector<float> GPUbenchmark<chunk_t>::runConcurrent(void (*kernel)(chunk_t*, T...),
                                                        int nChunks,
                                                        int nLaunches,
                                                        int dimStreams,
                                                        int nBlocks,
                                                        int nThreads,
                                                        T&... args)
{
  std::vector<hipEvent_t> starts(nChunks), stops(nChunks);
  std::vector<hipStream_t> streams(dimStreams);

  std::vector<float> results(nChunks);
  GPUCHECK(hipSetDevice(mOptions.deviceId));

  for (auto iStream{0}; iStream < dimStreams; ++iStream) {
    GPUCHECK(hipStreamCreate(&(streams.at(iStream)))); // round-robin on stream pool
  }

  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    GPUCHECK(hipEventCreate(&(starts[iChunk])));
    GPUCHECK(hipEventCreate(&(stops[iChunk])));
  }

  // Warm up on every chunk
  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    chunk_t* chunkPtr = getPartPtr<chunk_t>(mState.scratchPtr, mState.chunkReservedGB, iChunk);
    (*kernel)<<<nBlocks, nThreads, 0, streams[iChunk % dimStreams]>>>(chunkPtr, args...);
  }

  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    chunk_t* chunkPtr = getPartPtr<chunk_t>(mState.scratchPtr, mState.chunkReservedGB, iChunk);
    GPUCHECK(hipEventRecord(starts[iChunk], streams[iChunk % dimStreams]));
    for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) {
      (*kernel)<<<nBlocks, nThreads, 0, streams[iChunk % dimStreams]>>>(chunkPtr, args...);
    }
    GPUCHECK(hipEventRecord(stops[iChunk], streams[iChunk % dimStreams]));
  }

  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    GPUCHECK(hipEventSynchronize(stops[iChunk]));
    GPUCHECK(hipEventElapsedTime(&(results.at(iChunk)), starts[iChunk], stops[iChunk]));
    GPUCHECK(hipEventDestroy(starts[iChunk]));
    GPUCHECK(hipEventDestroy(stops[iChunk]));
  }

  for (auto iStream{0}; iStream < dimStreams; ++iStream) {
    GPUCHECK(hipStreamDestroy(streams[iStream]));
  }

  return results;
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::printDevices()
{
  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));

  for (int i = 0; i < deviceCnt; i++) {
    GPUCHECK(hipSetDevice(i));
    printDeviceProp(i);
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::globalInit()
{
  hipDeviceProp_t props;
  size_t free;

  // Fetch and store features
  GPUCHECK(hipGetDeviceProperties(&props, mOptions.deviceId));
  GPUCHECK(hipMemGetInfo(&free, &mState.totalMemory));
  GPUCHECK(hipSetDevice(mOptions.deviceId));

  mState.chunkReservedGB = mOptions.chunkReservedGB;
  mState.iterations = mOptions.kernelLaunches;
  mState.streams = mOptions.streams;
  mState.nMultiprocessors = props.multiProcessorCount;
  mState.nMaxThreadsPerBlock = props.maxThreadsPerMultiProcessor;
  mState.nMaxThreadsPerDimension = props.maxThreadsDim[0];
  mState.scratchSize = static_cast<long int>(mOptions.freeMemoryFractionToAllocate * free);
  std::cout << ">>> Running on: \033[1;31m" << props.name << "\e[0m" << std::endl;

  // Allocate scratch on GPU
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&mState.scratchPtr), mState.scratchSize));

  mState.computeScratchPtrs();
  GPUCHECK(hipMemset(mState.scratchPtr, 0, mState.scratchSize))

  std::cout << "    ├ Buffer type: \e[1m" << getType<chunk_t>() << "\e[0m" << std::endl
            << "    ├ Allocated: " << std::setprecision(2) << bytesToGB(mState.scratchSize) << "/" << std::setprecision(2) << bytesToGB(mState.totalMemory)
            << "(GB) [" << std::setprecision(3) << (100.f) * (mState.scratchSize / (float)mState.totalMemory) << "%]\n"
            << "    ├ Number of streams allocated: " << mState.getStreamsPoolSize() << "\n"
            << "    ├ Number of scratch chunks: " << mState.getMaxChunks() << " of " << mOptions.chunkReservedGB << "GB each\n"
            << "    └ Each chunk can store up to: " << mState.getChunkCapacity() << " elements" << std::endl
            << std::endl;
}

/// Read
template <class chunk_t>
void GPUbenchmark<chunk_t>::readInit()
{
  std::cout << ">>> Initializing (" << getType<chunk_t>() << ") read benchmarks with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  mState.hostReadResultsVector.resize(mState.getMaxChunks());
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&(mState.deviceReadResultsPtr)), mState.getMaxChunks() * sizeof(chunk_t)));
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::readSequential(SplitLevel sl)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("seq_read_SB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << "    ├ Sequential read single block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << 1 << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = runSequential(&gpu::readChunkSBKernel<chunk_t>,
                                      mState.getNKernelLaunches(),
                                      iChunk,
                                      1,        // nBlocks
                                      nThreads, // args...
                                      capacity);
          mResultWriter.get()->storeBenchmarkEntry(Test::Read, iChunk, result, mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }

    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("seq_read_MB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << "    ├ Sequential read multiple block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << dimGrid << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = runSequential(&gpu::readChunkMBKernel<chunk_t>,
                                      mState.getNKernelLaunches(),
                                      iChunk,
                                      dimGrid,
                                      nThreads, // args...
                                      capacity);
          mResultWriter.get()->storeBenchmarkEntry(Test::Read, iChunk, result, mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::readConcurrent(SplitLevel sl, int nRegions)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("conc_read_SB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ Concurrent read single block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << 1 << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        auto results = runConcurrent(&gpu::readChunkSBKernel<chunk_t>,
                                     mState.getMaxChunks(), // nStreams
                                     mState.getNKernelLaunches(),
                                     mState.getStreamsPoolSize(),
                                     1, // single Block
                                     nThreads,
                                     capacity);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(Test::Read, iResult, results[iResult], mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("conc_read_MB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto chunks{mState.getMaxChunks()};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto nBlocks{dimGrid / mState.getMaxChunks()};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ Concurrent read multiple blocks (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << nBlocks << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        auto results = runConcurrent(&gpu::readChunkMBKernel<chunk_t>,
                                     mState.getMaxChunks(), // nStreams
                                     mState.getNKernelLaunches(),
                                     mState.getStreamsPoolSize(),
                                     nBlocks,
                                     nThreads,
                                     capacity);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(Test::Read, iResult, results[iResult], mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::readFinalize()
{
  GPUCHECK(hipMemcpy(mState.hostReadResultsVector.data(), mState.deviceReadResultsPtr, mState.getMaxChunks() * sizeof(chunk_t), hipMemcpyDeviceToHost));
  GPUCHECK(hipFree(mState.deviceReadResultsPtr));
  std::cout << "    └\033[1;32m done\033[0m" << std::endl;
}

/// Write
template <class chunk_t>
void GPUbenchmark<chunk_t>::writeInit()
{
  std::cout << ">>> Initializing (" << getType<chunk_t>() << ") write benchmarks with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  mState.hostWriteResultsVector.resize(mState.getMaxChunks());
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&(mState.deviceWriteResultsPtr)), mState.getMaxChunks() * sizeof(chunk_t)));
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::writeSequential(SplitLevel sl)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("seq_write_SB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << "    ├ Sequential write single block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << 1 << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = runSequential(&gpu::writeChunkSBKernel<chunk_t>,
                                      mState.getNKernelLaunches(),
                                      iChunk,
                                      1, // nBlocks
                                      nThreads,
                                      capacity);
          mResultWriter.get()->storeBenchmarkEntry(Test::Write, iChunk, result, mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }

    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("seq_write_MB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << "    ├ Sequential write multiple block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << dimGrid << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = runSequential(&gpu::writeChunkMBKernel<chunk_t>,
                                      mState.getNKernelLaunches(),
                                      iChunk,
                                      dimGrid,
                                      nThreads,
                                      capacity);
          mResultWriter.get()->storeBenchmarkEntry(Test::Write, iChunk, result, mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::writeConcurrent(SplitLevel sl, int nRegions)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("conc_write_SB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ Concurrent write single block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << 1 << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        auto results = runConcurrent(&gpu::writeChunkSBKernel<chunk_t>,
                                     mState.getMaxChunks(), // nStreams
                                     mState.getNKernelLaunches(),
                                     mState.getStreamsPoolSize(),
                                     1, // nBlocks
                                     nThreads,
                                     capacity);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(Test::Write, iResult, results[iResult], mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("conc_write_MB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto nBlocks{dimGrid / mState.getMaxChunks()};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ Concurrent write multiple blocks (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << nBlocks << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        auto results = runConcurrent(&gpu::writeChunkMBKernel<chunk_t>,
                                     mState.getMaxChunks(), // nStreams
                                     mState.getNKernelLaunches(),
                                     mState.getStreamsPoolSize(),
                                     nBlocks,
                                     nThreads,
                                     capacity);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(Test::Write, iResult, results[iResult], mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::writeFinalize()
{
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMemcpy(mState.hostWriteResultsVector.data(), mState.deviceWriteResultsPtr, mState.getMaxChunks() * sizeof(chunk_t), hipMemcpyDeviceToHost));
  GPUCHECK(hipFree(mState.deviceWriteResultsPtr));
  std::cout << "    └\033[1;32m done\033[0m" << std::endl;
}

/// Copy
template <class chunk_t>
void GPUbenchmark<chunk_t>::copyInit()
{
  std::cout << ">>> Initializing (" << getType<chunk_t>() << ") copy benchmarks with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  mState.hostCopyInputsVector.resize(mState.getMaxChunks());
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&(mState.deviceCopyInputsPtr)), mState.getMaxChunks() * sizeof(chunk_t)));
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::copySequential(SplitLevel sl)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("seq_copy_SB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << "    ├ Sequential copy single block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << 1 << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = runSequential(&gpu::copyChunkSBKernel<chunk_t>,
                                      mState.getNKernelLaunches(),
                                      iChunk,
                                      1,
                                      nThreads,
                                      capacity);
          mResultWriter.get()->storeBenchmarkEntry(Test::Copy, iChunk, result, mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }

    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("seq_copy_MB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << "    ├ Sequential copy multiple block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << dimGrid << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = runSequential(&gpu::copyChunkMBKernel<chunk_t>,
                                      mState.getNKernelLaunches(),
                                      iChunk,
                                      dimGrid,
                                      nThreads,
                                      capacity);
          mResultWriter.get()->storeBenchmarkEntry(Test::Copy, iChunk, result, mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::copyConcurrent(SplitLevel sl, int nRegions)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("conc_copy_SB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ Concurrent copy single block (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << 1 << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        auto results = runConcurrent(&gpu::copyChunkSBKernel<chunk_t>,
                                     mState.getMaxChunks(), // nStreams
                                     mState.getNKernelLaunches(),
                                     mState.getStreamsPoolSize(),
                                     1, // nBlocks
                                     nThreads,
                                     capacity);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(Test::Copy, iResult, results[iResult], mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("conc_copy_MB", getType<chunk_t>(), mState.getMaxChunks());
      auto dimGrid{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto nBlocks{dimGrid / mState.getMaxChunks()};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getChunkCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {

        std::cout << "    ├ Concurrent copy multiple blocks (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
                  << "    │   · blocks per kernel: " << nBlocks << "/" << dimGrid << "\n"
                  << "    │   · threads per block: " << nThreads << "\n";
        auto results = runConcurrent(&gpu::copyChunkMBKernel<chunk_t>,
                                     mState.getMaxChunks(), // nStreams
                                     mState.getNKernelLaunches(),
                                     mState.getStreamsPoolSize(),
                                     nBlocks,
                                     nThreads,
                                     capacity);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(Test::Copy, iResult, results[iResult], mState.chunkReservedGB, mState.getNKernelLaunches());
        }
        mResultWriter.get()->snapshotBenchmark();
      }
      break;
    }
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::copyFinalize()
{
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipFree(mState.deviceCopyInputsPtr));
  std::cout << "    └\033[1;32m done\033[0m" << std::endl;
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::globalFinalize()
{
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipFree(mState.scratchPtr));
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::run()
{
  globalInit();

  for (auto& sl : mOptions.pools) {
    for (auto& test : mOptions.tests) {
      switch (test) {
        case Test::Read: {
          readInit();

          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Sequential) != mOptions.modes.end()) {
            readSequential(sl);
          }
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Concurrent) != mOptions.modes.end()) {
            readConcurrent(sl);
          }

          readFinalize();

          break;
        }
        case Test::Write: {
          writeInit();
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Sequential) != mOptions.modes.end()) {
            writeSequential(sl);
          }
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Concurrent) != mOptions.modes.end()) {
            writeConcurrent(sl);
          }

          writeFinalize();

          break;
        }
        case Test::Copy: {
          copyInit();
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Sequential) != mOptions.modes.end()) {
            copySequential(sl);
          }
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Concurrent) != mOptions.modes.end()) {
            copyConcurrent(sl);
          }

          copyFinalize();

          break;
        }
      }
    }
  }

  globalFinalize();
}

template class GPUbenchmark<char>;
template class GPUbenchmark<size_t>;
template class GPUbenchmark<int>;
// template class GPUbenchmark<uint4>;

} // namespace benchmark
} // namespace o2