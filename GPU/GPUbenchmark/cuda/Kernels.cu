#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Kernels.{cu, hip.cxx}
/// \author: mconcas@cern.ch

#include "../Shared/Kernels.h"
#include <chrono>
#include <cstdio>
#include <numeric>

// Memory partitioning schema
//
// |----------------------region 0-----------------|----------------------region 1-----------------| regions -> deafult: 2, to test lower and upper RAM
// |--chunk 0--|--chunk 1--|--chunk 2--|                  ***                          |--chunk n--| chunks  -> default size: 1GB (sing block pins)
// |__________________________________________scratch______________________________________________| scratch -> default size: 95% free GPU RAM

#define GPUCHECK(error)                                                                        \
  if (error != hipSuccess) {                                                                  \
    printf("%serror: '%s'(%d) at %s:%d%s\n", KRED, hipGetErrorString(error), error, __FILE__, \
           __LINE__, KNRM);                                                                    \
    failed("API returned error code.");                                                        \
  }

double bytesToconfig(size_t s) { return (double)s / (1024.0); }
double bytesToGB(size_t s) { return (double)s / GB; }

bool checkTestChunks(std::vector<std::pair<float, float>>& chunks, size_t availMemSizeGB)
{
  if (!chunks.size()) {
    return true;
  }

  bool check{false};

  sort(chunks.begin(), chunks.end());
  for (auto iChunk{0}; iChunk < chunks.size(); ++iChunk) { // Check boundaries
    if (chunks[iChunk].first + chunks[iChunk].second > availMemSizeGB) {
      check = false;
      break;
    }
    if (iChunk > 0) { // Check intersections
      if (chunks[iChunk].first < chunks[iChunk - 1].first + chunks[iChunk - 1].second) {
        check = false;
        break;
      }
    }
    check = true;
  }
  return check;
}

// CUDA does not support <type4> operations:
// https://forums.developer.nvidia.com/t/swizzling-float4-arithmetic-support/217
#ifndef __HIPCC__
inline __host__ __device__ void operator+=(int4& a, int4 b)
{
  a.x += b.x;
  a.y += b.y;
  a.z += b.z;
  a.w += b.w;
}
#endif

namespace o2
{
namespace benchmark
{

namespace gpu
{
////////////
// Kernels

// Read
template <class chunk_t>
__global__ void read_k(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  chunk_t sink{0};
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    sink += chunkPtr[i];
  }
  chunkPtr[threadIdx.x] = sink;
}

// Write
template <class chunk_t>
__global__ void write_k(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    chunkPtr[i] = 0;
  }
}

template <>
__global__ void write_k(
  int4* chunkPtr,
  size_t chunkSize)
{
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    chunkPtr[i] = {0, 1, 0, 0};
  };
}

// Copy
template <class chunk_t>
__global__ void copy_k(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  size_t offset = chunkSize / 2;
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < offset; i += blockDim.x * gridDim.x) {
    chunkPtr[i] = chunkPtr[offset + i];
  }
}

// Random read
template <class chunk_t>
__global__ void rand_read_k(
  chunk_t* chunkPtr,
  size_t chunkSize)
{
  chunk_t sink{0};
  BSDRnd r{};
  for (size_t i = threadIdx.x; i < chunkSize; i += blockDim.x) {
    sink = chunkPtr[i];
  }
  chunkPtr[threadIdx.x] = sink; // writing done once
}

// Distributed read
template <class chunk_t>
__global__ void read_dist_k(
  chunk_t** block_ptr,
  size_t* block_size)
{
  chunk_t sink{0};
  chunk_t* ptr = block_ptr[blockIdx.x];
  size_t n = block_size[blockIdx.x];
  for (size_t i = threadIdx.x; i < n; i += blockDim.x) {
    sink += ptr[i];
  }
  ptr[threadIdx.x] = sink;
}

// Distributed write
template <class chunk_t>
__global__ void write_dist_k(
  chunk_t** block_ptr,
  size_t* block_size)
{
  chunk_t* ptr = block_ptr[blockIdx.x];
  size_t n = block_size[blockIdx.x];
  for (size_t i = threadIdx.x; i < n; i += blockDim.x) {
    ptr[i] = 0;
  }
}

template <>
__global__ void write_dist_k(
  int4** block_ptr,
  size_t* block_size)
{
  int4* ptr = block_ptr[blockIdx.x];
  size_t n = block_size[blockIdx.x];
  for (size_t i = threadIdx.x; i < n; i += blockDim.x) {
    ptr[i] = {0, 1, 0, 0};
  }
}

// Distributed copy
template <class chunk_t>
__global__ void copy_dist_k(
  chunk_t** block_ptr,
  size_t* block_size)
{
  chunk_t* ptr = block_ptr[blockIdx.x];
  size_t n = block_size[blockIdx.x];
  size_t offset = n / 2;
  for (size_t i = threadIdx.x; i < offset; i += blockDim.x) {
    ptr[i] = ptr[offset + i];
  }
}

// Distributed Random read
template <class chunk_t>
__global__ void rand_read_dist_k(
  chunk_t** block_ptr,
  size_t* block_size)
{
}

} // namespace gpu

void printDeviceProp(int deviceId)
{
  const int w1 = 34;
  std::cout << std::left;
  std::cout << std::setw(w1)
            << "--------------------------------------------------------------------------------"
            << std::endl;
  std::cout << std::setw(w1) << "device#" << deviceId << std::endl;

  hipDeviceProp_t props;
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));

  std::cout << std::setw(w1) << "Name: " << props.name << std::endl;
  std::cout << std::setw(w1) << "pciBusID: " << props.pciBusID << std::endl;
  std::cout << std::setw(w1) << "pciDeviceID: " << props.pciDeviceID << std::endl;
  std::cout << std::setw(w1) << "pciDomainID: " << props.pciDomainID << std::endl;
  std::cout << std::setw(w1) << "multiProcessorCount: " << props.multiProcessorCount << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << std::setw(w1) << "isMultiGpuBoard: " << props.isMultiGpuBoard << std::endl;
  std::cout << std::setw(w1) << "clockRate: " << (float)props.clockRate / 1000.0 << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "memoryClockRate: " << (float)props.memoryClockRate / 1000.0 << " Mhz"
            << std::endl;
  std::cout << std::setw(w1) << "memoryBusWidth: " << props.memoryBusWidth << std::endl;
  std::cout << std::setw(w1) << "clockInstructionRate: " << (float)props.clockRate / 1000.0
            << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "totalGlobalMem: " << std::fixed << std::setprecision(2)
            << bytesToGB(props.totalGlobalMem) << " GB" << std::endl;
#if !defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToconfig(props.sharedMemPerMultiprocessor) << " config" << std::endl;
#endif
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToconfig(props.maxSharedMemoryPerMultiProcessor) << " config" << std::endl;
#endif
  std::cout << std::setw(w1) << "totalConstMem: " << props.totalConstMem << std::endl;
  std::cout << std::setw(w1) << "sharedMemPerBlock: " << (float)props.sharedMemPerBlock / 1024.0 << " config"
            << std::endl;
  std::cout << std::setw(w1) << "canMapHostMemory: " << props.canMapHostMemory << std::endl;
  std::cout << std::setw(w1) << "regsPerBlock: " << props.regsPerBlock << std::endl;
  std::cout << std::setw(w1) << "warpSize: " << props.warpSize << std::endl;
  std::cout << std::setw(w1) << "l2CacheSize: " << props.l2CacheSize << std::endl;
  std::cout << std::setw(w1) << "computeMode: " << props.computeMode << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.x: " << props.maxThreadsDim[0] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.y: " << props.maxThreadsDim[1] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.z: " << props.maxThreadsDim[2] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.x: " << props.maxGridSize[0] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.y: " << props.maxGridSize[1] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.z: " << props.maxGridSize[2] << std::endl;
  std::cout << std::setw(w1) << "major: " << props.major << std::endl;
  std::cout << std::setw(w1) << "minor: " << props.minor << std::endl;
  std::cout << std::setw(w1) << "concurrentKernels: " << props.concurrentKernels << std::endl;
  std::cout << std::setw(w1) << "cooperativeLaunch: " << props.cooperativeLaunch << std::endl;
  std::cout << std::setw(w1) << "cooperativeMultiDeviceLaunch: " << props.cooperativeMultiDeviceLaunch << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "arch.hasGlobalInt32Atomics: " << props.arch.hasGlobalInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalFloatAtomicExch: " << props.arch.hasGlobalFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt32Atomics: " << props.arch.hasSharedInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedFloatAtomicExch: " << props.arch.hasSharedFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasFloatAtomicAdd: " << props.arch.hasFloatAtomicAdd << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalInt64Atomics: " << props.arch.hasGlobalInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt64Atomics: " << props.arch.hasSharedInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasDoubles: " << props.arch.hasDoubles << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpVote: " << props.arch.hasWarpVote << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpBallot: " << props.arch.hasWarpBallot << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpShuffle: " << props.arch.hasWarpShuffle << std::endl;
  std::cout << std::setw(w1) << "arch.hasFunnelShift: " << props.arch.hasFunnelShift << std::endl;
  std::cout << std::setw(w1) << "arch.hasThreadFenceSystem: " << props.arch.hasThreadFenceSystem << std::endl;
  std::cout << std::setw(w1) << "arch.hasSyncThreadsExt: " << props.arch.hasSyncThreadsExt << std::endl;
  std::cout << std::setw(w1) << "arch.hasSurfaceFuncs: " << props.arch.hasSurfaceFuncs << std::endl;
  std::cout << std::setw(w1) << "arch.has3dGrid: " << props.arch.has3dGrid << std::endl;
  std::cout << std::setw(w1) << "arch.hasDynamicParallelism: " << props.arch.hasDynamicParallelism << std::endl;
  std::cout << std::setw(w1) << "gcnArchName: " << props.gcnArchName << std::endl;
#endif
  std::cout << std::setw(w1) << "isIntegrated: " << props.integrated << std::endl;
  std::cout << std::setw(w1) << "maxTexture1D: " << props.maxTexture1D << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.width: " << props.maxTexture2D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.height: " << props.maxTexture2D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.width: " << props.maxTexture3D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.height: " << props.maxTexture3D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.depth: " << props.maxTexture3D[2] << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "isLargeBar: " << props.isLargeBar << std::endl;
  std::cout << std::setw(w1) << "asicRevision: " << props.asicRevision << std::endl;
#endif

  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));
  std::cout << std::setw(w1) << "peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;
  std::cout << std::setw(w1) << "non-peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (!isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;

  size_t free, total;
  GPUCHECK(hipMemGetInfo(&free, &total));

  std::cout << std::fixed << std::setprecision(2);
  std::cout << std::setw(w1) << "memInfo.total: " << bytesToGB(total) << " GB" << std::endl;
  std::cout << std::setw(w1) << "memInfo.free:  " << bytesToGB(free) << " GB (" << std::setprecision(0)
            << (float)free / total * 100.0 << "%)" << std::endl;
}

template <class chunk_t>
template <typename... T>
float GPUbenchmark<chunk_t>::runSequential(void (*kernel)(chunk_t*, size_t, T...),
                                           std::pair<float, float>& chunk,
                                           int nLaunches,
                                           int nBlocks,
                                           int nThreads,
                                           T&... args) // run for each chunk
{
  float milliseconds{0.f};
  hipEvent_t start, stop;
  hipStream_t stream;
  GPUCHECK(hipStreamCreate(&stream));
  GPUCHECK(hipSetDevice(mOptions.deviceId));

  chunk_t* chunkPtr = getCustomPtr<chunk_t>(mState.scratchPtr, chunk.first);

  // Warm up
  (*kernel)<<<nBlocks, nThreads, 0, stream>>>(chunkPtr, getBufferCapacity<chunk_t>(chunk.second), args...);

  GPUCHECK(hipEventCreate(&start));
  GPUCHECK(hipEventCreate(&stop));

  GPUCHECK(hipEventRecord(start));
  for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) {                                                     // Schedule all the requested kernel launches
    (*kernel)<<<nBlocks, nThreads, 0, stream>>>(chunkPtr, getBufferCapacity<chunk_t>(chunk.second), args...); // NOLINT: clang-tidy false-positive
  }
  GPUCHECK(hipEventRecord(stop));      // record checkpoint
  GPUCHECK(hipEventSynchronize(stop)); // synchronize executions
  GPUCHECK(hipEventElapsedTime(&milliseconds, start, stop));
  GPUCHECK(hipEventDestroy(start));
  GPUCHECK(hipEventDestroy(stop));
  GPUCHECK(hipStreamDestroy(stream));

  return milliseconds;
}

template <class chunk_t>
template <typename... T>
std::vector<float> GPUbenchmark<chunk_t>::runConcurrent(void (*kernel)(chunk_t*, size_t, T...),
                                                        std::vector<std::pair<float, float>>& chunkRanges,
                                                        int nLaunches,
                                                        int dimStreams,
                                                        int nBlocks,
                                                        int nThreads,
                                                        T&... args)
{
  auto nChunks = chunkRanges.size();
  std::vector<float> results(nChunks + 1); // last spot is for the host time
  std::vector<hipEvent_t> starts(nChunks), stops(nChunks);
  std::vector<hipStream_t> streams(dimStreams);

  GPUCHECK(hipSetDevice(mOptions.deviceId));
  for (auto iStream{0}; iStream < dimStreams; ++iStream) {
    GPUCHECK(hipStreamCreate(&(streams.at(iStream)))); // round-robin on stream pool
  }
  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    GPUCHECK(hipEventCreate(&(starts[iChunk])));
    GPUCHECK(hipEventCreate(&(stops[iChunk])));
  }

  // Warm up on every chunk
  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    auto& chunk = chunkRanges[iChunk];
    chunk_t* chunkPtr = getCustomPtr<chunk_t>(mState.scratchPtr, chunk.first);
    (*kernel)<<<nBlocks, nThreads, 0, streams[iChunk % dimStreams]>>>(chunkPtr, getBufferCapacity<chunk_t>(chunk.second), args...);
  }
  auto start = std::chrono::high_resolution_clock::now();

  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    auto& chunk = chunkRanges[iChunk];
    chunk_t* chunkPtr = getCustomPtr<chunk_t>(mState.scratchPtr, chunk.first);
    GPUCHECK(hipEventRecord(starts[iChunk], streams[iChunk % dimStreams]));
    for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) {
      (*kernel)<<<nBlocks, nThreads, 0, streams[iChunk % dimStreams]>>>(chunkPtr, getBufferCapacity<chunk_t>(chunk.second), args...);
    }
    GPUCHECK(hipEventRecord(stops[iChunk], streams[iChunk % dimStreams]));
  }

  for (auto iChunk{0}; iChunk < nChunks; ++iChunk) {
    GPUCHECK(hipEventSynchronize(stops[iChunk]));
    GPUCHECK(hipEventElapsedTime(&(results.at(iChunk)), starts[iChunk], stops[iChunk]));
    GPUCHECK(hipEventDestroy(starts[iChunk]));
    GPUCHECK(hipEventDestroy(stops[iChunk]));
  }
  GPUCHECK(hipDeviceSynchronize());

  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> diff_t{end - start};

  for (auto iStream{0}; iStream < dimStreams; ++iStream) {
    GPUCHECK(hipStreamDestroy(streams[iStream]));
  }

  results[nChunks] = diff_t.count(); // register host time on latest spot
  return results;
}

template <class chunk_t>
template <typename... T>
float GPUbenchmark<chunk_t>::runDistributed(void (*kernel)(chunk_t**, size_t*, T...),
                                            std::vector<std::pair<float, float>>& chunkRanges,
                                            int nLaunches,
                                            int nBlocks,
                                            int nThreads,
                                            T&... args)
{
  std::vector<chunk_t*> chunkPtrs(chunkRanges.size()); // Pointers to the beginning of each chunk
  std::vector<chunk_t*> ptrPerBlocks(nBlocks);         // Pointers for each block
  std::vector<size_t> perBlockCapacity(nBlocks);       // Capacity of sub-buffer for block

  float totChunkGB{0.f};
  int totComputedBlocks{0};

  for (size_t iChunk{0}; iChunk < chunkRanges.size(); ++iChunk) {
    chunkPtrs[iChunk] = getCustomPtr<chunk_t>(mState.scratchPtr, chunkRanges[iChunk].first);
    totChunkGB += chunkRanges[iChunk].second;
  }
  int index{0};
  for (size_t iChunk{0}; iChunk < chunkRanges.size(); ++iChunk) {
    float percFromMem = chunkRanges[iChunk].second / totChunkGB;
    int blocksPerChunk = percFromMem * nBlocks;
    totComputedBlocks += blocksPerChunk;
    for (int iBlock{0}; iBlock < blocksPerChunk; ++iBlock, ++index) {
      float memPerBlock = chunkRanges[iChunk].second / blocksPerChunk;
      ptrPerBlocks[index] = getCustomPtr<chunk_t>(chunkPtrs[iChunk], iBlock * memPerBlock);
      perBlockCapacity[index] = getBufferCapacity<chunk_t>(memPerBlock);
    }
  }

  if (totComputedBlocks != nBlocks) {
    std::cout << "   │   - \033[1;33mWarning: Sum of used blocks (" << totComputedBlocks
              << ") is different from requested one (" << nBlocks << ")!\e[0m"
              << std::endl;
  }

  // Setup
  chunk_t** block_ptr;
  size_t* block_size;
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&block_ptr), 60 * sizeof(chunk_t*)));
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&block_size), 60 * sizeof(size_t)));
  GPUCHECK(hipMemcpy(block_ptr, ptrPerBlocks.data(), nBlocks * sizeof(chunk_t*), hipMemcpyHostToDevice));
  GPUCHECK(hipMemcpy(block_size, perBlockCapacity.data(), nBlocks * sizeof(size_t), hipMemcpyHostToDevice));

  float milliseconds{0.f};
  hipEvent_t start, stop;
  hipStream_t stream;

  GPUCHECK(hipStreamCreate(&stream));
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipEventCreate(&start));
  GPUCHECK(hipEventCreate(&stop));

  // Warm up
  (*kernel)<<<totComputedBlocks, nThreads, 0, stream>>>(block_ptr, block_size, args...);

  GPUCHECK(hipEventRecord(start));
  for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) {                                  // Schedule all the requested kernel launches
    (*kernel)<<<totComputedBlocks, nThreads, 0, stream>>>(block_ptr, block_size, args...); // NOLINT: clang-tidy false-positive
  }
  GPUCHECK(hipEventRecord(stop));      // record checkpoint
  GPUCHECK(hipEventSynchronize(stop)); // synchronize executions
  GPUCHECK(hipEventElapsedTime(&milliseconds, start, stop));
  GPUCHECK(hipEventDestroy(start));
  GPUCHECK(hipEventDestroy(stop));
  GPUCHECK(hipStreamDestroy(stream));
  return milliseconds;
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::printDevices()
{
  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));

  for (int i = 0; i < deviceCnt; i++) {
    GPUCHECK(hipSetDevice(i));
    printDeviceProp(i);
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::globalInit()
{
  hipDeviceProp_t props;
  size_t free;

  // Fetch and store features
  GPUCHECK(hipGetDeviceProperties(&props, mOptions.deviceId));
  GPUCHECK(hipMemGetInfo(&free, &mState.totalMemory));
  GPUCHECK(hipSetDevice(mOptions.deviceId));

  mState.chunkReservedGB = mOptions.chunkReservedGB;
  mState.iterations = mOptions.kernelLaunches;
  mState.streams = mOptions.streams;
  mState.testChunks = mOptions.testChunks;
  if (!checkTestChunks(mState.testChunks, mOptions.freeMemoryFractionToAllocate * free / GB)) {
    std::cerr << "Failed to configure memory chunks: check arbitrary chunks boundaries." << std::endl;
    exit(1);
  }
  mState.nMultiprocessors = props.multiProcessorCount;
  mState.nMaxThreadsPerBlock = props.maxThreadsPerMultiProcessor;
  mState.nMaxThreadsPerDimension = props.maxThreadsDim[0];
  mState.scratchSize = static_cast<long int>(mOptions.freeMemoryFractionToAllocate * free);

  if (mState.testChunks.empty()) {
    for (auto j{0}; j < mState.getMaxChunks() * mState.chunkReservedGB; j += mState.chunkReservedGB) {
      mState.testChunks.emplace_back(j, mState.chunkReservedGB);
    }
  }

  std::cout << " ◈ Running on: \033[1;31m" << props.name << "\e[0m" << std::endl;
  // Allocate scratch on GPU
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&mState.scratchPtr), mState.scratchSize));
  GPUCHECK(hipMemset(mState.scratchPtr, 0, mState.scratchSize))

  std::cout << "   ├ Buffer type: \e[1m" << getType<chunk_t>() << "\e[0m" << std::endl
            << "   ├ Allocated: " << std::setprecision(2) << bytesToGB(mState.scratchSize) << "/" << std::setprecision(2) << bytesToGB(mState.totalMemory)
            << "(GB) [" << std::setprecision(3) << (100.f) * (mState.scratchSize / (float)mState.totalMemory) << "%]\n"
            << "   └ Available streams: " << mState.getStreamsPoolSize() << "\n\n";
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::initTest(Test test)
{
  std::cout << " ◈ \033[1;33m" << getType<chunk_t>() << "\033[0m " << test << " benchmark with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  GPUCHECK(hipSetDevice(mOptions.deviceId));
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::runTest(Test test, Mode mode, KernelConfig config)
{
  mResultWriter.get()->addBenchmarkEntry(getTestName(mode, test, config), getType<chunk_t>(), mState.getMaxChunks());
  auto dimGrid{mState.nMultiprocessors};
  auto nBlocks{(config == KernelConfig::Single) ? 1 : (config == KernelConfig::Multi) ? dimGrid / mState.testChunks.size()
                                                                                      : (config == KernelConfig::All) ? dimGrid : mOptions.numBlocks};
  size_t nThreads;
  if (mOptions.numThreads < 0) {
    nThreads = std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock);
  } else {
    nThreads = mOptions.numThreads;
  }
  nThreads *= mOptions.threadPoolFraction;

  auto capacity{mState.getChunkCapacity()};

  void (*kernel)(chunk_t*, size_t);
  void (*kernel_distributed)(chunk_t**, size_t*);

  if (mode != Mode::Distributed) {
    switch (test) {
      case Test::Read: {
        kernel = &gpu::read_k<chunk_t>;
        break;
      }
      case Test::Write: {
        kernel = &gpu::write_k<chunk_t>;
        break;
      }
      case Test::Copy: {
        kernel = &gpu::copy_k<chunk_t>;
        break;
      }
    }
  } else {
    switch (test) {
      case Test::Read: {
        kernel_distributed = &gpu::read_dist_k<chunk_t>;
        break;
      }
      case Test::Write: {
        kernel_distributed = &gpu::write_dist_k<chunk_t>;
        break;
      }
      case Test::Copy: {
        kernel_distributed = &gpu::copy_dist_k<chunk_t>;
        break;
      }
    }
  }

  for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
    std::cout << "   ├ " << mode << " " << test << " " << config << " block(s) (" << measurement + 1 << "/" << mOptions.nTests << "): \n"
              << "   │   - blocks per kernel: " << nBlocks << "/" << dimGrid << "\n"
              << "   │   - threads per block: " << (int)nThreads << "\n";
    if (mode == Mode::Sequential) {
      std::cout << "   │   - per chunk throughput:\n";
      for (auto iChunk{0}; iChunk < mState.testChunks.size(); ++iChunk) { // loop over single chunks separately
        auto& chunk = mState.testChunks[iChunk];
        auto result = runSequential(kernel,
                                    chunk,
                                    mState.getNKernelLaunches(),
                                    nBlocks,
                                    nThreads);
        auto throughput = computeThroughput(test, result, chunk.second, mState.getNKernelLaunches());
        std::cout << "   │     " << ((mState.testChunks.size() - iChunk != 1) ? "├ " : "└ ") << iChunk + 1 << "/" << mState.testChunks.size()
                  << ": [" << chunk.first << "-" << chunk.first + chunk.second << ") \e[1m" << throughput << " GB/s \e[0m(" << result * 1e-3 << " s)\n";
        mResultWriter.get()->storeBenchmarkEntry(test, iChunk, result, chunk.second, mState.getNKernelLaunches());
      }
    } else if (mode == Mode::Concurrent) {
      std::cout << "   │   - per chunk throughput:\n";
      auto results = runConcurrent(kernel,
                                   mState.testChunks,
                                   mState.getNKernelLaunches(),
                                   mState.getStreamsPoolSize(),
                                   nBlocks,
                                   nThreads);
      float sum{0};
      for (auto iChunk{0}; iChunk < mState.testChunks.size(); ++iChunk) {
        auto& chunk = mState.testChunks[iChunk];
        auto throughput = computeThroughput(test, results[iChunk], chunk.second, mState.getNKernelLaunches());
        sum += throughput;
        std::cout << "   │     " << ((mState.testChunks.size() - iChunk != 1) ? "├ " : "└ ") << iChunk + 1 << "/" << mState.testChunks.size()
                  << ": [" << chunk.first << "-" << chunk.first + chunk.second << ") \e[1m" << throughput << " GB/s \e[0m(" << results[iChunk] * 1e-3 << " s)\n";
        mResultWriter.get()->storeBenchmarkEntry(test, iChunk, results[iChunk], chunk.second, mState.getNKernelLaunches());
      }
      if (mState.testChunks.size() > 1) {
        std::cout << "   │   - total throughput: \e[1m" << sum << " GB/s \e[0m" << std::endl;
      }

      // Add throughput computed via system time measurement
      float tot{0};
      for (auto& chunk : mState.testChunks) {
        tot += chunk.second;
      }

      std::cout << "   │   - total throughput with host time: \e[1m" << computeThroughput(test, results[mState.testChunks.size()], tot, mState.getNKernelLaunches())
                << " GB/s \e[0m (" << std::setw(2) << results[mState.testChunks.size()] / 1000 << " s)" << std::endl;
    } else if (mode == Mode::Distributed) {
      auto result = runDistributed(kernel_distributed,
                                   mState.testChunks,
                                   mState.getNKernelLaunches(),
                                   nBlocks,
                                   nThreads);
      float tot{0};
      for (auto& chunk : mState.testChunks) {
        tot += chunk.second;
      }
      auto throughput = computeThroughput(test, result, tot, mState.getNKernelLaunches());
      std::cout << "   │     └ throughput: \e[1m" << throughput << " GB/s \e[0m(" << result * 1e-3 << " s)\n";
      mResultWriter.get()->storeBenchmarkEntry(test, 0, result, tot, mState.getNKernelLaunches());
    }
    mResultWriter.get()->snapshotBenchmark();
  }
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::finalizeTest(Test test)
{
  std::cout << "   └\033[1;32m done\033[0m" << std::endl;
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::globalFinalize()
{
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipFree(mState.scratchPtr));
}

template <class chunk_t>
void GPUbenchmark<chunk_t>::run()
{
  globalInit();

  for (auto& test : mOptions.tests) {
    initTest(test);
    for (auto& mode : mOptions.modes) {
      for (auto& config : mOptions.pools) {
        runTest(test, mode, config);
      }
    }
    finalizeTest(test);
  }

  globalFinalize();
}

template class GPUbenchmark<char>;
template class GPUbenchmark<size_t>;
template class GPUbenchmark<int>;
template class GPUbenchmark<int4>;

} // namespace benchmark
} // namespace o2