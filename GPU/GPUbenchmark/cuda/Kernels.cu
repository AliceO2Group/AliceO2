#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Kernels.{cu, hip.cxx}
/// \author: mconcas@cern.ch

#include "../Shared/Kernels.h"
#if defined(__HIPCC__)
#include "hip/hip_runtime.h"
#endif
#include <stdio.h>

// Memory partition legend
//
// |----------------------region 0-----------------|----------------------region 1-----------------| regions -> deafult: 2, to test lower and upper RAM
// |--chunk 0--|--chunk 1--|--chunk 2--|                  ***                          |--chunk n--| chunks  -> default size: 1GB (single block pins)
// |__________________________________________scratch______________________________________________| scratch -> default size: 95% free GPU RAM

#define GPUCHECK(error)                                                                        \
  if (error != hipSuccess) {                                                                  \
    printf("%serror: '%s'(%d) at %s:%d%s\n", KRED, hipGetErrorString(error), error, __FILE__, \
           __LINE__, KNRM);                                                                    \
    failed("API returned error code.");                                                        \
  }

double bytesToKB(size_t s) { return (double)s / (1024.0); }
double bytesToGB(size_t s) { return (double)s / GB; }

int getCorrespondingRegionId(int Id, int nChunks, int nRegions = 1)
{
  return Id * nRegions / nChunks;
}

template <class T>
std::string getType()
{
  if (typeid(T).name() == typeid(char).name()) {
    return std::string{"char"};
  }
  if (typeid(T).name() == typeid(size_t).name()) {
    return std::string{"unsigned long"};
  }
  if (typeid(T).name() == typeid(int).name()) {
    return std::string{"int"};
  }
  if (typeid(T).name() == typeid(int4).name()) {
    return std::string{"int4"};
  }
  return std::string{"unknown"};
}

namespace o2
{
namespace benchmark
{
namespace gpu
{

///////////////////////////
// Device functions go here
template <class chunk_type>
GPUhd() chunk_type* getPartPtrOnScratch(chunk_type* scratchPtr, float chunkReservedGB, size_t partNumber)
{
  return reinterpret_cast<chunk_type*>(reinterpret_cast<char*>(scratchPtr) + static_cast<size_t>(GB * chunkReservedGB) * partNumber);
}

//////////////////
// Kernels go here
template <class chunk_type>
GPUg() void readChunkSBKernel(
  int chunkId,
  chunk_type* results,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  if (chunkId == blockIdx.x) { // runs only if blockIdx.x is allowed in given split
    for (size_t i = threadIdx.x; i < chunkSize; i += blockDim.x) {
      if (getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i] == static_cast<chunk_type>(1)) { // actual read operation is performed here
        results[chunkId] += getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i];               // this case should never happen and waves should be always in sync
      }
    }
  }
}

template <class chunk_type>
GPUg() void readChunkMBKernel(
  int chunkId,
  chunk_type* results,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    if (getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i] == static_cast<chunk_type>(1)) { // actual read operation is performed here
      results[chunkId] += getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i];               // this case should never happen and waves should be always in sync
    }
  }
}
} // namespace gpu

void printDeviceProp(int deviceId)
{
  const int w1 = 34;
  std::cout << std::left;
  std::cout << std::setw(w1)
            << "--------------------------------------------------------------------------------"
            << std::endl;
  std::cout << std::setw(w1) << "device#" << deviceId << std::endl;

  hipDeviceProp_t props;
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));

  std::cout << std::setw(w1) << "Name: " << props.name << std::endl;
  std::cout << std::setw(w1) << "pciBusID: " << props.pciBusID << std::endl;
  std::cout << std::setw(w1) << "pciDeviceID: " << props.pciDeviceID << std::endl;
  std::cout << std::setw(w1) << "pciDomainID: " << props.pciDomainID << std::endl;
  std::cout << std::setw(w1) << "multiProcessorCount: " << props.multiProcessorCount << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << std::setw(w1) << "isMultiGpuBoard: " << props.isMultiGpuBoard << std::endl;
  std::cout << std::setw(w1) << "clockRate: " << (float)props.clockRate / 1000.0 << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "memoryClockRate: " << (float)props.memoryClockRate / 1000.0 << " Mhz"
            << std::endl;
  std::cout << std::setw(w1) << "memoryBusWidth: " << props.memoryBusWidth << std::endl;
  std::cout << std::setw(w1) << "clockInstructionRate: " << (float)props.clockRate / 1000.0
            << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "totalGlobalMem: " << std::fixed << std::setprecision(2)
            << bytesToGB(props.totalGlobalMem) << " GB" << std::endl;
#if !defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.sharedMemPerMultiprocessor) << " KB" << std::endl;
#endif
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.maxSharedMemoryPerMultiProcessor) << " KB" << std::endl;
#endif
  std::cout << std::setw(w1) << "totalConstMem: " << props.totalConstMem << std::endl;
  std::cout << std::setw(w1) << "sharedMemPerBlock: " << (float)props.sharedMemPerBlock / 1024.0 << " KB"
            << std::endl;
  std::cout << std::setw(w1) << "canMapHostMemory: " << props.canMapHostMemory << std::endl;
  std::cout << std::setw(w1) << "regsPerBlock: " << props.regsPerBlock << std::endl;
  std::cout << std::setw(w1) << "warpSize: " << props.warpSize << std::endl;
  std::cout << std::setw(w1) << "l2CacheSize: " << props.l2CacheSize << std::endl;
  std::cout << std::setw(w1) << "computeMode: " << props.computeMode << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.x: " << props.maxThreadsDim[0] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.y: " << props.maxThreadsDim[1] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.z: " << props.maxThreadsDim[2] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.x: " << props.maxGridSize[0] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.y: " << props.maxGridSize[1] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.z: " << props.maxGridSize[2] << std::endl;
  std::cout << std::setw(w1) << "major: " << props.major << std::endl;
  std::cout << std::setw(w1) << "minor: " << props.minor << std::endl;
  std::cout << std::setw(w1) << "concurrentKernels: " << props.concurrentKernels << std::endl;
  std::cout << std::setw(w1) << "cooperativeLaunch: " << props.cooperativeLaunch << std::endl;
  std::cout << std::setw(w1) << "cooperativeMultiDeviceLaunch: " << props.cooperativeMultiDeviceLaunch << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "arch.hasGlobalInt32Atomics: " << props.arch.hasGlobalInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalFloatAtomicExch: " << props.arch.hasGlobalFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt32Atomics: " << props.arch.hasSharedInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedFloatAtomicExch: " << props.arch.hasSharedFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasFloatAtomicAdd: " << props.arch.hasFloatAtomicAdd << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalInt64Atomics: " << props.arch.hasGlobalInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt64Atomics: " << props.arch.hasSharedInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasDoubles: " << props.arch.hasDoubles << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpVote: " << props.arch.hasWarpVote << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpBallot: " << props.arch.hasWarpBallot << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpShuffle: " << props.arch.hasWarpShuffle << std::endl;
  std::cout << std::setw(w1) << "arch.hasFunnelShift: " << props.arch.hasFunnelShift << std::endl;
  std::cout << std::setw(w1) << "arch.hasThreadFenceSystem: " << props.arch.hasThreadFenceSystem << std::endl;
  std::cout << std::setw(w1) << "arch.hasSyncThreadsExt: " << props.arch.hasSyncThreadsExt << std::endl;
  std::cout << std::setw(w1) << "arch.hasSurfaceFuncs: " << props.arch.hasSurfaceFuncs << std::endl;
  std::cout << std::setw(w1) << "arch.has3dGrid: " << props.arch.has3dGrid << std::endl;
  std::cout << std::setw(w1) << "arch.hasDynamicParallelism: " << props.arch.hasDynamicParallelism << std::endl;
  std::cout << std::setw(w1) << "gcnArchName: " << props.gcnArchName << std::endl;
#endif
  std::cout << std::setw(w1) << "isIntegrated: " << props.integrated << std::endl;
  std::cout << std::setw(w1) << "maxTexture1D: " << props.maxTexture1D << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.width: " << props.maxTexture2D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.height: " << props.maxTexture2D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.width: " << props.maxTexture3D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.height: " << props.maxTexture3D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.depth: " << props.maxTexture3D[2] << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "isLargeBar: " << props.isLargeBar << std::endl;
  std::cout << std::setw(w1) << "asicRevision: " << props.asicRevision << std::endl;
#endif

  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));
  std::cout << std::setw(w1) << "peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;
  std::cout << std::setw(w1) << "non-peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (!isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;

  size_t free, total;
  GPUCHECK(hipMemGetInfo(&free, &total));

  std::cout << std::fixed << std::setprecision(2);
  std::cout << std::setw(w1) << "memInfo.total: " << bytesToGB(total) << " GB" << std::endl;
  std::cout << std::setw(w1) << "memInfo.free:  " << bytesToGB(free) << " GB (" << std::setprecision(0)
            << (float)free / total * 100.0 << "%)" << std::endl;
}

template <class chunk_type>
template <typename... T>
float GPUbenchmark<chunk_type>::benchmarkSync(void (*kernel)(T...),
                                              int nLaunches, int blocks, int threads, T&... args) // run for each chunk (id is passed in variadic args)
{
  hipEvent_t start, stop;
  GPUCHECK(hipEventCreate(&start));
  GPUCHECK(hipEventCreate(&stop));

  GPUCHECK(hipEventRecord(start));
  for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) { // Schedule all the requested kernel launches
    (*kernel)<<<blocks, threads, 0, 0>>>(args...);
  }
  GPUCHECK(hipEventRecord(stop)); // record checkpoint

  GPUCHECK(hipEventSynchronize(stop)); // synchronize executions
  float milliseconds{0.f};
  GPUCHECK(hipEventElapsedTime(&milliseconds, start, stop));

  return milliseconds;
}

template <class chunk_type>
template <typename... T>
std::vector<float> GPUbenchmark<chunk_type>::benchmarkAsync(void (*kernel)(int, T...),
                                                            int nStreams, int nLaunches, int blocks, int threads, T&... args)
{
  std::vector<hipEvent_t> starts(nStreams), stops(nStreams);
  std::vector<hipStream_t> streams(nStreams);
  std::vector<float> results(nStreams);

  for (auto iStream{0}; iStream < nStreams; ++iStream) { // one stream per chunk
    GPUCHECK(hipStreamCreate(&(streams.at(iStream))));
    GPUCHECK(hipEventCreate(&(starts[iStream])));
    GPUCHECK(hipEventCreate(&(stops[iStream])));
  }

  for (auto iStream{0}; iStream < nStreams; ++iStream) {
    GPUCHECK(hipEventRecord(starts[iStream], streams[iStream]));

    for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) { // consecutive launches on the same stream
      (*kernel)<<<blocks, threads, 0, streams[iStream]>>>(iStream, args...);
    }
    GPUCHECK(hipEventRecord(stops[iStream], streams[iStream]));
  }

  for (auto iStream{0}; iStream < nStreams; ++iStream) {
    GPUCHECK(hipEventSynchronize(stops[iStream]));
    GPUCHECK(hipEventElapsedTime(&(results.at(iStream)), starts[iStream], stops[iStream]));
  }

  return results;
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::printDevices()
{
  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));

  for (int i = 0; i < deviceCnt; i++) {
    GPUCHECK(hipSetDevice(i));
    printDeviceProp(i);
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::globalInit(const int deviceId)
{
  hipDeviceProp_t props;
  size_t free;

  // Fetch and store features
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));
  GPUCHECK(hipMemGetInfo(&free, &mState.totalMemory));

  mState.chunkReservedGB = mOptions.chunkReservedGB;
  mState.iterations = mOptions.kernelLaunches;
  mState.nMultiprocessors = props.multiProcessorCount;
  mState.nMaxThreadsPerBlock = props.maxThreadsPerMultiProcessor;
  mState.nMaxThreadsPerDimension = props.maxThreadsDim[0];
  mState.scratchSize = static_cast<long int>(mOptions.freeMemoryFractionToAllocate * free);
  std::cout << ">>> Running on: \033[1;31m" << props.name << "\e[0m" << std::endl;

  // Allocate scratch on GPU
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&mState.scratchPtr), mState.scratchSize));

  mState.computeScratchPtrs();
  GPUCHECK(hipMemset(mState.scratchPtr, 0, mState.scratchSize))

  std::cout << "    ├ Buffer type: \e[1m" << getType<chunk_type>() << "\e[0m" << std::endl
            << "    ├ Allocated: " << std::setprecision(2) << bytesToGB(mState.scratchSize) << "/" << std::setprecision(2) << bytesToGB(mState.totalMemory)
            << "(GB) [" << std::setprecision(3) << (100.f) * (mState.scratchSize / (float)mState.totalMemory) << "%]\n"
            << "    ├ Number of scratch chunks: " << mState.getMaxChunks() << " of " << mOptions.chunkReservedGB << "GB each\n"
            << "    └ Each chunk can store up to: " << mState.getPartitionCapacity() << " elements" << std::endl
            << std::endl;
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::readingInit()
{
  std::cout << ">>> Initializing read benchmarks with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  mState.hostReadingResultsVector.resize(mState.getMaxChunks());
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&(mState.deviceReadingResultsPtr)), mState.getMaxChunks() * sizeof(chunk_type)));
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::readingSequential(SplitLevel sl)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << ">>> Sequential read, one block per chunk (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::readChunkSBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceReadingResultsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mStreamer.get()->storeBenchmarkEntry("readSequentialSplitBlocks", std::to_string(iChunk), getType<chunk_type>(), result);
        }
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }

    case SplitLevel::Threads: {
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << ">>> Sequential read, splitting on threads (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::readChunkMBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceReadingResultsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mStreamer.get()->storeBenchmarkEntry("readSequentialSplitThreads", std::to_string(iChunk), getType<chunk_type>(), result);
        }
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::readingConcurrent(SplitLevel sl, int nRegions)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << ">>> Concurrent read, one block per chunk (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::readChunkSBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceReadingResultsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          auto region = getCorrespondingRegionId(iResult, mState.getMaxChunks(), nRegions);
          mStreamer.get()->storeEntryForRegion("conc_R_SB", std::to_string(region), getType<chunk_type>(), results[iResult]);
        }
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
    case SplitLevel::Threads: {
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << ">>> Concurrent read, one block per chunk (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::readChunkMBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceReadingResultsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          auto region = getCorrespondingRegionId(iResult, mState.getMaxChunks(), nRegions);
          mStreamer.get()->storeEntryForRegion("conc_R_MB", std::to_string(region), getType<chunk_type>(), results[iResult]);
        }
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::readingFinalize()
{
  GPUCHECK(hipMemcpy(mState.hostReadingResultsVector.data(), mState.deviceReadingResultsPtr, mState.getMaxChunks() * sizeof(chunk_type), hipMemcpyDeviceToHost));
  GPUCHECK(hipFree(mState.deviceReadingResultsPtr));
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::globalFinalize()
{
  GPUCHECK(hipFree(mState.scratchPtr));
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::run()
{
  globalInit(0);
  // Test calls go here:
  readingInit();
  // - Reading whole memory
  // readingSequential(SplitLevel::Threads);
  // readingSequential(SplitLevel::Blocks);

  // - Reading memory partitions
  readingConcurrent(SplitLevel::Blocks);
  readingFinalize();

  GPUbenchmark<chunk_type>::globalFinalize();
}

template class GPUbenchmark<char>;
// template class GPUbenchmark<uint4>;
template class GPUbenchmark<size_t>;
template class GPUbenchmark<int>;

} // namespace benchmark
} // namespace o2