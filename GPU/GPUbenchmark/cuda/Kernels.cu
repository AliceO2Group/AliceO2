#include "hip/hip_runtime.h"
// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Kernels.cu
/// \author: mconcas@cern.ch

#include <Kernels.h>
#include <Common.h>
#include <stdio.h>

#define GPUCHECK(error)                                                                        \
  if (error != hipSuccess) {                                                                  \
    printf("%serror: '%s'(%d) at %s:%d%s\n", KRED, hipGetErrorString(error), error, __FILE__, \
           __LINE__, KNRM);                                                                    \
    failed("API returned error code.");                                                        \
  }

#define CHECK(cmd)                                                                                         \
  {                                                                                                        \
    hipError_t error = cmd;                                                                               \
    if (error != hipSuccess) {                                                                            \
      fprintf(stderr, "error: '%s'(%d) at %s:%d\n", hipGetErrorString(error), error, __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                                  \
    }                                                                                                      \
  }

namespace o2
{
namespace benchmark
{
namespace gpu
{
// Kernels go here
/* 
 * Square each element in the array A and write to array C.
 */
template <typename T>
__global__ void
  vector_square(T* C_d, T* A_d, size_t N)
{
  size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
  size_t stride = blockDim.x * gridDim.x;

  for (size_t i = offset; i < N; i += stride) {
    C_d[i] = A_d[i] * A_d[i];
  }
}

// template <class buffer_type>
// GPUg() void readerKernel(
//   // buffer_type* buffer,
//   // size_t bufferSize)
// )
// {
//   printf("ciao");
//   // for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < bufferSize; i += blockDim.x * gridDim.x) {
//   //   if (i == 0) {
//   //       }
//   // }
// }
} // namespace gpu

void printDeviceProp(int deviceId)
{
  const int w1 = 34;
  std::cout << std::left;
  std::cout << std::setw(w1)
            << "--------------------------------------------------------------------------------"
            << std::endl;
  std::cout << std::setw(w1) << "device#" << deviceId << std::endl;

  hipDeviceProp_t props;
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));

  std::cout << std::setw(w1) << "Name: " << props.name << std::endl;
  std::cout << std::setw(w1) << "pciBusID: " << props.pciBusID << std::endl;
  std::cout << std::setw(w1) << "pciDeviceID: " << props.pciDeviceID << std::endl;
  std::cout << std::setw(w1) << "pciDomainID: " << props.pciDomainID << std::endl;
  std::cout << std::setw(w1) << "multiProcessorCount: " << props.multiProcessorCount << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << std::setw(w1) << "isMultiGpuBoard: " << props.isMultiGpuBoard << std::endl;
  std::cout << std::setw(w1) << "clockRate: " << (float)props.clockRate / 1000.0 << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "memoryClockRate: " << (float)props.memoryClockRate / 1000.0 << " Mhz"
            << std::endl;
  std::cout << std::setw(w1) << "memoryBusWidth: " << props.memoryBusWidth << std::endl;
  std::cout << std::setw(w1) << "clockInstructionRate: " << (float)props.clockRate / 1000.0
            << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "totalGlobalMem: " << std::fixed << std::setprecision(2)
            << bytesToGB(props.totalGlobalMem) << " GB" << std::endl;
#if !defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.sharedMemPerMultiprocessor) << " KB" << std::endl;
#endif
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.maxSharedMemoryPerMultiProcessor) << " KB" << std::endl;
#endif
  std::cout << std::setw(w1) << "totalConstMem: " << props.totalConstMem << std::endl;
  std::cout << std::setw(w1) << "sharedMemPerBlock: " << (float)props.sharedMemPerBlock / 1024.0 << " KB"
            << std::endl;
  std::cout << std::setw(w1) << "canMapHostMemory: " << props.canMapHostMemory << std::endl;
  std::cout << std::setw(w1) << "regsPerBlock: " << props.regsPerBlock << std::endl;
  std::cout << std::setw(w1) << "warpSize: " << props.warpSize << std::endl;
  std::cout << std::setw(w1) << "l2CacheSize: " << props.l2CacheSize << std::endl;
  std::cout << std::setw(w1) << "computeMode: " << props.computeMode << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.x: " << props.maxThreadsDim[0] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.y: " << props.maxThreadsDim[1] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.z: " << props.maxThreadsDim[2] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.x: " << props.maxGridSize[0] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.y: " << props.maxGridSize[1] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.z: " << props.maxGridSize[2] << std::endl;
  std::cout << std::setw(w1) << "major: " << props.major << std::endl;
  std::cout << std::setw(w1) << "minor: " << props.minor << std::endl;
  std::cout << std::setw(w1) << "concurrentKernels: " << props.concurrentKernels << std::endl;
  std::cout << std::setw(w1) << "cooperativeLaunch: " << props.cooperativeLaunch << std::endl;
  std::cout << std::setw(w1) << "cooperativeMultiDeviceLaunch: " << props.cooperativeMultiDeviceLaunch << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "arch.hasGlobalInt32Atomics: " << props.arch.hasGlobalInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalFloatAtomicExch: " << props.arch.hasGlobalFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt32Atomics: " << props.arch.hasSharedInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedFloatAtomicExch: " << props.arch.hasSharedFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasFloatAtomicAdd: " << props.arch.hasFloatAtomicAdd << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalInt64Atomics: " << props.arch.hasGlobalInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt64Atomics: " << props.arch.hasSharedInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasDoubles: " << props.arch.hasDoubles << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpVote: " << props.arch.hasWarpVote << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpBallot: " << props.arch.hasWarpBallot << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpShuffle: " << props.arch.hasWarpShuffle << std::endl;
  std::cout << std::setw(w1) << "arch.hasFunnelShift: " << props.arch.hasFunnelShift << std::endl;
  std::cout << std::setw(w1) << "arch.hasThreadFenceSystem: " << props.arch.hasThreadFenceSystem << std::endl;
  std::cout << std::setw(w1) << "arch.hasSyncThreadsExt: " << props.arch.hasSyncThreadsExt << std::endl;
  std::cout << std::setw(w1) << "arch.hasSurfaceFuncs: " << props.arch.hasSurfaceFuncs << std::endl;
  std::cout << std::setw(w1) << "arch.has3dGrid: " << props.arch.has3dGrid << std::endl;
  std::cout << std::setw(w1) << "arch.hasDynamicParallelism: " << props.arch.hasDynamicParallelism << std::endl;
  std::cout << std::setw(w1) << "gcnArchName: " << props.gcnArchName << std::endl;
#endif
  std::cout << std::setw(w1) << "isIntegrated: " << props.integrated << std::endl;
  std::cout << std::setw(w1) << "maxTexture1D: " << props.maxTexture1D << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.width: " << props.maxTexture2D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.height: " << props.maxTexture2D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.width: " << props.maxTexture3D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.height: " << props.maxTexture3D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.depth: " << props.maxTexture3D[2] << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "isLargeBar: " << props.isLargeBar << std::endl;
  std::cout << std::setw(w1) << "asicRevision: " << props.asicRevision << std::endl;
#endif

  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));
  std::cout << std::setw(w1) << "peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;
  std::cout << std::setw(w1) << "non-peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (!isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;

  size_t free, total;
  GPUCHECK(hipMemGetInfo(&free, &total));

  std::cout << std::fixed << std::setprecision(2);
  std::cout << std::setw(w1) << "memInfo.total: " << bytesToGB(total) << " GB" << std::endl;
  std::cout << std::setw(w1) << "memInfo.free:  " << bytesToGB(free) << " GB (" << std::setprecision(0)
            << (float)free / total * 100.0 << "%)" << std::endl;
}

template <class buffer_type>
template <typename... T>
float GPUbenchmark<buffer_type>::measure(void (GPUbenchmark<buffer_type>::*task)(T...), const char* taskName, T&&... args)
{
  float diff{0.f};
  auto start = std::chrono::high_resolution_clock::now();
  (this->*task)(std::forward<T>(args)...);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> diff_t{end - start};
  diff = diff_t.count();
  std::cout << std::setw(2) << ">>> " << taskName << " completed in: " << diff << " ms" << std::endl;
  return diff;
}

template <class buffer_type>
void GPUbenchmark<buffer_type>::printDevices()
{
  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));

  for (int i = 0; i < deviceCnt; i++) {
    GPUCHECK(hipSetDevice(i));
    printDeviceProp(i);
  }
}

template <class buffer_type>
void GPUbenchmark<buffer_type>::init(const int deviceId)
{
  hipDeviceProp_t props;
  size_t free;

  // Fetch and store traits
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));
  GPUCHECK(hipMemGetInfo(&free, &mState.totalMemory));

  mState.nMultiprocessors = props.multiProcessorCount;
  mState.nMaxThreadsPerBlock = props.maxThreadsPerMultiProcessor;
  mState.allocatedMemory = static_cast<long int>(FREE_MEMORY_FRACTION_TO_ALLOCATE * free);

  // Setup
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&mState.scratchPtr), mState.allocatedMemory));
}

template <class buffer_type>
void GPUbenchmark<buffer_type>::readingBenchmark()
{
  // dim3 nBlocks(mState.nMultiprocessors);
  // dim3 nThreads(mState.nMaxThreadsPerBlock);
  // gpu::readerKernel<buffer_type><<<1, 1>>>();
  float *A_d, *C_d;
  float *A_h, *C_h;
  size_t N = 1000000;
  size_t Nbytes = N * sizeof(float);

  hipDeviceProp_t props;
  CHECK(hipGetDeviceProperties(&props, 0 /*deviceID*/));
  printf("info: running on device %s\n", props.name);

  printf("info: allocate host mem (%6.2f MB)\n", 2 * Nbytes / 1024.0 / 1024.0);
  A_h = (float*)malloc(Nbytes);
  CHECK(A_h == 0 ? hipErrorOutOfMemory : hipSuccess);
  C_h = (float*)malloc(Nbytes);
  CHECK(C_h == 0 ? hipErrorOutOfMemory : hipSuccess);
  // Fill with Phi + i
  for (size_t i = 0; i < N; i++) {
    A_h[i] = 1.618f + i;
  }

  printf("info: allocate device mem (%6.2f MB)\n", 2 * Nbytes / 1024.0 / 1024.0);
  CHECK(hipMalloc(&A_d, Nbytes));
  CHECK(hipMalloc(&C_d, Nbytes));

  printf("info: copy Host2Device\n");
  CHECK(hipMemcpy(A_d, A_h, Nbytes, hipMemcpyHostToDevice));

  const unsigned blocks = 512;
  const unsigned threadsPerBlock = 256;

  printf("info: launch 'vector_square' kernel\n");
  gpu::vector_square<<<blocks, threadsPerBlock>>>(C_d, A_d, N);

  printf("info: copy Device2Host\n");
  CHECK(hipMemcpy(C_h, C_d, Nbytes, hipMemcpyDeviceToHost));

  printf("info: check result\n");
  for (size_t i = 0; i < N; i++) {
    if (C_h[i] != A_h[i] * A_h[i]) {
      CHECK(hipErrorUnknown);
    }
  }
  printf("PASSED!\n");
}

template <class buffer_type>
void GPUbenchmark<buffer_type>::finalize()
{
  GPUCHECK(hipFree(mState.scratchPtr));
}

template <class buffer_type>
void GPUbenchmark<buffer_type>::run()
{
  // printDevices();
  // measure(&GPUbenchmark<buffer_type>::init, "Init", 0);
  // std::cout << "  ├ Allocated " << mState.allocatedMemory << "/" << mState.totalMemory
  //           << " bytes (" << std::setprecision(3) << (100.f) * (mState.allocatedMemory / (float)mState.totalMemory) << "%)\n";
  // std::cout << "  └ Can do " << mState.getMaxSegments() << " of 1GB memory segments\n";
  // mState.computeBufferPointers();

  // for (auto& addr : mState.getBuffersPointers()) {
  //   std::cout << (void*)addr << std::endl;
  // }
  measure(&GPUbenchmark<buffer_type>::readingBenchmark, "Reading benchmark");
  // GPUbenchmark<buffer_type>::finalize();
}

template class GPUbenchmark<char>;

} // namespace benchmark
} // namespace o2