#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Kernels.{cu, hip.cxx}
/// \author: mconcas@cern.ch

#include "../Shared/Kernels.h"
#if defined(__HIPCC__)
#include "hip/hip_runtime.h"
#endif
#include <cstdio>

// Memory partitioning legend
//
// |----------------------region 0-----------------|----------------------region 1-----------------| regions -> deafult: 2, to test lower and upper RAM
// |--chunk 0--|--chunk 1--|--chunk 2--|                  ***                          |--chunk n--| chunks  -> default size: 1GB (sing block pins)
// |__________________________________________scratch______________________________________________| scratch -> default size: 95% free GPU RAM

#define GPUCHECK(error)                                                                        \
  if (error != hipSuccess) {                                                                  \
    printf("%serror: '%s'(%d) at %s:%d%s\n", KRED, hipGetErrorString(error), error, __FILE__, \
           __LINE__, KNRM);                                                                    \
    failed("API returned error code.");                                                        \
  }

double bytesToKB(size_t s) { return (double)s / (1024.0); }
double bytesToGB(size_t s) { return (double)s / GB; }

int getCorrespondingRegionId(int Id, int nChunks, int nRegions = 1)
{
  return Id * nRegions / nChunks;
}

template <class T>
std::string getType()
{
  if (typeid(T).name() == typeid(char).name()) {
    return std::string{"char"};
  }
  if (typeid(T).name() == typeid(size_t).name()) {
    return std::string{"unsigned_long"};
  }
  if (typeid(T).name() == typeid(int).name()) {
    return std::string{"int"};
  }
  if (typeid(T).name() == typeid(int4).name()) {
    return std::string{"int4"};
  }
  return std::string{"unknown"};
}

namespace o2
{
namespace benchmark
{
namespace gpu
{

///////////////////////////
// Device functions go here
template <class chunk_type>
__host__ __device__ inline chunk_type* getPartPtrOnScratch(chunk_type* scratchPtr, float chunkReservedGB, size_t partNumber)
{
  return reinterpret_cast<chunk_type*>(reinterpret_cast<char*>(scratchPtr) + static_cast<size_t>(GB * chunkReservedGB) * partNumber);
}

//////////////////
// Kernels go here
// Reading
template <class chunk_type>
__global__ void readChunkSBKernel(
  int chunkId,
  chunk_type* results,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  if (chunkId == blockIdx.x) { // runs only if blockIdx.x is allowed in given split
    chunk_type sink{0};
    chunk_type* ptr = getPartPtrOnScratch(scratch, chunkReservedGB, chunkId);
    for (size_t i = threadIdx.x; i < chunkSize; i += blockDim.x) {
      sink += ptr[i];
    }
    if (sink == static_cast<chunk_type>(1)) {
      results[chunkId] = sink;
    }
  }
}

template <class chunk_type>
__global__ void readChunkMBKernel(
  int chunkId,
  chunk_type* results,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    if (getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i] == static_cast<chunk_type>(1)) { // actual read operation is performed here
      results[chunkId] += getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i];               // this case should never happen and waves should be always in sync
    }
  }
}

// Writing
template <class chunk_type>
__global__ void writeChunkSBKernel(
  int chunkId,
  chunk_type* results,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  if (chunkId == blockIdx.x) { // runs only if blockIdx.x is allowed in given split
    for (size_t i = threadIdx.x; i < chunkSize; i += blockDim.x) {
      getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i] = 1;
    }
  }
}

template <class chunk_type>
__global__ void writeChunkMBKernel(
  int chunkId,
  chunk_type* results,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i] = 1;
  }
}

// Copying
template <class chunk_type>
__global__ void copyChunkSBKernel(
  int chunkId,
  chunk_type* inputs,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  if (chunkId == blockIdx.x) { // runs only if blockIdx.x is allowed in given split
    for (size_t i = threadIdx.x; i < chunkSize; i += blockDim.x) {
      getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i] = inputs[chunkId];
    }
  }
}

template <class chunk_type>
__global__ void copyChunkMBKernel(
  int chunkId,
  chunk_type* inputs,
  chunk_type* scratch,
  size_t chunkSize,
  float chunkReservedGB = 1.f)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < chunkSize; i += blockDim.x * gridDim.x) {
    getPartPtrOnScratch(scratch, chunkReservedGB, chunkId)[i] = inputs[chunkId];
  }
}

} // namespace gpu

void printDeviceProp(int deviceId)
{
  const int w1 = 34;
  std::cout << std::left;
  std::cout << std::setw(w1)
            << "--------------------------------------------------------------------------------"
            << std::endl;
  std::cout << std::setw(w1) << "device#" << deviceId << std::endl;

  hipDeviceProp_t props;
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));

  std::cout << std::setw(w1) << "Name: " << props.name << std::endl;
  std::cout << std::setw(w1) << "pciBusID: " << props.pciBusID << std::endl;
  std::cout << std::setw(w1) << "pciDeviceID: " << props.pciDeviceID << std::endl;
  std::cout << std::setw(w1) << "pciDomainID: " << props.pciDomainID << std::endl;
  std::cout << std::setw(w1) << "multiProcessorCount: " << props.multiProcessorCount << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << std::setw(w1) << "isMultiGpuBoard: " << props.isMultiGpuBoard << std::endl;
  std::cout << std::setw(w1) << "clockRate: " << (float)props.clockRate / 1000.0 << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "memoryClockRate: " << (float)props.memoryClockRate / 1000.0 << " Mhz"
            << std::endl;
  std::cout << std::setw(w1) << "memoryBusWidth: " << props.memoryBusWidth << std::endl;
  std::cout << std::setw(w1) << "clockInstructionRate: " << (float)props.clockRate / 1000.0
            << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "totalGlobalMem: " << std::fixed << std::setprecision(2)
            << bytesToGB(props.totalGlobalMem) << " GB" << std::endl;
#if !defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.sharedMemPerMultiprocessor) << " KB" << std::endl;
#endif
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.maxSharedMemoryPerMultiProcessor) << " KB" << std::endl;
#endif
  std::cout << std::setw(w1) << "totalConstMem: " << props.totalConstMem << std::endl;
  std::cout << std::setw(w1) << "sharedMemPerBlock: " << (float)props.sharedMemPerBlock / 1024.0 << " KB"
            << std::endl;
  std::cout << std::setw(w1) << "canMapHostMemory: " << props.canMapHostMemory << std::endl;
  std::cout << std::setw(w1) << "regsPerBlock: " << props.regsPerBlock << std::endl;
  std::cout << std::setw(w1) << "warpSize: " << props.warpSize << std::endl;
  std::cout << std::setw(w1) << "l2CacheSize: " << props.l2CacheSize << std::endl;
  std::cout << std::setw(w1) << "computeMode: " << props.computeMode << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.x: " << props.maxThreadsDim[0] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.y: " << props.maxThreadsDim[1] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.z: " << props.maxThreadsDim[2] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.x: " << props.maxGridSize[0] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.y: " << props.maxGridSize[1] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.z: " << props.maxGridSize[2] << std::endl;
  std::cout << std::setw(w1) << "major: " << props.major << std::endl;
  std::cout << std::setw(w1) << "minor: " << props.minor << std::endl;
  std::cout << std::setw(w1) << "concurrentKernels: " << props.concurrentKernels << std::endl;
  std::cout << std::setw(w1) << "cooperativeLaunch: " << props.cooperativeLaunch << std::endl;
  std::cout << std::setw(w1) << "cooperativeMultiDeviceLaunch: " << props.cooperativeMultiDeviceLaunch << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "arch.hasGlobalInt32Atomics: " << props.arch.hasGlobalInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalFloatAtomicExch: " << props.arch.hasGlobalFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt32Atomics: " << props.arch.hasSharedInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedFloatAtomicExch: " << props.arch.hasSharedFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasFloatAtomicAdd: " << props.arch.hasFloatAtomicAdd << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalInt64Atomics: " << props.arch.hasGlobalInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt64Atomics: " << props.arch.hasSharedInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasDoubles: " << props.arch.hasDoubles << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpVote: " << props.arch.hasWarpVote << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpBallot: " << props.arch.hasWarpBallot << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpShuffle: " << props.arch.hasWarpShuffle << std::endl;
  std::cout << std::setw(w1) << "arch.hasFunnelShift: " << props.arch.hasFunnelShift << std::endl;
  std::cout << std::setw(w1) << "arch.hasThreadFenceSystem: " << props.arch.hasThreadFenceSystem << std::endl;
  std::cout << std::setw(w1) << "arch.hasSyncThreadsExt: " << props.arch.hasSyncThreadsExt << std::endl;
  std::cout << std::setw(w1) << "arch.hasSurfaceFuncs: " << props.arch.hasSurfaceFuncs << std::endl;
  std::cout << std::setw(w1) << "arch.has3dGrid: " << props.arch.has3dGrid << std::endl;
  std::cout << std::setw(w1) << "arch.hasDynamicParallelism: " << props.arch.hasDynamicParallelism << std::endl;
  std::cout << std::setw(w1) << "gcnArchName: " << props.gcnArchName << std::endl;
#endif
  std::cout << std::setw(w1) << "isIntegrated: " << props.integrated << std::endl;
  std::cout << std::setw(w1) << "maxTexture1D: " << props.maxTexture1D << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.width: " << props.maxTexture2D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.height: " << props.maxTexture2D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.width: " << props.maxTexture3D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.height: " << props.maxTexture3D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.depth: " << props.maxTexture3D[2] << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "isLargeBar: " << props.isLargeBar << std::endl;
  std::cout << std::setw(w1) << "asicRevision: " << props.asicRevision << std::endl;
#endif

  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));
  std::cout << std::setw(w1) << "peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;
  std::cout << std::setw(w1) << "non-peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (!isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;

  size_t free, total;
  GPUCHECK(hipMemGetInfo(&free, &total));

  std::cout << std::fixed << std::setprecision(2);
  std::cout << std::setw(w1) << "memInfo.total: " << bytesToGB(total) << " GB" << std::endl;
  std::cout << std::setw(w1) << "memInfo.free:  " << bytesToGB(free) << " GB (" << std::setprecision(0)
            << (float)free / total * 100.0 << "%)" << std::endl;
}

template <class chunk_type>
template <typename... T>
float GPUbenchmark<chunk_type>::benchmarkSync(void (*kernel)(T...),
                                              int nLaunches, int blocks, int threads, T&... args) // run for each chunk (id is passed in variadic args)
{
  hipEvent_t start, stop;
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipEventCreate(&start));
  GPUCHECK(hipEventCreate(&stop));

  GPUCHECK(hipEventRecord(start));
  for (auto iLaunch{0}; iLaunch < nLaunches; ++iLaunch) { // Schedule all the requested kernel launches
    (*kernel)<<<blocks, threads, 0, 0>>>(args...);
  }
  GPUCHECK(hipEventRecord(stop)); // record checkpoint

  GPUCHECK(hipEventSynchronize(stop)); // synchronize executions
  float milliseconds{0.f};
  GPUCHECK(hipEventElapsedTime(&milliseconds, start, stop));
  GPUCHECK(hipEventDestroy(start));
  GPUCHECK(hipEventDestroy(stop));

  return milliseconds;
}

template <class chunk_type>
template <typename... T>
std::vector<float> GPUbenchmark<chunk_type>::benchmarkAsync(void (*kernel)(int, T...),
                                                            int nStreams, int nLaunches, int blocks, int threads, T&... args)
{
  std::vector<hipEvent_t> starts(nStreams), stops(nStreams);
  std::vector<hipStream_t> streams(nStreams);
  std::vector<float> results(nStreams);
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  for (auto iStream{0}; iStream < nStreams; ++iStream) { // one stream per chunk
    GPUCHECK(hipStreamCreate(&(streams.at(iStream))));
    GPUCHECK(hipEventCreate(&(starts[iStream])));
    GPUCHECK(hipEventCreate(&(stops[iStream])));
  }

  for (auto iStream{0}; iStream < nStreams; ++iStream) {
    GPUCHECK(hipEventRecord(starts[iStream], streams[iStream]));

    for (auto iLaunch{0}; iLaunch < 10 * nLaunches; ++iLaunch) { // 10x consecutive launches on the same stream
      (*kernel)<<<blocks, threads, 0, streams[iStream]>>>(iStream, args...);
    }
    GPUCHECK(hipEventRecord(stops[iStream], streams[iStream]));
  }

  for (auto iStream{0}; iStream < nStreams; ++iStream) {
    GPUCHECK(hipEventSynchronize(stops[iStream]));
    GPUCHECK(hipEventElapsedTime(&(results.at(iStream)), starts[iStream], stops[iStream]));
    GPUCHECK(hipEventDestroy(starts[iStream]));
    GPUCHECK(hipEventDestroy(stops[iStream]));
    GPUCHECK(hipStreamDestroy(streams[iStream]));
  }

  return results;
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::printDevices()
{
  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));

  for (int i = 0; i < deviceCnt; i++) {
    GPUCHECK(hipSetDevice(i));
    printDeviceProp(i);
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::globalInit()
{
  hipDeviceProp_t props;
  size_t free;

  // Fetch and store features
  GPUCHECK(hipGetDeviceProperties(&props, mOptions.deviceId));
  GPUCHECK(hipMemGetInfo(&free, &mState.totalMemory));
  GPUCHECK(hipSetDevice(mOptions.deviceId));

  mState.chunkReservedGB = mOptions.chunkReservedGB;
  mState.iterations = mOptions.kernelLaunches;
  mState.nMultiprocessors = props.multiProcessorCount;
  mState.nMaxThreadsPerBlock = props.maxThreadsPerMultiProcessor;
  mState.nMaxThreadsPerDimension = props.maxThreadsDim[0];
  mState.scratchSize = static_cast<long int>(mOptions.freeMemoryFractionToAllocate * free);
  std::cout << ">>> Running on: \033[1;31m" << props.name << "\e[0m" << std::endl;

  // Allocate scratch on GPU
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&mState.scratchPtr), mState.scratchSize));

  mState.computeScratchPtrs();
  GPUCHECK(hipMemset(mState.scratchPtr, 0, mState.scratchSize))

  std::cout << "    ├ Buffer type: \e[1m" << getType<chunk_type>() << "\e[0m" << std::endl
            << "    ├ Allocated: " << std::setprecision(2) << bytesToGB(mState.scratchSize) << "/" << std::setprecision(2) << bytesToGB(mState.totalMemory)
            << "(GB) [" << std::setprecision(3) << (100.f) * (mState.scratchSize / (float)mState.totalMemory) << "%]\n"
            << "    ├ Number of scratch chunks: " << mState.getMaxChunks() << " of " << mOptions.chunkReservedGB << "GB each\n"
            << "    └ Each chunk can store up to: " << mState.getPartitionCapacity() << " elements" << std::endl
            << std::endl;
}

/// Read
template <class chunk_type>
void GPUbenchmark<chunk_type>::readInit()
{
  std::cout << ">>> Initializing read benchmarks with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  mState.hostReadResultsVector.resize(mState.getMaxChunks());
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&(mState.deviceReadResultsPtr)), mState.getMaxChunks() * sizeof(chunk_type)));
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::readSequential(SplitLevel sl)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("seq_read_SB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << "    ├ (" << getType<chunk_type>() << ") Seq read, sing block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::readChunkSBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceReadResultsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mResultWriter.get()->storeBenchmarkEntry(iChunk, result);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }

    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("seq_read_MB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << "    ├ (" << getType<chunk_type>() << ") Seq read, mult block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::readChunkMBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceReadResultsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mResultWriter.get()->storeBenchmarkEntry(iChunk, result);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::readConcurrent(SplitLevel sl, int nRegions)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("conc_read_SB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ (" << getType<chunk_type>() << ") Conc read, sing block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::readChunkSBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceReadResultsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(iResult, results[iResult]);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("conc_read_MB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ (" << getType<chunk_type>() << ") Conc read, mult block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::readChunkMBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceReadResultsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(iResult, results[iResult]);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::readFinalize()
{
  GPUCHECK(hipMemcpy(mState.hostReadResultsVector.data(), mState.deviceReadResultsPtr, mState.getMaxChunks() * sizeof(chunk_type), hipMemcpyDeviceToHost));
  GPUCHECK(hipFree(mState.deviceReadResultsPtr));
  std::cout << "    └ done." << std::endl;
}

/// Write
template <class chunk_type>
void GPUbenchmark<chunk_type>::writeInit()
{
  std::cout << ">>> Initializing write benchmarks with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  mState.hostWriteResultsVector.resize(mState.getMaxChunks());
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&(mState.deviceWriteResultsPtr)), mState.getMaxChunks() * sizeof(chunk_type)));
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::writeSequential(SplitLevel sl)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("seq_write_SB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << "    ├ (" << getType<chunk_type>() << ") Seq write, sing block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::writeChunkSBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceWriteResultsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mResultWriter.get()->storeBenchmarkEntry(iChunk, result);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }

    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("seq_write_MB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << "    ├ (" << getType<chunk_type>() << ") Seq write, mult block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::writeChunkMBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceWriteResultsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mResultWriter.get()->storeBenchmarkEntry(iChunk, result);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::writeConcurrent(SplitLevel sl, int nRegions)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("conc_write_SB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ (" << getType<chunk_type>() << ") Conc write, sing block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::writeChunkSBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceWriteResultsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(iResult, results[iResult]);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("conc_write_MB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ (" << getType<chunk_type>() << ") Conc write, mult block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::writeChunkMBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceWriteResultsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(iResult, results[iResult]);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::writeFinalize()
{
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMemcpy(mState.hostWriteResultsVector.data(), mState.deviceWriteResultsPtr, mState.getMaxChunks() * sizeof(chunk_type), hipMemcpyDeviceToHost));
  GPUCHECK(hipFree(mState.deviceWriteResultsPtr));
  std::cout << "    └ done." << std::endl;
}

/// Copy
template <class chunk_type>
void GPUbenchmark<chunk_type>::copyInit()
{
  std::cout << ">>> Initializing copy benchmarks with \e[1m" << mOptions.nTests << "\e[0m runs and \e[1m" << mOptions.kernelLaunches << "\e[0m kernel launches" << std::endl;
  mState.hostCopyInputsVector.resize(mState.getMaxChunks());
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMalloc(reinterpret_cast<void**>(&(mState.deviceCopyInputsPtr)), mState.getMaxChunks() * sizeof(chunk_type)));
  GPUCHECK(hipMemset(mState.deviceCopyInputsPtr, 1, mState.getMaxChunks() * sizeof(chunk_type)));
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::copySequential(SplitLevel sl)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("seq_copy_SB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << "    ├ (" << getType<chunk_type>() << ") Seq copy, sing block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::copyChunkSBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceCopyInputsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mResultWriter.get()->storeBenchmarkEntry(iChunk, result);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }

    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("seq_copy_MB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) { // loop on the number of times we perform same measurement
        std::cout << std::setw(2) << "    ├ (" << getType<chunk_type>() << ") Seq copy, mult block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        for (auto iChunk{0}; iChunk < mState.getMaxChunks(); ++iChunk) { // loop over single chunks separately
          auto result = benchmarkSync(&gpu::copyChunkMBKernel<chunk_type>,
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      iChunk,
                                      mState.deviceCopyInputsPtr,
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
          mResultWriter.get()->storeBenchmarkEntry(iChunk, result);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::copyConcurrent(SplitLevel sl, int nRegions)
{
  switch (sl) {
    case SplitLevel::Blocks: {
      mResultWriter.get()->addBenchmarkEntry("conc_copy_SB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ (" << getType<chunk_type>() << ") Conc copy, sing block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::copyChunkSBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceCopyInputsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          mResultWriter.get()->storeBenchmarkEntry(iResult, results[iResult]);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
    case SplitLevel::Threads: {
      mResultWriter.get()->addBenchmarkEntry("conc_copy_MB", getType<chunk_type>(), mState.getMaxChunks());
      auto nBlocks{mState.nMultiprocessors};
      auto nThreads{std::min(mState.nMaxThreadsPerDimension, mState.nMaxThreadsPerBlock)};
      auto chunks{mState.getMaxChunks()};
      auto capacity{mState.getPartitionCapacity()};

      for (auto measurement{0}; measurement < mOptions.nTests; ++measurement) {
        std::cout << "    ├ (" << getType<chunk_type>() << ") Conc copy, mult block (" << measurement + 1 << "/" << mOptions.nTests << "):";
        auto results = benchmarkAsync(&gpu::copyChunkMBKernel<chunk_type>,
                                      mState.getMaxChunks(), // nStreams
                                      mState.getNKernelLaunches(),
                                      nBlocks,
                                      nThreads,
                                      mState.deviceCopyInputsPtr, // kernel arguments (chunkId is passed by wrapper)
                                      mState.scratchPtr,
                                      capacity,
                                      mState.chunkReservedGB);
        for (auto iResult{0}; iResult < results.size(); ++iResult) {
          auto region = getCorrespondingRegionId(iResult, nBlocks, nRegions);
          mResultWriter.get()->storeBenchmarkEntry(iResult, results[iResult]);
        }
        mResultWriter.get()->snapshotBenchmark();
        std::cout << "\033[1;32m complete\033[0m" << std::endl;
      }
      break;
    }
  }
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::copyFinalize()
{
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipMemcpy(mState.hostCopyInputsVector.data(), mState.deviceCopyInputsPtr, mState.getMaxChunks() * sizeof(chunk_type), hipMemcpyDeviceToHost));
  GPUCHECK(hipFree(mState.deviceCopyInputsPtr));
  std::cout << "    └ done." << std::endl;
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::globalFinalize()
{
  GPUCHECK(hipSetDevice(mOptions.deviceId));
  GPUCHECK(hipFree(mState.scratchPtr));
}

template <class chunk_type>
void GPUbenchmark<chunk_type>::run()
{
  globalInit();

  for (auto& sl : mOptions.pools) {
    for (auto& test : mOptions.tests) {
      switch (test) {
        case Test::Read: {
          readInit();

          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Sequential) != mOptions.modes.end()) {
            readSequential(sl);
          }
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Concurrent) != mOptions.modes.end()) {
            readConcurrent(sl);
          }

          readFinalize();

          break;
        }
        case Test::Write: {
          writeInit();
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Sequential) != mOptions.modes.end()) {
            writeSequential(sl);
          }
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Concurrent) != mOptions.modes.end()) {
            writeConcurrent(sl);
          }

          writeFinalize();

          break;
        }
        case Test::Copy: {
          copyInit();
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Sequential) != mOptions.modes.end()) {
            copySequential(sl);
          }
          if (std::find(mOptions.modes.begin(), mOptions.modes.end(), Mode::Concurrent) != mOptions.modes.end()) {
            copyConcurrent(sl);
          }

          copyFinalize();

          break;
        }
      }
    }
  }

  globalFinalize();
}

template class GPUbenchmark<char>;
template class GPUbenchmark<size_t>;
template class GPUbenchmark<int>;
// template class GPUbenchmark<uint4>;

} // namespace benchmark
} // namespace o2
