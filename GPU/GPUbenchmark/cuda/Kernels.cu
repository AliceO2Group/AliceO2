// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Kernels.cu
/// \author: mconcas@cern.ch

#include <Kernels.h>
#include <Common.h>

#define GPUCHECK(error)                                                                        \
  if (error != hipSuccess) {                                                                  \
    printf("%serror: '%s'(%d) at %s:%d%s\n", KRED, hipGetErrorString(error), error, __FILE__, \
           __LINE__, KNRM);                                                                    \
    failed("API returned error code.");                                                        \
  }

double bytesToKB(size_t s) { return (double)s / (1024.0); }
double bytesToGB(size_t s) { return (double)s / (1024.0 * 1024.0 * 1024.0); }

#define printLimit(w1, limit, units)                                          \
  {                                                                           \
    size_t val;                                                               \
    hipDeviceGetLimit(&val, limit);                                          \
    std::cout << setw(w1) << #limit ": " << val << " " << units << std::endl; \
  }

namespace o2
{
namespace benchmark
{
namespace gpu
{
// Kernels here
} // namespace gpu
void printDeviceProp(int deviceId)
{
  const int w1 = 34;
  std::cout << std::left;
  std::cout << std::setw(w1)
            << "--------------------------------------------------------------------------------"
            << std::endl;
  std::cout << std::setw(w1) << "device#" << deviceId << std::endl;

  hipDeviceProp_t props;
  GPUCHECK(hipGetDeviceProperties(&props, deviceId));

  std::cout << std::setw(w1) << "Name: " << props.name << std::endl;
  std::cout << std::setw(w1) << "pciBusID: " << props.pciBusID << std::endl;
  std::cout << std::setw(w1) << "pciDeviceID: " << props.pciDeviceID << std::endl;
  std::cout << std::setw(w1) << "pciDomainID: " << props.pciDomainID << std::endl;
  std::cout << std::setw(w1) << "multiProcessorCount: " << props.multiProcessorCount << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << std::setw(w1) << "isMultiGpuBoard: " << props.isMultiGpuBoard << std::endl;
  std::cout << std::setw(w1) << "clockRate: " << (float)props.clockRate / 1000.0 << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "memoryClockRate: " << (float)props.memoryClockRate / 1000.0 << " Mhz"
            << std::endl;
  std::cout << std::setw(w1) << "memoryBusWidth: " << props.memoryBusWidth << std::endl;
  std::cout << std::setw(w1) << "clockInstructionRate: " << (float)props.clockRate / 1000.0
            << " Mhz" << std::endl;
  std::cout << std::setw(w1) << "totalGlobalMem: " << std::fixed << std::setprecision(2)
            << bytesToGB(props.totalGlobalMem) << " GB" << std::endl;
#if !defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.sharedMemPerMultiprocessor) << " KB" << std::endl;
#endif
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "maxSharedMemoryPerMultiProcessor: " << std::fixed << std::setprecision(2)
            << bytesToKB(props.maxSharedMemoryPerMultiProcessor) << " KB" << std::endl;
#endif
  std::cout << std::setw(w1) << "totalConstMem: " << props.totalConstMem << std::endl;
  std::cout << std::setw(w1) << "sharedMemPerBlock: " << (float)props.sharedMemPerBlock / 1024.0 << " KB"
            << std::endl;
  std::cout << std::setw(w1) << "canMapHostMemory: " << props.canMapHostMemory << std::endl;
  std::cout << std::setw(w1) << "regsPerBlock: " << props.regsPerBlock << std::endl;
  std::cout << std::setw(w1) << "warpSize: " << props.warpSize << std::endl;
  std::cout << std::setw(w1) << "l2CacheSize: " << props.l2CacheSize << std::endl;
  std::cout << std::setw(w1) << "computeMode: " << props.computeMode << std::endl;
  std::cout << std::setw(w1) << "maxThreadsPerBlock: " << props.maxThreadsPerBlock << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.x: " << props.maxThreadsDim[0] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.y: " << props.maxThreadsDim[1] << std::endl;
  std::cout << std::setw(w1) << "maxThreadsDim.z: " << props.maxThreadsDim[2] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.x: " << props.maxGridSize[0] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.y: " << props.maxGridSize[1] << std::endl;
  std::cout << std::setw(w1) << "maxGridSize.z: " << props.maxGridSize[2] << std::endl;
  std::cout << std::setw(w1) << "major: " << props.major << std::endl;
  std::cout << std::setw(w1) << "minor: " << props.minor << std::endl;
  std::cout << std::setw(w1) << "concurrentKernels: " << props.concurrentKernels << std::endl;
  std::cout << std::setw(w1) << "cooperativeLaunch: " << props.cooperativeLaunch << std::endl;
  std::cout << std::setw(w1) << "cooperativeMultiDeviceLaunch: " << props.cooperativeMultiDeviceLaunch << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "arch.hasGlobalInt32Atomics: " << props.arch.hasGlobalInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalFloatAtomicExch: " << props.arch.hasGlobalFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt32Atomics: " << props.arch.hasSharedInt32Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedFloatAtomicExch: " << props.arch.hasSharedFloatAtomicExch
            << std::endl;
  std::cout << std::setw(w1) << "arch.hasFloatAtomicAdd: " << props.arch.hasFloatAtomicAdd << std::endl;
  std::cout << std::setw(w1) << "arch.hasGlobalInt64Atomics: " << props.arch.hasGlobalInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasSharedInt64Atomics: " << props.arch.hasSharedInt64Atomics << std::endl;
  std::cout << std::setw(w1) << "arch.hasDoubles: " << props.arch.hasDoubles << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpVote: " << props.arch.hasWarpVote << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpBallot: " << props.arch.hasWarpBallot << std::endl;
  std::cout << std::setw(w1) << "arch.hasWarpShuffle: " << props.arch.hasWarpShuffle << std::endl;
  std::cout << std::setw(w1) << "arch.hasFunnelShift: " << props.arch.hasFunnelShift << std::endl;
  std::cout << std::setw(w1) << "arch.hasThreadFenceSystem: " << props.arch.hasThreadFenceSystem << std::endl;
  std::cout << std::setw(w1) << "arch.hasSyncThreadsExt: " << props.arch.hasSyncThreadsExt << std::endl;
  std::cout << std::setw(w1) << "arch.hasSurfaceFuncs: " << props.arch.hasSurfaceFuncs << std::endl;
  std::cout << std::setw(w1) << "arch.has3dGrid: " << props.arch.has3dGrid << std::endl;
  std::cout << std::setw(w1) << "arch.hasDynamicParallelism: " << props.arch.hasDynamicParallelism << std::endl;
  std::cout << std::setw(w1) << "gcnArchName: " << props.gcnArchName << std::endl;
#endif
  std::cout << std::setw(w1) << "isIntegrated: " << props.integrated << std::endl;
  std::cout << std::setw(w1) << "maxTexture1D: " << props.maxTexture1D << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.width: " << props.maxTexture2D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture2D.height: " << props.maxTexture2D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.width: " << props.maxTexture3D[0] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.height: " << props.maxTexture3D[1] << std::endl;
  std::cout << std::setw(w1) << "maxTexture3D.depth: " << props.maxTexture3D[2] << std::endl;
#if defined(__HIPCC__)
  std::cout << std::setw(w1) << "isLargeBar: " << props.isLargeBar << std::endl;
  std::cout << std::setw(w1) << "asicRevision: " << props.asicRevision << std::endl;
#endif

  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));
  std::cout << std::setw(w1) << "peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;
  std::cout << std::setw(w1) << "non-peers: ";
  for (int i = 0; i < deviceCnt; i++) {
    int isPeer;
    GPUCHECK(hipDeviceCanAccessPeer(&isPeer, i, deviceId));
    if (!isPeer) {
      std::cout << "device#" << i << " ";
    }
  }
  std::cout << std::endl;

  size_t free, total;
  GPUCHECK(hipMemGetInfo(&free, &total));

  std::cout << std::fixed << std::setprecision(2);
  std::cout << std::setw(w1) << "memInfo.total: " << bytesToGB(total) << " GB" << std::endl;
  std::cout << std::setw(w1) << "memInfo.free:  " << bytesToGB(free) << " GB (" << std::setprecision(0)
            << (float)free / total * 100.0 << "%)" << std::endl;
}

void printDevices()
{
  int deviceCnt;
  GPUCHECK(hipGetDeviceCount(&deviceCnt));

  for (int i = 0; i < deviceCnt; i++) {
    GPUCHECK(hipSetDevice(i));
    printDeviceProp(i);
  }
}

void init()
{
  size_t free, total;
  GPUCHECK(hipMemGetInfo(&free, &total));

  void* devicePtr;
  GPUCHECK(hipMalloc(&devicePtr, total));
}

} // namespace benchmark
} // namespace o2