#include "hip/hip_runtime.h"
// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file testGPUsortCUDA.cu
/// \author Michael Lettrich

#define BOOST_TEST_MODULE Test GPUCommonAlgorithm Sorting CUDA
#define BOOST_TEST_MAIN
#define BOOST_TEST_DYN_LINK

#include <iostream>
#include <cstring>
#include <hip/hip_runtime.h>
#include <boost/test/unit_test.hpp>
#include "GPUCommonAlgorithm.h"

///////////////////////////////////////////////////////////////
// Test setup and tear down
///////////////////////////////////////////////////////////////

static constexpr float TOLERANCE = 10 * std::numeric_limits<float>::epsilon();

hipError_t cudaCheckError(hipError_t cudaErrorCode)
{
  if (cudaErrorCode != hipSuccess) {
    std::cerr << "ErrorCode " << cudaErrorCode << " " << hipGetErrorName(cudaErrorCode) << ": " << hipGetErrorString(cudaErrorCode) << std::endl;
    exit(-1);
  }
  return cudaErrorCode;
}

struct TestEnvironment {
  TestEnvironment() : size(101), data(nullptr), sorted(size)
  {
    cudaCheckError(hipMallocManaged(&data, size * sizeof(float)));

    // create an array of unordered floats with negative and positive values
    for (size_t i = 0; i < size; i++) {
      data[i] = size / 2.0 - i;
    }
    // create copy
    std::memcpy(sorted.data(), data, size * sizeof(float));
    // sort
    std::sort(sorted.begin(), sorted.end());
  }

  ~TestEnvironment()
  {
    hipFree(data);
  };

  const size_t size;
  float* data;
  std::vector<float> sorted;
};

template <typename T>
void testAlmostEqualArray(T* correct, T* testing, size_t size)
{
  for (size_t i = 0; i < size; i++) {
    if (std::fabs(correct[i]) < TOLERANCE) {
      BOOST_CHECK_SMALL(testing[i], TOLERANCE);
    } else {
      BOOST_CHECK_CLOSE(correct[i], testing[i], TOLERANCE);
    }
  }
}

///////////////////////////////////////////////////////////////

__global__ void sortInThread(float* data, size_t dataLength)
{
  // make sure only one thread is working on this.
  if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    o2::gpu::CAAlgo::sort(data, data + dataLength);
  }
}

__global__ void sortInThreadWithOperator(float* data, size_t dataLength)
{
  // make sure only one thread is working on this.
  if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    o2::gpu::CAAlgo::sort(data, data + dataLength, [](float a, float b) { return a < b; });
  }
}

///////////////////////////////////////////////////////////////

__global__ void sortInBlock(float* data, size_t dataLength)
{
  o2::gpu::CAAlgo::sortInBlock<float>(data, data + dataLength);
}

__global__ void sortInBlockWithOperator(float* data, size_t dataLength)
{
  o2::gpu::CAAlgo::sortInBlock(data, data + dataLength, [](float a, float b) { return a < b; });
}
///////////////////////////////////////////////////////////////

BOOST_AUTO_TEST_SUITE(TestsortInThread)

BOOST_FIXTURE_TEST_CASE(GPUsortThreadCUDA, TestEnvironment)
{
  sortInThread<<<1, 1>>>(data, size);
  BOOST_CHECK_EQUAL(cudaCheckError(hipDeviceSynchronize()), hipSuccess);
  testAlmostEqualArray(sorted.data(), data, size);
}

BOOST_FIXTURE_TEST_CASE(GPUsortThreadOperatorCUDA, TestEnvironment)
{
  sortInThreadWithOperator<<<1, 1>>>(data, size);
  BOOST_CHECK_EQUAL(cudaCheckError(hipDeviceSynchronize()), hipSuccess);
  testAlmostEqualArray(sorted.data(), data, size);
}

BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE(TestsortInBlock)

BOOST_FIXTURE_TEST_CASE(GPUsortBlockCUDA, TestEnvironment)
{
  sortInBlock<<<1, 128>>>(data, size);
  BOOST_CHECK_EQUAL(cudaCheckError(hipDeviceSynchronize()), hipSuccess);
  testAlmostEqualArray(sorted.data(), data, size);
}

BOOST_FIXTURE_TEST_CASE(GPUsortBlockOperatorCUDA, TestEnvironment)
{
  sortInBlockWithOperator<<<1, 128>>>(data, size);
  BOOST_CHECK_EQUAL(cudaCheckError(hipDeviceSynchronize()), hipSuccess);
  testAlmostEqualArray(sorted.data(), data, size);
}

BOOST_AUTO_TEST_SUITE_END()
