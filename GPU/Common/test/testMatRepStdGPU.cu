#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file testGPUSMatrixImp.cu
/// \author Matteo Concas

#define BOOST_TEST_MODULE Test GPUSMatrixImpl
#ifdef __HIPCC__
#define GPUPLATFORM "HIP"
#include "hip/hip_runtime.h"
#else
#define GPUPLATFORM "CUDA"
#include <hip/hip_runtime.h>
#endif

#include <boost/test/unit_test.hpp>
#include <iostream>

#include <MathUtils/SMatrixGPU.h>
#include <Math/SMatrix.h>

template <typename T>
void discardResult(const T&) {}

void prologue() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess || !deviceCount) {
        std::cerr << "No " << GPUPLATFORM << " devices found" << std::endl;
        return;
    }

    for (int iDevice = 0; iDevice < deviceCount; ++iDevice) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, iDevice);
        std::cout << GPUPLATFORM << " Device " << iDevice << ": " << deviceProp.name << std::endl;
    }
}

using Mat3DGPU = o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepStdGPU<float, 3, 3>>;
using Mat3D = ROOT::Math::SMatrix<float, 3, 3, ROOT::Math::MatRepStd<float, 3, 3>>;

template <typename MatrixType>
__device__ void printMatrix(const MatrixType& matrix, const char* name) {
    printf("%s(0,0) = %f, %s(0,1) = %f, %s(0,2) = %f\n", name, matrix(0, 0), name, matrix(0, 1), name, matrix(0, 2));
    printf("%s(1,0) = %f, %s(1,1) = %f, %s(1,2) = %f\n", name, matrix(1, 0), name, matrix(1, 1), name, matrix(1, 2));
    printf("%s(2,0) = %f, %s(2,1) = %f, %s(2,2) = %f\n", name, matrix(2, 0), name, matrix(2, 1), name, matrix(2, 2));
}

// Invert test
template <typename T, int D>
__global__ void invertMatrixKernel(o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepStdGPU<float, 3, 3>>* matrix) {
    Mat3DGPU smat2 = *matrix;

    printMatrix(*matrix, "A");
    printMatrix(smat2, "B");

    printf("\nInverting A...\n");
    matrix->Invert();

    printMatrix(*matrix, "A");

    printf("\nC = (A^-1) * B...\n");
    auto smat3 = (*matrix) * smat2;

    printMatrix(smat3, "C");

    printf("\nEvaluating...\n");
    Mat3DGPU tmp;
    o2::math_utils::Assign::Evaluate(tmp, smat3);

    printMatrix(tmp, "A");
    *matrix = tmp;
}

struct GPUSMatrixImplFixture {
    GPUSMatrixImplFixture() : i(3), SMatrix3D_d(nullptr), SMatrix3D_h() {
        prologue();
        SMatrix3D_h(0, 0) = 1;
        SMatrix3D_h(1, 1) = 2;
        SMatrix3D_h(2, 2) = 3;
        SMatrix3D_h(0, 1) = 4;
        SMatrix3D_h(0, 2) = 5;
        SMatrix3D_h(1, 2) = 6;

        hipError_t error = hipMalloc(&SMatrix3D_d, sizeof(Mat3DGPU));
        if (error != hipSuccess) {
            std::cerr << "hipMalloc failed: " << hipGetErrorString(error) << std::endl;
        }

        error = hipMemcpy(SMatrix3D_d, &SMatrix3D_h, sizeof(Mat3DGPU), hipMemcpyHostToDevice);
        if (error != hipSuccess) {
            std::cerr << "hipMemcpy failed: " << hipGetErrorString(error) << std::endl;
        }

        std::cout << "sizeof(Mat3DGPU) = " << sizeof(Mat3DGPU) << std::endl;
        std::cout << "sizeof(Mat3D) = " << sizeof(Mat3D) << std::endl;
    }

    ~GPUSMatrixImplFixture() {
        hipFree(SMatrix3D_d);
    }

    int i;
    Mat3DGPU* SMatrix3D_d; // device ptr
    Mat3D SMatrix3D_h;
};

BOOST_FIXTURE_TEST_CASE(DummyFixtureUsage, GPUSMatrixImplFixture) {
    invertMatrixKernel<float, 3><<<1, 1>>>(SMatrix3D_d);
    hipDeviceSynchronize();

    hipMemcpy(&SMatrix3D_h, SMatrix3D_d, sizeof(Mat3DGPU), hipMemcpyDeviceToHost);

    Mat3D identity;
    identity(0, 0) = 1;
    identity(1, 1) = 1;
    identity(2, 2) = 1;
    BOOST_TEST(SMatrix3D_h == identity);
}

// Transpose test
template <typename T>
__global__ void testTransposeTwiceKernel(o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepStdGPU<float, 3, 3>>* matrix) {
    auto transposedOnce = o2::math_utils::Transpose(*matrix);
    auto transposedTwice = o2::math_utils::Transpose(transposedOnce);

    *matrix = transposedTwice;
}

BOOST_FIXTURE_TEST_CASE(TestMatrixDoubleTranspose, GPUSMatrixImplFixture) {
    testTransposeTwiceKernel<<<1, 1>>>(SMatrix3D_d);
    hipDeviceSynchronize();
    hipError_t error = hipMemcpy(&SMatrix3D_h, SMatrix3D_d, sizeof(Mat3DGPU), hipMemcpyDeviceToHost);
    BOOST_REQUIRE(error == hipSuccess);

    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            BOOST_TEST(SMatrix3D_h(i, j) == (i * 3 + j + 1));
        }
    }

    // Test on CPU
    Mat3D cpuMatrix = SMatrix3D_h;
    Mat3D transposedOnce = ROOT::Math::Transpose(cpuMatrix);
    Mat3D transposedTwice = ROOT::Math::Transpose(transposedOnce);

    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            BOOST_TEST(cpuMatrix(i, j) == transposedTwice(i, j));
        }
    }
}

// Multiplication test
template <typename T>
__global__ void testMatrixMultiplicationKernel(
    o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepStdGPU<float, 3, 3>>* matrixA,
    o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepStdGPU<float, 3, 3>>* matrixB,
    o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepStdGPU<float, 3, 3>>* result)
{
    *result = (*matrixA) * (*matrixB);
}

BOOST_FIXTURE_TEST_CASE(TestMatrixMultiplication, GPUSMatrixImplFixture) {
    Mat3DGPU *matrixB_d, *result_d;
    Mat3D matrixB_h, result_h;

    matrixB_h(0, 0) = 9; matrixB_h(0, 1) = 8; matrixB_h(0, 2) = 7;
    matrixB_h(1, 0) = 6; matrixB_h(1, 1) = 5; matrixB_h(1, 2) = 4;
    matrixB_h(2, 0) = 3; matrixB_h(2, 1) = 2; matrixB_h(2, 2) = 1;

    hipMalloc(&matrixB_d, sizeof(Mat3DGPU));
    hipMalloc(&result_d, sizeof(Mat3DGPU));
    hipMemcpy(matrixB_d, &matrixB_h, sizeof(Mat3DGPU), hipMemcpyHostToDevice);

    testMatrixMultiplicationKernel<<<1, 1>>>(SMatrix3D_d, matrixB_d, result_d);
    hipDeviceSynchronize();

    hipMemcpy(&result_h, result_d, sizeof(Mat3DGPU), hipMemcpyDeviceToHost);

    Mat3D expected;
    expected(0, 0) = 26; expected(0, 1) = 20; expected(0, 2) = 14;
    expected(1, 0) = 57; expected(1, 1) = 44; expected(1, 2) = 31;
    expected(2, 0) = 96; expected(2, 1) = 74; expected(2, 2) = 52;

    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j) {
            BOOST_TEST(result_h(i, j) == expected(i, j));
        }
    }

    hipFree(matrixB_d);
    hipFree(result_d);
}
