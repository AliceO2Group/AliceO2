#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

/// \file testGPUSMatrixImp.cu
/// \author Matteo Concas

#define BOOST_TEST_MODULE Test GPUSMatrixImpl
#ifdef __HIPCC__
#define GPUPLATFORM "HIP"
#include "hip/hip_runtime.h"
#else
#define GPUPLATFORM "CUDA"
#include <hip/hip_runtime.h>
#endif

#include <boost/test/unit_test.hpp>
#include <iostream>

#include <MathUtils/SMatrixGPU.h>
#include <Math/SMatrix.h>

template <typename T>
void discardResult(const T&)
{
}

void prologue()
{
  int deviceCount;
  discardResult(hipGetDeviceCount(&deviceCount));
  if (!deviceCount) {
    std::cerr << "No " << GPUPLATFORM << " devices found" << std::endl;
  }
  for (int iDevice = 0; iDevice < deviceCount; ++iDevice) {
    hipDeviceProp_t deviceProp;
    discardResult(hipGetDeviceProperties(&deviceProp, iDevice));
    std::cout << GPUPLATFORM << " Device " << iDevice << ": " << deviceProp.name << std::endl;
  }
}

using MatSym3DGPU = o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepSymGPU<float, 3>>;
using MatSym3D = ROOT::Math::SMatrix<float, 3, 3, ROOT::Math::MatRepSym<float, 3>>;

template <typename T, int D>
__global__ void invertSymMatrixKernel(o2::math_utils::SMatrixGPU<float, 3, 3, o2::math_utils::MatRepSymGPU<float, 3>>* matrix)
{
  MatSym3DGPU smat2 = *matrix;

  printf("A(0,0) = %f, A(0,1) = %f, A(0,2) = %f\n", (*matrix)(0, 0), (*matrix)(0, 1), (*matrix)(0, 2));
  printf("A(1,0) = %f, A(1,1) = %f, A(1,2) = %f\n", (*matrix)(1, 0), (*matrix)(1, 1), (*matrix)(1, 2));
  printf("A(2,0) = %f, A(2,1) = %f, A(2,2) = %f\n", (*matrix)(2, 0), (*matrix)(2, 1), (*matrix)(2, 2));

  printf("B(0,0) = %f, B(0,1) = %f, B(0,2) = %f\n", smat2(0, 0), smat2(0, 1), smat2(0, 2));
  printf("B(1,0) = %f, B(1,1) = %f, B(1,2) = %f\n", smat2(1, 0), smat2(1, 1), smat2(1, 2));
  printf("B(2,0) = %f, B(2,1) = %f, B(2,2) = %f\n", smat2(2, 0), smat2(2, 1), smat2(2, 2));

  printf("\nInverting A...\n");
  matrix->Invert();

  printf("A(0,0) = %f, A(0,1) = %f, A(0,2) = %f\n", (*matrix)(0, 0), (*matrix)(0, 1), (*matrix)(0, 2));
  printf("A(1,0) = %f, A(1,1) = %f, A(1,2) = %f\n", (*matrix)(1, 0), (*matrix)(1, 1), (*matrix)(1, 2));
  printf("A(2,0) = %f, A(2,1) = %f, A(2,2) = %f\n", (*matrix)(2, 0), (*matrix)(2, 1), (*matrix)(2, 2));

  printf("\nC = (A^-1) * B...\n");
  auto smat3 = (*matrix) * smat2;

  printf("C(0,0) = %f, C(0,1) = %f, C(0,2) = %f\n", smat3(0, 0), smat3(0, 1), smat3(0, 2));
  printf("C(1,0) = %f, C(1,1) = %f, C(1,2) = %f\n", smat3(1, 0), smat3(1, 1), smat3(1, 2));
  printf("C(2,0) = %f, C(2,1) = %f, C(2,2) = %f\n", smat3(2, 0), smat3(2, 1), smat3(2, 2));

  printf("\nEvaluating...\n");
  MatSym3DGPU tmp;
  o2::math_utils::AssignSym::Evaluate(tmp, smat3);

  printf("A(0,0) = %f, A(0,1) = %f, A(0,2) = %f\n", tmp(0, 0), tmp(0, 1), tmp(0, 2));
  printf("A(1,0) = %f, A(1,1) = %f, A(1,2) = %f\n", tmp(1, 0), tmp(1, 1), tmp(1, 2));
  printf("A(2,0) = %f, A(2,1) = %f, A(2,2) = %f\n", tmp(2, 0), tmp(2, 1), tmp(2, 2));
  (*matrix) = tmp;
}

struct GPUSMatrixImplFixture {
  GPUSMatrixImplFixture() : SMatrix3D_d(nullptr)
  {
    prologue();

    SMatrix3D_h(0, 0) = 1;
    SMatrix3D_h(1, 1) = 2;
    SMatrix3D_h(2, 2) = 3;
    SMatrix3D_h(0, 1) = 4;
    SMatrix3D_h(0, 2) = 5;
    SMatrix3D_h(1, 2) = 6;

    discardResult(hipMalloc(&SMatrix3D_d, sizeof(MatSym3DGPU)));
    discardResult(hipMemcpy(SMatrix3D_d, &SMatrix3D_h, sizeof(MatSym3DGPU), hipMemcpyHostToDevice));

    std::cout << "sizeof(MatSym3DGPU) = " << sizeof(MatSym3DGPU) << std::endl;
    std::cout << "sizeof(MatSym3D) = " << sizeof(MatSym3D) << std::endl;
    i = 3;
  }

  ~GPUSMatrixImplFixture()
  {
    discardResult(hipFree(SMatrix3D_d));
  }

  int i;
  MatSym3DGPU* SMatrix3D_d; // device ptr
  MatSym3D SMatrix3D_h;
};

BOOST_FIXTURE_TEST_CASE(DummyFixtureUsage, GPUSMatrixImplFixture)
{
  invertSymMatrixKernel<float, 3><<<1, 1>>>(SMatrix3D_d);
  discardResult(hipDeviceSynchronize());

  discardResult(hipMemcpy(&SMatrix3D_h, SMatrix3D_d, sizeof(MatSym3DGPU), hipMemcpyDeviceToHost));

  MatSym3D identity;
  identity(0, 0) = 1;
  identity(1, 1) = 1;
  identity(2, 2) = 1;
  BOOST_TEST(SMatrix3D_h == identity);
}