#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>
#define GPUCA_GPUTYPE_PASCAL

#include "AliGPUReconstructionCUDA.h"
#include "AliGPUReconstructionCUDAInternals.h"
#include "AliGPUReconstructionCommon.h"

__constant__ uint4 gGPUConstantMemBuffer[(sizeof(AliGPUCAConstantMem) + sizeof(uint4) - 1) / sizeof(uint4)];
__constant__ char& gGPUConstantMemBufferChar = (char&) gGPUConstantMemBuffer;
__constant__ AliGPUCAConstantMem& gGPUConstantMem = (AliGPUCAConstantMem&) gGPUConstantMemBufferChar;

#ifdef GPUCA_GPU_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

#ifdef HAVE_O2HEADERS
#include "ITStrackingCUDA/TrackerTraitsNV.h"
#else
namespace o2 { namespace ITS { class TrackerTraitsNV : public TrackerTraits {}; }}
#endif

#define DEVICE_KERNELS_PRE
#include "AliGPUDeviceKernels.h"

template <class TProcess, typename... Args> GPUg() void runKernelCUDA(int iSlice, Args... args)
{
	AliGPUTPCTracker &tracker = gGPUConstantMem.tpcTrackers[iSlice];
	GPUshared() typename TProcess::AliGPUTPCSharedMemory smem;
	TProcess::Thread(get_num_groups(0), get_local_size(0), get_group_id(0), get_local_id(0), smem, tracker, args...);
}

template <class TProcess, typename... Args> GPUg() void runKernelCUDAMulti(int firstSlice, int nSliceCount, Args... args)
{
	const int iSlice = nSliceCount * (get_group_id(0) + (get_num_groups(0) % nSliceCount != 0 && nSliceCount * (get_group_id(0) + 1) % get_num_groups(0) != 0)) / get_num_groups(0);
	const int nSliceBlockOffset = get_num_groups(0) * iSlice / nSliceCount;
	const int sliceBlockId = get_group_id(0) - nSliceBlockOffset;
	const int sliceGridDim = get_num_groups(0) * (iSlice + 1) / nSliceCount - get_num_groups(0) * (iSlice) / nSliceCount;
	AliGPUTPCTracker &tracker = gGPUConstantMem.tpcTrackers[firstSlice + iSlice];
	GPUshared() typename TProcess::AliGPUTPCSharedMemory smem;
	TProcess::Thread(sliceGridDim, get_local_size(0), sliceBlockId, get_local_id(0), smem, tracker, args...);
}

template <class T, typename... Args> int AliGPUReconstructionCUDABackend::runKernelBackend(const krnlExec& x, const krnlRunRange& y, const krnlEvent& z, const Args&... args)
{
	if (x.device == krnlDeviceType::CPU) return AliGPUReconstructionCPU::runKernelBackend<T> (x, y, z, args...);
	if (z.evList) for (int k = 0;k < z.nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[x.stream], ((hipEvent_t*) z.evList)[k], 0));
	if (y.num <= 1)
	{
		runKernelCUDA<T> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>>(y.start, args...);
	}
	else
	{
		runKernelCUDAMulti<T> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>> (y.start, y.num, args...);
	}
	if (z.ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) z.ev, mInternals->CudaStreams[x.stream]));
	return 0;
}

AliGPUReconstructionCUDABackend::AliGPUReconstructionCUDABackend(const AliGPUCASettingsProcessing& cfg) : AliGPUReconstructionDeviceBase(cfg)
{
	mInternals = new AliGPUReconstructionCUDAInternals;
	mProcessingSettings.deviceType = CUDA;
	mITSTrackerTraits.reset(new o2::ITS::TrackerTraitsNV);
}

AliGPUReconstructionCUDABackend::~AliGPUReconstructionCUDABackend()
{
	mITSTrackerTraits.reset(nullptr); //Make sure we destroy the ITS tracker before we exit CUDA
	hipDeviceReset();
	delete mInternals;
}

AliGPUReconstruction* AliGPUReconstruction_Create_CUDA(const AliGPUCASettingsProcessing& cfg)
{
	return new AliGPUReconstructionCUDA(cfg);
}

int AliGPUReconstructionCUDABackend::InitDevice_Runtime()
{
	//Find best CUDA device, initialize and allocate memory

	hipDeviceProp_t hipDeviceProp_t;

	int count, bestDevice = -1;
	double bestDeviceSpeed = -1, deviceSpeed;
	if (GPUFailedMsgI(hipGetDeviceCount(&count)))
	{
		CAGPUError("Error getting CUDA Device Count");
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("Available CUDA devices:");
	const int reqVerMaj = 2;
	const int reqVerMin = 0;
	for (int i = 0;i < count;i++)
	{
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Examining device %d\n", i);
		size_t free, total;
		hipInit(0);
		hipDevice_t tmpDevice;
		hipDeviceGet(&tmpDevice, i);
		hipCtx_t tmpContext;
		hipCtxCreate(&tmpContext, 0, tmpDevice);
		if(hipMemGetInfo(&free, &total)) std::cout << "Error\n";
		hipCtxDestroy(tmpContext);
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained current memory usage for device %d\n", i);
		if (GPUFailedMsgI(hipGetDeviceProperties(&hipDeviceProp_t, i))) continue;
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained device properties for device %d\n", i);
		int deviceOK = true;
		const char* deviceFailure = "";
		if (hipDeviceProp_t.major >= 9) {deviceOK = false; deviceFailure = "Invalid Revision";}
		else if (hipDeviceProp_t.major < reqVerMaj || (hipDeviceProp_t.major == reqVerMaj && hipDeviceProp_t.minor < reqVerMin)) {deviceOK = false; deviceFailure = "Too low device revision";}
		else if (free < mDeviceMemorySize) {deviceOK = false; deviceFailure = "Insufficient GPU memory";}

		deviceSpeed = (double) hipDeviceProp_t.multiProcessorCount * (double) hipDeviceProp_t.clockRate * (double) hipDeviceProp_t.warpSize * (double) free * (double) hipDeviceProp_t.major * (double) hipDeviceProp_t.major;
		if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lld / %lld)%s %s", deviceOK ? " " : "[", i, hipDeviceProp_t.name, hipDeviceProp_t.major, hipDeviceProp_t.minor, (long long int) free, (long long int) hipDeviceProp_t.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
		if (!deviceOK) continue;
		if (deviceSpeed > bestDeviceSpeed)
		{
			bestDevice = i;
			bestDeviceSpeed = deviceSpeed;
		}
		else
		{
			if (mDeviceProcessingSettings.debugLevel >= 0) CAGPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
		}
	}
	if (bestDevice == -1)
	{
		CAGPUWarning("No %sCUDA Device available, aborting CUDA Initialisation", count ? "appropriate " : "");
		CAGPUImportant("Requiring Revision %d.%d, Mem: %lld", reqVerMaj, reqVerMin, (long long int) mDeviceMemorySize);
		return(1);
	}

	if (mDeviceProcessingSettings.deviceNum > -1)
	{
		if (mDeviceProcessingSettings.deviceNum < (signed) count)
		{
			bestDevice = mDeviceProcessingSettings.deviceNum;
		}
		else
		{
			CAGPUWarning("Requested device ID %d non existend, falling back to default device id %d", mDeviceProcessingSettings.deviceNum, bestDevice);
		}
	}
	fDeviceId = bestDevice;

	hipGetDeviceProperties(&hipDeviceProp_t ,fDeviceId );

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		CAGPUInfo("Using CUDA Device %s with Properties:", hipDeviceProp_t.name);
		CAGPUInfo("totalGlobalMem = %lld", (unsigned long long int) hipDeviceProp_t.totalGlobalMem);
		CAGPUInfo("sharedMemPerBlock = %lld", (unsigned long long int) hipDeviceProp_t.sharedMemPerBlock);
		CAGPUInfo("regsPerBlock = %d", hipDeviceProp_t.regsPerBlock);
		CAGPUInfo("warpSize = %d", hipDeviceProp_t.warpSize);
		CAGPUInfo("memPitch = %lld", (unsigned long long int) hipDeviceProp_t.memPitch);
		CAGPUInfo("maxThreadsPerBlock = %d", hipDeviceProp_t.maxThreadsPerBlock);
		CAGPUInfo("maxThreadsDim = %d %d %d", hipDeviceProp_t.maxThreadsDim[0], hipDeviceProp_t.maxThreadsDim[1], hipDeviceProp_t.maxThreadsDim[2]);
		CAGPUInfo("maxGridSize = %d %d %d", hipDeviceProp_t.maxGridSize[0], hipDeviceProp_t.maxGridSize[1], hipDeviceProp_t.maxGridSize[2]);
		CAGPUInfo("totalConstMem = %lld", (unsigned long long int) hipDeviceProp_t.totalConstMem);
		CAGPUInfo("major = %d", hipDeviceProp_t.major);
		CAGPUInfo("minor = %d", hipDeviceProp_t.minor);
		CAGPUInfo("clockRate = %d", hipDeviceProp_t.clockRate);
		CAGPUInfo("memoryClockRate = %d", hipDeviceProp_t.memoryClockRate);
		CAGPUInfo("multiProcessorCount = %d", hipDeviceProp_t.multiProcessorCount);
		CAGPUInfo("textureAlignment = %lld", (unsigned long long int) hipDeviceProp_t.textureAlignment);
	}
#ifdef GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE
	fConstructorBlockCount = hipDeviceProp_t.multiProcessorCount;
#else
	fConstructorBlockCount = hipDeviceProp_t.multiProcessorCount * GPUCA_GPU_BLOCK_COUNT_CONSTRUCTOR_MULTIPLIER;
#endif
	fConstructorThreadCount = GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;
	fSelectorBlockCount = hipDeviceProp_t.multiProcessorCount * GPUCA_GPU_BLOCK_COUNT_SELECTOR_MULTIPLIER;

	if (hipDeviceProp_t.major < 1 || (hipDeviceProp_t.major == 1 && hipDeviceProp_t.minor < 2))
	{
		CAGPUError( "Unsupported CUDA Device" );
		return(1);
	}

#ifdef GPUCA_GPU_USE_TEXTURES
	if (GPUCA_GPU_SLICE_DATA_MEMORY * NSLICES > (size_t) hipDeviceProp_t.maxTexture1DLinear)
	{
		CAGPUError("Invalid maximum texture size of device: %lld < %lld\n", (long long int) hipDeviceProp_t.maxTexture1DLinear, (long long int) (GPUCA_GPU_SLICE_DATA_MEMORY * NSLICES));
		return(1);
	}
#endif

	mNStreams = std::max(GPUCA_GPU_NUM_STREAMS, 3);

	if (hipCtxCreate(&mInternals->CudaContext, hipDeviceScheduleAuto, fDeviceId) != hipSuccess)
	{
		CAGPUError("Could not set CUDA Device!");
		return(1);
	}

	if (mDeviceMemorySize > hipDeviceProp_t.totalGlobalMem || GPUFailedMsgI(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize)))
	{
		CAGPUError("CUDA Memory Allocation Error");
		hipDeviceReset();
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) CAGPUInfo("GPU Memory used: %lld", (long long int) mDeviceMemorySize);
	if (GPUFailedMsgI(hipHostMalloc(&mHostMemoryBase, mHostMemorySize)))
	{
		CAGPUError("Error allocating Page Locked Host Memory");
		hipDeviceReset();
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) CAGPUInfo("Host Memory used: %lld", (long long int) mHostMemorySize);

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		memset(mHostMemoryBase, 0, mHostMemorySize);
		if (GPUFailedMsgI(hipMemset(mDeviceMemoryBase, 143, mDeviceMemorySize)))
		{
			CAGPUError("Error during CUDA memset");
			hipDeviceReset();
			return(1);
		}
	}

	for (int i = 0;i < mNStreams;i++)
	{
		if (GPUFailedMsgI(hipStreamCreate(&mInternals->CudaStreams[i])))
		{
			CAGPUError("Error creating CUDA Stream");
			hipDeviceReset();
			return(1);
		}
	}
	
	void* devPtrConstantMem;
	if (GPUFailedMsgI(hipGetSymbolAddress(&devPtrConstantMem, HIP_SYMBOL(gGPUConstantMemBuffer))))
	{
		CAGPUError("Error getting ptr to constant memory");
		hipDeviceReset();
		return 1;
	}
	mDeviceConstantMem = (AliGPUCAConstantMem*) devPtrConstantMem;
	
	hipEvent_t *events = (hipEvent_t*) &mEvents;
	for (unsigned int i = 0;i < sizeof(mEvents) / sizeof(hipEvent_t);i++)
	{
		if (GPUFailedMsgI(hipEventCreate(&events[i])))
		{
			CAGPUError("Error creating event");
			hipDeviceReset();
			return 1;
		}
	}

	ReleaseThreadContext();
	CAGPUInfo("CUDA Initialisation successfull (Device %d: %s, Thread %d, %lld/%lld bytes used)", fDeviceId, hipDeviceProp_t.name, fThreadId, (long long int) mHostMemorySize, (long long int) mDeviceMemorySize);

	return(0);
}

int AliGPUReconstructionCUDABackend::RunTPCTrackingSlices()
{
	int retVal = RunTPCTrackingSlices_internal();
	if (retVal) SynchronizeGPU();
	if (retVal >= 2)
	{
		ResetHelperThreads(retVal >= 3);
	}
	ReleaseThreadContext();
	return(retVal != 0);
}

int AliGPUReconstructionCUDABackend::RunTPCTrackingSlices_internal()
{
	//Primary reconstruction function
	if (fGPUStuck)
	{
		CAGPUWarning("This GPU is stuck, processing of tracking for this event is skipped!");
		return(1);
	}
	if (Reconstruct_Base_Init()) return(1);
	if (PrepareTextures()) return(2);

	//Copy Tracker Object to GPU Memory
	if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Copying Tracker objects to GPU");
	if (PrepareProfile()) return 2;
	
	WriteToConstantMemory((char*) &mDeviceConstantMem->param - (char*) mDeviceConstantMem, &mParam, sizeof(AliGPUCAParam), mNStreams - 1);
	WriteToConstantMemory((char*) mDeviceConstantMem->tpcTrackers - (char*) mDeviceConstantMem, mWorkersShadow->tpcTrackers, sizeof(AliGPUTPCTracker) * NSLICES, mNStreams - 1, &mEvents.init);
	
	for (int i = 0;i < mNStreams - 1;i++)
	{
		mStreamInit[i] = false;
	}
	mStreamInit[mNStreams - 1] = true;

	if (GPUDebug("Initialization (1)", 0, 0) RANDOM_ERROR)
	{
		return(2);
	}

	for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice++)
	{
		if (Reconstruct_Base_SliceInit(iSlice)) return(1);
		
		int useStream = (iSlice % mNStreams);
		//Initialize temporary memory where needed
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Copying Slice Data to GPU and initializing temporary memory");
		runKernel<AliGPUMemClean16>({fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT, useStream}, krnlRunRangeNone, {}, mWorkersShadow->tpcTrackers[iSlice].Data().HitWeights(), mWorkersShadow->tpcTrackers[iSlice].Data().NumberOfHitsPlusAlign() * sizeof(*mWorkersShadow->tpcTrackers[iSlice].Data().HitWeights()));
		if (GPUDebug("Initialization (2)", 2, iSlice) RANDOM_ERROR)
		{
			return(3);
		}

		//Copy Data to GPU Global Memory
		mWorkers->tpcTrackers[iSlice].StartTimer(0);
		TransferMemoryResourceLinkToGPU(mWorkers->tpcTrackers[iSlice].Data().MemoryResInput(), useStream);
		TransferMemoryResourceLinkToGPU(mWorkers->tpcTrackers[iSlice].Data().MemoryResRows(), useStream);
		TransferMemoryResourceLinkToGPU(mWorkers->tpcTrackers[iSlice].MemoryResCommon(), useStream);

		if (GPUDebug("Initialization (3)", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(0);

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Neighbours Finder (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(1);
		runKernel<AliGPUTPCNeighboursFinder>({GPUCA_ROW_COUNT, GPUCA_GPU_THREAD_COUNT_FINDER, useStream}, {iSlice}, {nullptr, mStreamInit[useStream] ? nullptr : &mEvents.init});
		if (GPUDebug("Neighbours finder", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(1);
		mStreamInit[useStream] = true;

		if (mDeviceProcessingSettings.keepAllMemory)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice].Data(), -1, true);
			memcpy(mWorkers->tpcTrackers[iSlice].LinkTmpMemory(), Res(mWorkers->tpcTrackers[iSlice].Data().MemoryResScratch()).Ptr(), Res(mWorkers->tpcTrackers[iSlice].Data().MemoryResScratch()).Size());
			if (mDeviceProcessingSettings.debugMask & 2) mWorkers->tpcTrackers[iSlice].DumpLinks(mDebugFile);
		}

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Neighbours Cleaner (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(2);
		runKernel<AliGPUTPCNeighboursCleaner>({GPUCA_ROW_COUNT - 2, GPUCA_GPU_THREAD_COUNT, useStream}, {iSlice});
		if (GPUDebug("Neighbours Cleaner", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(2);

		if (mDeviceProcessingSettings.debugLevel >= 4)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice].Data(), -1, true);
			if (mDeviceProcessingSettings.debugMask & 4) mWorkers->tpcTrackers[iSlice].DumpLinks(mDebugFile);
		}

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Start Hits Finder (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(3);
		runKernel<AliGPUTPCStartHitsFinder>({GPUCA_ROW_COUNT - 6, GPUCA_GPU_THREAD_COUNT, useStream}, {iSlice});
		if (GPUDebug("Start Hits Finder", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(3);

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Start Hits Sorter (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(4);
		runKernel<AliGPUTPCStartHitsSorter>({fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT, useStream}, {iSlice});
		if (GPUDebug("Start Hits Sorter", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(4);

		if (mDeviceProcessingSettings.debugLevel >= 2)
		{
			TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[iSlice].MemoryResCommon(), -1);
			if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Obtaining Number of Start Hits from GPU: %d (Slice %d)", *mWorkers->tpcTrackers[iSlice].NTracklets(), iSlice);
			if (*mWorkers->tpcTrackers[iSlice].NTracklets() > GPUCA_GPU_MAX_TRACKLETS RANDOM_ERROR)
			{
				CAGPUError("GPUCA_GPU_MAX_TRACKLETS constant insuffisant");
				return(3);
			}
		}

		if (mDeviceProcessingSettings.debugLevel >= 4 && *mWorkers->tpcTrackers[iSlice].NTracklets())
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice], -1, true);
			if (mDeviceProcessingSettings.debugMask & 32) mWorkers->tpcTrackers[iSlice].DumpStartHits(mDebugFile);
		}

#ifdef GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Tracklet Constructor (Slice %d/%d)", iSlice, NSLICES)
		mWorkers->tpcTrackers[iSlice].StartTimer(6);
		AliGPUTPCTrackletConstructorSingleSlice<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR, 0, mInternals->CudaStreams[useStream]>>>(iSlice);
		if (GPUDebug("Tracklet Constructor", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(6);
#endif
	}
	ReleaseEvent(&mEvents.init);

	for (int i = 0;i < mDeviceProcessingSettings.nDeviceHelperThreads;i++)
	{
		pthread_mutex_lock(&((pthread_mutex_t*) fHelperParams[i].fMutex)[1]);
	}

#ifdef GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE
	SynchronizeGPU();
#else
	if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Tracklet Constructor");
	mWorkers->tpcTrackers[0].StartTimer(6);
	AliGPUTPCTrackletConstructorGPU<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR>>>();
	if (GPUDebug("Tracklet Constructor", -1, 0) RANDOM_ERROR)
	{
		return(1);
	}
	mWorkers->tpcTrackers[0].StopTimer(6);
#endif //GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE
	ReleaseEvent(&mEvents.constructor);

	if (mDeviceProcessingSettings.debugLevel >= 4)
	{
		for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice++)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice], -1, true);
			CAGPUInfo("Obtained %d tracklets", *mWorkers->tpcTrackers[iSlice].NTracklets());
			if (mDeviceProcessingSettings.debugMask & 128) mWorkers->tpcTrackers[iSlice].DumpTrackletHits(mDebugFile);
		}
	}

	unsigned int runSlices = 0;
	int useStream = 0;
	int streamMap[NSLICES];
	for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice += runSlices)
	{
		if (runSlices < GPUCA_GPU_TRACKLET_SELECTOR_SLICE_COUNT) runSlices++;
		runSlices = CAMath::Min(runSlices, NSLICES - iSlice);
		if (fSelectorBlockCount < runSlices) runSlices = fSelectorBlockCount;

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running HLT Tracklet selector (Stream %d, Slice %d to %d)", useStream, iSlice, iSlice + runSlices);
		mWorkers->tpcTrackers[iSlice].StartTimer(7);
		runKernel<AliGPUTPCTrackletSelector>({fSelectorBlockCount, GPUCA_GPU_THREAD_COUNT_SELECTOR, useStream}, {iSlice, runSlices});
		if (GPUDebug("Tracklet Selector", useStream, iSlice) RANDOM_ERROR)
		{
			return(1);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(7);
		for (unsigned int k = iSlice;k < iSlice + runSlices;k++)
		{
			TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[k].MemoryResCommon(), useStream, &mEvents.selector[k]);
			streamMap[k] = useStream;
		}
		useStream++;
		if (useStream >= mNStreams) useStream = 0;
	}

	fSliceOutputReady = 0;

	if (Reconstruct_Base_StartGlobal()) return(1);

	bool transferRunning[NSLICES];
	for (unsigned int i = 0;i < NSLICES;i++) transferRunning[i] = true;
	unsigned int tmpSlice = 0;
	for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice++)
	{
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Transfering Tracks from GPU to Host");

		if (tmpSlice == iSlice) SynchronizeEvents(&mEvents.selector[iSlice]);
		while (tmpSlice < NSLICES && (tmpSlice == iSlice || IsEventDone(&mEvents.selector[tmpSlice])))
		{
			ReleaseEvent(&mEvents.selector[tmpSlice]);
			if (*mWorkers->tpcTrackers[tmpSlice].NTracks() > 0)
			{
				TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[tmpSlice].MemoryResTracks(), streamMap[tmpSlice]);
				TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[tmpSlice].MemoryResTrackHits(), streamMap[tmpSlice], &mEvents.selector[tmpSlice]);
			}
			else
			{
				transferRunning[tmpSlice] = false;
			}
			tmpSlice++;
		}

		if (mDeviceProcessingSettings.keepAllMemory)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice], -1, true);
			if (mDeviceProcessingSettings.debugMask & 256 && !mDeviceProcessingSettings.comparableDebutOutput) mWorkers->tpcTrackers[iSlice].DumpHitWeights(mDebugFile);
			if (mDeviceProcessingSettings.debugMask & 512) mWorkers->tpcTrackers[iSlice].DumpTrackHits(mDebugFile);
		}

		if (mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError RANDOM_ERROR)
		{
			const char* errorMsgs[] = GPUCA_GPU_ERROR_STRINGS;
			const char* errorMsg = (unsigned) mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError >= sizeof(errorMsgs) / sizeof(errorMsgs[0]) ? "UNKNOWN" : errorMsgs[mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError];
			CAGPUError("GPU Tracker returned Error Code %d (%s) in slice %d (Clusters %d)", mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError, errorMsg, iSlice, mWorkers->tpcTrackers[iSlice].Data().NumberOfHits());
			for (unsigned int iSlice2 = 0;iSlice2 < NSLICES;iSlice2++) if (transferRunning[iSlice2]) ReleaseEvent(&mEvents.selector[iSlice2]);
			return(3);
		}

		if (transferRunning[iSlice]) SynchronizeEvents(&mEvents.selector[iSlice]);
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Tracks Transfered: %d / %d", *mWorkers->tpcTrackers[iSlice].NTracks(), *mWorkers->tpcTrackers[iSlice].NTrackHits());
		if (Reconstruct_Base_FinishSlices(iSlice)) return(1);
	}
	for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice++) if (transferRunning[iSlice]) ReleaseEvent(&mEvents.selector[iSlice]);

	if (Reconstruct_Base_Finalize()) return(1);

	if (DoProfile()) return(1);
	
	if (mDeviceProcessingSettings.debugMask & 1024)
	{
		for (unsigned int i = 0;i < NSLICES;i++)
		{
			mWorkers->tpcTrackers[i].DumpOutput(stdout);
		}
	}

	return(0);
}

int AliGPUReconstructionCUDABackend::ExitDevice_Runtime()
{
	//Uninitialize CUDA
	ActivateThreadContext();

	SynchronizeGPU();

	GPUFailedMsgI(hipFree(mDeviceMemoryBase));
	mDeviceMemoryBase = nullptr;

	for (int i = 0;i < mNStreams;i++)
	{
		GPUFailedMsgI(hipStreamDestroy(mInternals->CudaStreams[i]));
	}

	GPUFailedMsgI(hipHostFree(mHostMemoryBase));
	mHostMemoryBase = nullptr;
	
	hipEvent_t *events = (hipEvent_t*) &mEvents;
	for (unsigned int i = 0;i < sizeof(mEvents) & sizeof(hipEvent_t);i++)
	{
		GPUFailedMsgI(hipEventDestroy(events[i]));
	}

	if (GPUFailedMsgI(hipDeviceReset()))
	{
		CAGPUError("Could not uninitialize GPU");
		return(1);
	}

	hipCtxDestroy(mInternals->CudaContext);

	CAGPUInfo("CUDA Uninitialized");
	return(0);
}

int AliGPUReconstructionCUDABackend::DoTRDGPUTracking()
{
#ifndef GPUCA_GPU_MERGER
	CAGPUError("GPUCA_GPU_MERGER compile flag not set");
	return(1);
#else
	ActivateThreadContext();
	SetupGPUProcessor(&mWorkers->trdTracker);
	mWorkersShadow->trdTracker.SetGeometry((AliGPUTRDGeometry*) mProcDevice.fTrdGeometry);

	GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), &mWorkersShadow->trdTracker, sizeof(mWorkersShadow->trdTracker), (char*) &mDeviceConstantMem->trdTracker - (char*) mDeviceConstantMem, hipMemcpyHostToDevice));

	TransferMemoryResourcesToGPU(&mWorkers->trdTracker);

	DoTrdTrackingGPU<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT_TRD>>>();
	SynchronizeGPU();

	TransferMemoryResourcesToHost(&mWorkers->trdTracker);
	SynchronizeGPU();

	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("GPU TRD tracker Finished");

	ReleaseThreadContext();
	return(0);
#endif
}

int AliGPUReconstructionCUDABackend::RefitMergedTracks(AliGPUTPCGMMerger* Merger, bool resetTimers)
{
#ifndef GPUCA_GPU_MERGER
	CAGPUError("GPUCA_GPU_MERGER compile flag not set");
	return(1);
#else

	HighResTimer timer;
	static double times[3] = {};
	static int nCount = 0;
	if (resetTimers)
	{
		for (unsigned int k = 0;k < sizeof(times) / sizeof(times[0]);k++) times[k] = 0;
		nCount = 0;
	}
	ActivateThreadContext();

	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("Running GPU Merger (%d/%d)", Merger->NOutputTrackClusters(), Merger->NClusters());
	timer.Start();

	SetupGPUProcessor(Merger);
	mWorkersShadow->tpcMerger.OverrideSliceTracker(mDeviceConstantMem->tpcTrackers);
	
	GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), &mWorkersShadow->tpcMerger, sizeof(mWorkersShadow->tpcMerger), (char*) &mDeviceConstantMem->tpcMerger - (char*) mDeviceConstantMem, hipMemcpyHostToDevice));
	TransferMemoryResourceLinkToGPU(Merger->MemoryResRefit());
	times[0] += timer.GetCurrentElapsedTime(true);
	
	RefitTracks<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT>>>(mWorkersShadow->tpcMerger.OutputTracks(), mWorkersShadow->tpcMerger.NOutputTracks(), mWorkersShadow->tpcMerger.Clusters());
	SynchronizeGPU();
	times[1] += timer.GetCurrentElapsedTime(true);
	
	TransferMemoryResourceLinkToHost(Merger->MemoryResRefit());
	SynchronizeGPU();
	times[2] += timer.GetCurrentElapsedTime();
	
	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("GPU Merger Finished");
	nCount++;

	if (mDeviceProcessingSettings.debugLevel > 0)
	{
		int copysize = 4 * Merger->NOutputTrackClusters() * sizeof(float) + Merger->NOutputTrackClusters() * sizeof(unsigned int) + Merger->NOutputTracks() * sizeof(AliGPUTPCGMMergedTrack) + 6 * sizeof(float) + sizeof(AliGPUCAParam);
		double speed = (double) copysize / times[0] * nCount / 1e9;
		printf("GPU Fit:\tCopy To:\t%'7d us (%6.3f GB/s)\n", (int) (times[0] * 1000000 / nCount), speed);
		printf("\t\tFit:\t\t%'7d us\n", (int) (times[1] * 1000000 / nCount));
		speed = (double) copysize / times[2] * nCount / 1e9;
		printf("\t\tCopy From:\t%'7d us (%6.3f GB/s)\n", (int) (times[2] * 1000000 / nCount), speed);
	}

	if (!GPUCA_TIMING_SUM)
	{
		for (int i = 0;i < 3;i++) times[i] = 0;
		nCount = 0;
	}

	ReleaseThreadContext();
	return(0);
#endif
}

void AliGPUReconstructionCUDABackend::TransferMemoryResourceToGPU(AliGPUMemoryResource* res, int stream, deviceEvent* ev, deviceEvent* evList, int nEvents)
{
	//if (evList == nullptr) nEvents = 0;
	if (mDeviceProcessingSettings.debugLevel >= 3) stream = -1;
	if (mDeviceProcessingSettings.debugLevel >= 3) printf("Copying to GPU: %s\n", res->Name());
	if (stream == -1)
	{
		GPUFailedMsg(hipMemcpy(res->PtrDevice(), res->Ptr(), res->Size(), hipMemcpyHostToDevice));
	}
	else
	{
		if (evList == nullptr) nEvents = 0;
		for (int k = 0;k < nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[stream], ((hipEvent_t*) evList)[k], 0));
		GPUFailedMsg(hipMemcpyAsync(res->PtrDevice(), res->Ptr(), res->Size(), hipMemcpyHostToDevice, mInternals->CudaStreams[stream]));
		if (ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
	}
}

void AliGPUReconstructionCUDABackend::TransferMemoryResourceToHost(AliGPUMemoryResource* res, int stream, deviceEvent* ev, deviceEvent* evList, int nEvents)
{
	//if (evList == nullptr) nEvents = 0;
	if (mDeviceProcessingSettings.debugLevel >= 3) stream = -1;
	if (mDeviceProcessingSettings.debugLevel >= 3) printf("Copying to Host: %s\n", res->Name());
	if (stream == -1)
	{
		GPUFailedMsg(hipMemcpy(res->Ptr(), res->PtrDevice(), res->Size(), hipMemcpyDeviceToHost));
	}
	else
	{
		if (evList == nullptr) nEvents = 0;
		for (int k = 0;k < nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[stream], ((hipEvent_t*) evList)[k], 0));
		GPUFailedMsg(hipMemcpyAsync(res->Ptr(), res->PtrDevice(), res->Size(), hipMemcpyDeviceToHost, mInternals->CudaStreams[stream]));
		if (ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
	}
}

void AliGPUReconstructionCUDABackend::WriteToConstantMemory(size_t offset, const void* src, size_t size, int stream, deviceEvent* ev)
{
	if (stream == -1) GPUFailedMsg(hipMemcpyToSymbol(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice));
	else GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice, mInternals->CudaStreams[stream]));
	if (ev && stream != -1) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
}

void AliGPUReconstructionCUDABackend::ReleaseEvent(deviceEvent* ev) {}

int AliGPUReconstructionCUDABackend::GPUMergerAvailable() const
{
#ifdef GPUCA_GPU_MERGER
	return(1);
#else
	return(0);
#endif
}

void AliGPUReconstructionCUDABackend::ActivateThreadContext()
{
	hipCtxPushCurrent(mInternals->CudaContext);
}
void AliGPUReconstructionCUDABackend::ReleaseThreadContext()
{
	hipCtxPopCurrent(&mInternals->CudaContext);
}

void AliGPUReconstructionCUDABackend::SynchronizeGPU()
{
	GPUFailedMsg(hipDeviceSynchronize());
}

void AliGPUReconstructionCUDABackend::SynchronizeStream(int stream)
{
	GPUFailedMsg(hipStreamSynchronize(mInternals->CudaStreams[stream]));
}

void AliGPUReconstructionCUDABackend::SynchronizeEvents(deviceEvent* evList, int nEvents)
{
	for (int i = 0;i < nEvents;i++)
	{
		GPUFailedMsg(hipEventSynchronize(((hipEvent_t*) evList)[i]));
	}
}

int AliGPUReconstructionCUDABackend::IsEventDone(deviceEvent* evList, int nEvents)
{
	for (int i = 0;i < nEvents;i++)
	{
		hipError_t retVal = hipEventSynchronize(((hipEvent_t*) evList)[i]);
		if (retVal == hipErrorNotReady) return 0;
		GPUFailedMsg(retVal);
	}
	return(1);
}

int AliGPUReconstructionCUDABackend::GPUDebug(const char* state, int stream, int slice)
{
	//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards, in case of debugmode
	if (mDeviceProcessingSettings.debugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		CAGPUError("Cuda Error %s while running kernel (%s) (Stream %d; Slice %d/%d)", hipGetErrorString(cuErr), state, stream, slice, NSLICES);
		return(1);
	}
	if (GPUFailedMsgI(hipDeviceSynchronize()))
	{
		CAGPUError("CUDA Error while synchronizing (%s) (Stream %d; Slice %d/%d)", state, stream, slice, NSLICES);
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("GPU Sync Done");
	return(0);
}

int AliGPUReconstructionCUDABackend::PrepareTextures()
{
#ifdef GPUCA_GPU_USE_TEXTURES
	hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
	size_t offset;
	GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSLICES * GPUCA_GPU_SLICE_DATA_MEMORY));
	hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
	GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSLICES * GPUCA_GPU_SLICE_DATA_MEMORY));
#endif
	return(0);
}

int AliGPUReconstructionCUDABackend::PrepareProfile()
{
#ifdef GPUCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* tmpMem;
	GPUFailedMsg(hipMalloc(&tmpMem, 100000000));
	mWorkersShadow->tpcTrackers[0].fStageAtSync = tmpMem;
	GPUFailedMsg(hipMemset(mWorkersShadow->tpcTrackers[0].StageAtSync(), 0, 100000000));
#endif
	return 0;
}

int AliGPUReconstructionCUDABackend::DoProfile()
{
#ifdef GPUCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* stageAtSync = (char*) malloc(100000000);
	GPUFailedMsg(hipMemcpy(stageAtSync, mWorkersShadow->tpcTrackers[0].StageAtSync(), 100 * 1000 * 1000, hipMemcpyDeviceToHost));
	hipFree(mWorkersShadow->tpcTrackers[0].StageAtSync());

	FILE* fp = fopen("profile.txt", "w+");
	FILE* fp2 = fopen("profile.bmp", "w+b");
	int nEmptySync = 0, fEmpty;

	const int bmpheight = 8192;
	BITMAPFILEHEADER bmpFH;
	BITMAPINFOHEADER bmpIH;
	ZeroMemory(&bmpFH, sizeof(bmpFH));
	ZeroMemory(&bmpIH, sizeof(bmpIH));

	bmpFH.bfType = 19778; //"BM"
	bmpFH.bfSize = sizeof(bmpFH) + sizeof(bmpIH) + (fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1) * bmpheight ;
	bmpFH.bfOffBits = sizeof(bmpFH) + sizeof(bmpIH);

	bmpIH.biSize = sizeof(bmpIH);
	bmpIH.biWidth = fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1;
	bmpIH.biHeight = bmpheight;
	bmpIH.biPlanes = 1;
	bmpIH.biBitCount = 32;

	fwrite(&bmpFH, 1, sizeof(bmpFH), fp2);
	fwrite(&bmpIH, 1, sizeof(bmpIH), fp2);

	for (int i = 0;i < bmpheight * fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;i += fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR)
	{
		fEmpty = 1;
		for (int j = 0;j < fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;j++)
		{
			fprintf(fp, "%d\t", stageAtSync[i + j]);
			int color = 0;
			if (stageAtSync[i + j] == 1) color = RGB(255, 0, 0);
			if (stageAtSync[i + j] == 2) color = RGB(0, 255, 0);
			if (stageAtSync[i + j] == 3) color = RGB(0, 0, 255);
			if (stageAtSync[i + j] == 4) color = RGB(255, 255, 0);
			fwrite(&color, 1, sizeof(int), fp2);
			if (j > 0 && j % 32 == 0)
			{
				color = RGB(255, 255, 255);
				fwrite(&color, 1, 4, fp2);
			}
			if (stageAtSync[i + j]) fEmpty = 0;
		}
		fprintf(fp, "\n");
		if (fEmpty) nEmptySync++;
		else nEmptySync = 0;
		//if (nEmptySync == GPUCA_GPU_SCHED_ROW_STEP + 2) break;
	}

	fclose(fp);
	fclose(fp2);
	free(stageAtSync);
#endif
	return 0;
}
