#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>
#define GPUCA_GPUTYPE_PASCAL

#include "AliGPUReconstructionCUDA.h"
#include "AliGPUReconstructionCUDAInternals.h"
#include "AliGPUReconstructionCommon.h"

#define get_global_id(dim) (blockIdx.x * blockDim.x + threadIdx.x)
#define get_global_size(dim) (blockDim.x * gridDim.x)
#define get_num_groups(dim) (gridDim.x)
#define get_local_id(dim) (threadIdx.x)
#define get_local_size(dim) (blockDim.x)
#define get_group_id(dim) (blockIdx.x)

__constant__ uint4 gGPUConstantMemBuffer[(sizeof(AliGPUCAConstantMem) + sizeof(uint4) - 1) / sizeof(uint4)];
__constant__ char& gGPUConstantMemBufferChar = (char&) gGPUConstantMemBuffer;
__constant__ AliGPUCAConstantMem& gGPUConstantMem = (AliGPUCAConstantMem&) gGPUConstantMemBufferChar;

#ifdef GPUCA_GPU_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

#ifdef HAVE_O2HEADERS
#include "ITStrackingCUDA/TrackerTraitsNV.h"
#else
namespace o2 { namespace ITS { class TrackerTraitsNV : public TrackerTraits {}; }}
#endif

#include "AliGPUDeviceKernels.h"

template <class TProcess, typename... Args> GPUg() void runKernelCUDA(int iSlice, Args... args)
{
	AliGPUTPCTracker &tracker = gGPUConstantMem.tpcTrackers[iSlice];
	GPUshared() typename TProcess::AliGPUTPCSharedMemory smem;

	for (int iSync = 0; iSync <= TProcess::NThreadSyncPoints(); iSync++)
	{
		GPUsync();
		TProcess::Thread(get_num_groups(0), get_local_size(0), get_group_id(0), get_local_id(0), iSync, smem, tracker, args...);
	}
}

template <class TProcess, typename... Args> GPUg() void runKernelCUDAMulti(int firstSlice, int nSliceCount, Args... args)
{
	const int iSlice = nSliceCount * (get_group_id(0) + (get_num_groups(0) % nSliceCount != 0 && nSliceCount * (get_group_id(0) + 1) % get_num_groups(0) != 0)) / get_num_groups(0);
	const int nSliceBlockOffset = get_num_groups(0) * iSlice / nSliceCount;
	const int sliceBlockId = get_group_id(0) - nSliceBlockOffset;
	const int sliceGridDim = get_num_groups(0) * (iSlice + 1) / nSliceCount - get_num_groups(0) * (iSlice) / nSliceCount;
	AliGPUTPCTracker &tracker = gGPUConstantMem.tpcTrackers[firstSlice + iSlice];
	GPUshared() typename TProcess::AliGPUTPCSharedMemory smem;

	for (int iSync = 0; iSync <= TProcess::NThreadSyncPoints(); iSync++)
	{
		GPUsync();
		TProcess::Thread(sliceGridDim, get_local_size(0), sliceBlockId, get_local_id(0), iSync, smem, tracker, args...);
	}
}

template <class T, typename... Args> int AliGPUReconstructionCUDABackend::runKernelBackend(const krnlExec& x, const krnlRunRange& y, const Args&... args)
{
	if (x.device == krnlDeviceType::CPU) return AliGPUReconstructionCPU::runKernelBackend<T> (x, y, args...);
	if (y.num <= 1)
	{
		runKernelCUDA<T> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>>(y.start, args...);
	}
	else
	{
		runKernelCUDAMulti<T> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>> (y.start, y.num, args...);
	}
	return 0;
}

AliGPUReconstructionCUDABackend::AliGPUReconstructionCUDABackend(const AliGPUCASettingsProcessing& cfg) : AliGPUReconstructionDeviceBase(cfg)
{
	mInternals = new AliGPUReconstructionCUDAInternals;
	mProcessingSettings.deviceType = CUDA;
	mITSTrackerTraits.reset(new o2::ITS::TrackerTraitsNV);
}

AliGPUReconstructionCUDABackend::~AliGPUReconstructionCUDABackend()
{
	mITSTrackerTraits.reset(nullptr); //Make sure we destroy the ITS tracker before we exit CUDA
	hipDeviceReset();
	delete mInternals;
}

AliGPUReconstruction* AliGPUReconstruction_Create_CUDA(const AliGPUCASettingsProcessing& cfg)
{
	return new AliGPUReconstructionCUDA(cfg);
}

int AliGPUReconstructionCUDABackend::InitDevice_Runtime()
{
	//Find best CUDA device, initialize and allocate memory

	hipDeviceProp_t hipDeviceProp_t;

	int count, bestDevice = -1;
	double bestDeviceSpeed = -1, deviceSpeed;
	if (GPUFailedMsg(hipGetDeviceCount(&count)))
	{
		CAGPUError("Error getting CUDA Device Count");
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("Available CUDA devices:");
	const int reqVerMaj = 2;
	const int reqVerMin = 0;
	for (int i = 0;i < count;i++)
	{
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Examining device %d\n", i);
		size_t free, total;
		hipInit(0);
		hipDevice_t tmpDevice;
		hipDeviceGet(&tmpDevice, i);
		hipCtx_t tmpContext;
		hipCtxCreate(&tmpContext, 0, tmpDevice);
		if(hipMemGetInfo(&free, &total)) std::cout << "Error\n";
		hipCtxDestroy(tmpContext);
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained current memory usage for device %d\n", i);
		if (GPUFailedMsg(hipGetDeviceProperties(&hipDeviceProp_t, i))) continue;
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained device properties for device %d\n", i);
		int deviceOK = true;
		const char* deviceFailure = "";
		if (hipDeviceProp_t.major >= 9) {deviceOK = false; deviceFailure = "Invalid Revision";}
		else if (hipDeviceProp_t.major < reqVerMaj || (hipDeviceProp_t.major == reqVerMaj && hipDeviceProp_t.minor < reqVerMin)) {deviceOK = false; deviceFailure = "Too low device revision";}
		else if (free < mDeviceMemorySize) {deviceOK = false; deviceFailure = "Insufficient GPU memory";}

		deviceSpeed = (double) hipDeviceProp_t.multiProcessorCount * (double) hipDeviceProp_t.clockRate * (double) hipDeviceProp_t.warpSize * (double) free * (double) hipDeviceProp_t.major * (double) hipDeviceProp_t.major;
		if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lld / %lld)%s %s", deviceOK ? " " : "[", i, hipDeviceProp_t.name, hipDeviceProp_t.major, hipDeviceProp_t.minor, (long long int) free, (long long int) hipDeviceProp_t.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
		if (!deviceOK) continue;
		if (deviceSpeed > bestDeviceSpeed)
		{
			bestDevice = i;
			bestDeviceSpeed = deviceSpeed;
		}
		else
		{
			if (mDeviceProcessingSettings.debugLevel >= 0) CAGPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
		}
	}
	if (bestDevice == -1)
	{
		CAGPUWarning("No %sCUDA Device available, aborting CUDA Initialisation", count ? "appropriate " : "");
		CAGPUImportant("Requiring Revision %d.%d, Mem: %lld", reqVerMaj, reqVerMin, (long long int) mDeviceMemorySize);
		return(1);
	}

	if (mDeviceProcessingSettings.deviceNum > -1)
	{
		if (mDeviceProcessingSettings.deviceNum < (signed) count)
		{
			bestDevice = mDeviceProcessingSettings.deviceNum;
		}
		else
		{
			CAGPUWarning("Requested device ID %d non existend, falling back to default device id %d", mDeviceProcessingSettings.deviceNum, bestDevice);
		}
	}
	fDeviceId = bestDevice;

	hipGetDeviceProperties(&hipDeviceProp_t ,fDeviceId );

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		CAGPUInfo("Using CUDA Device %s with Properties:", hipDeviceProp_t.name);
		CAGPUInfo("totalGlobalMem = %lld", (unsigned long long int) hipDeviceProp_t.totalGlobalMem);
		CAGPUInfo("sharedMemPerBlock = %lld", (unsigned long long int) hipDeviceProp_t.sharedMemPerBlock);
		CAGPUInfo("regsPerBlock = %d", hipDeviceProp_t.regsPerBlock);
		CAGPUInfo("warpSize = %d", hipDeviceProp_t.warpSize);
		CAGPUInfo("memPitch = %lld", (unsigned long long int) hipDeviceProp_t.memPitch);
		CAGPUInfo("maxThreadsPerBlock = %d", hipDeviceProp_t.maxThreadsPerBlock);
		CAGPUInfo("maxThreadsDim = %d %d %d", hipDeviceProp_t.maxThreadsDim[0], hipDeviceProp_t.maxThreadsDim[1], hipDeviceProp_t.maxThreadsDim[2]);
		CAGPUInfo("maxGridSize = %d %d %d", hipDeviceProp_t.maxGridSize[0], hipDeviceProp_t.maxGridSize[1], hipDeviceProp_t.maxGridSize[2]);
		CAGPUInfo("totalConstMem = %lld", (unsigned long long int) hipDeviceProp_t.totalConstMem);
		CAGPUInfo("major = %d", hipDeviceProp_t.major);
		CAGPUInfo("minor = %d", hipDeviceProp_t.minor);
		CAGPUInfo("clockRate = %d", hipDeviceProp_t.clockRate);
		CAGPUInfo("memoryClockRate = %d", hipDeviceProp_t.memoryClockRate);
		CAGPUInfo("multiProcessorCount = %d", hipDeviceProp_t.multiProcessorCount);
		CAGPUInfo("textureAlignment = %lld", (unsigned long long int) hipDeviceProp_t.textureAlignment);
	}
#ifdef GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE
	fConstructorBlockCount = hipDeviceProp_t.multiProcessorCount;
#else
	fConstructorBlockCount = hipDeviceProp_t.multiProcessorCount * GPUCA_GPU_BLOCK_COUNT_CONSTRUCTOR_MULTIPLIER;
#endif
	fConstructorThreadCount = GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;
	fSelectorBlockCount = hipDeviceProp_t.multiProcessorCount * GPUCA_GPU_BLOCK_COUNT_SELECTOR_MULTIPLIER;

	if (hipDeviceProp_t.major < 1 || (hipDeviceProp_t.major == 1 && hipDeviceProp_t.minor < 2))
	{
		CAGPUError( "Unsupported CUDA Device" );
		return(1);
	}

#ifdef GPUCA_GPU_USE_TEXTURES
	if (GPUCA_GPU_SLICE_DATA_MEMORY * NSLICES > (size_t) hipDeviceProp_t.maxTexture1DLinear)
	{
		CAGPUError("Invalid maximum texture size of device: %lld < %lld\n", (long long int) hipDeviceProp_t.maxTexture1DLinear, (long long int) (GPUCA_GPU_SLICE_DATA_MEMORY * NSLICES));
		return(1);
	}
#endif

	int nStreams = GPUCA_GPU_NUM_STREAMS == 0 ? 3 : GPUCA_GPU_NUM_STREAMS;
	if (nStreams < 3)
	{
		CAGPUError("Invalid number of streams");
		return(1);
	}

	if (hipCtxCreate(&mInternals->CudaContext, hipDeviceScheduleAuto, fDeviceId) != hipSuccess)
	{
		CAGPUError("Could not set CUDA Device!");
		return(1);
	}

	if (mDeviceMemorySize > hipDeviceProp_t.totalGlobalMem || GPUFailedMsg(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize)))
	{
		CAGPUError("CUDA Memory Allocation Error");
		hipDeviceReset();
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) CAGPUInfo("GPU Memory used: %lld", (long long int) mDeviceMemorySize);
	if (GPUFailedMsg(hipHostMalloc(&mHostMemoryBase, mHostMemorySize)))
	{
		hipFree(mDeviceMemoryBase);
		hipDeviceReset();
		CAGPUError("Error allocating Page Locked Host Memory");
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) CAGPUInfo("Host Memory used: %lld", (long long int) mHostMemorySize);

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		memset(mHostMemoryBase, 0, mHostMemorySize);
		if (GPUFailedMsg(hipMemset(mDeviceMemoryBase, 143, mDeviceMemorySize)))
		{
			hipFree(mDeviceMemoryBase);
			hipDeviceReset();
			CAGPUError("Error during CUDA memset");
			return(1);
		}
	}

	mInternals->CudaStreams = (hipStream_t*) malloc(nStreams * sizeof(hipStream_t));
	for (int i = 0;i < nStreams;i++)
	{
		if (GPUFailedMsg(hipStreamCreate(&mInternals->CudaStreams[i])))
		{
			hipFree(mDeviceMemoryBase);
			hipHostFree(mHostMemoryBase);
			hipDeviceReset();
			CAGPUError("Error creating CUDA Stream");
			return(1);
		}
	}
	
	void* devPtrConstantMem;
	if (GPUFailedMsg(hipGetSymbolAddress(&devPtrConstantMem, HIP_SYMBOL(gGPUConstantMemBuffer))))
	{
		CAGPUError("Error getting ptr to constant memory");
		ResetHelperThreads(0);
		return 1;
	}
	mDeviceConstantMem = (AliGPUCAConstantMem*) devPtrConstantMem;

	ReleaseThreadContext();
	CAGPUInfo("CUDA Initialisation successfull (Device %d: %s, Thread %d, %lld/%lld bytes used)", fDeviceId, hipDeviceProp_t.name, fThreadId, (long long int) mHostMemorySize, (long long int) mDeviceMemorySize);

	return(0);
}

int AliGPUReconstructionCUDABackend::GPUSync(const char* state, int stream, int slice)
{
	//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards

	if (mDeviceProcessingSettings.debugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		CAGPUError("Cuda Error %s while running kernel (%s) (Stream %d; %d/%d)", hipGetErrorString(cuErr), state, stream, slice, NSLICES);
		return(1);
	}
	if (SynchronizeGPU())
	{
		CAGPUError("CUDA Error while synchronizing (%s) (Stream %d; %d/%d)", state, stream, slice, NSLICES);
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("GPU Sync Done");
	return(0);
}

__global__ void PreInitRowBlocks(int* const SliceDataHitWeights, int nSliceDataHits)
{
	//Initialize GPU RowBlocks and HitWeights
	int4* const sliceDataHitWeights4 = (int4*) SliceDataHitWeights;
	const int stride = get_global_size(0);
	int4 i0;
	i0.x = i0.y = i0.z = i0.w = 0;
	for (int i = get_global_id(0);i < nSliceDataHits * sizeof(int) / sizeof(int4);i += stride)
		sliceDataHitWeights4[i] = i0;
}

int AliGPUReconstructionCUDABackend::RunTPCTrackingSlices()
{
	int retVal = RunTPCTrackingSlices_internal();
	if (retVal) SynchronizeGPU();
	if (retVal >= 2)
	{
		ResetHelperThreads(retVal >= 3);
	}
	ReleaseThreadContext();
	return(retVal != 0);
}

int AliGPUReconstructionCUDABackend::RunTPCTrackingSlices_internal()
{
	//Primary reconstruction function
	if (fGPUStuck)
	{
		CAGPUWarning("This GPU is stuck, processing of tracking for this event is skipped!");
		return(1);
	}
	if (Reconstruct_Base_Init()) return(1);
	if (PrepareTextures()) return(2);

	//Copy Tracker Object to GPU Memory
	if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Copying Tracker objects to GPU");
	if (PrepareProfile()) return 2;
	if (GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), &mParam, sizeof(AliGPUCAParam), (char*) &mDeviceConstantMem->param - (char*) mDeviceConstantMem, hipMemcpyHostToDevice, mInternals->CudaStreams[0])))
	{
		CAGPUError("Error writing to constant memory");
		return(2);
	}
	
	if (GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), mWorkersShadow->tpcTrackers, sizeof(AliGPUTPCTracker) * NSLICES, (char*) mDeviceConstantMem->tpcTrackers - (char*) mDeviceConstantMem, hipMemcpyHostToDevice, mInternals->CudaStreams[0])))
	{
		CAGPUError("Error writing to constant memory");
		return(2);
	}
	
	bool globalSymbolDone = false;
	if (GPUSync("Initialization (1)", 0, 0) RANDOM_ERROR)
	{
		return(2);
	}

	for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice++)
	{
		if (Reconstruct_Base_SliceInit(iSlice)) return(1);
		
		int useStream = GPUCA_GPU_NUM_STREAMS == 0 ? (iSlice & 1) : (iSlice % GPUCA_GPU_NUM_STREAMS);
		//Initialize temporary memory where needed
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Copying Slice Data to GPU and initializing temporary memory");
		PreInitRowBlocks<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT, 0, mInternals->CudaStreams[GPUCA_GPU_NUM_STREAMS == 0 ? 2 : useStream]>>>(mWorkersShadow->tpcTrackers[iSlice].Data().HitWeights(), mWorkers->tpcTrackers[iSlice].Data().NumberOfHitsPlusAlign());
		if (GPUSync("Initialization (2)", 2, iSlice) RANDOM_ERROR)
		{
			return(3);
		}

		//Copy Data to GPU Global Memory
		mWorkers->tpcTrackers[iSlice].StartTimer(0);
		if (TransferMemoryResourceLinkToGPU(mWorkers->tpcTrackers[iSlice].Data().MemoryResInput(), useStream) ||
			TransferMemoryResourceLinkToGPU(mWorkers->tpcTrackers[iSlice].Data().MemoryResRows(), useStream) ||
			TransferMemoryResourceLinkToGPU(mWorkers->tpcTrackers[iSlice].MemoryResCommon(), useStream))
		{
			CAGPUError("Error copying data to GPU");
			return(3);
		}

		if (GPUCA_GPU_NUM_STREAMS && useStream && globalSymbolDone == false)
		{
			hipStreamSynchronize(mInternals->CudaStreams[0]);
			globalSymbolDone = true;
		}

		if (GPUSync("Initialization (3)", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(0);

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Neighbours Finder (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(1);
		runKernel<AliGPUTPCNeighboursFinder>({GPUCA_ROW_COUNT, GPUCA_GPU_THREAD_COUNT_FINDER, useStream}, {iSlice});
		if (GPUSync("Neighbours finder", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(1);

		if (mDeviceProcessingSettings.keepAllMemory)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice].Data(), -1, true);
			memcpy(mWorkers->tpcTrackers[iSlice].LinkTmpMemory(), Res(mWorkers->tpcTrackers[iSlice].Data().MemoryResScratch()).Ptr(), Res(mWorkers->tpcTrackers[iSlice].Data().MemoryResScratch()).Size());
			if (mDeviceProcessingSettings.debugMask & 2) mWorkers->tpcTrackers[iSlice].DumpLinks(mDebugFile);
		}

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Neighbours Cleaner (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(2);
		runKernel<AliGPUTPCNeighboursCleaner>({GPUCA_ROW_COUNT - 2, GPUCA_GPU_THREAD_COUNT, useStream}, {iSlice});
		if (GPUSync("Neighbours Cleaner", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(2);

		if (mDeviceProcessingSettings.debugLevel >= 4)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice].Data(), -1, true);
			if (mDeviceProcessingSettings.debugMask & 4) mWorkers->tpcTrackers[iSlice].DumpLinks(mDebugFile);
		}

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Start Hits Finder (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(3);
		runKernel<AliGPUTPCStartHitsFinder>({GPUCA_ROW_COUNT - 6, GPUCA_GPU_THREAD_COUNT, useStream}, {iSlice});
		
		if (GPUSync("Start Hits Finder", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(3);

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Start Hits Sorter (Slice %d/%d)", iSlice, NSLICES);
		mWorkers->tpcTrackers[iSlice].StartTimer(4);
		runKernel<AliGPUTPCStartHitsSorter>({fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT, useStream}, {iSlice});
		if (GPUSync("Start Hits Sorter", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(4);

		if (mDeviceProcessingSettings.debugLevel >= 2)
		{
			TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[iSlice].MemoryResCommon(), -1);
			if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Obtaining Number of Start Hits from GPU: %d (Slice %d)", *mWorkers->tpcTrackers[iSlice].NTracklets(), iSlice);
			if (*mWorkers->tpcTrackers[iSlice].NTracklets() > GPUCA_GPU_MAX_TRACKLETS RANDOM_ERROR)
			{
				CAGPUError("GPUCA_GPU_MAX_TRACKLETS constant insuffisant");
				return(3);
			}
		}

		if (mDeviceProcessingSettings.debugLevel >= 4 && *mWorkers->tpcTrackers[iSlice].NTracklets())
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice], -1, true);
			if (mDeviceProcessingSettings.debugMask & 32) mWorkers->tpcTrackers[iSlice].DumpStartHits(mDebugFile);
		}

#ifdef GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Tracklet Constructor (Slice %d/%d)", iSlice, NSLICES)
		mWorkers->tpcTrackers[iSlice].StartTimer(6);
		AliGPUTPCTrackletConstructorSingleSlice<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR, 0, mInternals->CudaStreams[useStream]>>>(iSlice);
		if (GPUSync("Tracklet Constructor", useStream, iSlice) RANDOM_ERROR)
		{
			return(3);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(6);
#endif
	}

	for (int i = 0;i < mDeviceProcessingSettings.nDeviceHelperThreads;i++)
	{
		pthread_mutex_lock(&((pthread_mutex_t*) fHelperParams[i].fMutex)[1]);
	}

#ifdef GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE
	SynchronizeGPU();
#else
	if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running GPU Tracklet Constructor");
	mWorkers->tpcTrackers[0].StartTimer(6);
	AliGPUTPCTrackletConstructorGPU<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR>>>();
	if (GPUSync("Tracklet Constructor", -1, 0) RANDOM_ERROR)
	{
		return(1);
	}
	mWorkers->tpcTrackers[0].StopTimer(6);
#endif //GPUCA_GPU_CONSTRUCTOR_SINGLE_SLICE

	if (mDeviceProcessingSettings.debugLevel >= 4)
	{
		for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice++)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice], -1, true);
			CAGPUInfo("Obtained %d tracklets", *mWorkers->tpcTrackers[iSlice].NTracklets());
			if (mDeviceProcessingSettings.debugMask & 128) mWorkers->tpcTrackers[iSlice].DumpTrackletHits(mDebugFile);
		}
	}

	unsigned int runSlices = 0;
	int useStream = 0;
	int streamMap[NSLICES];
	for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice += runSlices)
	{
		if (runSlices < GPUCA_GPU_TRACKLET_SELECTOR_SLICE_COUNT) runSlices++;
		runSlices = CAMath::Min(runSlices, NSLICES - iSlice);
		if (fSelectorBlockCount < runSlices) runSlices = fSelectorBlockCount;
		if (GPUCA_GPU_NUM_STREAMS && useStream + 1 == GPUCA_GPU_NUM_STREAMS) runSlices = NSLICES - iSlice;
		if (fSelectorBlockCount < runSlices)
		{
			CAGPUError("Insufficient number of blocks for tracklet selector");
			return(1);
		}

		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Running HLT Tracklet selector (Stream %d, Slice %d to %d)", useStream, iSlice, iSlice + runSlices);
		mWorkers->tpcTrackers[iSlice].StartTimer(7);
		runKernel<AliGPUTPCTrackletSelector>({fSelectorBlockCount, GPUCA_GPU_THREAD_COUNT_SELECTOR, useStream}, {iSlice, runSlices});

		if (GPUSync("Tracklet Selector", iSlice, iSlice) RANDOM_ERROR)
		{
			return(1);
		}
		mWorkers->tpcTrackers[iSlice].StopTimer(7);
		for (unsigned int k = iSlice;k < iSlice + runSlices;k++)
		{
			if (TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[k].MemoryResCommon(), useStream) RANDOM_ERROR)
			{
				return(3);
			}
			streamMap[k] = useStream;
		}
		useStream++;
	}

	fSliceOutputReady = 0;

	if (Reconstruct_Base_StartGlobal()) return(1);

	unsigned int tmpSlice = 0;
	for (unsigned int iSlice = 0;iSlice < NSLICES;iSlice++)
	{
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Transfering Tracks from GPU to Host");

		while (tmpSlice < NSLICES && (tmpSlice == iSlice ? hipStreamSynchronize(mInternals->CudaStreams[streamMap[tmpSlice]]) : hipStreamQuery(mInternals->CudaStreams[streamMap[tmpSlice]])) == (hipError_t) hipSuccess)
		{
			if (*mWorkers->tpcTrackers[tmpSlice].NTracks() > 0)
			{
				useStream = GPUCA_GPU_NUM_STREAMS ? streamMap[tmpSlice] : tmpSlice;
				TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[tmpSlice].MemoryResTracks(), useStream);
				TransferMemoryResourceLinkToHost(mWorkers->tpcTrackers[tmpSlice].MemoryResTrackHits(), useStream);
			}
			tmpSlice++;
		}

		useStream = GPUCA_GPU_NUM_STREAMS ? streamMap[iSlice] : iSlice;
		if (GPUFailedMsg(hipStreamSynchronize(mInternals->CudaStreams[useStream])) RANDOM_ERROR)
		{
			return(3);
		}

		if (mDeviceProcessingSettings.keepAllMemory)
		{
			TransferMemoryResourcesToHost(&mWorkers->tpcTrackers[iSlice], -1, true);
			if (mDeviceProcessingSettings.debugMask & 256 && !mDeviceProcessingSettings.comparableDebutOutput) mWorkers->tpcTrackers[iSlice].DumpHitWeights(mDebugFile);
			if (mDeviceProcessingSettings.debugMask & 512) mWorkers->tpcTrackers[iSlice].DumpTrackHits(mDebugFile);
		}

		if (mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError RANDOM_ERROR)
		{
			const char* errorMsgs[] = GPUCA_GPU_ERROR_STRINGS;
			const char* errorMsg = (unsigned) mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError >= sizeof(errorMsgs) / sizeof(errorMsgs[0]) ? "UNKNOWN" : errorMsgs[mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError];
			CAGPUError("GPU Tracker returned Error Code %d (%s) in slice %d (Clusters %d)", mWorkers->tpcTrackers[iSlice].GPUParameters()->fGPUError, errorMsg, iSlice, mWorkers->tpcTrackers[iSlice].Data().NumberOfHits());

			return(3);
		}
		if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("Tracks Transfered: %d / %d", *mWorkers->tpcTrackers[iSlice].NTracks(), *mWorkers->tpcTrackers[iSlice].NTrackHits());

		if (Reconstruct_Base_FinishSlices(iSlice)) return(1);
	}
	if (Reconstruct_Base_Finalize()) return(1);

	if (DoProfile()) return(1);
	
	if (mDeviceProcessingSettings.debugMask & 1024)
	{
		for (unsigned int i = 0;i < NSLICES;i++)
		{
			mWorkers->tpcTrackers[i].DumpOutput(stdout);
		}
	}

	return(0);
}

int AliGPUReconstructionCUDABackend::ExitDevice_Runtime()
{
	//Uninitialize CUDA
	ActivateThreadContext();

	SynchronizeGPU();
	if (mDeviceMemoryBase)
	{
		hipFree(mDeviceMemoryBase);
		mDeviceMemoryBase = nullptr;
	}
	if (mHostMemoryBase)
	{
		int nStreams = GPUCA_GPU_NUM_STREAMS == 0 ? 3 : GPUCA_GPU_NUM_STREAMS;
		for (int i = 0;i < nStreams;i++)
		{
			hipStreamDestroy(mInternals->CudaStreams[i]);
		}
		free(mInternals->CudaStreams);
		hipHostFree(mHostMemoryBase);
		mHostMemoryBase = nullptr;
	}

	if (GPUFailedMsg(hipDeviceReset()))
	{
		CAGPUError("Could not uninitialize GPU");
		return(1);
	}

	hipCtxDestroy(mInternals->CudaContext);

	CAGPUInfo("CUDA Uninitialized");
	return(0);
}

int AliGPUReconstructionCUDABackend::DoTRDGPUTracking()
{
#ifndef GPUCA_GPU_MERGER
	CAGPUError("GPUCA_GPU_MERGER compile flag not set");
	return(1);
#else
	hipCtxPushCurrent(mInternals->CudaContext);
	SetupGPUProcessor(&mWorkers->trdTracker);
	mWorkersShadow->trdTracker.SetGeometry((AliGPUTRDGeometry*) mProcDevice.fTrdGeometry);

	GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), &mWorkersShadow->trdTracker, sizeof(mWorkersShadow->trdTracker), (char*) &mDeviceConstantMem->trdTracker - (char*) mDeviceConstantMem, hipMemcpyHostToDevice));

	TransferMemoryResourcesToGPU(&mWorkers->trdTracker);

	DoTrdTrackingGPU<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT_TRD>>>();
	GPUFailedMsg(hipDeviceSynchronize());

	TransferMemoryResourcesToHost(&mWorkers->trdTracker);
	GPUFailedMsg(hipDeviceSynchronize());

	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("GPU TRD tracker Finished");


	hipCtxPopCurrent((hipCtx_t*) &mInternals->CudaContext);
	return(0);
#endif
}

int AliGPUReconstructionCUDABackend::RefitMergedTracks(AliGPUTPCGMMerger* Merger, bool resetTimers)
{
#ifndef GPUCA_GPU_MERGER
	CAGPUError("GPUCA_GPU_MERGER compile flag not set");
	return(1);
#else

	HighResTimer timer;
	static double times[3] = {};
	static int nCount = 0;
	if (resetTimers)
	{
		for (unsigned int k = 0;k < sizeof(times) / sizeof(times[0]);k++) times[k] = 0;
		nCount = 0;
	}
	ActivateThreadContext();

	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("Running GPU Merger (%d/%d)", Merger->NOutputTrackClusters(), Merger->NClusters());
	timer.Start();

	SetupGPUProcessor(Merger);
	mWorkersShadow->tpcMerger.OverrideSliceTracker(mDeviceConstantMem->tpcTrackers);
	
	GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), &mWorkersShadow->tpcMerger, sizeof(mWorkersShadow->tpcMerger), (char*) &mDeviceConstantMem->tpcMerger - (char*) mDeviceConstantMem, hipMemcpyHostToDevice));
	TransferMemoryResourceLinkToGPU(Merger->MemoryResRefit());
	times[0] += timer.GetCurrentElapsedTime(true);
	
	RefitTracks<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT>>>(mWorkersShadow->tpcMerger.OutputTracks(), mWorkersShadow->tpcMerger.NOutputTracks(), mWorkersShadow->tpcMerger.Clusters());
	if (SynchronizeGPU()) return(1);
	times[1] += timer.GetCurrentElapsedTime(true);
	
	TransferMemoryResourceLinkToHost(Merger->MemoryResRefit());
	if (SynchronizeGPU()) return(1);
	times[2] += timer.GetCurrentElapsedTime();
	
	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("GPU Merger Finished");
	nCount++;

	if (mDeviceProcessingSettings.debugLevel > 0)
	{
		int copysize = 4 * Merger->NOutputTrackClusters() * sizeof(float) + Merger->NOutputTrackClusters() * sizeof(unsigned int) + Merger->NOutputTracks() * sizeof(AliGPUTPCGMMergedTrack) + 6 * sizeof(float) + sizeof(AliGPUCAParam);
		double speed = (double) copysize / times[0] * nCount / 1e9;
		printf("GPU Fit:\tCopy To:\t%'7d us (%6.3f GB/s)\n", (int) (times[0] * 1000000 / nCount), speed);
		printf("\t\tFit:\t\t%'7d us\n", (int) (times[1] * 1000000 / nCount));
		speed = (double) copysize / times[2] * nCount / 1e9;
		printf("\t\tCopy From:\t%'7d us (%6.3f GB/s)\n", (int) (times[2] * 1000000 / nCount), speed);
	}

	if (!GPUCA_TIMING_SUM)
	{
		for (int i = 0;i < 3;i++) times[i] = 0;
		nCount = 0;
	}

	ReleaseThreadContext();
	return(0);
#endif
}

int AliGPUReconstructionCUDABackend::TransferMemoryResourceToGPU(AliGPUMemoryResource* res, int stream, int nEvents, deviceEvent* evList, deviceEvent* ev)
{
	if (mDeviceProcessingSettings.debugLevel >= 3) stream = -1;
	if (mDeviceProcessingSettings.debugLevel >= 3) printf("Copying to GPU: %s\n", res->Name());
	if (stream == -1) return GPUFailedMsg(hipMemcpy(res->PtrDevice(), res->Ptr(), res->Size(), hipMemcpyHostToDevice));
	else return GPUFailedMsg(hipMemcpyAsync(res->PtrDevice(), res->Ptr(), res->Size(), hipMemcpyHostToDevice, mInternals->CudaStreams[stream]));
}

int AliGPUReconstructionCUDABackend::TransferMemoryResourceToHost(AliGPUMemoryResource* res, int stream, int nEvents, deviceEvent* evList, deviceEvent* ev)
{
	if (mDeviceProcessingSettings.debugLevel >= 3) stream = -1;
	if (mDeviceProcessingSettings.debugLevel >= 3) printf("Copying to Host: %s\n", res->Name());
	if (stream == -1) return GPUFailedMsg(hipMemcpy(res->Ptr(), res->PtrDevice(), res->Size(), hipMemcpyDeviceToHost));
	return GPUFailedMsg(hipMemcpyAsync(res->Ptr(), res->PtrDevice(), res->Size(), hipMemcpyDeviceToHost, mInternals->CudaStreams[stream]));
}

int AliGPUReconstructionCUDABackend::GPUMergerAvailable() const
{
#ifdef GPUCA_GPU_MERGER
	return(1);
#else
	return(0);
#endif
}

void AliGPUReconstructionCUDABackend::ActivateThreadContext()
{
	hipCtxPushCurrent(mInternals->CudaContext);
}
void AliGPUReconstructionCUDABackend::ReleaseThreadContext()
{
	hipCtxPopCurrent(&mInternals->CudaContext);
}

int AliGPUReconstructionCUDABackend::SynchronizeGPU()
{
	GPUFailedMsg(hipDeviceSynchronize());
	return(0);
}

int AliGPUReconstructionCUDABackend::PrepareTextures()
{
#ifdef GPUCA_GPU_USE_TEXTURES
	hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
	size_t offset;
	if (GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSLICES * GPUCA_GPU_SLICE_DATA_MEMORY)) || offset RANDOM_ERROR)
	{
		CAGPUError("Error binding CUDA Texture cahit2 (Offset %d)", (int) offset);
		return(2);
	}
	hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
	if (GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSLICES * GPUCA_GPU_SLICE_DATA_MEMORY)) || offset RANDOM_ERROR)
	{
		CAGPUError("Error binding CUDA Texture calink (Offset %d)", (int) offset);
		return(2);
	}
#endif
	return(0);
}

int AliGPUReconstructionCUDABackend::PrepareProfile()
{
#ifdef GPUCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* tmpMem;
	if (GPUFailedMsg(hipMalloc(&tmpMem, 100000000)))
	{
		CAGPUError("Error allocating CUDA profile memory");
		return(2);
	}
	mWorkersShadow->tpcTrackers[0].fStageAtSync = tmpMem;
	if (GPUFailedMsg(hipMemset(mWorkersShadow->tpcTrackers[0].StageAtSync(), 0, 100000000)))
	{
		CAGPUError("Error clearing stageatsync");
		return(2);
	}
#endif
	return 0;
}

int AliGPUReconstructionCUDABackend::DoProfile()
{
#ifdef GPUCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* stageAtSync = (char*) malloc(100000000);
	GPUFailedMsg(hipMemcpy(stageAtSync, mWorkersShadow->tpcTrackers[0].StageAtSync(), 100 * 1000 * 1000, hipMemcpyDeviceToHost));
	hipFree(mWorkersShadow->tpcTrackers[0].StageAtSync());

	FILE* fp = fopen("profile.txt", "w+");
	FILE* fp2 = fopen("profile.bmp", "w+b");
	int nEmptySync = 0, fEmpty;

	const int bmpheight = 8192;
	BITMAPFILEHEADER bmpFH;
	BITMAPINFOHEADER bmpIH;
	ZeroMemory(&bmpFH, sizeof(bmpFH));
	ZeroMemory(&bmpIH, sizeof(bmpIH));

	bmpFH.bfType = 19778; //"BM"
	bmpFH.bfSize = sizeof(bmpFH) + sizeof(bmpIH) + (fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1) * bmpheight ;
	bmpFH.bfOffBits = sizeof(bmpFH) + sizeof(bmpIH);

	bmpIH.biSize = sizeof(bmpIH);
	bmpIH.biWidth = fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1;
	bmpIH.biHeight = bmpheight;
	bmpIH.biPlanes = 1;
	bmpIH.biBitCount = 32;

	fwrite(&bmpFH, 1, sizeof(bmpFH), fp2);
	fwrite(&bmpIH, 1, sizeof(bmpIH), fp2);

	for (int i = 0;i < bmpheight * fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;i += fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR)
	{
		fEmpty = 1;
		for (int j = 0;j < fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;j++)
		{
			fprintf(fp, "%d\t", stageAtSync[i + j]);
			int color = 0;
			if (stageAtSync[i + j] == 1) color = RGB(255, 0, 0);
			if (stageAtSync[i + j] == 2) color = RGB(0, 255, 0);
			if (stageAtSync[i + j] == 3) color = RGB(0, 0, 255);
			if (stageAtSync[i + j] == 4) color = RGB(255, 255, 0);
			fwrite(&color, 1, sizeof(int), fp2);
			if (j > 0 && j % 32 == 0)
			{
				color = RGB(255, 255, 255);
				fwrite(&color, 1, 4, fp2);
			}
			if (stageAtSync[i + j]) fEmpty = 0;
		}
		fprintf(fp, "\n");
		if (fEmpty) nEmptySync++;
		else nEmptySync = 0;
		//if (nEmptySync == GPUCA_GPU_SCHED_ROW_STEP + 2) break;
	}

	fclose(fp);
	fclose(fp2);
	free(stageAtSync);
#endif
	return 0;
}
