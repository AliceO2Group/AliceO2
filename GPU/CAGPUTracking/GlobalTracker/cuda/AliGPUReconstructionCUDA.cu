#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>
#define GPUCA_GPUTYPE_PASCAL

#include "AliGPUReconstructionCUDA.h"
#include "AliGPUReconstructionCUDAInternals.h"
#include "AliGPUReconstructionCommon.h"

__constant__ uint4 gGPUConstantMemBuffer[(sizeof(AliGPUCAConstantMem) + sizeof(uint4) - 1) / sizeof(uint4)];
__constant__ char& gGPUConstantMemBufferChar = (char&) gGPUConstantMemBuffer;
__constant__ AliGPUCAConstantMem& gGPUConstantMem = (AliGPUCAConstantMem&) gGPUConstantMemBufferChar;

#ifdef GPUCA_GPU_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

#ifdef HAVE_O2HEADERS
#include "ITStrackingCUDA/TrackerTraitsNV.h"
#else
namespace o2 { namespace ITS { class TrackerTraitsNV : public TrackerTraits {}; }}
#endif

#define DEVICE_KERNELS_PRE
#include "AliGPUDeviceKernels.h"

template <class TProcess, int I, typename... Args> GPUg() void runKernelCUDA(int iSlice, Args... args)
{
	AliGPUTPCTracker &tracker = gGPUConstantMem.tpcTrackers[iSlice];
	GPUshared() typename TProcess::AliGPUTPCSharedMemory smem;
	TProcess::template Thread<I>(get_num_groups(0), get_local_size(0), get_group_id(0), get_local_id(0), smem, tracker, args...);
}

template <class TProcess, int I, typename... Args> GPUg() void runKernelCUDAMulti(int firstSlice, int nSliceCount, Args... args)
{
	const int iSlice = nSliceCount * (get_group_id(0) + (get_num_groups(0) % nSliceCount != 0 && nSliceCount * (get_group_id(0) + 1) % get_num_groups(0) != 0)) / get_num_groups(0);
	const int nSliceBlockOffset = get_num_groups(0) * iSlice / nSliceCount;
	const int sliceBlockId = get_group_id(0) - nSliceBlockOffset;
	const int sliceGridDim = get_num_groups(0) * (iSlice + 1) / nSliceCount - get_num_groups(0) * (iSlice) / nSliceCount;
	AliGPUTPCTracker &tracker = gGPUConstantMem.tpcTrackers[firstSlice + iSlice];
	GPUshared() typename TProcess::AliGPUTPCSharedMemory smem;
	TProcess::template Thread<I>(sliceGridDim, get_local_size(0), sliceBlockId, get_local_id(0), smem, tracker, args...);
}

template <class T, int I, typename... Args> int AliGPUReconstructionCUDABackend::runKernelBackend(const krnlExec& x, const krnlRunRange& y, const krnlEvent& z, const Args&... args)
{
	if (x.device == krnlDeviceType::CPU) return AliGPUReconstructionCPU::runKernelBackend<T, I> (x, y, z, args...);
	if (z.evList) for (int k = 0;k < z.nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[x.stream], ((hipEvent_t*) z.evList)[k], 0));
	if (y.num <= 1)
	{
		runKernelCUDA<T, I> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>>(y.start, args...);
	}
	else
	{
		runKernelCUDAMulti<T, I> <<<x.nBlocks, x.nThreads, 0, mInternals->CudaStreams[x.stream]>>> (y.start, y.num, args...);
	}
	if (z.ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) z.ev, mInternals->CudaStreams[x.stream]));
	return 0;
}

AliGPUReconstructionCUDABackend::AliGPUReconstructionCUDABackend(const AliGPUCASettingsProcessing& cfg) : AliGPUReconstructionDeviceBase(cfg)
{
	mInternals = new AliGPUReconstructionCUDAInternals;
	mProcessingSettings.deviceType = CUDA;
	mITSTrackerTraits.reset(new o2::ITS::TrackerTraitsNV);
}

AliGPUReconstructionCUDABackend::~AliGPUReconstructionCUDABackend()
{
	mITSTrackerTraits.reset(nullptr); //Make sure we destroy the ITS tracker before we exit CUDA
	GPUFailedMsgI(hipDeviceReset());
	delete mInternals;
}

AliGPUReconstruction* AliGPUReconstruction_Create_CUDA(const AliGPUCASettingsProcessing& cfg)
{
	return new AliGPUReconstructionCUDA(cfg);
}

int AliGPUReconstructionCUDABackend::InitDevice_Runtime()
{
	//Find best CUDA device, initialize and allocate memory

	hipDeviceProp_t hipDeviceProp_t;

	int count, bestDevice = -1;
	double bestDeviceSpeed = -1, deviceSpeed;
	if (GPUFailedMsgI(hipGetDeviceCount(&count)))
	{
		CAGPUError("Error getting CUDA Device Count");
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("Available CUDA devices:");
	const int reqVerMaj = 2;
	const int reqVerMin = 0;
	for (int i = 0;i < count;i++)
	{
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Examining device %d\n", i);
		size_t free, total;
		hipInit(0);
		hipDevice_t tmpDevice;
		hipDeviceGet(&tmpDevice, i);
		hipCtx_t tmpContext;
		hipCtxCreate(&tmpContext, 0, tmpDevice);
		if(hipMemGetInfo(&free, &total)) std::cout << "Error\n";
		hipCtxDestroy(tmpContext);
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained current memory usage for device %d\n", i);
		if (GPUFailedMsgI(hipGetDeviceProperties(&hipDeviceProp_t, i))) continue;
		if (mDeviceProcessingSettings.debugLevel >= 4) printf("Obtained device properties for device %d\n", i);
		int deviceOK = true;
		const char* deviceFailure = "";
		if (hipDeviceProp_t.major >= 9) {deviceOK = false; deviceFailure = "Invalid Revision";}
		else if (hipDeviceProp_t.major < reqVerMaj || (hipDeviceProp_t.major == reqVerMaj && hipDeviceProp_t.minor < reqVerMin)) {deviceOK = false; deviceFailure = "Too low device revision";}
		else if (free < mDeviceMemorySize) {deviceOK = false; deviceFailure = "Insufficient GPU memory";}

		deviceSpeed = (double) hipDeviceProp_t.multiProcessorCount * (double) hipDeviceProp_t.clockRate * (double) hipDeviceProp_t.warpSize * (double) free * (double) hipDeviceProp_t.major * (double) hipDeviceProp_t.major;
		if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUImportant("Device %s%2d: %s (Rev: %d.%d - Mem Avail %lld / %lld)%s %s", deviceOK ? " " : "[", i, hipDeviceProp_t.name, hipDeviceProp_t.major, hipDeviceProp_t.minor, (long long int) free, (long long int) hipDeviceProp_t.totalGlobalMem, deviceOK ? " " : " ]", deviceOK ? "" : deviceFailure);
		if (!deviceOK) continue;
		if (deviceSpeed > bestDeviceSpeed)
		{
			bestDevice = i;
			bestDeviceSpeed = deviceSpeed;
		}
		else
		{
			if (mDeviceProcessingSettings.debugLevel >= 0) CAGPUInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
		}
	}
	if (bestDevice == -1)
	{
		CAGPUWarning("No %sCUDA Device available, aborting CUDA Initialisation", count ? "appropriate " : "");
		CAGPUImportant("Requiring Revision %d.%d, Mem: %lld", reqVerMaj, reqVerMin, (long long int) mDeviceMemorySize);
		return(1);
	}

	if (mDeviceProcessingSettings.deviceNum > -1)
	{
		if (mDeviceProcessingSettings.deviceNum < (signed) count)
		{
			bestDevice = mDeviceProcessingSettings.deviceNum;
		}
		else
		{
			CAGPUWarning("Requested device ID %d non existend, falling back to default device id %d", mDeviceProcessingSettings.deviceNum, bestDevice);
		}
	}
	fDeviceId = bestDevice;

	GPUFailedMsgI(hipGetDeviceProperties(&hipDeviceProp_t ,fDeviceId));

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		CAGPUInfo("Using CUDA Device %s with Properties:", hipDeviceProp_t.name);
		CAGPUInfo("totalGlobalMem = %lld", (unsigned long long int) hipDeviceProp_t.totalGlobalMem);
		CAGPUInfo("sharedMemPerBlock = %lld", (unsigned long long int) hipDeviceProp_t.sharedMemPerBlock);
		CAGPUInfo("regsPerBlock = %d", hipDeviceProp_t.regsPerBlock);
		CAGPUInfo("warpSize = %d", hipDeviceProp_t.warpSize);
		CAGPUInfo("memPitch = %lld", (unsigned long long int) hipDeviceProp_t.memPitch);
		CAGPUInfo("maxThreadsPerBlock = %d", hipDeviceProp_t.maxThreadsPerBlock);
		CAGPUInfo("maxThreadsDim = %d %d %d", hipDeviceProp_t.maxThreadsDim[0], hipDeviceProp_t.maxThreadsDim[1], hipDeviceProp_t.maxThreadsDim[2]);
		CAGPUInfo("maxGridSize = %d %d %d", hipDeviceProp_t.maxGridSize[0], hipDeviceProp_t.maxGridSize[1], hipDeviceProp_t.maxGridSize[2]);
		CAGPUInfo("totalConstMem = %lld", (unsigned long long int) hipDeviceProp_t.totalConstMem);
		CAGPUInfo("major = %d", hipDeviceProp_t.major);
		CAGPUInfo("minor = %d", hipDeviceProp_t.minor);
		CAGPUInfo("clockRate = %d", hipDeviceProp_t.clockRate);
		CAGPUInfo("memoryClockRate = %d", hipDeviceProp_t.memoryClockRate);
		CAGPUInfo("multiProcessorCount = %d", hipDeviceProp_t.multiProcessorCount);
		CAGPUInfo("textureAlignment = %lld", (unsigned long long int) hipDeviceProp_t.textureAlignment);
	}

	fThreadCount = GPUCA_GPU_THREAD_COUNT;
	fBlockCount = hipDeviceProp_t.multiProcessorCount;
	fConstructorBlockCount = hipDeviceProp_t.multiProcessorCount * (mDeviceProcessingSettings.trackletConstructorInPipeline ? 1 : GPUCA_GPU_BLOCK_COUNT_CONSTRUCTOR_MULTIPLIER);
	fSelectorBlockCount = hipDeviceProp_t.multiProcessorCount * GPUCA_GPU_BLOCK_COUNT_SELECTOR_MULTIPLIER;
	fConstructorThreadCount = GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;
	fSelectorThreadCount = GPUCA_GPU_THREAD_COUNT_SELECTOR;
	fFinderThreadCount = GPUCA_GPU_THREAD_COUNT_FINDER;
	fTRDThreadCount = GPUCA_GPU_THREAD_COUNT_TRD;

	if (hipDeviceProp_t.major < 1 || (hipDeviceProp_t.major == 1 && hipDeviceProp_t.minor < 2))
	{
		CAGPUError( "Unsupported CUDA Device" );
		return(1);
	}

#ifdef GPUCA_GPU_USE_TEXTURES
	if (GPUCA_GPU_SLICE_DATA_MEMORY * NSLICES > (size_t) hipDeviceProp_t.maxTexture1DLinear)
	{
		CAGPUError("Invalid maximum texture size of device: %lld < %lld\n", (long long int) hipDeviceProp_t.maxTexture1DLinear, (long long int) (GPUCA_GPU_SLICE_DATA_MEMORY * NSLICES));
		return(1);
	}
#endif

	mNStreams = std::max(mDeviceProcessingSettings.nStreams, 3);

	if (hipCtxCreate(&mInternals->CudaContext, hipDeviceScheduleAuto, fDeviceId) != hipSuccess)
	{
		CAGPUError("Could not set CUDA Device!");
		return(1);
	}

	if (mDeviceMemorySize > hipDeviceProp_t.totalGlobalMem || GPUFailedMsgI(hipMalloc(&mDeviceMemoryBase, mDeviceMemorySize)))
	{
		CAGPUError("CUDA Memory Allocation Error");
		GPUFailedMsgI(hipDeviceReset());
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) CAGPUInfo("GPU Memory used: %lld", (long long int) mDeviceMemorySize);
	if (GPUFailedMsgI(hipHostMalloc(&mHostMemoryBase, mHostMemorySize)))
	{
		CAGPUError("Error allocating Page Locked Host Memory");
		GPUFailedMsgI(hipDeviceReset());
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 1) CAGPUInfo("Host Memory used: %lld", (long long int) mHostMemorySize);

	if (mDeviceProcessingSettings.debugLevel >= 1)
	{
		memset(mHostMemoryBase, 0, mHostMemorySize);
		if (GPUFailedMsgI(hipMemset(mDeviceMemoryBase, 143, mDeviceMemorySize)))
		{
			CAGPUError("Error during CUDA memset");
			GPUFailedMsgI(hipDeviceReset());
			return(1);
		}
	}

	for (int i = 0;i < mNStreams;i++)
	{
		if (GPUFailedMsgI(hipStreamCreate(&mInternals->CudaStreams[i])))
		{
			CAGPUError("Error creating CUDA Stream");
			GPUFailedMsgI(hipDeviceReset());
			return(1);
		}
	}
	
	void* devPtrConstantMem;
	if (GPUFailedMsgI(hipGetSymbolAddress(&devPtrConstantMem, HIP_SYMBOL(gGPUConstantMemBuffer))))
	{
		CAGPUError("Error getting ptr to constant memory");
		GPUFailedMsgI(hipDeviceReset());
		return 1;
	}
	mDeviceConstantMem = (AliGPUCAConstantMem*) devPtrConstantMem;
	
	hipEvent_t *events = (hipEvent_t*) &mEvents;
	for (unsigned int i = 0;i < sizeof(mEvents) / sizeof(hipEvent_t);i++)
	{
		if (GPUFailedMsgI(hipEventCreate(&events[i])))
		{
			CAGPUError("Error creating event");
			GPUFailedMsgI(hipDeviceReset());
			return 1;
		}
	}

	ReleaseThreadContext();
	CAGPUInfo("CUDA Initialisation successfull (Device %d: %s, Thread %d, %lld/%lld bytes used)", fDeviceId, hipDeviceProp_t.name, fThreadId, (long long int) mHostMemorySize, (long long int) mDeviceMemorySize);

	return(0);
}

int AliGPUReconstructionCUDABackend::ExitDevice_Runtime()
{
	//Uninitialize CUDA
	ActivateThreadContext();

	SynchronizeGPU();

	GPUFailedMsgI(hipFree(mDeviceMemoryBase));
	mDeviceMemoryBase = nullptr;

	for (int i = 0;i < mNStreams;i++)
	{
		GPUFailedMsgI(hipStreamDestroy(mInternals->CudaStreams[i]));
	}

	GPUFailedMsgI(hipHostFree(mHostMemoryBase));
	mHostMemoryBase = nullptr;
	
	hipEvent_t *events = (hipEvent_t*) &mEvents;
	for (unsigned int i = 0;i < sizeof(mEvents) / sizeof(hipEvent_t);i++)
	{
		GPUFailedMsgI(hipEventDestroy(events[i]));
	}

	if (GPUFailedMsgI(hipDeviceReset()))
	{
		CAGPUError("Could not uninitialize GPU");
		return(1);
	}

	hipCtxDestroy(mInternals->CudaContext);

	CAGPUInfo("CUDA Uninitialized");
	return(0);
}

int AliGPUReconstructionCUDABackend::DoTRDGPUTracking()
{
#ifndef GPUCA_GPU_MERGER
	CAGPUError("GPUCA_GPU_MERGER compile flag not set");
	return(1);
#else
	ActivateThreadContext();
	SetupGPUProcessor(&mWorkers->trdTracker);
	mWorkersShadow->trdTracker.SetGeometry((AliGPUTRDGeometry*) mProcDevice.fTrdGeometry);

	GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), &mWorkersShadow->trdTracker, sizeof(mWorkersShadow->trdTracker), (char*) &mDeviceConstantMem->trdTracker - (char*) mDeviceConstantMem, hipMemcpyHostToDevice));

	TransferMemoryResourcesToGPU(&mWorkers->trdTracker);

	DoTrdTrackingGPU<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT_TRD>>>();
	SynchronizeGPU();

	TransferMemoryResourcesToHost(&mWorkers->trdTracker);
	SynchronizeGPU();

	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("GPU TRD tracker Finished");

	ReleaseThreadContext();
	return(0);
#endif
}

int AliGPUReconstructionCUDABackend::RefitMergedTracks(AliGPUTPCGMMerger* Merger, bool resetTimers)
{
#ifndef GPUCA_GPU_MERGER
	CAGPUError("GPUCA_GPU_MERGER compile flag not set");
	return(1);
#else

	HighResTimer timer;
	static double times[3] = {};
	static int nCount = 0;
	if (resetTimers)
	{
		for (unsigned int k = 0;k < sizeof(times) / sizeof(times[0]);k++) times[k] = 0;
		nCount = 0;
	}
	ActivateThreadContext();

	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("Running GPU Merger (%d/%d)", Merger->NOutputTrackClusters(), Merger->NClusters());
	timer.Start();

	SetupGPUProcessor(Merger);
	mWorkersShadow->tpcMerger.OverrideSliceTracker(mDeviceConstantMem->tpcTrackers);
	
	GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), &mWorkersShadow->tpcMerger, sizeof(mWorkersShadow->tpcMerger), (char*) &mDeviceConstantMem->tpcMerger - (char*) mDeviceConstantMem, hipMemcpyHostToDevice));
	TransferMemoryResourceLinkToGPU(Merger->MemoryResRefit());
	times[0] += timer.GetCurrentElapsedTime(true);
	
	RefitTracks<<<fConstructorBlockCount, GPUCA_GPU_THREAD_COUNT>>>(mWorkersShadow->tpcMerger.OutputTracks(), mWorkersShadow->tpcMerger.NOutputTracks(), mWorkersShadow->tpcMerger.Clusters());
	SynchronizeGPU();
	times[1] += timer.GetCurrentElapsedTime(true);
	
	TransferMemoryResourceLinkToHost(Merger->MemoryResRefit());
	SynchronizeGPU();
	times[2] += timer.GetCurrentElapsedTime();
	
	if (mDeviceProcessingSettings.debugLevel >= 2) CAGPUInfo("GPU Merger Finished");
	nCount++;

	if (mDeviceProcessingSettings.debugLevel > 0)
	{
		int copysize = 4 * Merger->NOutputTrackClusters() * sizeof(float) + Merger->NOutputTrackClusters() * sizeof(unsigned int) + Merger->NOutputTracks() * sizeof(AliGPUTPCGMMergedTrack) + 6 * sizeof(float) + sizeof(AliGPUCAParam);
		double speed = (double) copysize / times[0] * nCount / 1e9;
		printf("GPU Fit:\tCopy To:\t%'7d us (%6.3f GB/s)\n", (int) (times[0] * 1000000 / nCount), speed);
		printf("\t\tFit:\t\t%'7d us\n", (int) (times[1] * 1000000 / nCount));
		speed = (double) copysize / times[2] * nCount / 1e9;
		printf("\t\tCopy From:\t%'7d us (%6.3f GB/s)\n", (int) (times[2] * 1000000 / nCount), speed);
	}

	if (!GPUCA_TIMING_SUM)
	{
		for (int i = 0;i < 3;i++) times[i] = 0;
		nCount = 0;
	}

	ReleaseThreadContext();
	return(0);
#endif
}

void AliGPUReconstructionCUDABackend::TransferMemoryResourceToGPU(AliGPUMemoryResource* res, int stream, deviceEvent* ev, deviceEvent* evList, int nEvents)
{
	//if (evList == nullptr) nEvents = 0;
	if (mDeviceProcessingSettings.debugLevel >= 3) stream = -1;
	if (mDeviceProcessingSettings.debugLevel >= 3) printf("Copying to GPU: %s\n", res->Name());
	if (stream == -1)
	{
		GPUFailedMsg(hipMemcpy(res->PtrDevice(), res->Ptr(), res->Size(), hipMemcpyHostToDevice));
	}
	else
	{
		if (evList == nullptr) nEvents = 0;
		for (int k = 0;k < nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[stream], ((hipEvent_t*) evList)[k], 0));
		GPUFailedMsg(hipMemcpyAsync(res->PtrDevice(), res->Ptr(), res->Size(), hipMemcpyHostToDevice, mInternals->CudaStreams[stream]));
		if (ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
	}
}

void AliGPUReconstructionCUDABackend::TransferMemoryResourceToHost(AliGPUMemoryResource* res, int stream, deviceEvent* ev, deviceEvent* evList, int nEvents)
{
	//if (evList == nullptr) nEvents = 0;
	if (mDeviceProcessingSettings.debugLevel >= 3) stream = -1;
	if (mDeviceProcessingSettings.debugLevel >= 3) printf("Copying to Host: %s\n", res->Name());
	if (stream == -1)
	{
		GPUFailedMsg(hipMemcpy(res->Ptr(), res->PtrDevice(), res->Size(), hipMemcpyDeviceToHost));
	}
	else
	{
		if (evList == nullptr) nEvents = 0;
		for (int k = 0;k < nEvents;k++) GPUFailedMsg(hipStreamWaitEvent(mInternals->CudaStreams[stream], ((hipEvent_t*) evList)[k], 0));
		GPUFailedMsg(hipMemcpyAsync(res->Ptr(), res->PtrDevice(), res->Size(), hipMemcpyDeviceToHost, mInternals->CudaStreams[stream]));
		if (ev) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
	}
}

void AliGPUReconstructionCUDABackend::WriteToConstantMemory(size_t offset, const void* src, size_t size, int stream, deviceEvent* ev)
{
	if (stream == -1) GPUFailedMsg(hipMemcpyToSymbol(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice));
	else GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gGPUConstantMemBuffer), src, size, offset, hipMemcpyHostToDevice, mInternals->CudaStreams[stream]));
	if (ev && stream != -1) GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
}

void AliGPUReconstructionCUDABackend::ReleaseEvent(deviceEvent* ev) {}

void AliGPUReconstructionCUDABackend::RecordMarker(deviceEvent* ev, int stream)
{
	GPUFailedMsg(hipEventRecord(*(hipEvent_t*) ev, mInternals->CudaStreams[stream]));
}

int AliGPUReconstructionCUDABackend::GPUMergerAvailable() const
{
#ifdef GPUCA_GPU_MERGER
	return(1);
#else
	return(0);
#endif
}

void AliGPUReconstructionCUDABackend::ActivateThreadContext()
{
	hipCtxPushCurrent(mInternals->CudaContext);
}
void AliGPUReconstructionCUDABackend::ReleaseThreadContext()
{
	hipCtxPopCurrent(&mInternals->CudaContext);
}

void AliGPUReconstructionCUDABackend::SynchronizeGPU()
{
	GPUFailedMsg(hipDeviceSynchronize());
}

void AliGPUReconstructionCUDABackend::SynchronizeStream(int stream)
{
	GPUFailedMsg(hipStreamSynchronize(mInternals->CudaStreams[stream]));
}

void AliGPUReconstructionCUDABackend::SynchronizeEvents(deviceEvent* evList, int nEvents)
{
	for (int i = 0;i < nEvents;i++)
	{
		GPUFailedMsg(hipEventSynchronize(((hipEvent_t*) evList)[i]));
	}
}

int AliGPUReconstructionCUDABackend::IsEventDone(deviceEvent* evList, int nEvents)
{
	for (int i = 0;i < nEvents;i++)
	{
		hipError_t retVal = hipEventSynchronize(((hipEvent_t*) evList)[i]);
		if (retVal == hipErrorNotReady) return 0;
		GPUFailedMsg(retVal);
	}
	return(1);
}

int AliGPUReconstructionCUDABackend::GPUDebug(const char* state, int stream, int slice)
{
	//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards, in case of debugmode
	if (mDeviceProcessingSettings.debugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		CAGPUError("Cuda Error %s while running kernel (%s) (Stream %d; Slice %d/%d)", hipGetErrorString(cuErr), state, stream, slice, NSLICES);
		return(1);
	}
	if (GPUFailedMsgI(hipDeviceSynchronize()))
	{
		CAGPUError("CUDA Error while synchronizing (%s) (Stream %d; Slice %d/%d)", state, stream, slice, NSLICES);
		return(1);
	}
	if (mDeviceProcessingSettings.debugLevel >= 3) CAGPUInfo("GPU Sync Done");
	return(0);
}

int AliGPUReconstructionCUDABackend::PrepareTextures()
{
#ifdef GPUCA_GPU_USE_TEXTURES
	hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
	size_t offset;
	GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu2, NSLICES * GPUCA_GPU_SLICE_DATA_MEMORY));
	hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
	GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, mWorkersShadow->tpcTrackers[0].Data().Memory(), &channelDescu, NSLICES * GPUCA_GPU_SLICE_DATA_MEMORY));
#endif
	return(0);
}

int AliGPUReconstructionCUDABackend::PrepareProfile()
{
#ifdef GPUCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* tmpMem;
	GPUFailedMsg(hipMalloc(&tmpMem, 100000000));
	mWorkersShadow->tpcTrackers[0].fStageAtSync = tmpMem;
	GPUFailedMsg(hipMemset(mWorkersShadow->tpcTrackers[0].StageAtSync(), 0, 100000000));
#endif
	return 0;
}

int AliGPUReconstructionCUDABackend::DoProfile()
{
#ifdef GPUCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* stageAtSync = (char*) malloc(100000000);
	GPUFailedMsg(hipMemcpy(stageAtSync, mWorkersShadow->tpcTrackers[0].StageAtSync(), 100 * 1000 * 1000, hipMemcpyDeviceToHost));
	hipFree(mWorkersShadow->tpcTrackers[0].StageAtSync());

	FILE* fp = fopen("profile.txt", "w+");
	FILE* fp2 = fopen("profile.bmp", "w+b");
	int nEmptySync = 0, fEmpty;

	const int bmpheight = 8192;
	BITMAPFILEHEADER bmpFH;
	BITMAPINFOHEADER bmpIH;
	ZeroMemory(&bmpFH, sizeof(bmpFH));
	ZeroMemory(&bmpIH, sizeof(bmpIH));

	bmpFH.bfType = 19778; //"BM"
	bmpFH.bfSize = sizeof(bmpFH) + sizeof(bmpIH) + (fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1) * bmpheight ;
	bmpFH.bfOffBits = sizeof(bmpFH) + sizeof(bmpIH);

	bmpIH.biSize = sizeof(bmpIH);
	bmpIH.biWidth = fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1;
	bmpIH.biHeight = bmpheight;
	bmpIH.biPlanes = 1;
	bmpIH.biBitCount = 32;

	fwrite(&bmpFH, 1, sizeof(bmpFH), fp2);
	fwrite(&bmpIH, 1, sizeof(bmpIH), fp2);

	for (int i = 0;i < bmpheight * fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;i += fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR)
	{
		fEmpty = 1;
		for (int j = 0;j < fConstructorBlockCount * GPUCA_GPU_THREAD_COUNT_CONSTRUCTOR;j++)
		{
			fprintf(fp, "%d\t", stageAtSync[i + j]);
			int color = 0;
			if (stageAtSync[i + j] == 1) color = RGB(255, 0, 0);
			if (stageAtSync[i + j] == 2) color = RGB(0, 255, 0);
			if (stageAtSync[i + j] == 3) color = RGB(0, 0, 255);
			if (stageAtSync[i + j] == 4) color = RGB(255, 255, 0);
			fwrite(&color, 1, sizeof(int), fp2);
			if (j > 0 && j % 32 == 0)
			{
				color = RGB(255, 255, 255);
				fwrite(&color, 1, 4, fp2);
			}
			if (stageAtSync[i + j]) fEmpty = 0;
		}
		fprintf(fp, "\n");
		if (fEmpty) nEmptySync++;
		else nEmptySync = 0;
		//if (nEmptySync == GPUCA_GPU_SCHED_ROW_STEP + 2) break;
	}

	fclose(fp);
	fclose(fp2);
	free(stageAtSync);
#endif
	return 0;
}
