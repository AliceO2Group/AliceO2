#include "hip/hip_runtime.h"
// **************************************************************************
// This file is property of and copyright by the ALICE HLT Project          *
// ALICE Experiment at CERN, All rights reserved.                           *
//                                                                          *
// Primary Authors: Sergey Gorbunov <sergey.gorbunov@kip.uni-heidelberg.de> *
//                  Ivan Kisel <kisel@kip.uni-heidelberg.de>                *
//					David Rohr <drohr@kip.uni-heidelberg.de>				*
//                  for The ALICE HLT Project.                              *
//                                                                          *
// Permission to use, copy, modify and distribute this software and its     *
// documentation strictly for non-commercial purposes is hereby granted     *
// without fee, provided that the above copyright notice appears in all     *
// copies and that both the copyright notice and this permission notice     *
// appear in the supporting documentation. The authors make no claims       *
// about the suitability of this software for any purpose. It is            *
// provided "as is" without express or implied warranty.                    *
//                                                                          *
//***************************************************************************

#include "AliHLTTPCCADef.h"
#include "AliHLTTPCCAGPUConfig.h"

#include <sm_11_atomic_functions.h>
#include <sm_12_atomic_functions.h>

#include <iostream>

//Disable assertions since they produce errors in GPU Code
#ifdef assert
#undef assert
#endif
#define assert(param)

#include "AliHLTTPCCAGPUTracker.h"

__constant__ float4 gAliHLTTPCCATracker[HLTCA_GPU_TRACKER_CONSTANT_MEM / sizeof( float4 )];
#ifdef HLTCA_GPU_TEXTURE_FETCH
texture<ushort2, 1, hipReadModeElementType> texRefu2;
texture<unsigned short, 1, hipReadModeElementType> texRefu;
texture<signed short, 1, hipReadModeElementType> texRefs;
#endif

#include "AliHLTTPCCAHit.h"

//Include CXX Files, GPUd() macro will then produce CUDA device code out of the tracker source code
#include "AliHLTTPCCATrackParam.cxx"
#include "AliHLTTPCCATrack.cxx" 

#include "AliHLTTPCCATrackletSelector.cxx"

#include "AliHLTTPCCAHitArea.cxx"
#include "AliHLTTPCCAGrid.cxx"
#include "AliHLTTPCCARow.cxx"
#include "AliHLTTPCCAParam.cxx"
#include "AliHLTTPCCATracker.cxx"

#include "AliHLTTPCCAOutTrack.cxx"

#include "AliHLTTPCCAProcess.h"

#include "AliHLTTPCCANeighboursFinder.cxx"

#include "AliHLTTPCCANeighboursCleaner.cxx"
#include "AliHLTTPCCAStartHitsFinder.cxx"
#include "AliHLTTPCCAStartHitsSorter.cxx"
#include "AliHLTTPCCATrackletConstructor.cxx"
#include "AliHLTTPCCASliceOutput.cxx"

#include "MemoryAssignmentHelpers.h"

//Find best CUDA device, initialize and allocate memory
int AliHLTTPCCAGPUTracker::InitGPU(int sliceCount, int forceDeviceID)
{
	hipDeviceProp_t fCudaDeviceProp;

#ifndef CUDA_DEVICE_EMULATION
	int count, bestDevice = -1, bestDeviceSpeed = 0;
	if (CUDA_FAILED_MSG(hipGetDeviceCount(&count)))
	{
		std::cout << "Error getting CUDA Device Count" << std::endl;
		return(1);
	}
	if (fDebugLevel >= 2) std::cout << "Available CUDA devices: ";
	for (int i = 0;i < count;i++)
	{
		hipGetDeviceProperties(&fCudaDeviceProp, i);
		if (fDebugLevel >= 2) std::cout << fCudaDeviceProp.name << " (" << i << ")     ";
		if (fCudaDeviceProp.major < 9 && !(fCudaDeviceProp.major < 1 || (fCudaDeviceProp.major == 1 && fCudaDeviceProp.minor < 2)) && fCudaDeviceProp.multiProcessorCount * fCudaDeviceProp.clockRate > bestDeviceSpeed)
		{
			bestDevice = i;
			bestDeviceSpeed = fCudaDeviceProp.multiProcessorCount * fCudaDeviceProp.clockRate;
		}
	}
	if (fDebugLevel >= 2) std::cout << std::endl;

	if (bestDevice == -1)
	{
		std::cout << "No CUDA Device available, aborting CUDA Initialisation" << std::endl;
		return(1);
	}

  int cudaDevice;
  if (forceDeviceID == -1)
	  cudaDevice = bestDevice;
  else
	  cudaDevice = forceDeviceID;
#else
	int cudaDevice = 0;
#endif

  hipGetDeviceProperties(&fCudaDeviceProp ,cudaDevice ); 

  if (fDebugLevel >= 1)
  {
	  std::cout<<"CUDA Device Properties: "<<std::endl;
	  std::cout<<"name = "<<fCudaDeviceProp.name<<std::endl;
	  std::cout<<"totalGlobalMem = "<<fCudaDeviceProp.totalGlobalMem<<std::endl;
	  std::cout<<"sharedMemPerBlock = "<<fCudaDeviceProp.sharedMemPerBlock<<std::endl;
	  std::cout<<"regsPerBlock = "<<fCudaDeviceProp.regsPerBlock<<std::endl;
	  std::cout<<"warpSize = "<<fCudaDeviceProp.warpSize<<std::endl;
	  std::cout<<"memPitch = "<<fCudaDeviceProp.memPitch<<std::endl;
	  std::cout<<"maxThreadsPerBlock = "<<fCudaDeviceProp.maxThreadsPerBlock<<std::endl;
	  std::cout<<"maxThreadsDim = "<<fCudaDeviceProp.maxThreadsDim[0]<<" "<<fCudaDeviceProp.maxThreadsDim[1]<<" "<<fCudaDeviceProp.maxThreadsDim[2]<<std::endl;
	  std::cout<<"maxGridSize = "  <<fCudaDeviceProp.maxGridSize[0]<<" "<<fCudaDeviceProp.maxGridSize[1]<<" "<<fCudaDeviceProp.maxGridSize[2]<<std::endl;
	  std::cout<<"totalConstMem = "<<fCudaDeviceProp.totalConstMem<<std::endl;
	  std::cout<<"major = "<<fCudaDeviceProp.major<<std::endl;
	  std::cout<<"minor = "<<fCudaDeviceProp.minor<<std::endl;
	  std::cout<<"clockRate = "<<fCudaDeviceProp.clockRate<<std::endl;
	  std::cout<<"textureAlignment = "<<fCudaDeviceProp.textureAlignment<<std::endl;
  }

  if (fCudaDeviceProp.major < 1 || (fCudaDeviceProp.major == 1 && fCudaDeviceProp.minor < 2))
  {
	  std::cout << "Unsupported CUDA Device\n";
	  return(1);
  }

  if (CUDA_FAILED_MSG(hipSetDevice(cudaDevice)))
  {
	  std::cout << "Could not set CUDA Device!\n";
	  return(1);
  }

  if (fgkNSlices * AliHLTTPCCATracker::CommonMemorySize() > HLTCA_GPU_COMMON_MEMORY)
  {
	  std::cout << "Insufficiant Common Memory\n";
	  hipDeviceReset();
	  return(1);
  }

  if (fgkNSlices * (HLTCA_ROW_COUNT + 1) * sizeof(AliHLTTPCCARow) > HLTCA_GPU_ROWS_MEMORY)
  {
	  std::cout << "Insufficiant Row Memory\n";
	  hipDeviceReset();
	  return(1);
  }

  fGPUMemSize = HLTCA_GPU_ROWS_MEMORY + HLTCA_GPU_COMMON_MEMORY + sliceCount * (HLTCA_GPU_SLICE_DATA_MEMORY + HLTCA_GPU_GLOBAL_MEMORY);
  if (fGPUMemSize > fCudaDeviceProp.totalGlobalMem || CUDA_FAILED_MSG(hipMalloc(&fGPUMemory, (size_t) fGPUMemSize)))
  {
	  std::cout << "CUDA Memory Allocation Error\n";
	  hipDeviceReset();
	  return(1);
  }
  if (fDebugLevel >= 1) std::cout << "GPU Memory used: " << fGPUMemSize << std::endl;
  int HostMemSize = HLTCA_GPU_ROWS_MEMORY + HLTCA_GPU_COMMON_MEMORY + sliceCount * (HLTCA_GPU_SLICE_DATA_MEMORY + HLTCA_GPU_TRACKS_MEMORY) + HLTCA_GPU_TRACKER_OBJECT_MEMORY;
  if (CUDA_FAILED_MSG(hipHostMalloc(&fHostLockedMemory, HostMemSize)))
  {
	  hipFree(fGPUMemory);
	  hipDeviceReset();
	  std::cout << "Error allocating Page Locked Host Memory";
	  return(1);
  }
  if (fDebugLevel >= 1) std::cout << "Host Memory used: " << HostMemSize << std::endl;

  if (fDebugLevel >= 1)
  {
	  CUDA_FAILED_MSG(hipMemset(fGPUMemory, 143, (size_t) fGPUMemSize));
  }
  std::cout << "CUDA Initialisation successfull\n";

  //Don't run constructor / destructor here, this will be just local memcopy of Tracker in GPU Memory
  if (sizeof(AliHLTTPCCATracker) * sliceCount > HLTCA_GPU_TRACKER_OBJECT_MEMORY)
  {
	  std::cout << "Insufficiant Tracker Object Memory\n";
	  return(1);
  }
  fSliceCount = sliceCount;
  fGpuTracker = (AliHLTTPCCATracker*) TrackerMemory(fHostLockedMemory, 0);

  for (int i = 0;i < fgkNSlices;i++)
  {
    fSlaveTrackers[i].SetGPUTracker();
	fSlaveTrackers[i].SetGPUTrackerCommonMemory((char*) CommonMemory(fHostLockedMemory, i));
	fSlaveTrackers[i].pData()->SetGPUSliceDataMemory(SliceDataMemory(fHostLockedMemory, i), RowMemory(fHostLockedMemory, i));
  }

  pCudaStreams = malloc(CAMath::Max(3, fSliceCount) * sizeof(hipStream_t));
  hipStream_t* const cudaStreams = (hipStream_t*) pCudaStreams;
  for (int i = 0;i < CAMath::Max(3, fSliceCount);i++)
  {
	if (CUDA_FAILED_MSG(hipStreamCreate(&cudaStreams[i])))
	{
		std::cout << "Error creating CUDA Stream" << std::endl;
		return(1);
	}
  }

#if defined(HLTCA_STANDALONE) & !defined(CUDA_DEVICE_EMULATION)
  if (fDebugLevel < 2)
  {
	  //Do one initial run for Benchmark reasons
	  const int useDebugLevel = fDebugLevel;
	  fDebugLevel = 0;
	  AliHLTTPCCAClusterData tmpCluster;
	  AliHLTTPCCASliceOutput tmpOutput;
	  AliHLTTPCCAParam tmpParam;
	  tmpParam.SetNRows(HLTCA_ROW_COUNT);
	  fSlaveTrackers[0].SetParam(tmpParam);
	  Reconstruct(&tmpOutput, &tmpCluster, 0, 1);
	  fDebugLevel = useDebugLevel;
  }
#endif
  return(0);
}

//Macro to align Pointers.
//Will align to start at 1 MB segments, this should be consistent with every alignment in the tracker
//(As long as every single data structure is <= 1 MB)
template <class T> inline T* AliHLTTPCCAGPUTracker::alignPointer(T* ptr, int alignment)
{
	size_t adr = (size_t) ptr;
	if (adr % alignment)
	{
		adr += alignment - (adr % alignment);
	}
	return((T*) adr);
}

//Check for CUDA Error and in the case of an error display the corresponding error string
bool AliHLTTPCCAGPUTracker::CUDA_FAILED_MSG(hipError_t error)
{
	if (error == hipSuccess) return(false);
	printf("CUDA Error: %d / %s\n", error, hipGetErrorString(error));
	return(true);
}

//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards
int AliHLTTPCCAGPUTracker::CUDASync(char* state)
{
	if (fDebugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		printf("Cuda Error %s while invoking kernel (%s)\n", hipGetErrorString(cuErr), state);
		return(1);
	}
	if (CUDA_FAILED_MSG(hipDeviceSynchronize()))
	{
		printf("CUDA Error while synchronizing (%s)\n", state);
		return(1);
	}
	if (fDebugLevel >= 5) printf("CUDA Sync Done\n");
	return(0);
}

void AliHLTTPCCAGPUTracker::SetDebugLevel(int dwLevel, std::ostream *NewOutFile)
{
	fDebugLevel = dwLevel;
	if (NewOutFile) fOutFile = NewOutFile;
}

int AliHLTTPCCAGPUTracker::SetGPUTrackerOption(char* OptionName, int OptionValue)
{
	if (strcmp(OptionName, "SimpleSched") == 0)
	{
		fOptionSimpleSched = OptionValue;
	}
	else
	{
		printf("Unknown Option: %s\n", OptionName);
		return(1);
	}
	return(0);
}

#ifdef HLTCA_STANDALONE
void AliHLTTPCCAGPUTracker::StandalonePerfTime(int iSlice, int i)
{
  if (fDebugLevel >= 1)
  {
	  AliHLTTPCCAStandaloneFramework::StandaloneQueryTime( fSlaveTrackers[iSlice].PerfTimer(i));
  }
}
#else
void AliHLTTPCCAGPUTracker::StandalonePerfTime(int /*iSlice*/, int /*i*/) {}
#endif

void AliHLTTPCCAGPUTracker::DumpRowBlocks(AliHLTTPCCATracker* tracker, int iSlice, bool check)
{
	if (fDebugLevel >= 4)
	{
		*fOutFile << "RowBlock Tracklets" << std::endl;
	
		int4* RowBlockPos = (int4*) malloc(sizeof(int4) * (tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * 2);
		int* RowBlockTracklets = (int*) malloc(sizeof(int) * (tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * HLTCA_GPU_MAX_TRACKLETS * 2);
		uint2* BlockStartingTracklet = (uint2*) malloc(sizeof(uint2) * HLTCA_GPU_BLOCK_COUNT);
		CUDA_FAILED_MSG(hipMemcpy(RowBlockPos, fGpuTracker[iSlice].RowBlockPos(), sizeof(int4) * (tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * 2, hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(RowBlockTracklets, fGpuTracker[iSlice].RowBlockTracklets(), sizeof(int) * (tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * HLTCA_GPU_MAX_TRACKLETS * 2, hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(BlockStartingTracklet, fGpuTracker[iSlice].BlockStartingTracklet(), sizeof(uint2) * HLTCA_GPU_BLOCK_COUNT, hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(tracker[iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemorySize(), hipMemcpyDeviceToHost));

		int k = tracker[iSlice].GPUParameters()->fScheduleFirstDynamicTracklet;
		for (int i = 0; i < tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1;i++)
		{
			*fOutFile << "Rowblock: " << i << ", up " << RowBlockPos[i].y << "/" << RowBlockPos[i].x << ", down " << 
				RowBlockPos[tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1 + i].y << "/" << RowBlockPos[tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1 + i].x << endl << "Phase 1: ";
			for (int j = 0;j < RowBlockPos[i].x;j++)
			{
				//Use Tracker Object to calculate Offset instead of fGpuTracker, since *fNTracklets of fGpuTracker points to GPU Mem!
				*fOutFile << RowBlockTracklets[(tracker[iSlice].RowBlockTracklets(0, i) - tracker[iSlice].RowBlockTracklets(0, 0)) + j] << ", ";
				if (check && RowBlockTracklets[(tracker[iSlice].RowBlockTracklets(0, i) - tracker[iSlice].RowBlockTracklets(0, 0)) + j] != k)
				{
					printf("Wrong starting Row Block %d, entry %d, is %d, should be %d\n", i, j, RowBlockTracklets[(tracker[iSlice].RowBlockTracklets(0, i) - tracker[iSlice].RowBlockTracklets(0, 0)) + j], k);
				}
				k++;
				if (RowBlockTracklets[(tracker[iSlice].RowBlockTracklets(0, i) - tracker[iSlice].RowBlockTracklets(0, 0)) + j] == -1)
				{
					printf("Error, -1 Tracklet found\n");
				}
			}
			*fOutFile << endl << "Phase 2: ";
			for (int j = 0;j < RowBlockPos[tracker[iSlice].Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1 + i].x;j++)
			{
				*fOutFile << RowBlockTracklets[(tracker[iSlice].RowBlockTracklets(1, i) - tracker[iSlice].RowBlockTracklets(0, 0)) + j] << ", ";
			}
			*fOutFile << endl;
		}

		if (check)
		{
			*fOutFile << "Starting Threads: (First Dynamic: " << tracker[iSlice].GPUParameters()->fScheduleFirstDynamicTracklet << ")" << std::endl;
			for (int i = 0;i < HLTCA_GPU_BLOCK_COUNT;i++)
			{
				*fOutFile << i << ": " << BlockStartingTracklet[i].x << " - " << BlockStartingTracklet[i].y << std::endl;
			}
		}

		free(RowBlockPos);
		free(RowBlockTracklets);
		free(BlockStartingTracklet);
	}
}

__global__ void PreInitRowBlocks(int4* const RowBlockPos, int* const RowBlockTracklets, int* const SliceDataHitWeights, int nSliceDataHits)
{
	int4* const RowBlockTracklets4 = (int4*) RowBlockTracklets;
	int4* const SliceDataHitWeights4 = (int4*) SliceDataHitWeights;
	const int stride = blockDim.x * gridDim.x;
	int4 i0, i1;
	i0.x = i0.y = i0.z = i0.w = 0;
	i1.x = i1.y = i1.z = i1.w = -1;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;i < sizeof(int4) * 2 * (HLTCA_ROW_COUNT / HLTCA_GPU_SCHED_ROW_STEP + 1) / sizeof(int4);i += stride)
		RowBlockPos[i] = i0;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;i < sizeof(int) * (HLTCA_ROW_COUNT / HLTCA_GPU_SCHED_ROW_STEP + 1) * HLTCA_GPU_MAX_TRACKLETS * 2 / sizeof(int4);i += stride)
		RowBlockTracklets4[i] = i1;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;i < nSliceDataHits * sizeof(int) / sizeof(int4);i += stride)
		SliceDataHitWeights4[i] = i0;
}

//Primary reconstruction function
int AliHLTTPCCAGPUTracker::Reconstruct(AliHLTTPCCASliceOutput* pOutput, AliHLTTPCCAClusterData* pClusterData, int firstSlice, int sliceCountLocal)
{
	hipStream_t* const cudaStreams = (hipStream_t*) pCudaStreams;

	if (sliceCountLocal == -1) sliceCountLocal = this->fSliceCount;

	if (sliceCountLocal * sizeof(AliHLTTPCCATracker) > HLTCA_GPU_TRACKER_CONSTANT_MEM)
	{
		printf("Insuffissant constant memory (Required %d, Available %d, Tracker %d, Param %d, SliceData %d)\n", sliceCountLocal * (int) sizeof(AliHLTTPCCATracker), (int) HLTCA_GPU_TRACKER_CONSTANT_MEM, (int) sizeof(AliHLTTPCCATracker), (int) sizeof(AliHLTTPCCAParam), (int) sizeof(AliHLTTPCCASliceData));
		return(1);
	}

	if (fDebugLevel >= 4)
	{
		for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
		{
			*fOutFile << endl << endl << "Slice: " << fSlaveTrackers[firstSlice + iSlice].Param().ISlice() << endl;
		}
	}

	memcpy(fGpuTracker, &fSlaveTrackers[firstSlice], sizeof(AliHLTTPCCATracker) * sliceCountLocal);

	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
	{
		if (fDebugLevel >= 2) printf("\n\nInitialising GPU Tracker (Slice %d)\n", fSlaveTrackers[firstSlice + iSlice].Param().ISlice());

		//Make this a GPU Tracker
		fGpuTracker[iSlice].SetGPUTracker();
		fGpuTracker[iSlice].SetGPUTrackerCommonMemory((char*) CommonMemory(fGPUMemory, iSlice));
		fGpuTracker[iSlice].pData()->SetGPUSliceDataMemory(SliceDataMemory(fGPUMemory, iSlice), RowMemory(fGPUMemory, iSlice));
		fGpuTracker[iSlice].pData()->SetPointers(&pClusterData[iSlice], false);

		//Set Pointers to GPU Memory
		char* tmpMem = (char*) GlobalMemory(fGPUMemory, iSlice);

		if (fDebugLevel >= 5) printf("Initialising GPU Hits Memory\n");
		tmpMem = fGpuTracker[iSlice].SetGPUTrackerHitsMemory(tmpMem, pClusterData[iSlice].NumberOfClusters(), fOptionSimpleSched);
		tmpMem = alignPointer(tmpMem, 1024 * 1024);

		if (fDebugLevel >= 5) printf("Initialising GPU Tracklet Memory\n");
		tmpMem = fGpuTracker[iSlice].SetGPUTrackerTrackletsMemory(tmpMem, HLTCA_GPU_MAX_TRACKLETS /* *fSlaveTrackers[firstSlice + iSlice].NTracklets()*/, fOptionSimpleSched);
		tmpMem = alignPointer(tmpMem, 1024 * 1024);

		if (fDebugLevel >= 5) printf("Initialising GPU Track Memory\n");
		tmpMem = fGpuTracker[iSlice].SetGPUTrackerTracksMemory(tmpMem, HLTCA_GPU_MAX_TRACKS /* *fSlaveTrackers[firstSlice + iSlice].NTracklets()*/, pClusterData[iSlice].NumberOfClusters());
		tmpMem = alignPointer(tmpMem, 1024 * 1024);

		if (fGpuTracker[iSlice].TrackMemorySize() >= HLTCA_GPU_TRACKS_MEMORY)
		{
			printf("Insufficiant Track Memory\n");
			return(1);
		}

		if (tmpMem - (char*) GlobalMemory(fGPUMemory, iSlice) > HLTCA_GPU_GLOBAL_MEMORY)
		{
			printf("Insufficiant Global Memory\n");
			return(1);
		}

		//Initialize Startup Constants
		*fSlaveTrackers[firstSlice + iSlice].NTracklets() = 0;
		*fSlaveTrackers[firstSlice + iSlice].NTracks() = 0;
		*fSlaveTrackers[firstSlice + iSlice].NTrackHits() = 0;
		fGpuTracker[iSlice].GPUParametersConst()->fGPUFixedBlockCount = HLTCA_GPU_BLOCK_COUNT * (iSlice + 1) / sliceCountLocal - HLTCA_GPU_BLOCK_COUNT * (iSlice) / sliceCountLocal;
		if (fDebugLevel >= 5) printf("Blocks for Slice %d: %d\n", iSlice, fGpuTracker[iSlice].GPUParametersConst()->fGPUFixedBlockCount);
		fGpuTracker[iSlice].GPUParametersConst()->fGPUiSlice = iSlice;
		fGpuTracker[iSlice].GPUParametersConst()->fGPUnSlices = sliceCountLocal;
		fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fGPUError = 0;
#ifdef HLTCA_GPU_SCHED_FIXED_START
		fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fNextTracklet = fGpuTracker[iSlice].GPUParametersConst()->fGPUFixedBlockCount * HLTCA_GPU_THREAD_COUNT;
#else
		fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fNextTracklet = 0;
#endif

		fGpuTracker[iSlice].pData()->GPUTextureBase() = fGpuTracker[0].SliceDataMemory();
	}

#ifdef HLTCA_GPU_TEXTURE_FETCH
		hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<ushort2>();
		size_t offset;
//		if (pClusterData[iSlice].NumberOfClusters() && (CUDA_FAILED_MSG(hipBindTexture(&offset, &texRef, fGpuTracker[iSlice].pData()->HitData(), &channelDesc, fGpuTracker[iSlice].Data().NumberOfHitsPlusAlign() * sizeof(ushort2))) || offset))
		if (CUDA_FAILED_MSG(hipBindTexture(&offset, &texRefu2, fGpuTracker[0].SliceDataMemory(), &channelDescu2, sliceCountLocal * HLTCA_GPU_SLICE_DATA_MEMORY)) || offset)
		{
			printf("Error binding CUDA Texture (Offset %d)\n", (size_t) offset);
			return(1);
		}
		hipChannelFormatDesc channelDescu = hipCreateChannelDesc<unsigned short>();
		if (CUDA_FAILED_MSG(hipBindTexture(&offset, &texRefu, fGpuTracker[0].SliceDataMemory(), &channelDescu, sliceCountLocal * HLTCA_GPU_SLICE_DATA_MEMORY)) || offset)
		{
			printf("Error binding CUDA Texture (Offset %d)\n", (size_t) offset);
			return(1);
		}
		hipChannelFormatDesc channelDescs = hipCreateChannelDesc<signed short>();
		if (CUDA_FAILED_MSG(hipBindTexture(&offset, &texRefs, fGpuTracker[0].SliceDataMemory(), &channelDescs, sliceCountLocal * HLTCA_GPU_SLICE_DATA_MEMORY)) || offset)
		{
			printf("Error binding CUDA Texture (Offset %d)\n", (size_t) offset);
			return(1);
		}
#endif

	//Copy Tracker Object to GPU Memory
#ifdef HLTCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	if (CUDA_FAILED_MSG(hipMalloc(&fGpuTracker[0].fStageAtSync, 100000000))) return(1);
	CUDA_FAILED_MSG(hipMemset(fGpuTracker[0].fStageAtSync, 0, 100000000));
#endif
	CUDA_FAILED_MSG(hipMemcpyToSymbolAsync(HIP_SYMBOL(gAliHLTTPCCATracker), fGpuTracker, sizeof(AliHLTTPCCATracker) * sliceCountLocal, 0, hipMemcpyHostToDevice, cudaStreams[0]));

	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
	{
		StandalonePerfTime(firstSlice + iSlice, 0);

		if (!fOptionSimpleSched)
		{
			PreInitRowBlocks<<<30, 256, 0, cudaStreams[2]>>>(fGpuTracker[iSlice].RowBlockPos(), fGpuTracker[iSlice].RowBlockTracklets(), fGpuTracker[iSlice].SliceDataHitWeights(), fSlaveTrackers[firstSlice + iSlice].Data().NumberOfHitsPlusAlign());
		}
		else
		{
			CUDA_FAILED_MSG(hipMemset(fGpuTracker[iSlice].SliceDataHitWeights(), 0, fSlaveTrackers[firstSlice + iSlice].Data().NumberOfHitsPlusAlign() * sizeof(int)));
		}

		//Initialize GPU Slave Tracker
		fSlaveTrackers[firstSlice + iSlice].pData()->SetGPUSliceDataMemory(SliceDataMemory(fHostLockedMemory, iSlice), RowMemory(fHostLockedMemory, firstSlice + iSlice));
		fSlaveTrackers[firstSlice + iSlice].ReadEvent(&pClusterData[iSlice]);
		if (fSlaveTrackers[firstSlice + iSlice].Data().MemorySize() > HLTCA_GPU_SLICE_DATA_MEMORY)
		{
			printf("Insufficiant Slice Data Memory\n");
			return(1);
		}

		/*if (fSlaveTrackers[firstSlice + iSlice].CheckEmptySlice())
		{
			if (fDebugLevel >= 5) printf("Slice Empty, not running GPU Tracker\n");
			if (sliceCountLocal == 1)
				return(0);
		}*/

		if (fDebugLevel >= 5) printf("Initialising Slice Tracker (CPU) Output Memory\n");

		if (fDebugLevel >= 4)
		{
			fSlaveTrackers[firstSlice + iSlice].TrackletMemory() = reinterpret_cast<char*> ( new uint4 [ fGpuTracker[iSlice].TrackletMemorySize()/sizeof( uint4 ) + 100] );
			fSlaveTrackers[firstSlice + iSlice].SetPointersTracklets( HLTCA_GPU_MAX_TRACKLETS );
			fSlaveTrackers[firstSlice + iSlice].HitMemory() = reinterpret_cast<char*> ( new uint4 [ fGpuTracker[iSlice].HitMemorySize()/sizeof( uint4 ) + 100] );
			fSlaveTrackers[firstSlice + iSlice].SetPointersHits( pClusterData[iSlice].NumberOfClusters() );
		}

		//Copy Data to GPU Global Memory
		CUDA_FAILED_MSG(hipMemcpyAsync(fGpuTracker[iSlice].CommonMemory(), fSlaveTrackers[firstSlice + iSlice].CommonMemory(), fSlaveTrackers[firstSlice + iSlice].CommonMemorySize(), hipMemcpyHostToDevice, cudaStreams[iSlice & 1]));
		CUDA_FAILED_MSG(hipMemcpyAsync(fGpuTracker[iSlice].SliceDataMemory(), fSlaveTrackers[firstSlice + iSlice].SliceDataMemory(), fSlaveTrackers[firstSlice + iSlice].SliceDataMemorySize(), hipMemcpyHostToDevice, cudaStreams[iSlice & 1]));

#ifdef SLICE_DATA_EXTERN_ROWS
		CUDA_FAILED_MSG(hipMemcpyAsync(fGpuTracker[iSlice].SliceDataRows(), fSlaveTrackers[firstSlice + iSlice].SliceDataRows(), (HLTCA_ROW_COUNT + 1) * sizeof(AliHLTTPCCARow), hipMemcpyHostToDevice, cudaStreams[iSlice & 1]));
#endif

		if (CUDASync("Initialization")) return(1);
		StandalonePerfTime(firstSlice + iSlice, 1);

		if (fDebugLevel >= 5) printf("Running GPU Neighbours Finder\n");
		AliHLTTPCCAProcess<AliHLTTPCCANeighboursFinder> <<<fSlaveTrackers[firstSlice + iSlice].Param().NRows(), 256, 0, cudaStreams[iSlice & 1]>>>(iSlice);

		if (CUDASync("Neighbours finder")) return 1;

		StandalonePerfTime(firstSlice + iSlice, 2);

		if (fDebugLevel >= 4)
		{
			*fOutFile << "Neighbours Finder:" << endl;
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].SliceDataMemory(), fGpuTracker[iSlice].SliceDataMemory(), fSlaveTrackers[firstSlice + iSlice].SliceDataMemorySize(), hipMemcpyDeviceToHost));
			fSlaveTrackers[firstSlice + iSlice].DumpLinks(*fOutFile);
		}

		if (fDebugLevel >= 5) printf("Running GPU Neighbours Cleaner\n");
		AliHLTTPCCAProcess<AliHLTTPCCANeighboursCleaner> <<<fSlaveTrackers[firstSlice + iSlice].Param().NRows()-2, 256, 0, cudaStreams[iSlice & 1]>>>(iSlice);
		if (CUDASync("Neighbours Cleaner")) return 1;

		StandalonePerfTime(firstSlice + iSlice, 3);

		if (fDebugLevel >= 4)
		{
			*fOutFile << "Neighbours Cleaner:" << endl;
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].SliceDataMemory(), fGpuTracker[iSlice].SliceDataMemory(), fSlaveTrackers[firstSlice + iSlice].SliceDataMemorySize(), hipMemcpyDeviceToHost));
			fSlaveTrackers[firstSlice + iSlice].DumpLinks(*fOutFile);
		}

		if (fDebugLevel >= 5) printf("Running GPU Start Hits Finder\n");
		AliHLTTPCCAProcess<AliHLTTPCCAStartHitsFinder> <<<fSlaveTrackers[firstSlice + iSlice].Param().NRows()-6, 256, 0, cudaStreams[iSlice & 1]>>>(iSlice);
		if (CUDASync("Start Hits Finder")) return 1;

		StandalonePerfTime(firstSlice + iSlice, 4);

		if (!fOptionSimpleSched)
		{
			if (fDebugLevel >= 5) printf("Running GPU Start Hits Sorter\n");
			AliHLTTPCCAProcess<AliHLTTPCCAStartHitsSorter> <<<30, 256, 0, cudaStreams[iSlice & 1]>>>(iSlice);
			if (CUDASync("Start Hits Sorter")) return 1;
		}

		StandalonePerfTime(firstSlice + iSlice, 5);

		if (fDebugLevel >= 2)
		{
			if (fDebugLevel >= 5) printf("Obtaining Number of Start Hits from GPU: ");
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugLevel >= 5) printf("%d\n", *fSlaveTrackers[firstSlice + iSlice].NTracklets());
			else if (fDebugLevel >= 2) printf("%3d ", *fSlaveTrackers[firstSlice + iSlice].NTracklets());

			if (*fSlaveTrackers[firstSlice + iSlice].NTracklets() > HLTCA_GPU_MAX_TRACKLETS)
			{
				printf("HLTCA_GPU_MAX_TRACKLETS constant insuffisant\n");
				return(1);
			}
		}

		if (!fOptionSimpleSched && fDebugLevel >= 4)
		{
			*fOutFile << "Start Hits Tmp: (" << *fSlaveTrackers[firstSlice + iSlice].NTracklets() << ")" << endl;
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].TrackletStartHits(), fGpuTracker[iSlice].TrackletTmpStartHits(), pClusterData[iSlice].NumberOfClusters() * sizeof(AliHLTTPCCAHitId), hipMemcpyDeviceToHost));
			fSlaveTrackers[firstSlice + iSlice].DumpStartHits(*fOutFile);
			uint3* tmpMemory = (uint3*) malloc(sizeof(uint3) * fSlaveTrackers[firstSlice + iSlice].Param().NRows());
			CUDA_FAILED_MSG(hipMemcpy(tmpMemory, fGpuTracker[iSlice].RowStartHitCountOffset(), fSlaveTrackers[firstSlice + iSlice].Param().NRows() * sizeof(uint3), hipMemcpyDeviceToHost));
			*fOutFile << "Start Hits Sort Vector:" << std::endl;
			for (int i = 0;i < fSlaveTrackers[firstSlice + iSlice].Param().NRows();i++)
			{
				*fOutFile << "Row: " << i << ", Len: " << tmpMemory[i].x << ", Offset: " << tmpMemory[i].y << ", New Offset: " << tmpMemory[i].z << std::endl;
			}
			free(tmpMemory);
		}

		if (fDebugLevel >= 4)
		{
			*fOutFile << "Start Hits: (" << *fSlaveTrackers[firstSlice + iSlice].NTracklets() << ")" << endl;
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].HitMemory(), fGpuTracker[iSlice].HitMemory(), fSlaveTrackers[firstSlice + iSlice].HitMemorySize(), hipMemcpyDeviceToHost));
			fSlaveTrackers[firstSlice + iSlice].DumpStartHits(*fOutFile);
		}

		StandalonePerfTime(firstSlice + iSlice, 6);
		
		fSlaveTrackers[firstSlice + iSlice].SetGPUTrackerTracksMemory((char*) TracksMemory(fHostLockedMemory, iSlice), HLTCA_GPU_MAX_TRACKS, pClusterData[iSlice].NumberOfClusters());
	}

	StandalonePerfTime(firstSlice, 7);
	if (fOptionSimpleSched)
	{
		AliHLTTPCCATrackletConstructorNewGPUSimple<<<HLTCA_GPU_BLOCK_COUNT, HLTCA_GPU_THREAD_COUNT>>>();
		if (CUDASync("Tracklet Constructor Simple Sched")) return 1;
	}
	else
	{
#ifdef HLTCA_GPU_PREFETCHDATA
		for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
		{
			if (fSlaveTrackers[firstSlice + iSlice].Data().GPUSharedDataReq() * sizeof(ushort_v) > ALIHLTTPCCATRACKLET_CONSTRUCTOR_TEMP_MEM / 4 * sizeof(uint4))
			{
				printf("Insufficiant GPU shared Memory, required: %d, available %d\n", fSlaveTrackers[firstSlice + iSlice].Data().GPUSharedDataReq() * sizeof(ushort_v), ALIHLTTPCCATRACKLET_CONSTRUCTOR_TEMP_MEM / 4 * sizeof(uint4));
				return(1);
			}
			if (fDebugLevel >= 1)
			{
				static int infoShown = 0;
				if (!infoShown)
				{
					printf("GPU Shared Memory Cache Size: %d\n", 2 * fSlaveTrackers[firstSlice + iSlice].Data().GPUSharedDataReq() * sizeof(ushort_v));
					infoShown = 1;
				}
			}
		}
#endif

		if (fDebugLevel >= 5) printf("Running GPU Tracklet Constructor\n");

		for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
		{
			AliHLTTPCCATrackletConstructorInit<<<HLTCA_GPU_MAX_TRACKLETS /* *fSlaveTrackers[firstSlice + iSlice].NTracklets() */ / HLTCA_GPU_THREAD_COUNT + 1, HLTCA_GPU_THREAD_COUNT>>>(iSlice);
			if (CUDASync("Tracklet Initializer")) return 1;
			DumpRowBlocks(fSlaveTrackers, iSlice);
		}

		AliHLTTPCCATrackletConstructorNewGPU<<<HLTCA_GPU_BLOCK_COUNT, HLTCA_GPU_THREAD_COUNT>>>();
		if (CUDASync("Tracklet Constructor (new)")) return 1;
		for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
		{
			DumpRowBlocks(&fSlaveTrackers[firstSlice], iSlice, false);
		}
	}
	
	StandalonePerfTime(firstSlice, 8);

	if (fDebugLevel >= 4)
	{
		for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
		{
			*fOutFile << "Tracklet Hits:" << endl;
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugLevel >= 5)
			{
				printf("Obtained %d tracklets\n", *fSlaveTrackers[firstSlice + iSlice].NTracklets());
			}
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].TrackletMemory(), fGpuTracker[iSlice].TrackletMemory(), fGpuTracker[iSlice].TrackletMemorySize(), hipMemcpyDeviceToHost));
			CUDA_FAILED_MSG(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].HitMemory(), fGpuTracker[iSlice].HitMemory(), fGpuTracker[iSlice].HitMemorySize(), hipMemcpyDeviceToHost));
			fSlaveTrackers[firstSlice + iSlice].DumpTrackletHits(*fOutFile);
		}
	}

	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice += HLTCA_GPU_TRACKLET_SELECTOR_SLICE_COUNT)
	{
		AliHLTTPCCAProcessMulti<AliHLTTPCCATrackletSelector><<<HLTCA_GPU_BLOCK_COUNT, HLTCA_GPU_THREAD_COUNT, 0, cudaStreams[iSlice]>>>(iSlice, CAMath::Min(HLTCA_GPU_TRACKLET_SELECTOR_SLICE_COUNT, sliceCountLocal - iSlice));
	}
	if (CUDASync("Tracklet Selector")) return 1;
	StandalonePerfTime(firstSlice, 9);

	CUDA_FAILED_MSG(hipMemcpyAsync(fSlaveTrackers[firstSlice + 0].CommonMemory(), fGpuTracker[0].CommonMemory(), fGpuTracker[0].CommonMemorySize(), hipMemcpyDeviceToHost, cudaStreams[0]));
	for (int iSliceTmp = 0;iSliceTmp <= sliceCountLocal;iSliceTmp++)
	{
		if (iSliceTmp < sliceCountLocal)
		{
			int iSlice = iSliceTmp;
			if (fDebugLevel >= 5) printf("Transfering Tracks from GPU to Host ");
			hipStreamSynchronize(cudaStreams[iSlice]);
			CUDA_FAILED_MSG(hipMemcpyAsync(fSlaveTrackers[firstSlice + iSlice].Tracks(), fGpuTracker[iSlice].Tracks(), sizeof(AliHLTTPCCATrack) * *fSlaveTrackers[firstSlice + iSlice].NTracks(), hipMemcpyDeviceToHost, cudaStreams[iSlice]));
			CUDA_FAILED_MSG(hipMemcpyAsync(fSlaveTrackers[firstSlice + iSlice].TrackHits(), fGpuTracker[iSlice].TrackHits(), sizeof(AliHLTTPCCAHitId) * *fSlaveTrackers[firstSlice + iSlice].NTrackHits(), hipMemcpyDeviceToHost, cudaStreams[iSlice]));
			if (iSlice + 1 < sliceCountLocal)
				CUDA_FAILED_MSG(hipMemcpyAsync(fSlaveTrackers[firstSlice + iSlice + 1].CommonMemory(), fGpuTracker[iSlice + 1].CommonMemory(), fGpuTracker[iSlice + 1].CommonMemorySize(), hipMemcpyDeviceToHost, cudaStreams[iSlice + 1]));
		}

		if (iSliceTmp)
		{
			int iSlice = iSliceTmp - 1;
			hipStreamSynchronize(cudaStreams[iSlice]);

			if (fDebugLevel >= 4)
			{
				*fOutFile << "Track Hits: (" << *fSlaveTrackers[firstSlice + iSlice].NTracks() << ")" << endl;
				fSlaveTrackers[firstSlice + iSlice].DumpTrackHits(*fOutFile);
			}

			if (fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fGPUError)
			{
				printf("GPU Tracker returned Error Code %d\n", fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fGPUError);
				return(1);
			}
			if (fDebugLevel >= 5) printf("%d / %d\n", *fSlaveTrackers[firstSlice + iSlice].NTracks(), *fSlaveTrackers[firstSlice + iSlice].NTrackHits());

			fSlaveTrackers[firstSlice + iSlice].SetOutput(&pOutput[iSlice]);
			fSlaveTrackers[firstSlice + iSlice].WriteOutput();

			if (fDebugLevel >= 4)
			{
				delete[] fSlaveTrackers[firstSlice + iSlice].HitMemory();
				delete[] fSlaveTrackers[firstSlice + iSlice].TrackletMemory();
			}
		}
	}

	StandalonePerfTime(firstSlice, 10);

	if (fDebugLevel >= 5) printf("GPU Reconstruction finished\n");

#ifdef HLTCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* stageAtSync = (char*) malloc(100000000);
	CUDA_FAILED_MSG(hipMemcpy(stageAtSync, fGpuTracker[0].fStageAtSync, 100 * 1000 * 1000, hipMemcpyDeviceToHost));
	hipFree(fGpuTracker[0].fStageAtSync);

	FILE* fp = fopen("profile.txt", "w+");
	FILE* fp2 = fopen("profile.bmp", "w+b");
	int nEmptySync = 0, fEmpty;

	const int bmpheight = 1000;
	BITMAPFILEHEADER bmpFH;
	BITMAPINFOHEADER bmpIH;
	ZeroMemory(&bmpFH, sizeof(bmpFH));
	ZeroMemory(&bmpIH, sizeof(bmpIH));
	
	bmpFH.bfType = 19778; //"BM"
	bmpFH.bfSize = sizeof(bmpFH) + sizeof(bmpIH) + (HLTCA_GPU_BLOCK_COUNT * HLTCA_GPU_THREAD_COUNT / 32 * 33 - 1) * bmpheight ;
	bmpFH.bfOffBits = sizeof(bmpFH) + sizeof(bmpIH);

	bmpIH.biSize = sizeof(bmpIH);
	bmpIH.biWidth = HLTCA_GPU_BLOCK_COUNT * HLTCA_GPU_THREAD_COUNT / 32 * 33 - 1;
	bmpIH.biHeight = bmpheight;
	bmpIH.biPlanes = 1;
	bmpIH.biBitCount = 32;

	fwrite(&bmpFH, 1, sizeof(bmpFH), fp2);
	fwrite(&bmpIH, 1, sizeof(bmpIH), fp2); 	

	for (int i = 0;i < bmpheight * HLTCA_GPU_BLOCK_COUNT * HLTCA_GPU_THREAD_COUNT;i += HLTCA_GPU_BLOCK_COUNT * HLTCA_GPU_THREAD_COUNT)
	{
		fEmpty = 1;
		for (int j = 0;j < HLTCA_GPU_BLOCK_COUNT * HLTCA_GPU_THREAD_COUNT;j++)
		{
			fprintf(fp, "%d\t", stageAtSync[i + j]);
			int color = 0;
			if (stageAtSync[i + j] == 1) color = RGB(255, 0, 0);
			if (stageAtSync[i + j] == 2) color = RGB(0, 255, 0);
			if (stageAtSync[i + j] == 3) color = RGB(0, 0, 255);
			if (stageAtSync[i + j] == 4) color = RGB(255, 255, 0);
			fwrite(&color, 1, sizeof(int), fp2);
			if (j > 0 && j % 32 == 0)
			{
				color = RGB(255, 255, 255);
				fwrite(&color, 1, 4, fp2);
			}
			if (stageAtSync[i + j]) fEmpty = 0;
		}
		fprintf(fp, "\n");
		if (fEmpty) nEmptySync++;
		else nEmptySync = 0;
		//if (nEmptySync == HLTCA_GPU_SCHED_ROW_STEP + 2) break;
	}

	fclose(fp);
	fclose(fp2);
	free(stageAtSync);
#endif 

	return(0);
}

int AliHLTTPCCAGPUTracker::InitializeSliceParam(int iSlice, AliHLTTPCCAParam &param)
{
	fSlaveTrackers[iSlice].Initialize(param);
	if (fSlaveTrackers[iSlice].Param().NRows() != HLTCA_ROW_COUNT)
	{
		printf("Error, Slice Tracker %d Row Count of %d exceeds Constant of %d\n", iSlice, fSlaveTrackers[iSlice].Param().NRows(), HLTCA_ROW_COUNT);
		return(1);
	}
	return(0);
}

int AliHLTTPCCAGPUTracker::ExitGPU()
{
	hipDeviceSynchronize();
	if (fGPUMemory)
	{
		hipFree(fGPUMemory);
		fGPUMemory = NULL;
	}
	if (fHostLockedMemory)
	{
		for (int i = 0;i < CAMath::Max(3, fSliceCount);i++)
		{
			hipStreamDestroy(((hipStream_t*) pCudaStreams)[i]);
		}
		free(pCudaStreams);
		fGpuTracker = NULL;
		hipHostFree(fHostLockedMemory);
	}

	if (CUDA_FAILED_MSG(hipDeviceReset()))
	{
		printf("Could not uninitialize GPU\n");
		return(1);
	}
	printf("CUDA Uninitialized\n");
	return(0);
}
