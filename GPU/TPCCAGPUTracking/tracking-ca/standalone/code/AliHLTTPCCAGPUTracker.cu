#include "hip/hip_runtime.h"
// **************************************************************************
// This file is property of and copyright by the ALICE HLT Project          *
// ALICE Experiment at CERN, All rights reserved.                           *
//                                                                          *
// Primary Authors: Sergey Gorbunov <sergey.gorbunov@kip.uni-heidelberg.de> *
//                  Ivan Kisel <kisel@kip.uni-heidelberg.de>                *
//					David Rohr <drohr@kip.uni-heidelberg.de>				*
//                  for The ALICE HLT Project.                              *
//                                                                          *
// Permission to use, copy, modify and distribute this software and its     *
// documentation strictly for non-commercial purposes is hereby granted     *
// without fee, provided that the above copyright notice appears in all     *
// copies and that both the copyright notice and this permission notice     *
// appear in the supporting documentation. The authors make no claims       *
// about the suitability of this software for any purpose. It is            *
// provided "as is" without express or implied warranty.                    *
//                                                                          *
//***************************************************************************

#include "AliHLTTPCCAGPUConfig.h"

#include <cutil.h>
#include <cutil_inline_runtime.h>
#include <sm_11_atomic_functions.h>
#include <sm_12_atomic_functions.h>

#include <iostream>

//Disable assertions since they produce errors in GPU Code
#ifdef assert
#undef assert
#endif
#define assert(param)

#include "AliHLTTPCCAGPUTracker.h"

#ifdef BUILD_GPU

//Include CXX Files, GPUd() macro will then produce CUDA device code out of the tracker source code
#include "AliHLTTPCCATrackParam.cxx"
#include "AliHLTTPCCATrack.cxx" 

#include "AliHLTTPCCATrackletSelector.cxx"

#include "AliHLTTPCCAHitArea.cxx"
#include "AliHLTTPCCAGrid.cxx"
#include "AliHLTTPCCARow.cxx"
#include "AliHLTTPCCAParam.cxx"
#include "AliHLTTPCCATracker.cxx"

#include "AliHLTTPCCAOutTrack.cxx"

#include "AliHLTTPCCAProcess.h"

#include "AliHLTTPCCANeighboursFinder.cxx"

#include "AliHLTTPCCANeighboursCleaner.cxx"
#include "AliHLTTPCCAStartHitsFinder.cxx"
#include "AliHLTTPCCATrackletConstructor.cxx"
#include "AliHLTTPCCASliceOutput.cxx"

#endif

//Find best CUDA device, initialize and allocate memory
int AliHLTTPCCAGPUTracker::InitGPU()
{
#ifdef BUILD_GPU
	int cudaDevice = cutGetMaxGflopsDeviceId();
	hipSetDevice(cudaDevice);
	hipDeviceProp_t fCudaDeviceProp;

  hipGetDeviceProperties(&fCudaDeviceProp ,cudaDevice ); 

  if (fDebugLevel >= 1)
  {
	  std::cout<<"CUDA Device Properties: "<<std::endl;
	  std::cout<<"name = "<<fCudaDeviceProp.name<<std::endl;
	  std::cout<<"totalGlobalMem = "<<fCudaDeviceProp.totalGlobalMem<<std::endl;
	  std::cout<<"sharedMemPerBlock = "<<fCudaDeviceProp.sharedMemPerBlock<<std::endl;
	  std::cout<<"regsPerBlock = "<<fCudaDeviceProp.regsPerBlock<<std::endl;
	  std::cout<<"warpSize = "<<fCudaDeviceProp.warpSize<<std::endl;
	  std::cout<<"memPitch = "<<fCudaDeviceProp.memPitch<<std::endl;
	  std::cout<<"maxThreadsPerBlock = "<<fCudaDeviceProp.maxThreadsPerBlock<<std::endl;
	  std::cout<<"maxThreadsDim = "<<fCudaDeviceProp.maxThreadsDim[0]<<" "<<fCudaDeviceProp.maxThreadsDim[1]<<" "<<fCudaDeviceProp.maxThreadsDim[2]<<std::endl;
	  std::cout<<"maxGridSize = "  <<fCudaDeviceProp.maxGridSize[0]<<" "<<fCudaDeviceProp.maxGridSize[1]<<" "<<fCudaDeviceProp.maxGridSize[2]<<std::endl;
	  std::cout<<"totalConstMem = "<<fCudaDeviceProp.totalConstMem<<std::endl;
	  std::cout<<"major = "<<fCudaDeviceProp.major<<std::endl;
	  std::cout<<"minor = "<<fCudaDeviceProp.minor<<std::endl;
	  std::cout<<"clockRate = "<<fCudaDeviceProp.clockRate<<std::endl;
	  std::cout<<"textureAlignment = "<<fCudaDeviceProp.textureAlignment<<std::endl;
  }

  if (fCudaDeviceProp.major < 1 || (fCudaDeviceProp.major == 1 && fCudaDeviceProp.minor < 2))
  {
	  std::cout << "Unsupported CUDA Device\n";
	  return(1);
  }

  fGPUMemSize = (long long int) fCudaDeviceProp.totalGlobalMem - 400 * 1024 * 1024;
  if (fGPUMemSize > 1024 * 1024 * 1024) fGPUMemSize = 1024 * 1024 * 1024;
  if (CUDA_FAILED_MSG(hipMalloc(&fGPUMemory, (size_t) fGPUMemSize)))
  {
	  std::cout << "CUDA Memory Allocation Error\n";
	  return(1);
  }
  std::cout << "CUDA Initialisation successfull\n";
#endif

	return(0);
}

//Macro to align Pointers.
//Will align to start at 1 MB segments, this should be consistent with every alignment in the tracker
//(As long as every single data structure is <= 1 MB)
template <class T> inline T* AliHLTTPCCAGPUTracker::alignPointer(T* ptr, int alignment)
{
	size_t adr = (size_t) ptr;
	if (adr % alignment)
	{
		adr += alignment - (adr % alignment);
	}
	return((T*) adr);
}

//Check for CUDA Error and in the case of an error display the corresponding error string
bool AliHLTTPCCAGPUTracker::CUDA_FAILED_MSG(hipError_t error)
{
	if (error == hipSuccess) return(false);
	printf("CUDA Error: %d / %s\n", error, hipGetErrorString(error));
	return(true);
}

//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards
int AliHLTTPCCAGPUTracker::CUDASync()
{
	if (fDebugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		printf("Cuda Error %s while invoking kernel\n", hipGetErrorString(cuErr));
		return(1);
	}
	if (CUDA_FAILED_MSG(hipDeviceSynchronize()))
	{
		printf("CUDA Error while synchronizing\n");
		return(1);
	}
	if (fDebugLevel >= 4) printf("CUDA Sync Done\n");
	return(0);
}

void AliHLTTPCCAGPUTracker::SetDebugLevel(int dwLevel, std::ostream *NewOutFile)
{
	fDebugLevel = dwLevel;
	if (NewOutFile) fOutFile = NewOutFile;
}

int AliHLTTPCCAGPUTracker::SetGPUTrackerOption(char* OptionName, int OptionValue)
{
	if (strcmp(OptionName, "SingleBlock") == 0)
	{
		fOptionSingleBlock = OptionValue;
	}
	else
	{
		printf("Unknown Option: %s\n", OptionName);
		return(1);
	}
	return(0);
}

void AliHLTTPCCAGPUTracker::StandalonePerfTime(int i)
{
#ifdef HLTCA_STANDALONE
  if (fDebugLevel >= 2)
  {
	  fGpuTracker.StandaloneQueryTime( fGpuTracker.PerfTimer(i));
  }
#endif
}

//Primary reconstruction function
int AliHLTTPCCAGPUTracker::Reconstruct(AliHLTTPCCATracker* tracker)
{
    int nThreads;
    int nBlocks;
	int size;

	int cudaDevice;
	hipDeviceProp_t fCudaDeviceProp;
	hipGetDevice(&cudaDevice);
	hipGetDeviceProperties(&fCudaDeviceProp, cudaDevice);
	

	StandalonePerfTime(0);

	if (tracker->CheckEmptySlice())
	{
		if (fDebugLevel >= 4) printf("Slice Empty, not running GPU Tracker\n");
		return(0);
	}

	if (fDebugLevel >= 3)
	{
		*fOutFile << endl << endl << "Slice: " << tracker->Param().ISlice() << endl;
	}

	if (fDebugLevel >= 4) printf("\n\nInitialising GPU Tracker\n");
	memcpy(&fGpuTracker, tracker, sizeof(AliHLTTPCCATracker));
	char* tmpMem = alignPointer((char*) fGPUMemory, 1024 * 1024);
	fGpuTracker.SetGPUTracker();

	if (fDebugLevel >= 4) printf("Initialising GPU Common Memory\n");
	tmpMem = fGpuTracker.SetGPUTrackerCommonMemory(tmpMem);
	tmpMem = alignPointer(tmpMem, 1024 * 1024);

	if (fDebugLevel >= 4) printf("Initialising GPU Hits Memory\n");
	tmpMem = fGpuTracker.SetGPUTrackerHitsMemory(tmpMem, tracker->NHitsTotal());
	tmpMem = alignPointer(tmpMem, 1024 * 1024);

	if (fDebugLevel >= 4) printf("Initialising GPU Slice Data Memory\n");
	tmpMem = fGpuTracker.SetGPUSliceDataMemory(tmpMem, fGpuTracker.ClusterData());
	tmpMem = alignPointer(tmpMem, 1024 * 1024);
	if (tmpMem - (char*) fGPUMemory > fGPUMemSize)
	{
		printf("Out of CUDA Memory\n");
		return(1);
	}
	
	CUDA_FAILED_MSG(hipMemcpy(fGpuTracker.CommonMemory(), tracker->CommonMemory(), tracker->CommonMemorySize(), hipMemcpyHostToDevice));
	CUDA_FAILED_MSG(hipMemcpy(fGpuTracker.SliceDataMemory(), tracker->SliceDataMemory(), tracker->SliceDataMemorySize(), hipMemcpyHostToDevice));
	CUDA_FAILED_MSG(hipMemcpyToSymbol(HIP_SYMBOL(gAliHLTTPCCATracker), &fGpuTracker, sizeof(AliHLTTPCCATracker)));

	StandalonePerfTime(1);

	if (fDebugLevel >= 4) printf("Running GPU Neighbours Finder\n");
	AliHLTTPCCAProcess<AliHLTTPCCANeighboursFinder> <<<fGpuTracker.Param().NRows(), 256>>>();
	if (CUDASync()) return 1;

	StandalonePerfTime(2);

	if (fDebugLevel >= 3)
	{
		*fOutFile << "Neighbours Finder:" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->SliceDataMemory(), fGpuTracker.SliceDataMemory(), tracker->SliceDataMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpLinks(*fOutFile);
    }

	if (fDebugLevel >= 4) printf("Running GPU Neighbours Cleaner\n");
	AliHLTTPCCAProcess<AliHLTTPCCANeighboursCleaner> <<<fGpuTracker.Param().NRows()-2, 256>>>();
	if (CUDASync()) return 1;

	StandalonePerfTime(3);

	if (fDebugLevel >= 3)
	{
		*fOutFile << "Neighbours Cleaner:" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->SliceDataMemory(), fGpuTracker.SliceDataMemory(), tracker->SliceDataMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpLinks(*fOutFile);
    }

	if (fDebugLevel >= 4) printf("Running GPU Start Hits Finder\n");
	AliHLTTPCCAProcess<AliHLTTPCCAStartHitsFinder> <<<fGpuTracker.Param().NRows()-4, 256>>>();
	if (CUDASync()) return 1;

	StandalonePerfTime(4);

	if (fDebugLevel >= 4) printf("Obtaining Number of Start Hits from GPU: ");
	CUDA_FAILED_MSG(hipMemcpy(tracker->CommonMemory(), fGpuTracker.CommonMemory(), tracker->CommonMemorySize(), hipMemcpyDeviceToHost));
	if (fDebugLevel >= 4) printf("%d\n", *tracker->NTracklets());
	else if (fDebugLevel >= 2) printf("%3d ", *tracker->NTracklets());

	if (fDebugLevel >= 3)
	{
		*fOutFile << "Start Hits: (" << *tracker->NTracklets() << ")" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->HitMemory(), fGpuTracker.HitMemory(), tracker->HitMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpStartHits(*fOutFile);
    }

	/*tracker->RunNeighboursFinder();
	tracker->RunNeighboursCleaner();
	tracker->RunStartHitsFinder();*/

	if (fDebugLevel >= 4) printf("Initialising GPU Track Memory\n");
	tmpMem = fGpuTracker.SetGPUTrackerTracksMemory(tmpMem, *tracker->NTracklets(), tracker->NHitsTotal());
	tmpMem = alignPointer(tmpMem, 1024 * 1024);
	if (tmpMem - (char*) fGPUMemory > fGPUMemSize)
	{
		printf("Out of CUDA Memory\n");
		return(1);
	}

	tracker->ClearSliceDataHitWeights();
	CUDA_FAILED_MSG(hipMemcpy(fGpuTracker.SliceDataHitWeights(), tracker->SliceDataHitWeights(), tracker->NHitsTotal() * sizeof(int), hipMemcpyHostToDevice));
	CUDA_FAILED_MSG(hipMemcpyToSymbol(HIP_SYMBOL(gAliHLTTPCCATracker), &fGpuTracker, sizeof(AliHLTTPCCATracker)));

	if (fDebugLevel >= 4) printf("Initialising Slice Tracker (CPU) Track Memory\n");
	tracker->TrackMemory() = reinterpret_cast<char*> ( new uint4 [ fGpuTracker.TrackMemorySize()/sizeof( uint4 ) + 100] );
    tracker->SetPointersTracks( *tracker->NTracklets(), tracker->NHitsTotal() );

/*	tracker->RunTrackletConstructor();
	if (fDebugLevel >= 3)
	{
		*fOutFile << "Tracklet Hits:" << endl;
		tracker->DumpTrackletHits(*fOutFile);
	}*/

	StandalonePerfTime(5);

	int nMemThreads = TRACKLET_CONSTRUCTOR_NMEMTHREDS;
    nThreads = 256;//96;
    nBlocks = *tracker->NTracklets()/nThreads + 1;
    if( nBlocks<30 ){
		nBlocks = HLTCA_GPU_BLOCK_COUNT;
		nThreads = (*tracker->NTracklets())/nBlocks+1;
		if( nThreads%32 ) nThreads = (nThreads/32+1)*32;
	}
	if (nThreads + nMemThreads > fCudaDeviceProp.maxThreadsPerBlock || (nThreads + nMemThreads) * HLTCA_GPU_REGS > fCudaDeviceProp.regsPerBlock)
	{
		printf("Invalid CUDA Kernel Configuration %d blocks %d threads %d memthreads\n", nBlocks, nThreads, nMemThreads);
		return(1);
	}

	if (fDebugLevel >= 4) printf("Running GPU Tracklet Constructor\n");
	if (!fOptionSingleBlock)
	{
		AliHLTTPCCAProcess1<AliHLTTPCCATrackletConstructor> <<<nBlocks, nMemThreads+nThreads>>>(); 
	}
	else
	{
		AliHLTTPCCAProcess1<AliHLTTPCCATrackletConstructor> <<<1, TRACKLET_CONSTRUCTOR_NMEMTHREDS + *tracker->NTracklets()>>>();
	}
	if (CUDASync()) return 1;

	StandalonePerfTime(6);

	if (fDebugLevel >= 3)
	{
		*fOutFile << "Tracklet Hits:" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->NTracklets(), fGpuTracker.NTracklets(), sizeof(int), hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(tracker->Tracklets(), fGpuTracker.Tracklets(), fGpuTracker.TrackMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpTrackletHits(*fOutFile);
    }

	//tracker->RunTrackletSelector();
	

	nThreads = 128;
	nBlocks = *tracker->NTracklets()/nThreads + 1;
	if( nBlocks<30 ){
	  nBlocks = HLTCA_GPU_BLOCK_COUNT;  
	  nThreads = *tracker->NTracklets()/nBlocks+1;
	  nThreads = (nThreads/32+1)*32;
	}
	if (nThreads > fCudaDeviceProp.maxThreadsPerBlock || (nThreads) * HLTCA_GPU_REGS > fCudaDeviceProp.regsPerBlock)
	{
		printf("Invalid CUDA Kernel Configuration %d blocks %d threads\n", nBlocks, nThreads);
		return(1);
	}

	if (fDebugLevel >= 4) printf("Running GPU Tracklet Selector\n");
	if (!fOptionSingleBlock)
	{
		AliHLTTPCCAProcess<AliHLTTPCCATrackletSelector><<<nBlocks, nThreads>>>();
	}
	else
	{
		AliHLTTPCCAProcess<AliHLTTPCCATrackletSelector><<<1, *tracker->NTracklets()>>>();
	}
	if (CUDASync()) return 1;

	StandalonePerfTime(7);

	if (fDebugLevel >= 4) printf("Transfering Tracks from GPU to Host ");
	CUDA_FAILED_MSG(hipMemcpy(tracker->NTracks(), fGpuTracker.NTracks(), sizeof(int), hipMemcpyDeviceToHost));
	CUDA_FAILED_MSG(hipMemcpy(tracker->NTrackHits(), fGpuTracker.NTrackHits(), sizeof(int), hipMemcpyDeviceToHost));
	if (fDebugLevel >= 4) printf("%d / %d\n", *tracker->NTracks(), *tracker->NTrackHits());
	size = sizeof(AliHLTTPCCATrack) * *tracker->NTracks();
	CUDA_FAILED_MSG(hipMemcpy(tracker->Tracks(), fGpuTracker.Tracks(), size, hipMemcpyDeviceToHost));
	size = sizeof(AliHLTTPCCAHitId) * *tracker->NTrackHits();
	if (CUDA_FAILED_MSG(hipMemcpy(tracker->TrackHits(), fGpuTracker.TrackHits(), size, hipMemcpyDeviceToHost)))
	{
		printf("CUDA Error during Reconstruction\n");
		return(1);
	}

	if (fDebugLevel >= 3)
	{
		*fOutFile << "Track Hits: (" << *tracker->NTracks() << ")" << endl;
		tracker->DumpTrackHits(*fOutFile);
    }

	if (fDebugLevel >= 4) printf("Running WriteOutput\n");
	tracker->WriteOutput();

	StandalonePerfTime(8);

	if (fDebugLevel >= 4) printf("GPU Reconstruction finished\n");
	
	return(0);
}

int AliHLTTPCCAGPUTracker::ExitGPU()
{
	hipFree(fGPUMemory);
	return(0);
}
