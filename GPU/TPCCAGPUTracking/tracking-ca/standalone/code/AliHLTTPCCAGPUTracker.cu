#include "hip/hip_runtime.h"
// **************************************************************************
// This file is property of and copyright by the ALICE HLT Project          *
// ALICE Experiment at CERN, All rights reserved.                           *
//                                                                          *
// Primary Authors: Sergey Gorbunov <sergey.gorbunov@kip.uni-heidelberg.de> *
//                  Ivan Kisel <kisel@kip.uni-heidelberg.de>                *
//					David Rohr <drohr@kip.uni-heidelberg.de>				*
//                  for The ALICE HLT Project.                              *
//                                                                          *
// Permission to use, copy, modify and distribute this software and its     *
// documentation strictly for non-commercial purposes is hereby granted     *
// without fee, provided that the above copyright notice appears in all     *
// copies and that both the copyright notice and this permission notice     *
// appear in the supporting documentation. The authors make no claims       *
// about the suitability of this software for any purpose. It is            *
// provided "as is" without express or implied warranty.                    *
//                                                                          *
//***************************************************************************

#include "AliHLTTPCCADef.h"
#include "AliHLTTPCCAGPUConfig.h"

//#include <cutil.h>
#ifndef CUDA_DEVICE_EMULATION
//#include <cutil_inline_runtime.h>
#else
//#include <cutil_inline.h>
#endif
#include <sm_11_atomic_functions.h>
#include <sm_12_atomic_functions.h>

#include <iostream>

//Disable assertions since they produce errors in GPU Code
#ifdef assert
#undef assert
#endif
#define assert(param)

#include "AliHLTTPCCAGPUTracker.h"

#include "AliHLTTPCCAHit.h"

//Include CXX Files, GPUd() macro will then produce CUDA device code out of the tracker source code
#include "AliHLTTPCCATrackParam.cxx"
#include "AliHLTTPCCATrack.cxx" 

#include "AliHLTTPCCATrackletSelector.cxx"

#include "AliHLTTPCCAHitArea.cxx"
#include "AliHLTTPCCAGrid.cxx"
#include "AliHLTTPCCARow.cxx"
#include "AliHLTTPCCAParam.cxx"
#include "AliHLTTPCCATracker.cxx"

#include "AliHLTTPCCAOutTrack.cxx"

#include "AliHLTTPCCAProcess.h"

#include "AliHLTTPCCANeighboursFinder.cxx"

#include "AliHLTTPCCANeighboursCleaner.cxx"
#include "AliHLTTPCCAStartHitsFinder.cxx"
#include "AliHLTTPCCAStartHitsSorter.cxx"
#include "AliHLTTPCCATrackletConstructor.cxx"
#include "AliHLTTPCCASliceOutput.cxx"

#include "MemoryAssignmentHelpers.h"

//Find best CUDA device, initialize and allocate memory
int AliHLTTPCCAGPUTracker::InitGPU(int forceDeviceID)
{
	hipDeviceProp_t fCudaDeviceProp;


	int count, bestDevice, bestDeviceSpeed = 0;
	hipGetDeviceCount(&count);
	if (fDebugLevel >= 2) std::cout << "Available CUDA devices: ";
	for (int i = 0;i < count;i++)
	{
		hipGetDeviceProperties(&fCudaDeviceProp, i);
		if (fDebugLevel >= 2) std::cout << fCudaDeviceProp.name << " (" << i << ")     ";
		if (fCudaDeviceProp.multiProcessorCount * fCudaDeviceProp.clockRate > bestDeviceSpeed)
		{
			bestDevice = i;
			bestDeviceSpeed = fCudaDeviceProp.multiProcessorCount * fCudaDeviceProp.clockRate;
		}
	}
	if (fDebugLevel >= 2) std::cout << std::endl;

  int cudaDevice;
  if (forceDeviceID == -1)
	  cudaDevice = bestDevice;
  else
	  cudaDevice = forceDeviceID;
  hipSetDevice(cudaDevice);

  hipGetDeviceProperties(&fCudaDeviceProp ,cudaDevice ); 

  if (fDebugLevel >= 1)
  {
	  std::cout<<"CUDA Device Properties: "<<std::endl;
	  std::cout<<"name = "<<fCudaDeviceProp.name<<std::endl;
	  std::cout<<"totalGlobalMem = "<<fCudaDeviceProp.totalGlobalMem<<std::endl;
	  std::cout<<"sharedMemPerBlock = "<<fCudaDeviceProp.sharedMemPerBlock<<std::endl;
	  std::cout<<"regsPerBlock = "<<fCudaDeviceProp.regsPerBlock<<std::endl;
	  std::cout<<"warpSize = "<<fCudaDeviceProp.warpSize<<std::endl;
	  std::cout<<"memPitch = "<<fCudaDeviceProp.memPitch<<std::endl;
	  std::cout<<"maxThreadsPerBlock = "<<fCudaDeviceProp.maxThreadsPerBlock<<std::endl;
	  std::cout<<"maxThreadsDim = "<<fCudaDeviceProp.maxThreadsDim[0]<<" "<<fCudaDeviceProp.maxThreadsDim[1]<<" "<<fCudaDeviceProp.maxThreadsDim[2]<<std::endl;
	  std::cout<<"maxGridSize = "  <<fCudaDeviceProp.maxGridSize[0]<<" "<<fCudaDeviceProp.maxGridSize[1]<<" "<<fCudaDeviceProp.maxGridSize[2]<<std::endl;
	  std::cout<<"totalConstMem = "<<fCudaDeviceProp.totalConstMem<<std::endl;
	  std::cout<<"major = "<<fCudaDeviceProp.major<<std::endl;
	  std::cout<<"minor = "<<fCudaDeviceProp.minor<<std::endl;
	  std::cout<<"clockRate = "<<fCudaDeviceProp.clockRate<<std::endl;
	  std::cout<<"textureAlignment = "<<fCudaDeviceProp.textureAlignment<<std::endl;
  }

  if (fCudaDeviceProp.major < 1 || (fCudaDeviceProp.major == 1 && fCudaDeviceProp.minor < 2))
  {
	  std::cout << "Unsupported CUDA Device\n";
	  return(1);
  }

  fGPUMemSize = (long long int) fCudaDeviceProp.totalGlobalMem - 400 * 1024 * 1024;
  if (fGPUMemSize > 1024 * 1024 * 1024) fGPUMemSize = 1024 * 1024 * 1024;
  if (CUDA_FAILED_MSG(hipMalloc(&fGPUMemory, (size_t) fGPUMemSize)))
  {
	  std::cout << "CUDA Memory Allocation Error\n";
	  return(1);
  }
  if (fDebugLevel >= 1)
  {
	  CUDA_FAILED_MSG(hipMemset(fGPUMemory, 255, (size_t) fGPUMemSize));
  }
  std::cout << "CUDA Initialisation successfull\n";

	return(0);
}

//Macro to align Pointers.
//Will align to start at 1 MB segments, this should be consistent with every alignment in the tracker
//(As long as every single data structure is <= 1 MB)
template <class T> inline T* AliHLTTPCCAGPUTracker::alignPointer(T* ptr, int alignment)
{
	size_t adr = (size_t) ptr;
	if (adr % alignment)
	{
		adr += alignment - (adr % alignment);
	}
	return((T*) adr);
}

//Check for CUDA Error and in the case of an error display the corresponding error string
bool AliHLTTPCCAGPUTracker::CUDA_FAILED_MSG(hipError_t error)
{
	if (error == hipSuccess) return(false);
	printf("CUDA Error: %d / %s\n", error, hipGetErrorString(error));
	return(true);
}

//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards
int AliHLTTPCCAGPUTracker::CUDASync(char* state)
{
	if (fDebugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		printf("Cuda Error %s while invoking kernel (%s)\n", hipGetErrorString(cuErr), state);
		return(1);
	}
	if (CUDA_FAILED_MSG(hipDeviceSynchronize()))
	{
		printf("CUDA Error while synchronizing (%s)\n", state);
		return(1);
	}
	if (fDebugLevel >= 5) printf("CUDA Sync Done\n");
	return(0);
}

void AliHLTTPCCAGPUTracker::SetDebugLevel(int dwLevel, std::ostream *NewOutFile)
{
	fDebugLevel = dwLevel;
	if (NewOutFile) fOutFile = NewOutFile;
}

int AliHLTTPCCAGPUTracker::SetGPUTrackerOption(char* OptionName, int OptionValue)
{
	if (strcmp(OptionName, "SingleBlock") == 0)
	{
		fOptionSingleBlock = OptionValue;
	}
	else if (strcmp(OptionName, "AdaptSched") == 0)
	{
		fOptionAdaptiveSched = OptionValue;
	}
	else
	{
		printf("Unknown Option: %s\n", OptionName);
		return(1);
	}
	return(0);
}

void AliHLTTPCCAGPUTracker::StandalonePerfTime(int i)
{
#ifdef HLTCA_STANDALONE
  if (fDebugLevel >= 1)
  {
	  fGpuTracker.StandaloneQueryTime( fGpuTracker.PerfTimer(i));
  }
#endif
}

void AliHLTTPCCAGPUTracker::DumpRowBlocks(AliHLTTPCCATracker* tracker, bool check)
{
	if (fDebugLevel >= 4)
	{
		*fOutFile << "RowBlock Tracklets" << std::endl;
	
		int3* RowBlockPos = (int3*) malloc(sizeof(int3) * (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * 2);
		int* RowBlockTracklets = (int*) malloc(sizeof(int) * (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * HLTCA_GPU_MAX_TRACKLETS * 2);
		uint2* BlockStartingTracklet = (uint2*) malloc(sizeof(uint2) * HLTCA_GPU_BLOCK_COUNT);
		CUDA_FAILED_MSG(hipMemcpy(RowBlockPos, fGpuTracker.RowBlockPos(), sizeof(int3) * (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * 2, hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(RowBlockTracklets, fGpuTracker.RowBlockTracklets(), sizeof(int) * (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * HLTCA_GPU_MAX_TRACKLETS * 2, hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(BlockStartingTracklet, fGpuTracker.BlockStartingTracklet(), sizeof(uint2) * HLTCA_GPU_BLOCK_COUNT, hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(tracker->CommonMemory(), fGpuTracker.CommonMemory(), tracker->CommonMemorySize(), hipMemcpyDeviceToHost));

		int k = tracker->GPUParameters()->fScheduleFirstDynamicTracklet;
		for (int i = 0; i < tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1;i++)
		{
			*fOutFile << "Rowblock: " << i << ", up " << RowBlockPos[i].y << "/" << RowBlockPos[i].x << ", down " << 
				RowBlockPos[tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1 + i].y << "/" << RowBlockPos[tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1 + i].x << endl << "Phase 1: ";
			for (int j = 0;j < RowBlockPos[i].x;j++)
			{
				//Use Tracker Object to calculate Offset instead of fGpuTracker, since *fNTracklets of fGpuTracker points to GPU Mem!
				*fOutFile << RowBlockTracklets[(tracker->RowBlockTracklets(0, i) - tracker->RowBlockTracklets(0, 0)) + j] << ", ";
				if (check && RowBlockTracklets[(tracker->RowBlockTracklets(0, i) - tracker->RowBlockTracklets(0, 0)) + j] != k)
				{
					printf("Wrong starting Row Block %d, entry %d, is %d, should be %d\n", i, j, RowBlockTracklets[(tracker->RowBlockTracklets(0, i) - tracker->RowBlockTracklets(0, 0)) + j], k);
				}
				k++;
				if (RowBlockTracklets[(tracker->RowBlockTracklets(0, i) - tracker->RowBlockTracklets(0, 0)) + j] == -1)
				{
					printf("Error, -1 Tracklet found\n");
				}
			}
			*fOutFile << endl << "Phase 2: ";
			for (int j = 0;j < RowBlockPos[tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1 + i].x;j++)
			{
				*fOutFile << RowBlockTracklets[(tracker->RowBlockTracklets(1, i) - tracker->RowBlockTracklets(0, 0)) + j] << ", ";
			}
			*fOutFile << endl;
		}

		if (check)
		{
			*fOutFile << "Starting Threads: (First Dynamic: " << tracker->GPUParameters()->fScheduleFirstDynamicTracklet << ")" << std::endl;
			for (int i = 0;i < HLTCA_GPU_BLOCK_COUNT;i++)
			{
				*fOutFile << i << ": " << BlockStartingTracklet[i].x << " - " << BlockStartingTracklet[i].y << std::endl;
			}
		}

		free(RowBlockPos);
		free(RowBlockTracklets);
		free(BlockStartingTracklet);
	}
}

//Primary reconstruction function
int AliHLTTPCCAGPUTracker::Reconstruct(AliHLTTPCCATracker* tracker)
{
    int nThreads;
    int nBlocks;
	int size;

	int cudaDevice;
	hipDeviceProp_t fCudaDeviceProp;
	hipGetDevice(&cudaDevice);
	hipGetDeviceProperties(&fCudaDeviceProp, cudaDevice);
	
	if (tracker->CheckEmptySlice())
	{
		if (fDebugLevel >= 5) printf("Slice Empty, not running GPU Tracker\n");
		return(0);
	}

	if (fDebugLevel >= 4)
	{
		*fOutFile << endl << endl << "Slice: " << tracker->Param().ISlice() << endl;
	}

	if (fDebugLevel >= 5) printf("\n\nInitialising GPU Tracker\n");
	memcpy(&fGpuTracker, tracker, sizeof(AliHLTTPCCATracker));

	StandalonePerfTime(0);

	char* tmpMem = alignPointer((char*) fGPUMemory, 1024 * 1024);
	fGpuTracker.SetGPUTracker();

	if (fDebugLevel >= 5) printf("Initialising GPU Common Memory\n");
	tmpMem = fGpuTracker.SetGPUTrackerCommonMemory(tmpMem);
	tmpMem = alignPointer(tmpMem, 1024 * 1024);

	if (fDebugLevel >= 5) printf("Initialising GPU Hits Memory\n");
	tmpMem = fGpuTracker.SetGPUTrackerHitsMemory(tmpMem, tracker->NHitsTotal());
	tmpMem = alignPointer(tmpMem, 1024 * 1024);

	if (fDebugLevel >= 5) printf("Initialising GPU Slice Data Memory\n");
	tmpMem = fGpuTracker.SetGPUSliceDataMemory(tmpMem, fGpuTracker.ClusterData());
	tmpMem = alignPointer(tmpMem, 1024 * 1024);
	if (tmpMem - (char*) fGPUMemory > fGPUMemSize)
	{
		printf("Out of CUDA Memory\n");
		return(1);
	}

#ifdef HLTCA_STANDALONE
	if (fDebugLevel >= 2)
	{
		if (CUDA_FAILED_MSG(hipMalloc((void**) &fGpuTracker.fGPUDebugMem, 100 * 1024 * 1024)))
		{
			printf("Out of CUDA Memory\n");
			return(1);
		}
		CUDA_FAILED_MSG(hipMemset(fGpuTracker.fGPUDebugMem, 0, 100 * 1024 * 1024));
	}
#endif

	if (fDebugLevel >= 5) printf("Initialising GPU Track Memory\n");
	tmpMem = fGpuTracker.SetGPUTrackerTracksMemory(tmpMem, HLTCA_GPU_MAX_TRACKLETS /**tracker->NTracklets()*/, tracker->NHitsTotal());
	tmpMem = alignPointer(tmpMem, 1024 * 1024);
	if (tmpMem - (char*) fGPUMemory > fGPUMemSize)
	{
		printf("Out of CUDA Memory\n");
		return(1);
	}

	if (fDebugLevel >= 1)
	{
		static int showMemInfo = true;
		if (showMemInfo)
			printf("GPU Memory used: %d bytes\n", (int) (tmpMem - (char*) fGPUMemory));
		showMemInfo = false;
	}

	*tracker->NTracklets() = 0;
	tracker->GPUParameters()->fStaticStartingTracklets = 1;
	tracker->GPUParameters()->fGPUError = 0;
	tracker->GPUParameters()->fGPUSchedCollisions = 0;

	CUDA_FAILED_MSG(hipMemcpy(fGpuTracker.CommonMemory(), tracker->CommonMemory(), tracker->CommonMemorySize(), hipMemcpyHostToDevice));
	CUDA_FAILED_MSG(hipMemcpy(fGpuTracker.SliceDataMemory(), tracker->SliceDataMemory(), tracker->SliceDataMemorySize(), hipMemcpyHostToDevice));
	CUDA_FAILED_MSG(hipMemset(fGpuTracker.RowBlockPos(), 0, sizeof(int3) * 2 * (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1)));
	CUDA_FAILED_MSG(hipMemset(fGpuTracker.RowBlockTracklets(), -1, sizeof(int) * (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * HLTCA_GPU_MAX_TRACKLETS * 2));
	CUDA_FAILED_MSG(hipMemcpyToSymbol(HIP_SYMBOL(gAliHLTTPCCATracker), &fGpuTracker, sizeof(AliHLTTPCCATracker)));

	StandalonePerfTime(1);

	if (fDebugLevel >= 5) printf("Running GPU Neighbours Finder\n");
	AliHLTTPCCAProcess<AliHLTTPCCANeighboursFinder> <<<fGpuTracker.Param().NRows(), 256>>>();
	if (CUDASync("Neighbours finder")) return 1;

	StandalonePerfTime(2);

	if (fDebugLevel >= 4)
	{
		*fOutFile << "Neighbours Finder:" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->SliceDataMemory(), fGpuTracker.SliceDataMemory(), tracker->SliceDataMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpLinks(*fOutFile);
    }

	if (fDebugLevel >= 5) printf("Running GPU Neighbours Cleaner\n");
	AliHLTTPCCAProcess<AliHLTTPCCANeighboursCleaner> <<<fGpuTracker.Param().NRows()-2, 256>>>();
	if (CUDASync("Neighbours Cleaner")) return 1;

	StandalonePerfTime(3);

	if (fDebugLevel >= 4)
	{
		*fOutFile << "Neighbours Cleaner:" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->SliceDataMemory(), fGpuTracker.SliceDataMemory(), tracker->SliceDataMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpLinks(*fOutFile);
    }

	if (fDebugLevel >= 5) printf("Running GPU Start Hits Finder\n");
	AliHLTTPCCAProcess<AliHLTTPCCAStartHitsFinder> <<<fGpuTracker.Param().NRows()-4, 256>>>();
	if (CUDASync("Start Hits Finder")) return 1;

	StandalonePerfTime(4);

#ifdef HLTCA_GPU_SORT_STARTHITS
	if (fDebugLevel >= 5) printf("Running GPU Start Hits Sorter\n");
	AliHLTTPCCAProcess<AliHLTTPCCAStartHitsSorter> <<<30, 256>>>();
	if (CUDASync("Start Hits Sorter")) return 1;
#endif

	StandalonePerfTime(5);

	if (fDebugLevel >= 5) printf("Obtaining Number of Start Hits from GPU: ");
	CUDA_FAILED_MSG(hipMemcpy(tracker->CommonMemory(), fGpuTracker.CommonMemory(), tracker->CommonMemorySize(), hipMemcpyDeviceToHost));
	if (fDebugLevel >= 5) printf("%d\n", *tracker->NTracklets());
	else if (fDebugLevel >= 2) printf("%3d ", *tracker->NTracklets());

#ifdef HLTCA_GPU_SORT_STARTHITS
	if (fDebugLevel >= 4)
	{
		*fOutFile << "Start Hits Tmp: (" << *tracker->NTracklets() << ")" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->TrackletStartHits(), fGpuTracker.TrackletTmpStartHits(), tracker->NHitsTotal() * sizeof(AliHLTTPCCAHit), hipMemcpyDeviceToHost));
		tracker->DumpStartHits(*fOutFile);
		uint3* tmpMem = (uint3*) malloc(sizeof(uint3) * tracker->Param().NRows());
		CUDA_FAILED_MSG(hipMemcpy(tmpMem, fGpuTracker.RowStartHitCountOffset(), tracker->Param().NRows() * sizeof(uint3), hipMemcpyDeviceToHost));
		*fOutFile << "Start Hits Sort Vector:" << std::endl;
		for (int i = 0;i < tracker->Param().NRows();i++)
		{
			*fOutFile << "Row: " << i << ", Len: " << tmpMem[i].x << ", Offset: " << tmpMem[i].y << ", New Offset: " << tmpMem[i].z << std::endl;
		}
		free(tmpMem);
    }
#endif

	if (fDebugLevel >= 4)
	{
		*fOutFile << "Start Hits: (" << *tracker->NTracklets() << ")" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->HitMemory(), fGpuTracker.HitMemory(), tracker->HitMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpStartHits(*fOutFile);
    }

	/*tracker->RunNeighboursFinder();
	tracker->RunNeighboursCleaner();
	tracker->RunStartHitsFinder();*/

	if (*tracker->NTracklets() > HLTCA_GPU_MAX_TRACKLETS)
	{
		printf("HLTCA_GPU_MAX_TRACKLETS constant insuffisant\n");
		return(1);
	}

	CUDA_FAILED_MSG(hipMemset(fGpuTracker.SliceDataHitWeights(), 0, tracker->NHitsTotal() * sizeof(int)));
	//tracker->ClearSliceDataHitWeights();
	//CUDA_FAILED_MSG(hipMemcpy(fGpuTracker.SliceDataHitWeights(), tracker->SliceDataHitWeights(), tracker->NHitsTotal() * sizeof(int), hipMemcpyHostToDevice));

	if (fDebugLevel >= 5) printf("Initialising Slice Tracker (CPU) Track Memory\n");
	tracker->TrackMemory() = reinterpret_cast<char*> ( new uint4 [ fGpuTracker.TrackMemorySize()/sizeof( uint4 ) + 100] );
    tracker->SetPointersTracks( *tracker->NTracklets(), tracker->NHitsTotal() );

/*	tracker->RunTrackletConstructor();
	if (fDebugLevel >= 4)
	{
		*fOutFile << "Tracklet Hits:" << endl;
		tracker->DumpTrackletHits(*fOutFile);
	}*/

	StandalonePerfTime(6);

#ifdef HLTCA_GPU_PREFETCHDATA
	if (tracker->Data().GPUSharedDataReq() * sizeof(ushort_v) > ALIHLTTPCCATRACKLET_CONSTRUCTOR_TEMP_MEM / 4 * sizeof(uint4))
	{
		printf("Insufficiant GPU shared Memory, required: %d, available %d\n", tracker->Data().GPUSharedDataReq() * sizeof(ushort_v), ALIHLTTPCCATRACKLET_CONSTRUCTOR_TEMP_MEM / 4 * sizeof(uint4));
		return(1);
	}
#endif

	int nMemThreads = TRACKLET_CONSTRUCTOR_NMEMTHREDS;
    nThreads = HLTCA_GPU_THREAD_COUNT - nMemThreads;//96;
    nBlocks = *tracker->NTracklets()/nThreads + 1;
    if( nBlocks<30 ){
		nBlocks = HLTCA_GPU_BLOCK_COUNT;
		nThreads = (*tracker->NTracklets())/nBlocks+1;
		if( nThreads%32 ) nThreads = (nThreads/32+1)*32;
	}
	if (nThreads + nMemThreads > fCudaDeviceProp.maxThreadsPerBlock || (nThreads + nMemThreads) * HLTCA_GPU_REGS > fCudaDeviceProp.regsPerBlock)
	{
		printf("Invalid CUDA Kernel Configuration %d blocks %d threads %d memthreads\n", nBlocks, nThreads, nMemThreads);
		return(1);
	}

#ifdef HLTCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	if (CUDA_FAILED_MSG(hipMalloc((void**) &fGpuTracker.fStageAtSync, nBlocks * nThreads * (4 + 159*2 + 1 + 1 + 1) * sizeof(int))) ||
		CUDA_FAILED_MSG(hipMalloc((void**) &fGpuTracker.fThreadTimes, 30 * 256 * sizeof(int))))
	{
		return(1);
	}
	CUDA_FAILED_MSG(hipMemset(fGpuTracker.fStageAtSync, 0, nBlocks * nThreads * (4 + 159*2 + 1 + 1 + 1) * sizeof(int)));
	int* StageAtSync = (int*) malloc(nBlocks * nThreads * (4 + 159*2 + 1 + 1 + 1) * sizeof(int));
	int* ThreadTimes = (int*) malloc(30 * 256 * sizeof(int));
	CUDA_FAILED_MSG(hipMemcpyToSymbol(HIP_SYMBOL(gAliHLTTPCCATracker), &fGpuTracker, sizeof(AliHLTTPCCATracker)));
#endif

	if (fDebugLevel >= 5) printf("Running GPU Tracklet Constructor\n");

	if (fOptionAdaptiveSched)
	{
		AliHLTTPCCATrackletConstructorInit<<<*tracker->NTracklets() / HLTCA_GPU_THREAD_COUNT + 1, HLTCA_GPU_THREAD_COUNT>>>();
		if (CUDASync("Tracklet Initializer")) return 1;
		StandalonePerfTime(7);

		DumpRowBlocks(tracker);
#ifdef HLTCA_GPU_SCHED_HOST_SYNC
		for (int i = 0;i < (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * 2;i++)
		{
			if (fDebugLevel >= 4) *fOutFile << "Scheduled Tracklet Constructor Iteration " << i << std::endl;
			AliHLTTPCCATrackletConstructorNew<<<HLTCA_GPU_BLOCK_COUNT, HLTCA_GPU_THREAD_COUNT>>>();
			if (CUDASync("Tracklet Constructor (new)")) return 1;
			AliHLTTPCCATrackletConstructorUpdateRowBlockPos<<<HLTCA_GPU_BLOCK_COUNT, (tracker->Param().NRows() / HLTCA_GPU_SCHED_ROW_STEP + 1) * 2 / HLTCA_GPU_BLOCK_COUNT + 1>>>();
			if (CUDASync("Tracklet Constructor (update)")) return 1;
			DumpRowBlocks(tracker, false);
		}
#else
		AliHLTTPCCATrackletConstructorNew<<<HLTCA_GPU_BLOCK_COUNT, HLTCA_GPU_THREAD_COUNT>>>();
		DumpRowBlocks(tracker, false);
#endif
	}
	else if (!fOptionSingleBlock)
	{
		StandalonePerfTime(7);
		//AliHLTTPCCAProcess1<AliHLTTPCCATrackletConstructor> <<<nBlocks, nMemThreads+nThreads>>>(); 
	}
	else
	{
		StandalonePerfTime(7);
		//AliHLTTPCCAProcess1<AliHLTTPCCATrackletConstructor> <<<1, TRACKLET_CONSTRUCTOR_NMEMTHREDS + *tracker->NTracklets()>>>();
	}
	if (CUDASync("Tracklet Constructor")) return 1;

#ifdef HLTCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	printf("Saving Profile\n");
	CUDA_FAILED_MSG(hipMemcpy(StageAtSync, fGpuTracker.fStageAtSync, nBlocks * nThreads * (4 + 159*2 + 1 + 1 + 1) * sizeof(int), hipMemcpyDeviceToHost));
	CUDA_FAILED_MSG(hipMemcpy(ThreadTimes, fGpuTracker.fThreadTimes, 30 * 256 * sizeof(int), hipMemcpyDeviceToHost));

	FILE *fp = fopen("profile.txt", "w+"), *fp2 = fopen("profile.bmp", "w+b"), *fp3 = fopen("times.txt", "w+");
	if (fp == NULL || fp2 == NULL || fp3 == NULL)
	{
		printf("Error opening Profile File\n");
		return(1);
	}
	BITMAPFILEHEADER bmpFH;
	BITMAPINFOHEADER bmpIH;
	ZeroMemory(&bmpFH, sizeof(bmpFH));
	ZeroMemory(&bmpIH, sizeof(bmpIH));
	
	bmpFH.bfType = 19778; //"BM"
	bmpFH.bfSize = sizeof(bmpFH) + sizeof(bmpIH) + nBlocks * nThreads * (4 + 159*2 + 1 + 1 + 1) * 4;
	bmpFH.bfOffBits = sizeof(bmpFH) + sizeof(bmpIH);

	bmpIH.biSize = sizeof(bmpIH);
	bmpIH.biWidth = nBlocks * nThreads + nBlocks * nThreads / 32;
	if (nBlocks * nThreads % 32 == 0) bmpIH.biWidth--;
	bmpIH.biHeight = 4 + 159*2 + 1 + 1 + 1;
	bmpIH.biPlanes = 1;
	bmpIH.biBitCount = 32;

	fwrite(&bmpFH, 1, sizeof(bmpFH), fp2);
	fwrite(&bmpIH, 1, sizeof(bmpIH), fp2);

	for (int i = 0;i < 4 + 159*2 + 1 + 1 + 1;i++)
	{
		for (int j = 0;j < nBlocks * nThreads;j++)
		{
			fprintf(fp, "%d\t", StageAtSync[i * nBlocks * nThreads + j]);
			int color = 0;
			if (StageAtSync[i * nBlocks * nThreads + j] == 1) color = RGB(255, 0, 0);
			if (StageAtSync[i * nBlocks * nThreads + j] == 2) color = RGB(0, 255, 0);
			if (StageAtSync[i * nBlocks * nThreads + j] == 3) color = RGB(0, 0, 255);
			if (StageAtSync[i * nBlocks * nThreads + j] == 4) color = RGB(255, 255, 0);
			fwrite(&color, 1, 4, fp2);
			if (j > 0 && j % 32 == 0)
			{
				color = RGB(255, 255, 255);
				fwrite(&color, 1, 4, fp2);
			}
		}
		fprintf(fp, "\n");
	}

	for (int i = 0;i < 30;i++)
	{
		for (int j = 0;j < 256;j++)
		{
			fprintf(fp3, "%d\t", ThreadTimes[i * 256 + j]);
		}
		fprintf(fp3, "\n");
	}
	fclose(fp);
	fclose(fp2);
	fclose(fp3);

	hipFree(fGpuTracker.fStageAtSync);
	hipFree(fGpuTracker.fThreadTimes);
	free(StageAtSync);
	free(ThreadTimes);
#endif

	StandalonePerfTime(8);

	if (fDebugLevel >= 4)
	{
		*fOutFile << "Tracklet Hits:" << endl;
		CUDA_FAILED_MSG(hipMemcpy(tracker->NTracklets(), fGpuTracker.NTracklets(), sizeof(int), hipMemcpyDeviceToHost));
		CUDA_FAILED_MSG(hipMemcpy(tracker->Tracklets(), fGpuTracker.Tracklets(), fGpuTracker.TrackMemorySize(), hipMemcpyDeviceToHost));
		tracker->DumpTrackletHits(*fOutFile);
    }

	//tracker->RunTrackletSelector();
	

	nThreads = HLTCA_GPU_THREAD_COUNT;
	nBlocks = *tracker->NTracklets()/nThreads + 1;
	if( nBlocks<30 ){
	  nBlocks = HLTCA_GPU_BLOCK_COUNT;  
	  nThreads = *tracker->NTracklets()/nBlocks+1;
	  nThreads = (nThreads/32+1)*32;
	}
	if (nThreads > fCudaDeviceProp.maxThreadsPerBlock || (nThreads) * HLTCA_GPU_REGS > fCudaDeviceProp.regsPerBlock)
	{
		printf("Invalid CUDA Kernel Configuration %d blocks %d threads\n", nBlocks, nThreads);
		return(1);
	}

	if (fDebugLevel >= 5) printf("Running GPU Tracklet Selector\n");
	if (!fOptionSingleBlock)
	{
		AliHLTTPCCAProcess<AliHLTTPCCATrackletSelector><<<nBlocks, nThreads>>>();
	}
	else
	{
		AliHLTTPCCAProcess<AliHLTTPCCATrackletSelector><<<1, *tracker->NTracklets()>>>();
	}
	if (CUDASync("Tracklet Selector")) return 1;

	StandalonePerfTime(9);

	if (fDebugLevel >= 5) printf("Transfering Tracks from GPU to Host ");
	CUDA_FAILED_MSG(hipMemcpy(tracker->CommonMemory(), fGpuTracker.CommonMemory(), tracker->CommonMemorySize(), hipMemcpyDeviceToHost));
	if (tracker->GPUParameters()->fGPUError)
	{
		printf("GPU Tracker returned Error Code %d\n", tracker->GPUParameters()->fGPUError);
		return(1);
	}
	if (tracker->GPUParameters()->fGPUSchedCollisions)
		printf("Collisions: %d\n", tracker->GPUParameters()->fGPUSchedCollisions);
	if (fDebugLevel >= 5) printf("%d / %d\n", *tracker->NTracks(), *tracker->NTrackHits());
	size = sizeof(AliHLTTPCCATrack) * *tracker->NTracks();
	CUDA_FAILED_MSG(hipMemcpy(tracker->Tracks(), fGpuTracker.Tracks(), size, hipMemcpyDeviceToHost));
	size = sizeof(AliHLTTPCCAHitId) * *tracker->NTrackHits();
	if (CUDA_FAILED_MSG(hipMemcpy(tracker->TrackHits(), fGpuTracker.TrackHits(), size, hipMemcpyDeviceToHost)))
	{
		printf("CUDA Error during Reconstruction\n");
		return(1);
	}

	if (fDebugLevel >= 4)
	{
		*fOutFile << "Track Hits: (" << *tracker->NTracks() << ")" << endl;
		tracker->DumpTrackHits(*fOutFile);
    }

	if (fDebugLevel >= 5) printf("Running WriteOutput\n");
	tracker->WriteOutput();

	StandalonePerfTime(10);

	if (fDebugLevel >= 5) printf("GPU Reconstruction finished\n");

#ifdef HLTCA_STANDALONE
	if (fDebugLevel >= 2)
	{
		std::ofstream tmpout("tmpdebug.out");
		int* GPUDebug = (int*) malloc(100 * 1024 * 1024);
		CUDA_FAILED_MSG(hipMemcpy(GPUDebug, fGpuTracker.fGPUDebugMem, 100 * 1024 * 1024, hipMemcpyDeviceToHost));
		free(GPUDebug);
		hipFree(fGpuTracker.fGPUDebugMem);
		tmpout.close();
	}
#endif

	
	return(0);
}

int AliHLTTPCCAGPUTracker::ExitGPU()
{
	hipFree(fGPUMemory);
	return(0);
}
