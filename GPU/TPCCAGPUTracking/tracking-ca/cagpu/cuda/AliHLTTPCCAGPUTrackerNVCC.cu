#include "hip/hip_runtime.h"
// **************************************************************************
// This file is property of and copyright by the ALICE HLT Project          *
// ALICE Experiment at CERN, All rights reserved.                           *
//                                                                          *
// Primary Authors: Sergey Gorbunov <sergey.gorbunov@kip.uni-heidelberg.de> *
//                  Ivan Kisel <kisel@kip.uni-heidelberg.de>                *
//					David Rohr <drohr@kip.uni-heidelberg.de>				*
//                  for The ALICE HLT Project.                              *
//                                                                          *
// Permission to use, copy, modify and distribute this software and its     *
// documentation strictly for non-commercial purposes is hereby granted     *
// without fee, provided that the above copyright notice appears in all     *
// copies and that both the copyright notice and this permission notice     *
// appear in the supporting documentation. The authors make no claims       *
// about the suitability of this software for any purpose. It is            *
// provided "as is" without express or implied warranty.                    *
//                                                                          *
//***************************************************************************

#define PASCAL
#include "AliHLTTPCCAGPUTrackerNVCC.h"
#include "AliHLTTPCCAGPUTrackerCommon.h"
#define get_global_id(dim) (blockIdx.x * blockDim.x + threadIdx.x)
#define get_global_size(dim) (blockDim.x * gridDim.x)
#define get_num_groups(dim) (gridDim.x)
#define get_local_id(dim) (threadIdx.x)
#define get_local_size(dim) (blockDim.x)
#define get_group_id(dim) (blockIdx.x)

#include <hip/hip_runtime.h>
#include <sm_20_atomic_functions.h>

__constant__ float4 gAliHLTTPCCATracker[HLTCA_GPU_TRACKER_CONSTANT_MEM / sizeof( float4 )];
#ifdef HLTCA_GPU_USE_TEXTURES
texture<cahit2, hipTextureType1D, hipReadModeElementType> gAliTexRefu2;
texture<calink, hipTextureType1D, hipReadModeElementType> gAliTexRefu;
#endif

//Include CXX Files, GPUd() macro will then produce CUDA device code out of the tracker source code
#include "AliHLTTPCCATrackParam.cxx"
#include "AliHLTTPCCATrack.cxx" 

#include "AliHLTTPCCAHitArea.cxx"
#include "AliHLTTPCCAGrid.cxx"
#include "AliHLTTPCCARow.cxx"
#include "AliHLTTPCCAParam.cxx"
#include "AliHLTTPCCATracker.cxx"

#include "AliHLTTPCCAProcess.h"

#include "AliHLTTPCCATrackletSelector.cxx"
#include "AliHLTTPCCANeighboursFinder.cxx"
#include "AliHLTTPCCANeighboursCleaner.cxx"
#include "AliHLTTPCCAStartHitsFinder.cxx"
#include "AliHLTTPCCAStartHitsSorter.cxx"
#include "AliHLTTPCCATrackletConstructor.cxx"

#ifdef HLTCA_GPU_MERGER
#include "AliHLTTPCGMMerger.h"
#include "AliHLTTPCGMTrackParam.cxx"
#include "AliHLTTPCGMPhysicalTrackModel.cxx"
#include "AliHLTTPCGMPropagator.cxx"
#endif

ClassImp( AliHLTTPCCAGPUTrackerNVCC )

AliHLTTPCCAGPUTrackerNVCC::AliHLTTPCCAGPUTrackerNVCC() : fpCudaStreams(NULL)
{
	fCudaContext = (void*) new hipCtx_t;
};

AliHLTTPCCAGPUTrackerNVCC::~AliHLTTPCCAGPUTrackerNVCC()
{
	delete (hipCtx_t*) fCudaContext;
};

int AliHLTTPCCAGPUTrackerNVCC::InitGPU_Runtime(int sliceCount, int forceDeviceID)
{
	//Find best CUDA device, initialize and allocate memory

	hipDeviceProp_t fCudaDeviceProp;

#ifndef CUDA_DEVICE_EMULATION
	int count, bestDevice = -1;
	double bestDeviceSpeed = -1, deviceSpeed;
	if (GPUFailedMsg(hipGetDeviceCount(&count)))
	{
		HLTError("Error getting CUDA Device Count");
		return(1);
	}
	if (fDebugLevel >= 2) HLTInfo("Available CUDA devices:");
#if defined(FERMI) || defined(KEPLER)
	const int reqVerMaj = 2;
	const int reqVerMin = 0;
#else
	const int reqVerMaj = 1;
	const int reqVerMin = 2;
#endif
	for (int i = 0;i < count;i++)
	{
		if (fDebugLevel >= 4) printf("Examining device %d\n", i);
#if CUDA_VERSION > 3010
		size_t free, total;
#else
		unsigned int free, total;
#endif
		hipInit(0);
		hipDevice_t tmpDevice;
		hipDeviceGet(&tmpDevice, i);
		hipCtx_t tmpContext;
		hipCtxCreate(&tmpContext, 0, tmpDevice);
		if(hipMemGetInfo(&free, &total)) std::cout << "Error\n";
		hipCtxDestroy(tmpContext);
		if (fDebugLevel >= 4) printf("Obtained current memory usage for device %d\n", i);
		if (GPUFailedMsg(hipGetDeviceProperties(&fCudaDeviceProp, i))) continue;
		if (fDebugLevel >= 4) printf("Obtained device properties for device %d\n", i);
		int deviceOK = fCudaDeviceProp.major < 9 && (fCudaDeviceProp.major > reqVerMaj || (fCudaDeviceProp.major == reqVerMaj && fCudaDeviceProp.minor >= reqVerMin)) && (size_t) free >= (size_t) (fGPUMemSize + 100 * 1024 + 1024);

		if (fDebugLevel >= 0) HLTInfo("%s%2d: %s (Rev: %d.%d - Mem Avail %lld / %lld)%s", deviceOK ? " " : "[", i, fCudaDeviceProp.name, fCudaDeviceProp.major, fCudaDeviceProp.minor, (long long int) free, (long long int) fCudaDeviceProp.totalGlobalMem, deviceOK ? "" : " ]");
		deviceSpeed = (double) fCudaDeviceProp.multiProcessorCount * (double) fCudaDeviceProp.clockRate * (double) fCudaDeviceProp.warpSize * (double) free * (double) fCudaDeviceProp.major * (double) fCudaDeviceProp.major;
		if (deviceOK)
		{
			if (deviceSpeed > bestDeviceSpeed)
			{
				bestDevice = i;
				bestDeviceSpeed = deviceSpeed;
			}
			else
			{
				if (fDebugLevel >= 0) HLTInfo("Skipping: Speed %f < %f\n", deviceSpeed, bestDeviceSpeed);
			}
		}
	}
	if (bestDevice == -1)
	{
		HLTWarning("No %sCUDA Device available, aborting CUDA Initialisation", count ? "appropriate " : "");
		HLTInfo("Requiring Revision %d.%d, Mem: %lld, Multiprocessors: %d", reqVerMaj, reqVerMin, fGPUMemSize + 100 * 1024 * 1024, sliceCount);
		return(1);
	}

	if (forceDeviceID == -1)
		fCudaDevice = bestDevice;
	else
		fCudaDevice = forceDeviceID;
#else
	fCudaDevice = 0;
#endif

	hipGetDeviceProperties(&fCudaDeviceProp ,fCudaDevice ); 

	if (fDebugLevel >= 1)
	{
		HLTInfo("Using CUDA Device %s with Properties:", fCudaDeviceProp.name);
		HLTInfo("totalGlobalMem = %lld", (unsigned long long int) fCudaDeviceProp.totalGlobalMem);
		HLTInfo("sharedMemPerBlock = %lld", (unsigned long long int) fCudaDeviceProp.sharedMemPerBlock);
		HLTInfo("regsPerBlock = %d", fCudaDeviceProp.regsPerBlock);
		HLTInfo("warpSize = %d", fCudaDeviceProp.warpSize);
		HLTInfo("memPitch = %lld", (unsigned long long int) fCudaDeviceProp.memPitch);
		HLTInfo("maxThreadsPerBlock = %d", fCudaDeviceProp.maxThreadsPerBlock);
		HLTInfo("maxThreadsDim = %d %d %d", fCudaDeviceProp.maxThreadsDim[0], fCudaDeviceProp.maxThreadsDim[1], fCudaDeviceProp.maxThreadsDim[2]);
		HLTInfo("maxGridSize = %d %d %d", fCudaDeviceProp.maxGridSize[0], fCudaDeviceProp.maxGridSize[1], fCudaDeviceProp.maxGridSize[2]);
		HLTInfo("totalConstMem = %lld", (unsigned long long int) fCudaDeviceProp.totalConstMem);
		HLTInfo("major = %d", fCudaDeviceProp.major);
		HLTInfo("minor = %d", fCudaDeviceProp.minor);
		HLTInfo("clockRate = %d", fCudaDeviceProp.clockRate);
		HLTInfo("memoryClockRate = %d", fCudaDeviceProp.memoryClockRate);
		HLTInfo("multiProcessorCount = %d", fCudaDeviceProp.multiProcessorCount);
		HLTInfo("textureAlignment = %lld", (unsigned long long int) fCudaDeviceProp.textureAlignment);
	}
#ifdef HLTCA_GPU_CONSTRUCTOR_SINGLE_SLICE
	fConstructorBlockCount = fCudaDeviceProp.multiProcessorCount;
#else
	fConstructorBlockCount = fCudaDeviceProp.multiProcessorCount * HLTCA_GPU_BLOCK_COUNT_CONSTRUCTOR_MULTIPLIER;
#endif
	fConstructorThreadCount = HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR;
	fSelectorBlockCount = fCudaDeviceProp.multiProcessorCount * HLTCA_GPU_BLOCK_COUNT_SELECTOR_MULTIPLIER;

	if (fCudaDeviceProp.major < 1 || (fCudaDeviceProp.major == 1 && fCudaDeviceProp.minor < 2))
	{
		HLTError( "Unsupported CUDA Device" );
		return(1);
	}

#ifdef HLTCA_GPU_USE_TEXTURES
	if (HLTCA_GPU_SLICE_DATA_MEMORY * sliceCount > (size_t) fCudaDeviceProp.maxTexture1DLinear)
	{
		HLTError("Invalid maximum texture size of device: %lld < %lld\n", (long long int) fCudaDeviceProp.maxTexture1DLinear, (long long int) (HLTCA_GPU_SLICE_DATA_MEMORY * sliceCount));
		return(1);
	}
#endif

	int nStreams = HLTCA_GPU_NUM_STREAMS == 0 ? CAMath::Max(3, fSliceCount) : HLTCA_GPU_NUM_STREAMS;
	if (nStreams < 3)
	{
		HLTError("Invalid number of streams");
		return(1);
	}

	if (hipCtxCreate((hipCtx_t*) fCudaContext, hipDeviceScheduleAuto, fCudaDevice) != hipSuccess)
	{
		HLTError("Could not set CUDA Device!");
		return(1);
	}

	if (fGPUMemSize > fCudaDeviceProp.totalGlobalMem || GPUFailedMsg(hipMalloc(&fGPUMemory, (size_t) fGPUMemSize)))
	{
		HLTError("CUDA Memory Allocation Error");
		hipDeviceReset();
		return(1);
	}
	if (fDebugLevel >= 1) HLTInfo("GPU Memory used: %lld", fGPUMemSize);
	if (GPUFailedMsg(hipHostMalloc(&fHostLockedMemory, fHostMemSize)))
	{
		hipFree(fGPUMemory);
		hipDeviceReset();
		HLTError("Error allocating Page Locked Host Memory");
		return(1);
	}
	if (fDebugLevel >= 1) HLTInfo("Host Memory used: %lld", fHostMemSize);

	if (fDebugLevel >= 1)
	{
		GPUFailedMsg(hipMemset(fGPUMemory, 143, (size_t) fGPUMemSize));
	}
	
	fpCudaStreams = malloc(nStreams * sizeof(hipStream_t));
	hipStream_t* const cudaStreams = (hipStream_t*) fpCudaStreams;
	for (int i = 0;i < nStreams;i++)
	{
		if (GPUFailedMsg(hipStreamCreate(&cudaStreams[i])))
		{
			hipFree(fGPUMemory);
			hipHostFree(fHostLockedMemory);
			hipDeviceReset();
			HLTError("Error creating CUDA Stream");
			return(1);
		}
	}

	hipCtxPopCurrent((hipCtx_t*) fCudaContext);
	HLTInfo("CUDA Initialisation successfull (Device %d: %s, Thread %d, Max slices: %d, %lld bytes used)", fCudaDevice, fCudaDeviceProp.name, fThreadId, fSliceCount, fGPUMemSize);

	return(0);
}

bool AliHLTTPCCAGPUTrackerNVCC::GPUFailedMsgA(hipError_t error, const char* file, int line)
{
	//Check for CUDA Error and in the case of an error display the corresponding error string
	if (error == hipSuccess) return(false);
	HLTWarning("CUDA Error: %d / %s (%s:%d)", error, hipGetErrorString(error), file, line);
	return(true);
}

int AliHLTTPCCAGPUTrackerNVCC::GPUSync(const char* state, int stream, int slice)
{
	//Wait for CUDA-Kernel to finish and check for CUDA errors afterwards

	if (fDebugLevel == 0) return(0);
	hipError_t cuErr;
	cuErr = hipGetLastError();
	if (cuErr != hipSuccess)
	{
		HLTError("Cuda Error %s while running kernel (%s) (Stream %d; %d/%d)", hipGetErrorString(cuErr), state, stream, slice, fgkNSlices);
		return(1);
	}
	if (GPUFailedMsg(hipDeviceSynchronize()))
	{
		HLTError("CUDA Error while synchronizing (%s) (Stream %d; %d/%d)", state, stream, slice, fgkNSlices);
		return(1);
	}
	if (fDebugLevel >= 3) HLTInfo("GPU Sync Done");
	return(0);
}

__global__ void PreInitRowBlocks(int* const SliceDataHitWeights, int nSliceDataHits)
{
	//Initialize GPU RowBlocks and HitWeights
	int4* const sliceDataHitWeights4 = (int4*) SliceDataHitWeights;
	const int stride = get_global_size(0);
	int4 i0;
	i0.x = i0.y = i0.z = i0.w = 0;
	for (int i = get_global_id(0);i < nSliceDataHits * sizeof(int) / sizeof(int4);i += stride)
		sliceDataHitWeights4[i] = i0;
}

int AliHLTTPCCAGPUTrackerNVCC::Reconstruct(AliHLTTPCCASliceOutput** pOutput, AliHLTTPCCAClusterData* pClusterData, int firstSlice, int sliceCountLocal)
{
	//Primary reconstruction function

	hipStream_t* const cudaStreams = (hipStream_t*) fpCudaStreams;

	if (Reconstruct_Base_Init(pOutput, pClusterData, firstSlice, sliceCountLocal)) return(1);

#ifdef HLTCA_GPU_USE_TEXTURES
	hipChannelFormatDesc channelDescu2 = hipCreateChannelDesc<cahit2>();
	size_t offset;
	if (GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu2, fGpuTracker[0].Data().Memory(), &channelDescu2, sliceCountLocal * HLTCA_GPU_SLICE_DATA_MEMORY)) || offset RANDOM_ERROR)
	{
		HLTError("Error binding CUDA Texture cahit2 (Offset %d)", (int) offset);
		ResetHelperThreads(0);
		return(1);
	}
	hipChannelFormatDesc channelDescu = hipCreateChannelDesc<calink>();
	if (GPUFailedMsg(hipBindTexture(&offset, &gAliTexRefu, fGpuTracker[0].Data().Memory(), &channelDescu, sliceCountLocal * HLTCA_GPU_SLICE_DATA_MEMORY)) || offset RANDOM_ERROR)
	{
		HLTError("Error binding CUDA Texture calink (Offset %d)", (int) offset);
		ResetHelperThreads(0);
		return(1);
	}
#endif

	//Copy Tracker Object to GPU Memory
	if (fDebugLevel >= 3) HLTInfo("Copying Tracker objects to GPU");
#ifdef HLTCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* tmpMem;
	if (GPUFailedMsg(hipMalloc(&tmpMem, 100000000)))
	{
		HLTError("Error allocating CUDA profile memory");
		ResetHelperThreads(0);
		return(1);
	}
	fGpuTracker[0].fStageAtSync = tmpMem;
	GPUFailedMsg(hipMemset(fGpuTracker[0].StageAtSync(), 0, 100000000));
#endif
	GPUFailedMsg(hipMemcpyToSymbolAsync(HIP_SYMBOL(gAliHLTTPCCATracker), fGpuTracker, sizeof(AliHLTTPCCATracker) * sliceCountLocal, 0, hipMemcpyHostToDevice, cudaStreams[0]));
	bool globalSymbolDone = false;
	if (GPUSync("Initialization (1)", 0, firstSlice) RANDOM_ERROR)
	{
		ResetHelperThreads(0);
		return(1);
	}

	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
	{
		if (Reconstruct_Base_SliceInit(pClusterData, iSlice, firstSlice)) return(1);

		int useStream = HLTCA_GPU_NUM_STREAMS == 0 ? (iSlice & 1) : (iSlice % HLTCA_GPU_NUM_STREAMS);
		//Initialize temporary memory where needed
		if (fDebugLevel >= 3) HLTInfo("Copying Slice Data to GPU and initializing temporary memory");		
		PreInitRowBlocks<<<fConstructorBlockCount, HLTCA_GPU_THREAD_COUNT, 0, cudaStreams[HLTCA_GPU_NUM_STREAMS == 0 ? 2 : useStream]>>>(fGpuTracker[iSlice].Data().HitWeights(), fSlaveTrackers[firstSlice + iSlice].Data().NumberOfHitsPlusAlign());
		if (GPUSync("Initialization (2)", 2, iSlice + firstSlice) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			return(1);
		}

		//Copy Data to GPU Global Memory
		fSlaveTrackers[firstSlice + iSlice].StartTimer(0);
		GPUFailedMsg(hipMemcpyAsync(fGpuTracker[iSlice].CommonMemory(), fSlaveTrackers[firstSlice + iSlice].CommonMemory(), fSlaveTrackers[firstSlice + iSlice].CommonMemorySize(), hipMemcpyHostToDevice, cudaStreams[useStream]));
		GPUFailedMsg(hipMemcpyAsync(fGpuTracker[iSlice].Data().Memory(), fSlaveTrackers[firstSlice + iSlice].Data().Memory(), fSlaveTrackers[firstSlice + iSlice].Data().GpuMemorySize(), hipMemcpyHostToDevice, cudaStreams[useStream]));
		GPUFailedMsg(hipMemcpyAsync(fGpuTracker[iSlice].SliceDataRows(), fSlaveTrackers[firstSlice + iSlice].SliceDataRows(), (HLTCA_ROW_COUNT + 1) * sizeof(AliHLTTPCCARow), hipMemcpyHostToDevice, cudaStreams[useStream]));

		if (fDebugLevel >= 4)
		{
			if (fDebugLevel >= 5) HLTInfo("Allocating Debug Output Memory");
			fSlaveTrackers[firstSlice + iSlice].SetGPUTrackerTrackletsMemory(reinterpret_cast<char*> ( new uint4 [ fGpuTracker[iSlice].TrackletMemorySize()/sizeof( uint4 ) + 100] ), HLTCA_GPU_MAX_TRACKLETS);
			fSlaveTrackers[firstSlice + iSlice].SetGPUTrackerHitsMemory(reinterpret_cast<char*> ( new uint4 [ fGpuTracker[iSlice].HitMemorySize()/sizeof( uint4 ) + 100]), pClusterData[iSlice].NumberOfClusters() );
		}
		
		if (HLTCA_GPU_NUM_STREAMS && useStream && globalSymbolDone == false)
		{
			hipStreamSynchronize(cudaStreams[0]);
			globalSymbolDone = true;
		}

		if (GPUSync("Initialization (3)", useStream, iSlice + firstSlice) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			return(1);
		}
		fSlaveTrackers[firstSlice + iSlice].StopTimer(0);

		if (fDebugLevel >= 3) HLTInfo("Running GPU Neighbours Finder (Slice %d/%d)", iSlice, sliceCountLocal);
		fSlaveTrackers[firstSlice + iSlice].StartTimer(1);
		AliHLTTPCCAProcess<AliHLTTPCCANeighboursFinder> <<<fSlaveTrackers[firstSlice + iSlice].Param().NRows(), HLTCA_GPU_THREAD_COUNT_FINDER, 0, cudaStreams[useStream]>>>(iSlice);

		if (GPUSync("Neighbours finder", useStream, iSlice + firstSlice) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			return(1);
		}
		fSlaveTrackers[firstSlice + iSlice].StopTimer(1);

		if (fDebugLevel >= 4)
		{
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].Data().Memory(), fGpuTracker[iSlice].Data().Memory(), fSlaveTrackers[firstSlice + iSlice].Data().GpuMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugMask & 2) fSlaveTrackers[firstSlice + iSlice].DumpLinks(*fOutFile);
		}

		if (fDebugLevel >= 3) HLTInfo("Running GPU Neighbours Cleaner (Slice %d/%d)", iSlice, sliceCountLocal);
		fSlaveTrackers[firstSlice + iSlice].StartTimer(2);
		AliHLTTPCCAProcess<AliHLTTPCCANeighboursCleaner> <<<fSlaveTrackers[firstSlice + iSlice].Param().NRows()-2, HLTCA_GPU_THREAD_COUNT, 0, cudaStreams[useStream]>>>(iSlice);
		if (GPUSync("Neighbours Cleaner", useStream, iSlice + firstSlice) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			return(1);
		}
		fSlaveTrackers[firstSlice + iSlice].StopTimer(2);

		if (fDebugLevel >= 4)
		{
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].Data().Memory(), fGpuTracker[iSlice].Data().Memory(), fSlaveTrackers[firstSlice + iSlice].Data().GpuMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugMask & 4) fSlaveTrackers[firstSlice + iSlice].DumpLinks(*fOutFile);
		}

		if (fDebugLevel >= 3) HLTInfo("Running GPU Start Hits Finder (Slice %d/%d)", iSlice, sliceCountLocal);
		fSlaveTrackers[firstSlice + iSlice].StartTimer(3);
		AliHLTTPCCAProcess<AliHLTTPCCAStartHitsFinder> <<<fSlaveTrackers[firstSlice + iSlice].Param().NRows()-6, HLTCA_GPU_THREAD_COUNT, 0, cudaStreams[useStream]>>>(iSlice);
		if (GPUSync("Start Hits Finder", useStream, iSlice + firstSlice) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			return(1);
		}
		fSlaveTrackers[firstSlice + iSlice].StopTimer(3);

		if (fDebugLevel >= 3) HLTInfo("Running GPU Start Hits Sorter (Slice %d/%d)", iSlice, sliceCountLocal);
		fSlaveTrackers[firstSlice + iSlice].StartTimer(4);
		AliHLTTPCCAProcess<AliHLTTPCCAStartHitsSorter> <<<fConstructorBlockCount, HLTCA_GPU_THREAD_COUNT, 0, cudaStreams[useStream]>>>(iSlice);
		if (GPUSync("Start Hits Sorter", useStream, iSlice + firstSlice) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			return(1);
		}
		fSlaveTrackers[firstSlice + iSlice].StopTimer(4);

		if (fDebugLevel >= 2)
		{
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugLevel >= 3) HLTInfo("Obtaining Number of Start Hits from GPU: %d (Slice %d)", *fSlaveTrackers[firstSlice + iSlice].NTracklets(), iSlice);
			if (*fSlaveTrackers[firstSlice + iSlice].NTracklets() > HLTCA_GPU_MAX_TRACKLETS RANDOM_ERROR)
			{
				HLTError("HLTCA_GPU_MAX_TRACKLETS constant insuffisant");
				ResetHelperThreads(1);
				return(1);
			}
		}

		if (fDebugLevel >= 4 && *fSlaveTrackers[firstSlice + iSlice].NTracklets())
		{
#ifndef BITWISE_COMPATIBLE_DEBUG_OUTPUT
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].TrackletStartHits(), fGpuTracker[iSlice].TrackletTmpStartHits(), pClusterData[iSlice].NumberOfClusters() * sizeof(AliHLTTPCCAHitId), hipMemcpyDeviceToHost));
			if (fDebugMask & 8)
			{
				*fOutFile << "Temporary ";
				fSlaveTrackers[firstSlice + iSlice].DumpStartHits(*fOutFile);
			}
			uint3* tmpMemory = (uint3*) malloc(sizeof(uint3) * fSlaveTrackers[firstSlice + iSlice].Param().NRows());
			GPUFailedMsg(hipMemcpy(tmpMemory, fGpuTracker[iSlice].RowStartHitCountOffset(), fSlaveTrackers[firstSlice + iSlice].Param().NRows() * sizeof(uint3), hipMemcpyDeviceToHost));
			if (fDebugMask & 16)
			{
				*fOutFile << "Start Hits Sort Vector:" << std::endl;
				for (int i = 1;i < fSlaveTrackers[firstSlice + iSlice].Param().NRows() - 5;i++)
				{
					*fOutFile << "Row: " << i << ", Len: " << tmpMemory[i].x << ", Offset: " << tmpMemory[i].y << ", New Offset: " << tmpMemory[i].z << std::endl;
				}
			}
			free(tmpMemory);
#endif

			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].HitMemory(), fGpuTracker[iSlice].HitMemory(), fSlaveTrackers[firstSlice + iSlice].HitMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugMask & 32) fSlaveTrackers[firstSlice + iSlice].DumpStartHits(*fOutFile);
		}

		fSlaveTrackers[firstSlice + iSlice].SetGPUTrackerTracksMemory((char*) TracksMemory(fHostLockedMemory, iSlice), HLTCA_GPU_MAX_TRACKS, pClusterData[iSlice].NumberOfClusters());

#ifdef HLTCA_GPU_CONSTRUCTOR_SINGLE_SLICE
		fSlaveTrackers[firstSlice + iSlice].StartTimer(6);
		AliHLTTPCCATrackletConstructorSingleSlice<<<fConstructorBlockCount, HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR, 0, cudaStreams[useStream]>>>(firstSlice + iSlice);
		if (GPUSync("Tracklet Constructor", useStream, iSlice + firstSlice) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			return(1);
		}
		fSlaveTrackers[firstSlice + iSlice].StopTimer(6);
#endif
	}

	for (int i = 0;i < fNHelperThreads;i++)
	{
		pthread_mutex_lock(&((pthread_mutex_t*) fHelperParams[i].fMutex)[1]);
	}

#ifdef HLTCA_GPU_CONSTRUCTOR_SINGLE_SLICE
	hipDeviceSynchronize();
#else
	if (fDebugLevel >= 3) HLTInfo("Running GPU Tracklet Constructor");
	fSlaveTrackers[firstSlice].StartTimer(6);
	AliHLTTPCCATrackletConstructorGPU<<<fConstructorBlockCount, HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR>>>();
	if (GPUSync("Tracklet Constructor", -1, firstSlice) RANDOM_ERROR)
	{
		hipDeviceSynchronize();
		hipCtxPopCurrent((hipCtx_t*) fCudaContext);
		return(1);
	}
	fSlaveTrackers[firstSlice].StopTimer(6);
#endif //HLTCA_GPU_CONSTRUCTOR_SINGLE_SLICE

	if (fDebugLevel >= 4)
	{
		for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
		{
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugLevel >= 5)
			{
				HLTInfo("Obtained %d tracklets", *fSlaveTrackers[firstSlice + iSlice].NTracklets());
			}
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].TrackletMemory(), fGpuTracker[iSlice].TrackletMemory(), fGpuTracker[iSlice].TrackletMemorySize(), hipMemcpyDeviceToHost));
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].HitMemory(), fGpuTracker[iSlice].HitMemory(), fGpuTracker[iSlice].HitMemorySize(), hipMemcpyDeviceToHost));
			if (fDebugMask & 128) fSlaveTrackers[firstSlice + iSlice].DumpTrackletHits(*fOutFile);
			delete[] fSlaveTrackers[firstSlice + iSlice].TrackletMemory();
		}
	}

	int runSlices = 0;
	int useStream = 0;
	int streamMap[36];
	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice += runSlices)
	{
		if (runSlices < HLTCA_GPU_TRACKLET_SELECTOR_SLICE_COUNT) runSlices++;
		runSlices = CAMath::Min(runSlices, sliceCountLocal - iSlice);
		if (fSelectorBlockCount < runSlices) runSlices = fSelectorBlockCount;
		if (HLTCA_GPU_NUM_STREAMS && useStream + 1 == HLTCA_GPU_NUM_STREAMS) runSlices = sliceCountLocal - iSlice;
		if (fSelectorBlockCount < runSlices)
		{
			HLTError("Insufficient number of blocks for tracklet selector");
			hipCtxPopCurrent((hipCtx_t*) fCudaContext);
			return(1);
		}
		
		if (fDebugLevel >= 3) HLTInfo("Running HLT Tracklet selector (Stream %d, Slice %d to %d)", useStream, iSlice, iSlice + runSlices);
		fSlaveTrackers[firstSlice + iSlice].StartTimer(7);
		AliHLTTPCCAProcessMulti<AliHLTTPCCATrackletSelector><<<fSelectorBlockCount, HLTCA_GPU_THREAD_COUNT_SELECTOR, 0, cudaStreams[useStream]>>>(iSlice, runSlices);
		if (GPUSync("Tracklet Selector", iSlice, iSlice + firstSlice) RANDOM_ERROR)
		{
			hipDeviceSynchronize();
			hipCtxPopCurrent((hipCtx_t*) fCudaContext);
			return(1);
		}
		fSlaveTrackers[firstSlice + iSlice].StopTimer(7);
		for (int k = iSlice;k < iSlice + runSlices;k++) streamMap[k] = useStream;
		useStream++;
	}

	char *tmpMemoryGlobalTracking = NULL;
	fSliceOutputReady = 0;
	
	if (Reconstruct_Base_StartGlobal(pOutput, tmpMemoryGlobalTracking)) return(1);
	
	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
	{
		if (GPUFailedMsg(hipMemcpyAsync(fSlaveTrackers[firstSlice + iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemory(), fGpuTracker[iSlice].CommonMemorySize(), hipMemcpyDeviceToHost, cudaStreams[streamMap[iSlice]])) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			ActivateThreadContext();
			return(SelfHealReconstruct(pOutput, pClusterData, firstSlice, sliceCountLocal));
		}
	}

	int tmpSlice = 0;
	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
	{
		if (fDebugLevel >= 3) HLTInfo("Transfering Tracks from GPU to Host");

		while (tmpSlice < sliceCountLocal && (tmpSlice == iSlice ? hipStreamSynchronize(cudaStreams[streamMap[tmpSlice]]) : hipStreamQuery(cudaStreams[streamMap[tmpSlice]])) == (hipError_t) hipSuccess)
		{
			if (*fSlaveTrackers[firstSlice + tmpSlice].NTracks() > 0)
			{
				int useStream = HLTCA_GPU_NUM_STREAMS ? streamMap[tmpSlice] : tmpSlice;
				GPUFailedMsg(hipMemcpyAsync(fSlaveTrackers[firstSlice + tmpSlice].Tracks(), fGpuTracker[tmpSlice].Tracks(), sizeof(AliHLTTPCCATrack) * *fSlaveTrackers[firstSlice + tmpSlice].NTracks(), hipMemcpyDeviceToHost, cudaStreams[useStream]));
				GPUFailedMsg(hipMemcpyAsync(fSlaveTrackers[firstSlice + tmpSlice].TrackHits(), fGpuTracker[tmpSlice].TrackHits(), sizeof(AliHLTTPCCAHitId) * *fSlaveTrackers[firstSlice + tmpSlice].NTrackHits(), hipMemcpyDeviceToHost, cudaStreams[useStream]));
			}
			tmpSlice++;
		}

		int useStream = HLTCA_GPU_NUM_STREAMS ? streamMap[iSlice] : iSlice;
		if (GPUFailedMsg(hipStreamSynchronize(cudaStreams[useStream])) RANDOM_ERROR)
		{
			ResetHelperThreads(1);
			ActivateThreadContext();
			return(SelfHealReconstruct(pOutput, pClusterData, firstSlice, sliceCountLocal));
		}

		if (fDebugLevel >= 4)
		{
			GPUFailedMsg(hipMemcpy(fSlaveTrackers[firstSlice + iSlice].Data().HitWeights(), fGpuTracker[iSlice].Data().HitWeights(), fSlaveTrackers[firstSlice + iSlice].Data().NumberOfHitsPlusAlign() * sizeof(int), hipMemcpyDeviceToHost));
#ifndef BITWISE_COMPATIBLE_DEBUG_OUTPUT
			if (fDebugMask & 256) fSlaveTrackers[firstSlice + iSlice].DumpHitWeights(*fOutFile);
#endif
			if (fDebugMask & 512) fSlaveTrackers[firstSlice + iSlice].DumpTrackHits(*fOutFile);
		}

		if (fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fGPUError RANDOM_ERROR)
		{
			const char* errorMsgs[] = HLTCA_GPU_ERROR_STRINGS;
			const char* errorMsg = (unsigned) fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fGPUError >= sizeof(errorMsgs) / sizeof(errorMsgs[0]) ? "UNKNOWN" : errorMsgs[fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fGPUError];
			HLTError("GPU Tracker returned Error Code %d (%s) in slice %d (Clusters %d)", fSlaveTrackers[firstSlice + iSlice].GPUParameters()->fGPUError, errorMsg, firstSlice + iSlice, fSlaveTrackers[firstSlice + iSlice].Data().NumberOfHits());

			ResetHelperThreads(1);
			return(1);
		}
		if (fDebugLevel >= 3) HLTInfo("Tracks Transfered: %d / %d", *fSlaveTrackers[firstSlice + iSlice].NTracks(), *fSlaveTrackers[firstSlice + iSlice].NTrackHits());

		if (Reconstruct_Base_FinishSlices(pOutput, iSlice, firstSlice)) return(1);
		if (fDebugLevel >= 4)
		{
			delete[] fSlaveTrackers[firstSlice + iSlice].HitMemory();
		}
	}

	if (Reconstruct_Base_Finalize(pOutput, tmpMemoryGlobalTracking, firstSlice)) return(1);

	/*for (int i = firstSlice;i < firstSlice + sliceCountLocal;i++)
	{
		fSlaveTrackers[i].DumpOutput(stdout);
	}*/

	/*static int runnum = 0;
	std::ofstream tmpOut;
	char buffer[1024];
	sprintf(buffer, "GPUtracks%d.out", runnum++);
	tmpOut.open(buffer);
	for (int iSlice = 0;iSlice < sliceCountLocal;iSlice++)
	{
		fSlaveTrackers[firstSlice + iSlice].DumpTrackHits(tmpOut);
	}
	tmpOut.close();*/

#ifdef HLTCA_GPU_TRACKLET_CONSTRUCTOR_DO_PROFILE
	char* stageAtSync = (char*) malloc(100000000);
	GPUFailedMsg(hipMemcpy(stageAtSync, fGpuTracker[0].StageAtSync(), 100 * 1000 * 1000, hipMemcpyDeviceToHost));
	hipFree(fGpuTracker[0].StageAtSync());

	FILE* fp = fopen("profile.txt", "w+");
	FILE* fp2 = fopen("profile.bmp", "w+b");
	int nEmptySync = 0, fEmpty;

	const int bmpheight = 8192;
	BITMAPFILEHEADER bmpFH;
	BITMAPINFOHEADER bmpIH;
	ZeroMemory(&bmpFH, sizeof(bmpFH));
	ZeroMemory(&bmpIH, sizeof(bmpIH));

	bmpFH.bfType = 19778; //"BM"
	bmpFH.bfSize = sizeof(bmpFH) + sizeof(bmpIH) + (fConstructorBlockCount * HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1) * bmpheight ;
	bmpFH.bfOffBits = sizeof(bmpFH) + sizeof(bmpIH);

	bmpIH.biSize = sizeof(bmpIH);
	bmpIH.biWidth = fConstructorBlockCount * HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR / 32 * 33 - 1;
	bmpIH.biHeight = bmpheight;
	bmpIH.biPlanes = 1;
	bmpIH.biBitCount = 32;

	fwrite(&bmpFH, 1, sizeof(bmpFH), fp2);
	fwrite(&bmpIH, 1, sizeof(bmpIH), fp2); 	

	for (int i = 0;i < bmpheight * fConstructorBlockCount * HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR;i += fConstructorBlockCount * HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR)
	{
		fEmpty = 1;
		for (int j = 0;j < fConstructorBlockCount * HLTCA_GPU_THREAD_COUNT_CONSTRUCTOR;j++)
		{
			fprintf(fp, "%d\t", stageAtSync[i + j]);
			int color = 0;
			if (stageAtSync[i + j] == 1) color = RGB(255, 0, 0);
			if (stageAtSync[i + j] == 2) color = RGB(0, 255, 0);
			if (stageAtSync[i + j] == 3) color = RGB(0, 0, 255);
			if (stageAtSync[i + j] == 4) color = RGB(255, 255, 0);
			fwrite(&color, 1, sizeof(int), fp2);
			if (j > 0 && j % 32 == 0)
			{
				color = RGB(255, 255, 255);
				fwrite(&color, 1, 4, fp2);
			}
			if (stageAtSync[i + j]) fEmpty = 0;
		}
		fprintf(fp, "\n");
		if (fEmpty) nEmptySync++;
		else nEmptySync = 0;
		//if (nEmptySync == HLTCA_GPU_SCHED_ROW_STEP + 2) break;
	}

	fclose(fp);
	fclose(fp2);
	free(stageAtSync);
#endif 

	hipCtxPopCurrent((hipCtx_t*) fCudaContext);
	return(0);
}

__global__ void ClearPPHitWeights(int sliceCount)
{
	//Clear HitWeights

	for (int k = 0;k < sliceCount;k++)
	{
		AliHLTTPCCATracker &tracker = ((AliHLTTPCCATracker*) gAliHLTTPCCATracker)[k];
		int4* const pHitWeights = (int4*) tracker.Data().HitWeights();
		const int dwCount = tracker.Data().NumberOfHitsPlusAlign();
		const int stride = get_global_size(0);
		int4 i0;
		i0.x = i0.y = i0.z = i0.w = 0;

		for (int i = get_global_id(0);i < dwCount * sizeof(int) / sizeof(int4);i += stride)
		{
			pHitWeights[i] = i0;
		}
	}
}

int AliHLTTPCCAGPUTrackerNVCC::ExitGPU_Runtime()
{
	//Uninitialize CUDA
	hipCtxPushCurrent(*((hipCtx_t*) fCudaContext));

	hipDeviceSynchronize();
	if (fGPUMemory)
	{
		hipFree(fGPUMemory);
		fGPUMemory = NULL;
	}
	if (fHostLockedMemory)
	{
		int nStreams = HLTCA_GPU_NUM_STREAMS == 0 ? CAMath::Max(3, fSliceCount) : HLTCA_GPU_NUM_STREAMS;
		for (int i = 0;i < nStreams;i++)
		{
			hipStreamDestroy(((hipStream_t*) fpCudaStreams)[i]);
		}
		free(fpCudaStreams);
		fGpuTracker = NULL;
		hipHostFree(fHostLockedMemory);
	}

	if (GPUFailedMsg(hipDeviceReset()))
	{
		HLTError("Could not uninitialize GPU");
		return(1);
	}

	hipCtxDestroy(*((hipCtx_t*) fCudaContext));

	hipDeviceReset();

	HLTInfo("CUDA Uninitialized");
	fCudaInitialized = 0;
	return(0);
}

int AliHLTTPCCAGPUTrackerNVCC::RefitMergedTracks(AliHLTTPCGMMerger* Merger, bool resetTimers)
{
#ifndef HLTCA_GPU_MERGER
	HLTError("HLTCA_GPU_MERGER compile flag not set");
	return(1);
#else
	if (!fCudaInitialized)
	{
		HLTError("GPU Merger not initialized");
		return(1);
	}

	HighResTimer timer;
	static double times[3] = {};
	static int nCount = 0;
	if (resetTimers)
	{
		for (unsigned int k = 0;k < sizeof(times) / sizeof(times[0]);k++) times[k] = 0;
		nCount = 0;
	}
	char* gpumem = (char*) fGPUMergerMemory;
	AliHLTTPCGMMergedTrackHit *clusters;
	AliHLTTPCGMMergedTrack* tracks;
	AliHLTTPCGMPolynomialField* field;
	AliHLTTPCCAParam* param;

	AssignMemory(clusters, gpumem, Merger->NClusters());
	AssignMemory(tracks, gpumem, Merger->NOutputTracks());
	AssignMemory(field, gpumem, 1);
	AssignMemory(param, gpumem, 1);

	if ((size_t) (gpumem - (char*) fGPUMergerMemory) > (size_t) fGPUMergerMaxMemory)
	{
		HLTError("Insufficiant GPU Merger Memory");
	}

	hipCtxPushCurrent(*((hipCtx_t*) fCudaContext));

	if (fDebugLevel >= 2) HLTInfo("Running GPU Merger (%d/%d)", Merger->NOutputTrackClusters(), Merger->NClusters());
	timer.Start();
	GPUFailedMsg(hipMemcpy(clusters, Merger->Clusters(), Merger->NOutputTrackClusters() * sizeof(clusters[0]), hipMemcpyHostToDevice));
	GPUFailedMsg(hipMemcpy(tracks, Merger->OutputTracks(), Merger->NOutputTracks() * sizeof(AliHLTTPCGMMergedTrack), hipMemcpyHostToDevice));
	GPUFailedMsg(hipMemcpy(field, Merger->pField(), sizeof(AliHLTTPCGMPolynomialField), hipMemcpyHostToDevice));
	GPUFailedMsg(hipMemcpy(param, &Merger->SliceParam(), sizeof(AliHLTTPCCAParam), hipMemcpyHostToDevice));
	times[0] += timer.GetCurrentElapsedTime(true);
	RefitTracks<<<fConstructorBlockCount, HLTCA_GPU_THREAD_COUNT>>>(tracks, Merger->NOutputTracks(), field, clusters, param);
	GPUFailedMsg(hipDeviceSynchronize());
	times[1] += timer.GetCurrentElapsedTime(true);
	GPUFailedMsg(hipMemcpy(Merger->Clusters(), clusters, Merger->NOutputTrackClusters() * sizeof(clusters[0]), hipMemcpyDeviceToHost));
	GPUFailedMsg(hipMemcpy((void*) Merger->OutputTracks(), tracks, Merger->NOutputTracks() * sizeof(AliHLTTPCGMMergedTrack), hipMemcpyDeviceToHost));
	GPUFailedMsg(hipDeviceSynchronize());
	times[2] += timer.GetCurrentElapsedTime();
	if (fDebugLevel >= 2) HLTInfo("GPU Merger Finished");
	nCount++;

	if (fDebugLevel > 0)
	{
		int copysize = 4 * Merger->NOutputTrackClusters() * sizeof(float) + Merger->NOutputTrackClusters() * sizeof(unsigned int) + Merger->NOutputTracks() * sizeof(AliHLTTPCGMMergedTrack) + 6 * sizeof(float) + sizeof(AliHLTTPCCAParam);
		double speed = (double) copysize / times[0] * nCount / 1e9;
		printf("GPU Fit:\tCopy To:\t%1.0f us (%lf GB/s)\n", times[0] * 1000000 / nCount, speed);
		printf("\t\tFit:\t%1.0f us\n", times[1] * 1000000 / nCount);
		speed = (double) copysize / times[2] * nCount / 1e9;
		printf("\t\tCopy From:\t%1.0f us (%lf GB/s)\n", times[2] * 1000000 / nCount, speed);
	}
	
	if (!HLTCA_TIMING_SUM)
	{
		for (int i = 0;i < 3;i++) times[i] = 0;
		nCount = 0;
	}

	hipCtxPopCurrent((hipCtx_t*) fCudaContext);
	return(0);
#endif
}

int AliHLTTPCCAGPUTrackerNVCC::GPUMergerAvailable()
{
	return(1);
}

void AliHLTTPCCAGPUTrackerNVCC::ActivateThreadContext()
{
	hipCtxPushCurrent(*((hipCtx_t*) fCudaContext));
}
void AliHLTTPCCAGPUTrackerNVCC::ReleaseThreadContext()
{
	hipCtxPopCurrent((hipCtx_t*) fCudaContext);
}

void AliHLTTPCCAGPUTrackerNVCC::SynchronizeGPU()
{
	hipDeviceSynchronize();
}

AliHLTTPCCAGPUTracker* AliHLTTPCCAGPUTrackerNVCCCreate()
{
	return new AliHLTTPCCAGPUTrackerNVCC;
}

void AliHLTTPCCAGPUTrackerNVCCDestroy(AliHLTTPCCAGPUTracker* ptr)
{
	delete ptr;
}
