#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///

#include <array>
#include <sstream>
#include <iostream>

// #ifndef GPUCA_GPUCODE_GENRTC
// #include <hip/hip_cooperative_groups.h>
// #include "hipcub/hipcub.hpp"
// #endif

#include "ITStracking/Constants.h"
#include "ITStracking/Configuration.h"
#include "ITStracking/IndexTableUtils.h"
#include "ITStracking/MathUtils.h"
#include "ITStracking/TimeFrame.h"

#include "ITStrackingGPU/Context.h"
#include "ITStrackingGPU/Stream.h"
#include "ITStrackingGPU/Vector.h"
#include "ITStrackingGPU/TrackerTraitsGPU.h"

namespace o2
{
namespace its
{
using gpu::utils::host::checkGPUError;
using namespace constants::its2;

GPUd() const int4 getBinsRect(const Cluster& currentCluster, const int layerIndex,
                              const float z1, const float z2, float maxdeltaz, float maxdeltaphi)
{
  const float zRangeMin = o2::gpu::GPUCommonMath::Min(z1, z2) - maxdeltaz;
  const float phiRangeMin = currentCluster.phi - maxdeltaphi;
  const float zRangeMax = o2::gpu::GPUCommonMath::Max(z1, z2) + maxdeltaz;
  const float phiRangeMax = currentCluster.phi + maxdeltaphi;

  if (zRangeMax < -LayersZCoordinate()[layerIndex + 1] ||
      zRangeMin > LayersZCoordinate()[layerIndex + 1] || zRangeMin > zRangeMax) {

    return getEmptyBinsRect();
  }

  return int4{o2::gpu::GPUCommonMath::Max(0, getZBinIndex(layerIndex + 1, zRangeMin)),
              getPhiBinIndex(phiRangeMin),
              o2::gpu::GPUCommonMath::Min(ZBins - 1, getZBinIndex(layerIndex + 1, zRangeMax)),
              getPhiBinIndex(phiRangeMax)};
}

// template <int NLayers>
// void TrackerTraitsGPU<NLayers>::loadToDevice()
// {
//   mTimeFrameGPU.loadToDevice();
// }

namespace gpu
{

template <int NLayers>
struct StaticTrackingParameters {
  // StaticTrackingParameters<NLayers>& operator=(const StaticTrackingParameters<NLayers>& t);
  // int CellMinimumLevel();
  /// General parameters
  int ClusterSharing = 0;
  int MinTrackLength = NLayers;
  /// Trackleting cuts
  float TrackletMaxDeltaPhi = 0.3f;
  float TrackletMaxDeltaZ[NLayers - 1] = {0.1f, 0.1f, 0.3f, 0.3f, 0.3f, 0.3f};
  /// Cell finding cuts
  // float CellMaxDeltaTanLambda = 0.025f;
  // float CellMaxDCA[NLayers - 2] = {0.05f, 0.04f, 0.05f, 0.2f, 0.4f};
  // float CellMaxDeltaPhi = 0.14f;
  // float CellMaxDeltaZ[NLayers - 2] = {0.2f, 0.4f, 0.5f, 0.6f, 3.0f};
  // /// Neighbour finding cuts
  // float NeighbourMaxDeltaCurvature[NLayers - 3] = {0.008f, 0.0025f, 0.003f, 0.0035f};
  // float NeighbourMaxDeltaN[NLayers - 3] = {0.002f, 0.0090f, 0.002f, 0.005f};
};

template struct gpu::StaticTrackingParameters<7>;
__constant__ StaticTrackingParameters<7> kTrkPar;

// GPUd() void computeLayerTracklets(DeviceStoreNV& devStore, const int layerIndex,
//                                   Vector<Tracklet>& trackletsVector)
// {
//   const int currentClusterIndex = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);
//   int clusterTrackletsNum = 0;

//   if (currentClusterIndex < devStore.getClusters()[layerIndex].size()) {

//     Vector<Cluster> nextLayerClusters{devStore.getClusters()[layerIndex + 1].getWeakCopy()};
//     const Cluster currentCluster{devStore.getClusters()[layerIndex][currentClusterIndex]};

//     /*if (mUsedClustersTable[currentCluster.clusterId] != constants::its::UnusedIndex) {

//      continue;
//      }*/

//     const float tanLambda{(currentCluster.zCoordinate - devStore.getPrimaryVertex().z) / currentCluster.radius};
//     const float zAtRmin{tanLambda * (devStore.getRmin(layerIndex + 1) - currentCluster.radius) + currentCluster.zCoordinate};
//     const float zAtRmax{tanLambda * (devStore.getRmax(layerIndex + 1) - currentCluster.radius) + currentCluster.zCoordinate};

//     const int4 selectedBinsRect{getBinsRect(currentCluster, layerIndex, zAtRmin, zAtRmax,
//                                             kTrkPar.TrackletMaxDeltaZ[layerIndex], kTrkPar.TrackletMaxDeltaPhi)};

//     if (selectedBinsRect.x != 0 || selectedBinsRect.y != 0 || selectedBinsRect.z != 0 || selectedBinsRect.w != 0) {

//       const int nextLayerClustersNum{static_cast<int>(nextLayerClusters.size())};
//       int phiBinsNum{selectedBinsRect.w - selectedBinsRect.y + 1};

//       if (phiBinsNum < 0) {

//         phiBinsNum += constants::its2::PhiBins;
//       }

//       for (int iPhiBin{selectedBinsRect.y}, iPhiCount{0}; iPhiCount < phiBinsNum;
//            iPhiBin = ++iPhiBin == constants::its2::PhiBins ? 0 : iPhiBin, iPhiCount++) {

//         const int firstBinIndex{constants::its2::getBinIndex(selectedBinsRect.x, iPhiBin)};
//         const int firstRowClusterIndex = devStore.getIndexTables()[layerIndex][firstBinIndex];
//         const int maxRowClusterIndex = devStore.getIndexTables()[layerIndex][{firstBinIndex + selectedBinsRect.z - selectedBinsRect.x + 1}];

//         for (int iNextLayerCluster{firstRowClusterIndex};
//              iNextLayerCluster <= maxRowClusterIndex && iNextLayerCluster < nextLayerClustersNum; ++iNextLayerCluster) {

//           const Cluster& nextCluster{nextLayerClusters[iNextLayerCluster]};

//           const float deltaZ{o2::gpu::GPUCommonMath::Abs(
//             tanLambda * (nextCluster.radius - currentCluster.radius) + currentCluster.zCoordinate - nextCluster.zCoordinate)};
//           const float deltaPhi{o2::gpu::GPUCommonMath::Abs(currentCluster.phi - nextCluster.phi)};

//           if (deltaZ < kTrkPar.TrackletMaxDeltaZ[layerIndex] && (deltaPhi < kTrkPar.TrackletMaxDeltaPhi || o2::gpu::GPUCommonMath::Abs(deltaPhi - constants::math::TwoPi) < kTrkPar.TrackletMaxDeltaPhi)) {

//             cooperative_groups::coalesced_group threadGroup = cooperative_groups::coalesced_threads();
//             int currentIndex{};

//             if (threadGroup.thread_rank() == 0) {

//               currentIndex = trackletsVector.extend(threadGroup.size());
//             }

//             currentIndex = threadGroup.shfl(currentIndex, 0) + threadGroup.thread_rank();

//             trackletsVector.emplace(currentIndex, currentClusterIndex, iNextLayerCluster, currentCluster, nextCluster);
//             ++clusterTrackletsNum;
//           }
//         }
//       }

//       if (layerIndex > 0) {

//         devStore.getTrackletsPerClusterTable()[layerIndex - 1][currentClusterIndex] = clusterTrackletsNum;
//       }
//     }
//   }
// }

// GPUd() void computeLayerCells(DeviceStoreNV& devStore, const int layerIndex,
//                               Vector<Cell>& cellsVector)
// {
//   const int currentTrackletIndex = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);
//   const float3& primaryVertex = devStore.getPrimaryVertex();
//   int trackletCellsNum = 0;

//   if (currentTrackletIndex < devStore.getTracklets()[layerIndex].size()) {

//     const Tracklet& currentTracklet{devStore.getTracklets()[layerIndex][currentTrackletIndex]};
//     const int nextLayerClusterIndex{currentTracklet.secondClusterIndex};
//     const int nextLayerFirstTrackletIndex{
//       devStore.getTrackletsLookupTable()[layerIndex][nextLayerClusterIndex]};
//     const int nextLayerTrackletsNum{static_cast<int>(devStore.getTracklets()[layerIndex + 1].size())};

//     if (devStore.getTracklets()[layerIndex + 1][nextLayerFirstTrackletIndex].firstClusterIndex == nextLayerClusterIndex) {

//       const Cluster& firstCellCluster{
//         devStore.getClusters()[layerIndex][currentTracklet.firstClusterIndex]};
//       const Cluster& secondCellCluster{
//         devStore.getClusters()[layerIndex + 1][currentTracklet.secondClusterIndex]};
//       const float firstCellClusterQuadraticRCoordinate{firstCellCluster.radius * firstCellCluster.radius};
//       const float secondCellClusterQuadraticRCoordinate{secondCellCluster.radius * secondCellCluster.radius};
//       const float3 firstDeltaVector{secondCellCluster.xCoordinate - firstCellCluster.xCoordinate,
//                                     secondCellCluster.yCoordinate - firstCellCluster.yCoordinate, secondCellClusterQuadraticRCoordinate - firstCellClusterQuadraticRCoordinate};

//       for (int iNextLayerTracklet{nextLayerFirstTrackletIndex};
//            iNextLayerTracklet < nextLayerTrackletsNum && devStore.getTracklets()[layerIndex + 1][iNextLayerTracklet].firstClusterIndex == nextLayerClusterIndex; ++iNextLayerTracklet) {

//         const Tracklet& nextTracklet{devStore.getTracklets()[layerIndex + 1][iNextLayerTracklet]};
//         const float deltaTanLambda{o2::gpu::GPUCommonMath::Abs(currentTracklet.tanLambda - nextTracklet.tanLambda)};
//         const float deltaPhi{o2::gpu::GPUCommonMath::Abs(currentTracklet.phi - nextTracklet.phi)};

//         if (deltaTanLambda < kTrkPar.CellMaxDeltaTanLambda && (deltaPhi < kTrkPar.CellMaxDeltaPhi || o2::gpu::GPUCommonMath::Abs(deltaPhi - constants::math::TwoPi) < kTrkPar.CellMaxDeltaPhi)) {

//           const float averageTanLambda{0.5f * (currentTracklet.tanLambda + nextTracklet.tanLambda)};
//           const float directionZIntersection{-averageTanLambda * firstCellCluster.radius + firstCellCluster.zCoordinate};
//           const float deltaZ{o2::gpu::GPUCommonMath::Abs(directionZIntersection - primaryVertex.z)};

//           if (deltaZ < kTrkPar.CellMaxDeltaZ[layerIndex]) {

//             const Cluster& thirdCellCluster{
//               devStore.getClusters()[layerIndex + 2][nextTracklet.secondClusterIndex]};

//             const float thirdCellClusterQuadraticRCoordinate{thirdCellCluster.radius * thirdCellCluster.radius};

//             const float3 secondDeltaVector{thirdCellCluster.xCoordinate - firstCellCluster.xCoordinate,
//                                            thirdCellCluster.yCoordinate - firstCellCluster.yCoordinate, thirdCellClusterQuadraticRCoordinate - firstCellClusterQuadraticRCoordinate};

//             float3 cellPlaneNormalVector{math_utils::crossProduct(firstDeltaVector, secondDeltaVector)};

//             const float vectorNorm{o2::gpu::GPUCommonMath::Sqrt(
//               cellPlaneNormalVector.x * cellPlaneNormalVector.x + cellPlaneNormalVector.y * cellPlaneNormalVector.y + cellPlaneNormalVector.z * cellPlaneNormalVector.z)};

//             if (!(vectorNorm < constants::math::FloatMinThreshold || o2::gpu::GPUCommonMath::Abs(cellPlaneNormalVector.z) < constants::math::FloatMinThreshold)) {

//               const float inverseVectorNorm{1.0f / vectorNorm};
//               const float3 normalizedPlaneVector{cellPlaneNormalVector.x * inverseVectorNorm, cellPlaneNormalVector.y * inverseVectorNorm, cellPlaneNormalVector.z * inverseVectorNorm};
//               const float planeDistance{-normalizedPlaneVector.x * (secondCellCluster.xCoordinate - primaryVertex.x) - (normalizedPlaneVector.y * secondCellCluster.yCoordinate - primaryVertex.y) - normalizedPlaneVector.z * secondCellClusterQuadraticRCoordinate};
//               const float normalizedPlaneVectorQuadraticZCoordinate{normalizedPlaneVector.z * normalizedPlaneVector.z};
//               const float cellTrajectoryRadius{o2::gpu::GPUCommonMath::Sqrt(
//                 (1.0f - normalizedPlaneVectorQuadraticZCoordinate - 4.0f * planeDistance * normalizedPlaneVector.z) / (4.0f * normalizedPlaneVectorQuadraticZCoordinate))};
//               const float2 circleCenter{-0.5f * normalizedPlaneVector.x / normalizedPlaneVector.z, -0.5f * normalizedPlaneVector.y / normalizedPlaneVector.z};
//               const float distanceOfClosestApproach{o2::gpu::GPUCommonMath::Abs(
//                 cellTrajectoryRadius - o2::gpu::GPUCommonMath::Sqrt(circleCenter.x * circleCenter.x + circleCenter.y * circleCenter.y))};

//               if (distanceOfClosestApproach <= kTrkPar.CellMaxDCA[layerIndex]) {

//                 cooperative_groups::coalesced_group threadGroup = cooperative_groups::coalesced_threads();
//                 int currentIndex{};

//                 if (threadGroup.thread_rank() == 0) {

//                   currentIndex = cellsVector.extend(threadGroup.size());
//                 }

//                 currentIndex = threadGroup.shfl(currentIndex, 0) + threadGroup.thread_rank();

//                 cellsVector.emplace(currentIndex, currentTracklet.firstClusterIndex,
//                                     nextTracklet.firstClusterIndex, nextTracklet.secondClusterIndex, currentTrackletIndex,
//                                     iNextLayerTracklet, averageTanLambda);
//                 ++trackletCellsNum;
//               }
//             }
//           }
//         }
//       }

//       if (layerIndex > 0) {

//         devStore.getCellsPerTrackletTable()[layerIndex - 1][currentTrackletIndex] = trackletCellsNum;
//       }
//     }
//   }
// }

// GPUg() void layerTrackletsKernel(DeviceStoreNV& devStore, const int layerIndex,
//                                  Vector<Tracklet> trackletsVector)
// {
//   computeLayerTracklets(devStore, layerIndex, trackletsVector);
// }

// GPUg() void sortTrackletsKernel(DeviceStoreNV& devStore, const int layerIndex,
//                                 Vector<Tracklet> tempTrackletArray)
// {
//   const int currentTrackletIndex{static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x)};

//   if (currentTrackletIndex < tempTrackletArray.size()) {

//     const int firstClusterIndex = tempTrackletArray[currentTrackletIndex].firstClusterIndex;
//     const int offset = atomicAdd(&devStore.getTrackletsPerClusterTable()[layerIndex - 1][firstClusterIndex],
//                                  -1) -
//                        1;
//     const int startIndex = devStore.getTrackletsLookupTable()[layerIndex - 1][firstClusterIndex];

//     memcpy(&devStore.getTracklets()[layerIndex][startIndex + offset],
//            &tempTrackletArray[currentTrackletIndex], sizeof(Tracklet));
//   }
// }

// GPUg() void layerCellsKernel(DeviceStoreNV& devStore, const int layerIndex,
//                              Vector<Cell> cellsVector)
// {
//   computeLayerCells(devStore, layerIndex, cellsVector);
// }

// GPUg() void sortCellsKernel(DeviceStoreNV& devStore, const int layerIndex,
//                             Vector<Cell> tempCellsArray)
// {
//   const int currentCellIndex = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);

//   if (currentCellIndex < tempCellsArray.size()) {

//     const int firstTrackletIndex = tempCellsArray[currentCellIndex].getFirstTrackletIndex();
//     const int offset = atomicAdd(&devStore.getCellsPerTrackletTable()[layerIndex - 1][firstTrackletIndex],
//                                  -1) -
//                        1;
//     const int startIndex = devStore.getCellsLookupTable()[layerIndex - 1][firstTrackletIndex];

//     memcpy(&devStore.getCells()[layerIndex][startIndex + offset], &tempCellsArray[currentCellIndex],
//            sizeof(Cell));
//   }
// }

} // namespace gpu

// void TrackeTraitsGPU::adoptTimeFrame(TimeFrame* tf)
// {
//   mTimeFrameGPU = tf;
// }
// TrackerTraits* createTrackerTraitsGPU()
// {
//   return new TrackerTraitsGPU;
// }

template <int NLayers>
void TrackerTraitsGPU<NLayers>::computeLayerTracklets(const int iteration)
{
  //   PrimaryVertexContextNV* primaryVertexContext = static_cast<PrimaryVertexContextNV*>(nullptr); //TODO: FIX THIS with Time Frames

  checkGPUError(hipMemcpyToSymbol(HIP_SYMBOL(gpu::kTrkPar), &mTrkParams, sizeof(gpu::StaticTrackingParameters<7>)), __FILE__, __LINE__);
  //   std::array<size_t, constants::its2::CellsPerRoad> tempSize;
  //   std::array<int, constants::its2::CellsPerRoad> trackletsNum;
  //   std::array<gpu::Stream, constants::its2::TrackletsPerRoad> streamArray;

  //   for (int iLayer{0}; iLayer < constants::its2::CellsPerRoad; ++iLayer) {

  //     tempSize[iLayer] = 0;
  //     primaryVertexContext->getTempTrackletArray()[iLayer].reset(
  //       static_cast<int>(primaryVertexContext->getDeviceTracklets()[iLayer + 1].capacity()));

  //     hipcub::DeviceScan::ExclusiveSum(static_cast<void*>(NULL), tempSize[iLayer],
  //                                   primaryVertexContext->getDeviceTrackletsPerClustersTable()[iLayer].get(),
  //                                   primaryVertexContext->getDeviceTrackletsLookupTable()[iLayer].get(),
  //                                   primaryVertexContext->getClusters()[iLayer + 1].size());

  //     primaryVertexContext->getTempTableArray()[iLayer].reset(static_cast<int>(tempSize[iLayer]));
  //   }

  //   hipDeviceSynchronize();

  //   for (int iLayer{0}; iLayer < constants::its2::TrackletsPerRoad; ++iLayer) {

  //     const gpu::DeviceProperties& deviceProperties = gpu::Context::getInstance().getDeviceProperties();
  //     const int clustersNum{static_cast<int>(primaryVertexContext->getClusters()[iLayer].size())};
  //     dim3 threadsPerBlock{gpu::utils::host::getBlockSize(clustersNum, 1, 192)};
  //     dim3 blocksGrid{gpu::utils::host::getBlocksGrid(threadsPerBlock, clustersNum)};

  //     if (iLayer == 0) {

  //       gpu::layerTrackletsKernel<<<blocksGrid, threadsPerBlock, 0, streamArray[iLayer].get()>>>(primaryVertexContext->getDeviceContext(),
  //                                                                                                iLayer, primaryVertexContext->getDeviceTracklets()[iLayer].getWeakCopy());

  //     } else {

  //       gpu::layerTrackletsKernel<<<blocksGrid, threadsPerBlock, 0, streamArray[iLayer].get()>>>(primaryVertexContext->getDeviceContext(),
  //                                                                                                iLayer, primaryVertexContext->getTempTrackletArray()[iLayer - 1].getWeakCopy());
  //     }

  //     hipError_t error = hipGetLastError();

  //     if (error != hipSuccess) {

  //       std::ostringstream errorString{};
  //       errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << ")"
  //                   << std::endl;

  //       throw std::runtime_error{errorString.str()};
  //     }
  //   }

  //   hipDeviceSynchronize();

  //   for (int iLayer{0}; iLayer < constants::its2::CellsPerRoad; ++iLayer) {

  //     trackletsNum[iLayer] = primaryVertexContext->getTempTrackletArray()[iLayer].getSizeFromDevice();
  //     if (trackletsNum[iLayer] == 0) {
  //       continue;
  //     }
  //     primaryVertexContext->getDeviceTracklets()[iLayer + 1].resize(trackletsNum[iLayer]);

  //     hipcub::DeviceScan::ExclusiveSum(static_cast<void*>(primaryVertexContext->getTempTableArray()[iLayer].get()), tempSize[iLayer],
  //                                   primaryVertexContext->getDeviceTrackletsPerClustersTable()[iLayer].get(),
  //                                   primaryVertexContext->getDeviceTrackletsLookupTable()[iLayer].get(),
  //                                   primaryVertexContext->getClusters()[iLayer + 1].size(), streamArray[iLayer + 1].get());

  //     dim3 threadsPerBlock{gpu::utils::host::getBlockSize(trackletsNum[iLayer])};
  //     dim3 blocksGrid{gpu::utils::host::getBlocksGrid(threadsPerBlock, trackletsNum[iLayer])};

  //     gpu::sortTrackletsKernel<<<blocksGrid, threadsPerBlock, 0, streamArray[iLayer + 1].get()>>>(primaryVertexContext->getDeviceContext(),
  //                                                                                                 iLayer + 1, primaryVertexContext->getTempTrackletArray()[iLayer].getWeakCopy());

  //     hipError_t error = hipGetLastError();

  //     if (error != hipSuccess) {

  //       std::ostringstream errorString{};
  //       errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << ")"
  //                   << std::endl;

  //       throw std::runtime_error{errorString.str()};
  //     }
  //   }
}

// void TrackerTraitsGPU::computeLayerCells()
// {

//   PrimaryVertexContextNV* primaryVertexContext = static_cast<PrimaryVertexContextNV*>(nullptr); //TODO: FIX THIS with Time Frames
//   std::array<size_t, constants::its2::CellsPerRoad - 1> tempSize;
//   std::array<int, constants::its2::CellsPerRoad - 1> trackletsNum;
//   std::array<int, constants::its2::CellsPerRoad - 1> cellsNum;
//   std::array<gpu::Stream, constants::its2::CellsPerRoad> streamArray;

//   for (int iLayer{0}; iLayer < constants::its2::CellsPerRoad - 1; ++iLayer) {

//     tempSize[iLayer] = 0;
//     trackletsNum[iLayer] = primaryVertexContext->getDeviceTracklets()[iLayer + 1].getSizeFromDevice();
//     primaryVertexContext->getTempCellArray()[iLayer].reset(
//       static_cast<int>(primaryVertexContext->getDeviceCells()[iLayer + 1].capacity()));
//     if (trackletsNum[iLayer] == 0) {
//       continue;
//     }
//     hipcub::DeviceScan::ExclusiveSum(static_cast<void*>(NULL), tempSize[iLayer],
//                                   primaryVertexContext->getDeviceCellsPerTrackletTable()[iLayer].get(),
//                                   primaryVertexContext->getDeviceCellsLookupTable()[iLayer].get(), trackletsNum[iLayer]);

//     primaryVertexContext->getTempTableArray()[iLayer].reset(static_cast<int>(tempSize[iLayer]));
//   }

//   hipDeviceSynchronize();

//   for (int iLayer{0}; iLayer < constants::its2::CellsPerRoad; ++iLayer) {
//     const gpu::DeviceProperties& deviceProperties = gpu::Context::getInstance().getDeviceProperties();
//     const int trackletsSize = primaryVertexContext->getDeviceTracklets()[iLayer].getSizeFromDevice();
//     if (trackletsSize == 0) {
//       continue;
//     }
//     dim3 threadsPerBlock{gpu::utils::host::getBlockSize(trackletsSize)};
//     dim3 blocksGrid{gpu::utils::host::getBlocksGrid(threadsPerBlock, trackletsSize)};

//     if (iLayer == 0) {

//       gpu::layerCellsKernel<<<blocksGrid, threadsPerBlock, 0, streamArray[iLayer].get()>>>(primaryVertexContext->getDeviceContext(),
//                                                                                            iLayer, primaryVertexContext->getDeviceCells()[iLayer].getWeakCopy());

//     } else {

//       gpu::layerCellsKernel<<<blocksGrid, threadsPerBlock, 0, streamArray[iLayer].get()>>>(primaryVertexContext->getDeviceContext(),
//                                                                                            iLayer, primaryVertexContext->getTempCellArray()[iLayer - 1].getWeakCopy());
//     }

//     hipError_t error = hipGetLastError();

//     if (error != hipSuccess) {

//       std::ostringstream errorString{};
//       errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << ")"
//                   << std::endl;

//       throw std::runtime_error{errorString.str()};
//     }
//   }

//   hipDeviceSynchronize();

//   for (int iLayer{0}; iLayer < constants::its2::CellsPerRoad - 1; ++iLayer) {
//     cellsNum[iLayer] = primaryVertexContext->getTempCellArray()[iLayer].getSizeFromDevice();
//     if (cellsNum[iLayer] == 0) {
//       continue;
//     }
//     primaryVertexContext->getDeviceCells()[iLayer + 1].resize(cellsNum[iLayer]);

//     hipcub::DeviceScan::ExclusiveSum(static_cast<void*>(primaryVertexContext->getTempTableArray()[iLayer].get()), tempSize[iLayer],
//                                   primaryVertexContext->getDeviceCellsPerTrackletTable()[iLayer].get(),
//                                   primaryVertexContext->getDeviceCellsLookupTable()[iLayer].get(), trackletsNum[iLayer],
//                                   streamArray[iLayer + 1].get());

//     dim3 threadsPerBlock{gpu::utils::host::getBlockSize(trackletsNum[iLayer])};
//     dim3 blocksGrid{gpu::utils::host::getBlocksGrid(threadsPerBlock, trackletsNum[iLayer])};

//     gpu::sortCellsKernel<<<blocksGrid, threadsPerBlock, 0, streamArray[iLayer + 1].get()>>>(primaryVertexContext->getDeviceContext(),
//                                                                                             iLayer + 1, primaryVertexContext->getTempCellArray()[iLayer].getWeakCopy());

//     hipError_t error = hipGetLastError();

//     if (error != hipSuccess) {

//       std::ostringstream errorString{};
//       errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << ")"
//                   << std::endl;

//       throw std::runtime_error{errorString.str()};
//     }
//   }

//   hipDeviceSynchronize();

//   for (int iLayer{0}; iLayer < constants::its2::CellsPerRoad; ++iLayer) {

//     int cellsSize = 0;
//     if (iLayer == 0) {

//       cellsSize = primaryVertexContext->getDeviceCells()[iLayer].getSizeFromDevice();
//       if (cellsSize == 0) {
//         continue;
//       }
//     } else {

//       cellsSize = cellsNum[iLayer - 1];
//       if (cellsSize == 0) {
//         continue;
//       }
//       primaryVertexContext->getDeviceCellsLookupTable()[iLayer - 1].copyIntoVector(
//         primaryVertexContext->getCellsLookupTable()[iLayer - 1], trackletsNum[iLayer - 1]);
//     }

//     primaryVertexContext->getDeviceCells()[iLayer].copyIntoVector(primaryVertexContext->getCells()[iLayer], cellsSize);
//   }
// }

// void TrackerTraitsGPU::refitTracks(const std::vector<std::vector<TrackingFrameInfo>>& tf, std::vector<TrackITSExt>& tracks)
// {
//   PrimaryVertexContextNV* pvctx = static_cast<PrimaryVertexContextNV*>(nullptr); //TODO: FIX THIS with Time Frames

//   std::array<const Cell*, 5> cells;
//   for (int iLayer = 0; iLayer < 5; iLayer++) {
//     cells[iLayer] = pvctx->getDeviceCells()[iLayer].get();
//   }
//   std::array<const Cluster*, 7> clusters;
//   for (int iLayer = 0; iLayer < 7; iLayer++) {
//     clusters[iLayer] = pvctx->getDeviceClusters()[iLayer].get();
//   }
//   //TODO: restore this
//   // mChainRunITSTrackFit(*mChain, mPrimaryVertexContext->getRoads(), clusters, cells, tf, tracks);
// }
template class TrackerTraitsGPU<7>;
} // namespace its
} // namespace o2
