#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
//
/// \author matteo.concas@cern.ch

#include <iostream>
#include <sstream>
#include <fstream>
#include <array>
#include <cassert>
#include <thread>

#ifndef GPUCA_GPUCODE_GENRTC
#include <hipcub/hipcub.hpp>
#endif

#include "ITStracking/MathUtils.h"
#include "ITStracking/Configuration.h"
#include "ITStracking/ClusterLines.h"
#include "ITStracking/Tracklet.h"

#include "ITStrackingGPU/Utils.h"
#include "ITStrackingGPU/ClusterLinesGPU.h"
#include "ITStrackingGPU/VertexerTraitsGPU.h"

#include "GPUCommonArray.h"

#ifdef VTX_DEBUG
#include "TTree.h"
#include "TFile.h"
#endif

#include "ITStrackingGPU/TracerGPU.h"

namespace o2
{
namespace its
{
using constants::its::VertexerHistogramVolume;
using constants::math::TwoPi;
using gpu::utils::checkGPUError;
using math_utils::getNormalizedPhi;

using namespace constants::its2;
GPUd() const int4 getBinsRect(const Cluster& currentCluster, const int layerIndex,
                              const float z1, float maxdeltaz, float maxdeltaphi)
{
  const float zRangeMin = z1 - maxdeltaz;
  const float phiRangeMin = currentCluster.phi - maxdeltaphi;
  const float zRangeMax = z1 + maxdeltaz;
  const float phiRangeMax = currentCluster.phi + maxdeltaphi;

  if (zRangeMax < -LayersZCoordinate()[layerIndex + 1] ||
      zRangeMin > LayersZCoordinate()[layerIndex + 1] || zRangeMin > zRangeMax) {

    return getEmptyBinsRect();
  }

  return int4{o2::gpu::GPUCommonMath::Max(0, getZBinIndex(layerIndex + 1, zRangeMin)),
              getPhiBinIndex(phiRangeMin),
              o2::gpu::GPUCommonMath::Min(ZBins - 1, getZBinIndex(layerIndex + 1, zRangeMax)),
              getPhiBinIndex(phiRangeMax)};
}

GPUh() void gpuThrowOnError()
{
  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {
    std::ostringstream errorString{};
    errorString << GPU_ARCH << " API returned error  [" << hipGetErrorString(error) << "] (code " << error << ")" << std::endl;
    throw std::runtime_error{errorString.str()};
  }
}

VertexerTraitsGPU::VertexerTraitsGPU()
{
  setIsGPU(true);
}

VertexerTraitsGPU::~VertexerTraitsGPU()
{
  gpu::utils::gpuFree(mDeviceIndexTableUtils);
}

void VertexerTraitsGPU::initialise(const TrackingParameters& trackingParams)
{
  mTimeFrameGPU->initialise(0, trackingParams, 3, &mIndexTableUtils, &mTfGPUParams);
}

namespace gpu
{

template <typename... Args>
GPUd() void printOnThread(const unsigned int tId, const char* str, Args... args)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    printf(str, args...);
  }
}

template <typename... Args>
GPUd() void printOnBlock(const unsigned int bId, const char* str, Args... args)
{
  if (blockIdx.x == bId && threadIdx.x == 0) {
    printf(str, args...);
  }
}

GPUg() void printBufferOnThread(const int* v, size_t size, const int len = 150, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (int i{0}; i < size; ++i) {
      if (!(i % len)) {
        printf("\n start: ===>%d/%d\t", i, (int)size);
      }
      printf("%d\t", v[i]);
    }
    printf("\n");
  }
}

GPUg() void printBufferOnThreadF(const float* v, size_t size, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    printf("vector :");
    for (int i{0}; i < size; ++i) {
      printf("%.9f\t", v[i]);
    }
    printf("\n");
  }
}

GPUg() void resetTrackletsKernel(Tracklet* tracklets, const int nTracklets)
{
  for (int iCurrentLayerClusterIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentLayerClusterIndex < nTracklets; iCurrentLayerClusterIndex += blockDim.x * gridDim.x) {
    new (tracklets + iCurrentLayerClusterIndex) Tracklet{};
  }
}

GPUg() void dumpFoundTrackletsKernel(const Tracklet* tracklets, const int* nTracklet, const size_t nClustersMiddleLayer, const int maxTrackletsPerCluster)
{
  for (int iCurrentLayerClusterIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentLayerClusterIndex < nClustersMiddleLayer; iCurrentLayerClusterIndex += blockDim.x * gridDim.x) {
    const int stride{iCurrentLayerClusterIndex * maxTrackletsPerCluster};
    for (int iTracklet{0}; iTracklet < nTracklet[iCurrentLayerClusterIndex]; ++iTracklet) {
      auto& t = tracklets[stride + iTracklet];
      t.dump();
    }
  }
}

GPUg() void dumpMaximaKernel(const hipcub::KeyValuePair<int, int>* tmpVertexBins, const int threadId)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == threadId) {
    printf("XmaxBin: %d at index: %d | YmaxBin: %d at index: %d | ZmaxBin: %d at index: %d\n",
           tmpVertexBins[0].value, tmpVertexBins[0].key,
           tmpVertexBins[1].value, tmpVertexBins[1].key,
           tmpVertexBins[2].value, tmpVertexBins[2].key);
  }
}

template <TrackletMode Mode>
GPUg() void trackleterKernelSingleRof(
  const Cluster* clustersNextLayer,    // 0 2
  const Cluster* clustersCurrentLayer, // 1 1
  const int sizeNextLClusters,
  const int sizeCurrentLClusters,
  const int* indexTableNext,
  const float phiCut,
  Tracklet* Tracklets,
  int* foundTracklets,
  const IndexTableUtils* utils,
  const int rofId,
  const size_t maxTrackletsPerCluster = 1e2)
{
  const int phiBins{utils->getNphiBins()};
  const int zBins{utils->getNzBins()};
  // loop on layer1 clusters
  for (int iCurrentLayerClusterIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentLayerClusterIndex < sizeCurrentLClusters; iCurrentLayerClusterIndex += blockDim.x * gridDim.x) {
    if (iCurrentLayerClusterIndex < sizeCurrentLClusters) {
      unsigned int storedTracklets{0};
      const size_t stride{iCurrentLayerClusterIndex * maxTrackletsPerCluster};
      const Cluster& currentCluster = clustersCurrentLayer[iCurrentLayerClusterIndex];
      const int4 selectedBinsRect{VertexerTraits::getBinsRect(currentCluster, (int)Mode, 0.f, 50.f, phiCut / 2, *utils)};
      if (selectedBinsRect.x != 0 || selectedBinsRect.y != 0 || selectedBinsRect.z != 0 || selectedBinsRect.w != 0) {
        int phiBinsNum{selectedBinsRect.w - selectedBinsRect.y + 1};
        if (phiBinsNum < 0) {
          phiBinsNum += phiBins;
        }
        // loop on phi bins next layer
        for (unsigned int iPhiBin{(unsigned int)selectedBinsRect.y}, iPhiCount{0}; iPhiCount < (unsigned int)phiBinsNum; iPhiBin = ++iPhiBin == phiBins ? 0 : iPhiBin, iPhiCount++) {
          const int firstBinIndex{utils->getBinIndex(selectedBinsRect.x, iPhiBin)};
          const int firstRowClusterIndex{indexTableNext[firstBinIndex]};
          const int maxRowClusterIndex{indexTableNext[firstBinIndex + zBins]};
          // loop on clusters next layer
          for (int iNextLayerClusterIndex{firstRowClusterIndex}; iNextLayerClusterIndex < maxRowClusterIndex && iNextLayerClusterIndex < sizeNextLClusters; ++iNextLayerClusterIndex) {
            const Cluster& nextCluster = clustersNextLayer[iNextLayerClusterIndex];
            if (o2::gpu::GPUCommonMath::Abs(currentCluster.phi - nextCluster.phi) < phiCut) {
              if (storedTracklets < maxTrackletsPerCluster) {
                if constexpr (Mode == TrackletMode::Layer0Layer1) {
                  new (Tracklets + stride + storedTracklets) Tracklet{iNextLayerClusterIndex, iCurrentLayerClusterIndex, nextCluster, currentCluster, rofId, rofId};
                } else {
                  new (Tracklets + stride + storedTracklets) Tracklet{iCurrentLayerClusterIndex, iNextLayerClusterIndex, currentCluster, nextCluster, rofId, rofId};
                }
                ++storedTracklets;
              }
            }
          }
        }
      }
      foundTracklets[iCurrentLayerClusterIndex] = storedTracklets;
      if (storedTracklets >= maxTrackletsPerCluster) {
        printf("gpu tracklet finder: some lines will be left behind for cluster %d. valid: %u max: %zu\n", iCurrentLayerClusterIndex, storedTracklets, maxTrackletsPerCluster);
      }
    }
  }
}

template <TrackletMode Mode>
GPUg() void trackleterKernelMultipleRof(
  const Cluster* clustersNextLayer,    // 0 2
  const Cluster* clustersCurrentLayer, // 1 1
  const int* sizeNextLClusters,
  const int* sizeCurrentLClusters,
  const int* nextIndexTables,
  Tracklet* Tracklets,
  int* foundTracklets,
  const IndexTableUtils* utils,
  const unsigned int startRofId,
  const unsigned int rofSize,
  const float phiCut,
  const size_t maxTrackletsPerCluster = 1e2)
{
  const int phiBins{utils->getNphiBins()};
  const int zBins{utils->getNzBins()};
  for (unsigned int iRof{blockIdx.x}; iRof < rofSize; iRof += gridDim.x) {
    auto rof = iRof + startRofId;
    auto* clustersNextLayerRof = clustersNextLayer + (sizeNextLClusters[rof] - sizeNextLClusters[startRofId]);
    auto* clustersCurrentLayerRof = clustersCurrentLayer + (sizeCurrentLClusters[rof] - sizeCurrentLClusters[startRofId]);
    auto nClustersNextLayerRof = sizeNextLClusters[rof + 1] - sizeNextLClusters[rof];
    auto nClustersCurrentLayerRof = sizeCurrentLClusters[rof + 1] - sizeCurrentLClusters[rof];
    auto* indexTableNextRof = nextIndexTables + iRof * (phiBins * zBins + 1);
    auto* TrackletsRof = Tracklets + (sizeCurrentLClusters[rof] - sizeCurrentLClusters[startRofId]) * maxTrackletsPerCluster;
    auto* foundTrackletsRof = foundTracklets + (sizeCurrentLClusters[rof] - sizeCurrentLClusters[startRofId]);

    // single rof loop on layer1 clusters
    for (int iCurrentLayerClusterIndex = threadIdx.x; iCurrentLayerClusterIndex < nClustersCurrentLayerRof; iCurrentLayerClusterIndex += blockDim.x) {
      unsigned int storedTracklets{0};
      const size_t stride{iCurrentLayerClusterIndex * maxTrackletsPerCluster};
      const Cluster& currentCluster = clustersCurrentLayerRof[iCurrentLayerClusterIndex];
      const int4 selectedBinsRect{VertexerTraits::getBinsRect(currentCluster, (int)Mode, 0.f, 50.f, phiCut / 2, *utils)};
      if (selectedBinsRect.x != 0 || selectedBinsRect.y != 0 || selectedBinsRect.z != 0 || selectedBinsRect.w != 0) {
        int phiBinsNum{selectedBinsRect.w - selectedBinsRect.y + 1};
        if (phiBinsNum < 0) {
          phiBinsNum += phiBins;
        }
        // loop on phi bins next layer
        for (unsigned int iPhiBin{(unsigned int)selectedBinsRect.y}, iPhiCount{0}; iPhiCount < (unsigned int)phiBinsNum; iPhiBin = ++iPhiBin == phiBins ? 0 : iPhiBin, iPhiCount++) {
          const int firstBinIndex{utils->getBinIndex(selectedBinsRect.x, iPhiBin)};
          const int firstRowClusterIndex{indexTableNextRof[firstBinIndex]};
          const int maxRowClusterIndex{indexTableNextRof[firstBinIndex + zBins]};
          // loop on clusters next layer
          for (int iNextLayerClusterIndex{firstRowClusterIndex}; iNextLayerClusterIndex < maxRowClusterIndex && iNextLayerClusterIndex < nClustersNextLayerRof; ++iNextLayerClusterIndex) {
            const Cluster& nextCluster = clustersNextLayerRof[iNextLayerClusterIndex];
            if (o2::gpu::GPUCommonMath::Abs(currentCluster.phi - nextCluster.phi) < phiCut) {
              if (storedTracklets < maxTrackletsPerCluster) {
                if constexpr (Mode == TrackletMode::Layer0Layer1) {
                  new (TrackletsRof + stride + storedTracklets) Tracklet{iNextLayerClusterIndex, iCurrentLayerClusterIndex, nextCluster, currentCluster, static_cast<int>(rof), static_cast<int>(rof)};
                } else {
                  new (TrackletsRof + stride + storedTracklets) Tracklet{iCurrentLayerClusterIndex, iNextLayerClusterIndex, currentCluster, nextCluster, static_cast<int>(rof), static_cast<int>(rof)};
                }
                ++storedTracklets;
              }
            }
          }
        }
      }
      foundTrackletsRof[iCurrentLayerClusterIndex] = storedTracklets;
      // if (storedTracklets >= maxTrackletsPerCluster && storedTracklets - maxTrackletsPerCluster < 5) {
      //   printf("gpu tracklet finder: some lines will be left behind for cluster %d in rof: %d. valid: %u max: %lu (suppressing after 5 msgs)\n", iCurrentLayerClusterIndex, rof, storedTracklets, maxTrackletsPerCluster);
      // }
    }
  }
}

template <bool initRun>
GPUg() void trackletSelectionKernelSingleRof(
  const Cluster* clusters0,
  const Cluster* clusters1,
  const size_t nClustersMiddleLayer,
  Tracklet* tracklets01,
  Tracklet* tracklets12,
  const int* nFoundTracklet01,
  const int* nFoundTracklet12,
  unsigned char* usedTracklets,
  Line* lines,
  int* nFoundLines,
  int* nExclusiveFoundLines,
  const int maxTrackletsPerCluster = 1e2,
  const float tanLambdaCut = 0.025f,
  const float phiCut = 0.002f)
{
  for (int iCurrentLayerClusterIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentLayerClusterIndex < nClustersMiddleLayer; iCurrentLayerClusterIndex += blockDim.x * gridDim.x) {
    const int stride{iCurrentLayerClusterIndex * maxTrackletsPerCluster};
    int validTracklets{0};
    for (int iTracklet12{0}; iTracklet12 < nFoundTracklet12[iCurrentLayerClusterIndex]; ++iTracklet12) {
      for (int iTracklet01{0}; iTracklet01 < nFoundTracklet01[iCurrentLayerClusterIndex] && validTracklets < maxTrackletsPerCluster; ++iTracklet01) {
        const float deltaTanLambda{o2::gpu::GPUCommonMath::Abs(tracklets01[stride + iTracklet01].tanLambda - tracklets12[stride + iTracklet12].tanLambda)};
        const float deltaPhi{o2::gpu::GPUCommonMath::Abs(tracklets01[stride + iTracklet01].phi - tracklets12[stride + iTracklet12].phi)};
        if (!usedTracklets[stride + iTracklet01] && deltaTanLambda < tanLambdaCut && deltaPhi < phiCut && validTracklets != maxTrackletsPerCluster) {
          usedTracklets[stride + iTracklet01] = true;
          if constexpr (!initRun) {
            new (lines + nExclusiveFoundLines[iCurrentLayerClusterIndex] + validTracklets) Line{tracklets01[stride + iTracklet01], clusters0, clusters1};
          }
          ++validTracklets;
        }
      }
    }
    if constexpr (initRun) {
      nFoundLines[iCurrentLayerClusterIndex] = validTracklets;
      if (validTracklets >= maxTrackletsPerCluster) {
        printf("gpu tracklet selection: some lines will be left behind for cluster %d. valid: %d max: %d\n", iCurrentLayerClusterIndex, validTracklets, maxTrackletsPerCluster);
      }
    }
  }
}

template <bool initRun>
GPUg() void trackletSelectionKernelMultipleRof(
  const Cluster* clusters0,               // Clusters on layer 0
  const Cluster* clusters1,               // Clusters on layer 1
  const int* sizeClustersL0,              // Number of clusters on layer 0 per ROF
  const int* sizeClustersL1,              // Number of clusters on layer 1 per ROF
  Tracklet* tracklets01,                  // Tracklets on layer 0-1
  Tracklet* tracklets12,                  // Tracklets on layer 1-2
  const int* nFoundTracklets01,           // Number of tracklets found on layers 0-1
  const int* nFoundTracklets12,           // Number of tracklets found on layers 1-2
  unsigned char* usedTracklets,           // Used tracklets
  Line* lines,                            // Lines
  int* nFoundLines,                       // Number of found lines
  int* nExclusiveFoundLines,              // Number of found lines exclusive scan
  const unsigned int startRofId,          // Starting ROF ID
  const unsigned int rofSize,             // Number of ROFs to consider
  const int maxTrackletsPerCluster = 1e2, // Maximum number of tracklets per cluster
  const float tanLambdaCut = 0.025f,      // Cut on tan lambda
  const float phiCut = 0.002f)            // Cut on phi
{
  for (unsigned int iRof{blockIdx.x}; iRof < rofSize; iRof += gridDim.x) {
    auto rof = iRof + startRofId;
    auto* clustersL0Rof = clusters0 + (sizeClustersL0[rof] - sizeClustersL0[startRofId]);
    auto clustersL1offsetRof = sizeClustersL1[rof] - sizeClustersL1[startRofId];
    auto* clustersL1Rof = clusters1 + clustersL1offsetRof;
    auto nClustersL1Rof = sizeClustersL1[rof + 1] - sizeClustersL1[rof];
    auto* tracklets01Rof = tracklets01 + clustersL1offsetRof * maxTrackletsPerCluster;
    auto* tracklets12Rof = tracklets12 + clustersL1offsetRof * maxTrackletsPerCluster;
    auto* foundTracklets01Rof = nFoundTracklets01 + clustersL1offsetRof;
    auto* foundTracklets12Rof = nFoundTracklets12 + clustersL1offsetRof;
    auto* usedTrackletsRof = usedTracklets + clustersL1offsetRof * maxTrackletsPerCluster;
    auto* foundLinesRof = nFoundLines + clustersL1offsetRof;
    int* nExclusiveFoundLinesRof = nullptr;
    if constexpr (!initRun) {
      nExclusiveFoundLinesRof = nExclusiveFoundLines + clustersL1offsetRof;
    }
    for (int iClusterIndexLayer1 = threadIdx.x; iClusterIndexLayer1 < nClustersL1Rof; iClusterIndexLayer1 += blockDim.x) {
      const int stride{iClusterIndexLayer1 * maxTrackletsPerCluster};
      int validTracklets{0};
      for (int iTracklet12{0}; iTracklet12 < foundTracklets12Rof[iClusterIndexLayer1]; ++iTracklet12) {
        for (int iTracklet01{0}; iTracklet01 < foundTracklets01Rof[iClusterIndexLayer1] && validTracklets < maxTrackletsPerCluster; ++iTracklet01) {
          const float deltaTanLambda{o2::gpu::GPUCommonMath::Abs(tracklets01Rof[stride + iTracklet01].tanLambda - tracklets12Rof[stride + iTracklet12].tanLambda)};
          const float deltaPhi{o2::gpu::GPUCommonMath::Abs(tracklets01Rof[stride + iTracklet01].phi - tracklets12Rof[stride + iTracklet12].phi)};
          if (!usedTrackletsRof[stride + iTracklet01] && deltaTanLambda < tanLambdaCut && deltaPhi < phiCut && validTracklets != maxTrackletsPerCluster) {
            usedTrackletsRof[stride + iTracklet01] = true;
            if constexpr (!initRun) {
              new (lines + nExclusiveFoundLinesRof[iClusterIndexLayer1] + validTracklets) Line{tracklets01Rof[stride + iTracklet01], clustersL0Rof, clustersL1Rof};
            }
            ++validTracklets;
          }
        }
      }
      if constexpr (initRun) {
        foundLinesRof[iClusterIndexLayer1] = validTracklets;
        // if (validTracklets >= maxTrackletsPerCluster) {
        // printf("gpu tracklet selection: some lines will be left behind for cluster %d. valid: %d max: %d\n", iClusterIndexLayer1, validTracklets, maxTrackletsPerCluster);
        // }
      }
    }
  } // rof loop
}

GPUg() void lineClustererMultipleRof(
  const int* sizeClustersL1,     // Number of clusters on layer 1 per ROF
  Line* lines,                   // Lines
  int* nFoundLines,              // Number of found lines
  int* nExclusiveFoundLines,     // Number of found lines exclusive scan
  int* clusteredLines,           // Clustered lines
  const unsigned int startRofId, // Starting ROF ID
  const unsigned int rofSize,    // Number of ROFs to consider // Number of found lines exclusive scan
  const float pairCut)           // Selection on line pairs
{
  for (unsigned int iRof{threadIdx.x}; iRof < rofSize; iRof += blockDim.x) {
    auto rof = iRof + startRofId;
    auto clustersL1offsetRof = sizeClustersL1[rof] - sizeClustersL1[startRofId]; // starting cluster offset for this ROF
    auto nClustersL1Rof = sizeClustersL1[rof + 1] - sizeClustersL1[rof];         // number of clusters for this ROF
    auto linesOffsetRof = nExclusiveFoundLines[clustersL1offsetRof];             // starting line offset for this ROF
    // auto* foundLinesRof = nFoundLines + clustersL1offsetRof;
    auto nLinesRof = nExclusiveFoundLines[clustersL1offsetRof + nClustersL1Rof] - linesOffsetRof;
    // printf("rof: %d -> %d lines.\n", rof, nLinesRof);
    for (int iLine1 = 0; iLine1 < nLinesRof; ++iLine1) {
      auto absLine1Index = nExclusiveFoundLines[clustersL1offsetRof] + iLine1;
      if (clusteredLines[absLine1Index] > -1) {
        continue;
      }
      for (int iLine2 = iLine1 + 1; iLine2 < nLinesRof; ++iLine2) {
        auto absLine2Index = nExclusiveFoundLines[clustersL1offsetRof] + iLine2;
        if (clusteredLines[absLine2Index] > -1) {
          continue;
        }

        if (Line::getDCA(lines[absLine1Index], lines[absLine2Index]) < pairCut) {
          ClusterLinesGPU tmpClus{lines[absLine1Index], lines[absLine2Index]};
          float tmpVertex[3];
          tmpVertex[0] = tmpClus.getVertex()[0];
          tmpVertex[1] = tmpClus.getVertex()[1];
          tmpVertex[2] = tmpClus.getVertex()[2];
          if (tmpVertex[0] * tmpVertex[0] + tmpVertex[1] * tmpVertex[1] > 4.f) { // outside the beampipe, skip it
            break;
          }
          clusteredLines[absLine1Index] = iLine1; // We set local index of first line to contribute, so we can retrieve the cluster later
          clusteredLines[absLine2Index] = iLine1;
          for (int iLine3 = 0; iLine3 < nLinesRof; ++iLine3) {
            auto absLine3Index = nExclusiveFoundLines[clustersL1offsetRof] + iLine3;
            if (clusteredLines[absLine3Index] > -1) {
              continue;
            }
            if (Line::getDistanceFromPoint(lines[absLine3Index], tmpVertex) < pairCut) {
              clusteredLines[absLine3Index] = iLine1;
            }
          }
          break;
        }
      }
    }
  } // rof loop
}

GPUg() void computeCentroidsKernel(
  Line* lines,
  int* nFoundLines,
  int* nExclusiveFoundLines,
  const size_t nClustersMiddleLayer,
  float* centroids,
  const float lowHistX,
  const float highHistX,
  const float lowHistY,
  const float highHistY,
  const float pairCut)
{
  const int nLines = nExclusiveFoundLines[nClustersMiddleLayer - 1] + nFoundLines[nClustersMiddleLayer - 1];
  const int maxIterations{nLines * (nLines - 1) / 2};
  for (size_t currentThreadIndex = blockIdx.x * blockDim.x + threadIdx.x; currentThreadIndex < maxIterations; currentThreadIndex += blockDim.x * gridDim.x) {
    int iFirstLine = currentThreadIndex / nLines;
    int iSecondLine = currentThreadIndex % nLines;
    // All unique pairs
    if (iSecondLine <= iFirstLine) {
      iFirstLine = nLines - iFirstLine - 2;
      iSecondLine = nLines - iSecondLine - 1;
    }
    if (Line::getDCA(lines[iFirstLine], lines[iSecondLine]) < pairCut) {
      ClusterLinesGPU cluster{lines[iFirstLine], lines[iSecondLine]};
      if (cluster.getVertex()[0] * cluster.getVertex()[0] + cluster.getVertex()[1] * cluster.getVertex()[1] < 1.98f * 1.98f) {
        // printOnThread(0, "xCentr: %f, yCentr: %f \n", cluster.getVertex()[0], cluster.getVertex()[1]);
        centroids[2 * currentThreadIndex] = cluster.getVertex()[0];
        centroids[2 * currentThreadIndex + 1] = cluster.getVertex()[1];
      } else {
        // write values outside the histogram boundaries,
        // default behaviour is not to have them added to histogram later
        // (writing zeroes would be problematic)
        centroids[2 * currentThreadIndex] = 2 * lowHistX;
        centroids[2 * currentThreadIndex + 1] = 2 * lowHistY;
      }
    } else {
      // write values outside the histogram boundaries,
      // default behaviour is not to have them added to histogram later
      // (writing zeroes would be problematic)
      centroids[2 * currentThreadIndex] = 2 * highHistX;
      centroids[2 * currentThreadIndex + 1] = 2 * highHistY;
    }
  }
}

GPUg() void computeZCentroidsKernel(
  const int nLines,
  const hipcub::KeyValuePair<int, int>* tmpVtX,
  float* beamPosition,
  Line* lines,
  float* centroids,
  const int* histX, // X
  const float lowHistX,
  const float binSizeHistX,
  const int nBinsHistX,
  const int* histY, // Y
  const float lowHistY,
  const float binSizeHistY,
  const int nBinsHistY,
  const float lowHistZ, // Z
  const float pairCut,
  const int binOpeningX,
  const int binOpeningY)
{
  for (size_t currentThreadIndex = blockIdx.x * blockDim.x + threadIdx.x; currentThreadIndex < nLines; currentThreadIndex += blockDim.x * gridDim.x) {
    if (tmpVtX[0].value || tmpVtX[1].value) {
      float tmpX{lowHistX + tmpVtX[0].key * binSizeHistX + binSizeHistX / 2};
      int sumWX{tmpVtX[0].value};
      float wX{tmpX * tmpVtX[0].value};
      for (int iBin{o2::gpu::GPUCommonMath::Max(0, tmpVtX[0].key - binOpeningX)}; iBin < o2::gpu::GPUCommonMath::Min(tmpVtX[0].key + binOpeningX + 1, nBinsHistX - 1); ++iBin) {
        if (iBin != tmpVtX[0].key) {
          wX += (lowHistX + iBin * binSizeHistX + binSizeHistX / 2) * histX[iBin];
          sumWX += histX[iBin];
        }
      }
      float tmpY{lowHistY + tmpVtX[1].key * binSizeHistY + binSizeHistY / 2};
      int sumWY{tmpVtX[1].value};
      float wY{tmpY * tmpVtX[1].value};
      for (int iBin{o2::gpu::GPUCommonMath::Max(0, tmpVtX[1].key - binOpeningY)}; iBin < o2::gpu::GPUCommonMath::Min(tmpVtX[1].key + binOpeningY + 1, nBinsHistY - 1); ++iBin) {
        if (iBin != tmpVtX[1].key) {
          wY += (lowHistY + iBin * binSizeHistY + binSizeHistY / 2) * histY[iBin];
          sumWY += histY[iBin];
        }
      }
      beamPosition[0] = wX / sumWX;
      beamPosition[1] = wY / sumWY;
      float mockBeamPoint1[3] = {beamPosition[0], beamPosition[1], -1}; // get two points laying at different z, to create line object
      float mockBeamPoint2[3] = {beamPosition[0], beamPosition[1], 1};
      Line pseudoBeam = {mockBeamPoint1, mockBeamPoint2};
      if (Line::getDCA(lines[currentThreadIndex], pseudoBeam) < pairCut) {
        ClusterLinesGPU cluster{lines[currentThreadIndex], pseudoBeam};
        centroids[currentThreadIndex] = cluster.getVertex()[2];
      } else {
        centroids[currentThreadIndex] = 2 * lowHistZ;
      }
    }
  }
}

GPUg() void computeVertexKernel(
  hipcub::KeyValuePair<int, int>* tmpVertexBins,
  int* histZ, // Z
  const float lowHistZ,
  const float binSizeHistZ,
  const int nBinsHistZ,
  Vertex* vertices,
  float* beamPosition,
  const int vertIndex,
  const int minContributors,
  const int binOpeningZ)
{
  for (size_t currentThreadIndex = blockIdx.x * blockDim.x + threadIdx.x; currentThreadIndex < binOpeningZ; currentThreadIndex += blockDim.x * gridDim.x) {
    if (currentThreadIndex == 0) {
      if (tmpVertexBins[2].value > 1 && (tmpVertexBins[0].value || tmpVertexBins[1].value)) {
        float z{lowHistZ + tmpVertexBins[2].key * binSizeHistZ + binSizeHistZ / 2};
        float ex{0.f};
        float ey{0.f};
        float ez{0.f};
        int sumWZ{tmpVertexBins[2].value};
        float wZ{z * tmpVertexBins[2].value};
        for (int iBin{o2::gpu::GPUCommonMath::Max(0, tmpVertexBins[2].key - binOpeningZ)}; iBin < o2::gpu::GPUCommonMath::Min(tmpVertexBins[2].key + binOpeningZ + 1, nBinsHistZ - 1); ++iBin) {
          if (iBin != tmpVertexBins[2].key) {
            wZ += (lowHistZ + iBin * binSizeHistZ + binSizeHistZ / 2) * histZ[iBin];
            sumWZ += histZ[iBin];
          }
          histZ[iBin] = 0;
        }
        if (sumWZ > minContributors || vertIndex == 0) {
          new (vertices + vertIndex) Vertex{o2::math_utils::Point3D<float>(beamPosition[0], beamPosition[1], wZ / sumWZ), o2::gpu::gpustd::array<float, 6>{ex, 0, ey, 0, 0, ez}, static_cast<ushort>(sumWZ), 0};
        } else {
          new (vertices + vertIndex) Vertex{};
        }
      } else {
        new (vertices + vertIndex) Vertex{};
      }
    }
  }
}
} // namespace gpu

void VertexerTraitsGPU::updateVertexingParameters(const VertexingParameters& vrtPar, const TimeFrameGPUParameters& tfPar)
{
  mVrtParams = vrtPar;
  mTfGPUParams = tfPar;
  mIndexTableUtils.setTrackingParameters(vrtPar);
  mVrtParams.phiSpan = static_cast<int>(std::ceil(mIndexTableUtils.getNphiBins() * mVrtParams.phiCut /
                                                  constants::math::TwoPi));
  mVrtParams.zSpan = static_cast<int>(std::ceil(mVrtParams.zCut * mIndexTableUtils.getInverseZCoordinate(0)));
}

void VertexerTraitsGPU::computeTracklets()
{
  if (!mTimeFrameGPU->getClusters().size()) {
    return;
  }
  std::vector<std::thread> threads(mTimeFrameGPU->getNChunks());
  for (int chunkId{0}; chunkId < mTimeFrameGPU->getNChunks(); ++chunkId) {
    int rofPerChunk{mTimeFrameGPU->mNrof / (int)mTimeFrameGPU->getNChunks()};
    mTimeFrameGPU->getVerticesInChunks()[chunkId].clear();
    mTimeFrameGPU->getNVerticesInChunks()[chunkId].clear();
    mTimeFrameGPU->getLabelsInChunks()[chunkId].clear();
    auto doVertexReconstruction = [&, chunkId, rofPerChunk]() -> void {
      auto offset = chunkId * rofPerChunk;
      auto maxROF = offset + rofPerChunk;
      while (offset < maxROF) {
        auto rofs = mTimeFrameGPU->loadChunkData<gpu::Task::Vertexer>(chunkId, offset, maxROF);
        RANGE("chunk_gpu_vertexing", 1);
        gpu::trackleterKernelMultipleRof<TrackletMode::Layer0Layer1><<<rofs, 1024, 0, mTimeFrameGPU->getStream(chunkId).get()>>>(
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(0),         // const Cluster* clustersNextLayer,    // 0 2
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(1),         // const Cluster* clustersCurrentLayer, // 1 1
          mTimeFrameGPU->getDeviceROframesClusters(0),                   // const int* sizeNextLClusters,
          mTimeFrameGPU->getDeviceROframesClusters(1),                   // const int* sizeCurrentLClusters,
          mTimeFrameGPU->getChunk(chunkId).getDeviceIndexTables(0),      // const int* nextIndexTables,
          mTimeFrameGPU->getChunk(chunkId).getDeviceTracklets(0),        // Tracklet* Tracklets,
          mTimeFrameGPU->getChunk(chunkId).getDeviceNTrackletCluster(0), // int* foundTracklets,
          mTimeFrameGPU->getDeviceIndexTableUtils(),                     // const IndexTableUtils* utils,
          offset,                                                        // const unsigned int startRofId,
          rofs,                                                          // const unsigned int rofSize,
          mVrtParams.phiCut,                                             // const float phiCut,
          mVrtParams.maxTrackletsPerCluster);                            // const size_t maxTrackletsPerCluster = 1e2
        gpu::trackleterKernelMultipleRof<TrackletMode::Layer1Layer2><<<rofs, 1024, 0, mTimeFrameGPU->getStream(chunkId).get()>>>(
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(2),         // const Cluster* clustersNextLayer,    // 0 2
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(1),         // const Cluster* clustersCurrentLayer, // 1 1
          mTimeFrameGPU->getDeviceROframesClusters(2),                   // const int* sizeNextLClusters,
          mTimeFrameGPU->getDeviceROframesClusters(1),                   // const int* sizeCurrentLClusters,
          mTimeFrameGPU->getChunk(chunkId).getDeviceIndexTables(2),      // const int* nextIndexTables,
          mTimeFrameGPU->getChunk(chunkId).getDeviceTracklets(1),        // Tracklet* Tracklets,
          mTimeFrameGPU->getChunk(chunkId).getDeviceNTrackletCluster(1), // int* foundTracklets,
          mTimeFrameGPU->getDeviceIndexTableUtils(),                     // const IndexTableUtils* utils,
          offset,                                                        // const unsigned int startRofId,
          rofs,                                                          // const unsigned int rofSize,
          mVrtParams.phiCut,                                             // const float phiCut,
          mVrtParams.maxTrackletsPerCluster);                            // const size_t maxTrackletsPerCluster = 1e2

        // Tracklet selection
        gpu::trackletSelectionKernelMultipleRof<true><<<rofs, 1024, 0, mTimeFrameGPU->getStream(chunkId).get()>>>(
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(0),            // const Cluster* clusters0,               // Clusters on layer 0
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(1),            // const Cluster* clusters1,               // Clusters on layer 1
          mTimeFrameGPU->getDeviceROframesClusters(0),                      // const int* sizeClustersL0,              // Number of clusters on layer 0 per ROF
          mTimeFrameGPU->getDeviceROframesClusters(1),                      // const int* sizeClustersL1,              // Number of clusters on layer 1 per ROF
          mTimeFrameGPU->getChunk(chunkId).getDeviceTracklets(0),           // Tracklet* tracklets01,                  // Tracklets on layer 0-1
          mTimeFrameGPU->getChunk(chunkId).getDeviceTracklets(1),           // Tracklet* tracklets12,                  // Tracklets on layer 1-2
          mTimeFrameGPU->getChunk(chunkId).getDeviceNTrackletCluster(0),    // const int* nFoundTracklets01,           // Number of tracklets found on layers 0-1
          mTimeFrameGPU->getChunk(chunkId).getDeviceNTrackletCluster(1),    // const int* nFoundTracklet12,            // Number of tracklets found on layers 1-2
          mTimeFrameGPU->getChunk(chunkId).getDeviceUsedTracklets(),        // unsigned char* usedTracklets,           // Used tracklets
          mTimeFrameGPU->getChunk(chunkId).getDeviceLines(),                // Line* lines,                            // Lines
          mTimeFrameGPU->getChunk(chunkId).getDeviceNFoundLines(),          // int* nFoundLines,                       // Number of found lines
          mTimeFrameGPU->getChunk(chunkId).getDeviceNExclusiveFoundLines(), // int* nExclusiveFoundLines,              // Number of found lines exclusive scan
          offset,                                                           // const unsigned int startRofId,          // Starting ROF ID
          rofs,                                                             // const unsigned int rofSize,             // Number of ROFs to consider
          mVrtParams.maxTrackletsPerCluster,                                // const int maxTrackletsPerCluster = 1e2, // Maximum number of tracklets per cluster
          mVrtParams.tanLambdaCut,                                          // const float tanLambdaCut = 0.025f,      // Cut on tan lambda
          mVrtParams.phiCut);                                               // const float phiCut = 0.002f)            // Cut on phi

        discardResult(hipcub::DeviceScan::ExclusiveSum(mTimeFrameGPU->getChunk(chunkId).getDeviceCUBTmpBuffer(),
                                                    mTimeFrameGPU->getChunk(chunkId).getTimeFrameGPUParameters()->tmpCUBBufferSize,
                                                    mTimeFrameGPU->getChunk(chunkId).getDeviceNFoundLines(),
                                                    mTimeFrameGPU->getChunk(chunkId).getDeviceNExclusiveFoundLines(),
                                                    mTimeFrameGPU->getTotalClustersPerROFrange(offset, rofs, 1),
                                                    mTimeFrameGPU->getStream(chunkId).get()));

        // Reset used tracklets
        checkGPUError(hipMemsetAsync(mTimeFrameGPU->getChunk(chunkId).getDeviceUsedTracklets(),
                                      false,
                                      sizeof(unsigned char) * mVrtParams.maxTrackletsPerCluster * mTimeFrameGPU->getTotalClustersPerROFrange(offset, rofs, 1),
                                      mTimeFrameGPU->getStream(chunkId).get()),
                      __FILE__, __LINE__);

        gpu::trackletSelectionKernelMultipleRof<false><<<rofs, 1024, 0, mTimeFrameGPU->getStream(chunkId).get()>>>(
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(0),            // const Cluster* clusters0,               // Clusters on layer 0
          mTimeFrameGPU->getChunk(chunkId).getDeviceClusters(1),            // const Cluster* clusters1,               // Clusters on layer 1
          mTimeFrameGPU->getDeviceROframesClusters(0),                      // const int* sizeClustersL0,              // Number of clusters on layer 0 per ROF
          mTimeFrameGPU->getDeviceROframesClusters(1),                      // const int* sizeClustersL1,              // Number of clusters on layer 1 per ROF
          mTimeFrameGPU->getChunk(chunkId).getDeviceTracklets(0),           // Tracklet* tracklets01,                  // Tracklets on layer 0-1
          mTimeFrameGPU->getChunk(chunkId).getDeviceTracklets(1),           // Tracklet* tracklets12,                  // Tracklets on layer 1-2
          mTimeFrameGPU->getChunk(chunkId).getDeviceNTrackletCluster(0),    // const int* nFoundTracklets01,           // Number of tracklets found on layers 0-1
          mTimeFrameGPU->getChunk(chunkId).getDeviceNTrackletCluster(1),    // const int* nFoundTracklet12,            // Number of tracklets found on layers 1-2
          mTimeFrameGPU->getChunk(chunkId).getDeviceUsedTracklets(),        // unsigned char* usedTracklets,           // Used tracklets
          mTimeFrameGPU->getChunk(chunkId).getDeviceLines(),                // Line* lines,                            // Lines
          mTimeFrameGPU->getChunk(chunkId).getDeviceNFoundLines(),          // int* nFoundLines,                       // Number of found lines
          mTimeFrameGPU->getChunk(chunkId).getDeviceNExclusiveFoundLines(), // int* nExclusiveFoundLines,              // Number of found lines exclusive scan
          offset,                                                           // const unsigned int startRofId,          // Starting ROF ID
          rofs,                                                             // const unsigned int rofSize,             // Number of ROFs to consider
          mVrtParams.maxTrackletsPerCluster,                                // const int maxTrackletsPerCluster = 1e2, // Maximum number of tracklets per cluster
          mVrtParams.tanLambdaCut,                                          // const float tanLambdaCut = 0.025f,      // Cut on tan lambda
          mVrtParams.phiCut);                                               // const float phiCut = 0.002f)            // Cut on phi

        int nClusters = mTimeFrameGPU->getTotalClustersPerROFrange(offset, rofs, 1);
        int lastFoundLines;
        std::vector<int> exclusiveFoundLinesHost(nClusters + 1);

        // Obtain whole exclusive sum including nCluster+1 element  (nCluster+1)th element is the total number of found lines.
        checkGPUError(hipMemcpyAsync(exclusiveFoundLinesHost.data(), mTimeFrameGPU->getChunk(chunkId).getDeviceNExclusiveFoundLines(), (nClusters) * sizeof(int), hipMemcpyDeviceToHost, mTimeFrameGPU->getStream(chunkId).get()));
        checkGPUError(hipMemcpyAsync(&lastFoundLines, mTimeFrameGPU->getChunk(chunkId).getDeviceNFoundLines() + nClusters - 1, sizeof(int), hipMemcpyDeviceToHost, mTimeFrameGPU->getStream(chunkId).get()));
        exclusiveFoundLinesHost[nClusters] = exclusiveFoundLinesHost[nClusters - 1] + lastFoundLines;

        std::vector<Line> lines(exclusiveFoundLinesHost[nClusters]);

        checkGPUError(hipMemcpyAsync(lines.data(), mTimeFrameGPU->getChunk(chunkId).getDeviceLines(), sizeof(Line) * lines.size(), hipMemcpyDeviceToHost, mTimeFrameGPU->getStream(chunkId).get()));
        checkGPUError(hipStreamSynchronize(mTimeFrameGPU->getStream(chunkId).get()));

        // Compute vertices
        std::vector<ClusterLines> clusterLines;
        std::vector<bool> usedLines;
        for (int rofId{0}; rofId < rofs; ++rofId) {
          auto rof = offset + rofId;
          auto clustersL1offsetRof = mTimeFrameGPU->getROframeClusters(1)[rof] - mTimeFrameGPU->getROframeClusters(1)[offset]; // starting cluster offset for this ROF
          auto nClustersL1Rof = mTimeFrameGPU->getROframeClusters(1)[rof + 1] - mTimeFrameGPU->getROframeClusters(1)[rof];     // number of clusters for this ROF
          auto linesOffsetRof = exclusiveFoundLinesHost[clustersL1offsetRof];                                                  // starting line offset for this ROF
          auto nLinesRof = exclusiveFoundLinesHost[clustersL1offsetRof + nClustersL1Rof] - linesOffsetRof;
          gsl::span<const o2::its::Line> linesInRof(lines.data() + linesOffsetRof, static_cast<gsl::span<o2::its::Line>::size_type>(nLinesRof));

          usedLines.resize(linesInRof.size(), false);
          usedLines.assign(linesInRof.size(), false);
          clusterLines.clear();
          clusterLines.reserve(nClustersL1Rof);
          computeVerticesInRof(rof,
                               linesInRof,
                               usedLines,
                               clusterLines,
                               mTimeFrameGPU->getBeamXY(),
                               mTimeFrameGPU->getVerticesInChunks()[chunkId],
                               mTimeFrameGPU->getNVerticesInChunks()[chunkId],
                               mTimeFrameGPU->hasMCinformation() ? mTimeFrameGPU : nullptr,
                               mTimeFrameGPU->hasMCinformation() ? &mTimeFrameGPU->getLabelsInChunks()[chunkId] : nullptr);
        }
        offset += rofs;
      }
    };
    // Do work
    threads[chunkId] = std::thread(doVertexReconstruction);
  }
  for (auto& thread : threads) {
    thread.join();
  }
  for (int chunkId{0}; chunkId < mTimeFrameGPU->getNChunks(); ++chunkId) {
    int start{0};
    for (int rofId{0}; rofId < mTimeFrameGPU->getNVerticesInChunks()[chunkId].size(); ++rofId) {
      gsl::span<const Vertex> rofVerts{mTimeFrameGPU->getVerticesInChunks()[chunkId].data() + start, static_cast<gsl::span<Vertex>::size_type>(mTimeFrameGPU->getNVerticesInChunks()[chunkId][rofId])};
      mTimeFrameGPU->addPrimaryVertices(rofVerts);
      if (mTimeFrameGPU->hasMCinformation()) {
        mTimeFrameGPU->getVerticesLabels().emplace_back();
        // TODO: add MC labels
      }
      start += mTimeFrameGPU->getNVerticesInChunks()[chunkId][rofId];
    }
  }
  mTimeFrameGPU->wipe(3);
}

void VertexerTraitsGPU::computeTrackletMatching()
{
}

void VertexerTraitsGPU::computeVertices()
{
}

void VertexerTraitsGPU::computeVerticesHist()
{
}

VertexerTraits* createVertexerTraitsGPU()
{
  return new VertexerTraitsGPU;
}
} // namespace its
} // namespace o2
