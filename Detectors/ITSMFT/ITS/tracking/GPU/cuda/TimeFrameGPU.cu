#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

#include "ITStracking/Constants.h"

#include "ITStrackingGPU/Utils.h"
#include "ITStrackingGPU/TimeFrameGPU.h"
#include "ITStrackingGPU/TracerGPU.h"

#include <unistd.h>
#include <thread>

#ifndef __HIPCC__
#define THRUST_NAMESPACE thrust::cuda
#else
#define THRUST_NAMESPACE thrust::hip
#endif

namespace o2
{
namespace its
{
using constants::GB;
using constants::MB;

namespace gpu
{
using utils::checkGPUError;
/////////////////////////////////////////////////////////////////////////////////////////
// GpuChunk
/////////////////////////////////////////////////////////////////////////////////////////
template <int nLayers>
GpuTimeFrameChunk<nLayers>::~GpuTimeFrameChunk()
{
  if (mAllocated) {
    for (int i = 0; i < nLayers; ++i) {
      checkGPUError(hipFree(mClustersDevice[i]));
      checkGPUError(hipFree(mUsedClustersDevice[i]));
      checkGPUError(hipFree(mTrackingFrameInfoDevice[i]));
      checkGPUError(hipFree(mClusterExternalIndicesDevice[i]));
      checkGPUError(hipFree(mIndexTablesDevice[i]));
      if (i < nLayers - 1) {
        checkGPUError(hipFree(mTrackletsDevice[i]));
        checkGPUError(hipFree(mTrackletsLookupTablesDevice[i]));
        if (i < nLayers - 2) {
          checkGPUError(hipFree(mCellsDevice[i]));
          checkGPUError(hipFree(mCellsLookupTablesDevice[i]));
        }
      }
    }
    checkGPUError(hipFree(mCUBTmpBufferDevice));
    checkGPUError(hipFree(mFoundTrackletsDevice));
    checkGPUError(hipFree(mFoundCellsDevice));
  }
  mAllocated = false;
  LOGP(info, "Destroying GpuTimeFrameChunk");
}

template <int nLayers>
void GpuTimeFrameChunk<nLayers>::allocate(const size_t nrof, Stream& stream)
{
  RANGE("device_partition_allocation", 2);
  mNRof = nrof;
  for (int i = 0; i < nLayers; ++i) {
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mClustersDevice[i])), sizeof(Cluster) * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mUsedClustersDevice[i])), sizeof(unsigned char) * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mTrackingFrameInfoDevice[i])), sizeof(TrackingFrameInfo) * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mClusterExternalIndicesDevice[i])), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mIndexTablesDevice[i])), sizeof(int) * (256 * 128 + 1) * nrof, stream.get()));
    if (i < nLayers - 1) {
      checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mTrackletsLookupTablesDevice[i])), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
      checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mTrackletsDevice[i])), sizeof(Tracklet) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
      if (i < nLayers - 2) {
        checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mCellsLookupTablesDevice[i])), sizeof(int) * mTFGPUParams->validatedTrackletsCapacity * nrof, stream.get()));
        checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mCellsDevice[i])), sizeof(Cell) * mTFGPUParams->validatedTrackletsCapacity * nrof, stream.get()));
        if (i < 2) {
          checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mNTrackletsPerClusterDevice[i])), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
        }
      }
    }
  }
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mCUBTmpBufferDevice), mTFGPUParams->tmpCUBBufferSize * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mLinesDevice), sizeof(Line) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mNFoundLinesDevice), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mNExclusiveFoundLinesDevice), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof + 1, stream.get())); // + 1 for hipcub::DeviceScan::ExclusiveSum, to cover cases where we have maximum number of clusters per ROF
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mUsedTrackletsDevice), sizeof(unsigned char) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mClusteredLinesDevice), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mTFGPUParams->maxTrackletsPerCluster * nrof, stream.get()));

  /// Invariant allocations
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mFoundTrackletsDevice), (nLayers - 1) * sizeof(int) * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mFoundCellsDevice), (nLayers - 2) * sizeof(int) * nrof, stream.get()));

  mAllocated = true;
}

template <int nLayers>
void GpuTimeFrameChunk<nLayers>::reset(const Task task, Stream& stream)
{
  RANGE("buffer_reset", 0);
  if ((bool)task) { // Vertexer-only initialisation (cannot be constexpr: due to the presence of gpu raw calls can't be put in header)
    for (int i = 0; i < 2; i++) {
      auto thrustTrackletsBegin = thrust::device_ptr<Tracklet>(mTrackletsDevice[i]);
      auto thrustTrackletsEnd = thrustTrackletsBegin + mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * mNRof;
      thrust::fill(THRUST_NAMESPACE::par.on(stream.get()), thrustTrackletsBegin, thrustTrackletsEnd, Tracklet{});
      checkGPUError(hipMemsetAsync(mTrackletsLookupTablesDevice[i], 0, sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mNRof, stream.get()));
      checkGPUError(hipMemsetAsync(mNTrackletsPerClusterDevice[i], 0, sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mNRof, stream.get()));
    }
    checkGPUError(hipMemsetAsync(mUsedTrackletsDevice, false, sizeof(unsigned char) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * mNRof, stream.get()));
    checkGPUError(hipMemsetAsync(mClusteredLinesDevice, -1, sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mTFGPUParams->maxTrackletsPerCluster * mNRof, stream.get()));
  } else {
    for (int i = 0; i < nLayers - 1; ++i) {
      checkGPUError(hipMemsetAsync(mTrackletsLookupTablesDevice[i], 0, sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mNRof, stream.get()));
      auto thrustTrackletsBegin = thrust::device_ptr<Tracklet>(mTrackletsDevice[i]);
      auto thrustTrackletsEnd = thrustTrackletsBegin + mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * mNRof;
      thrust::fill(THRUST_NAMESPACE::par.on(stream.get()), thrustTrackletsBegin, thrustTrackletsEnd, Tracklet{});
      if (i < nLayers - 2) {
        checkGPUError(hipMemsetAsync(mCellsLookupTablesDevice[i], 0, sizeof(int) * mTFGPUParams->cellsLUTsize * mNRof, stream.get()));
      }
    }
    checkGPUError(hipMemsetAsync(mFoundCellsDevice, 0, (nLayers - 2) * sizeof(int), stream.get()));
  }
}

template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::computeScalingSizeBytes(const int nrof, const TimeFrameGPUParameters& config)
{
  size_t rofsize = nLayers * sizeof(int);                                                                      // number of clusters per ROF
  rofsize += nLayers * sizeof(Cluster) * config.clustersPerROfCapacity;                                        // clusters
  rofsize += nLayers * sizeof(unsigned char) * config.clustersPerROfCapacity;                                  // used clusters flags
  rofsize += nLayers * sizeof(TrackingFrameInfo) * config.clustersPerROfCapacity;                              // tracking frame info
  rofsize += nLayers * sizeof(int) * config.clustersPerROfCapacity;                                            // external cluster indices
  rofsize += nLayers * sizeof(int) * (256 * 128 + 1);                                                          // index tables
  rofsize += (nLayers - 1) * sizeof(int) * config.clustersPerROfCapacity;                                      // tracklets lookup tables
  rofsize += (nLayers - 1) * sizeof(Tracklet) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity; // tracklets
  rofsize += 2 * sizeof(int) * config.clustersPerROfCapacity;                                                  // tracklets found per cluster (vertexer)
  rofsize += sizeof(unsigned char) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity;            // used tracklets (vertexer)
  rofsize += (nLayers - 2) * sizeof(int) * config.validatedTrackletsCapacity;                                  // cells lookup tables
  rofsize += (nLayers - 2) * sizeof(Cell) * config.validatedTrackletsCapacity;                                 // cells
  rofsize += sizeof(Line) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity;                     // lines
  rofsize += sizeof(int) * config.clustersPerROfCapacity;                                                      // found lines
  rofsize += sizeof(int) * config.clustersPerROfCapacity;                                                      // found lines exclusive sum
  rofsize += sizeof(int) * config.clustersPerROfCapacity * config.maxTrackletsPerCluster;                      // lines used in clusterlines

  rofsize += (nLayers - 1) * sizeof(int); // total found tracklets
  rofsize += (nLayers - 2) * sizeof(int); // total found cells

  return rofsize * nrof;
}

template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::computeFixedSizeBytes(const TimeFrameGPUParameters& config)
{
  size_t total = config.tmpCUBBufferSize;                  // CUB tmp buffers
  total += sizeof(gpu::StaticTrackingParameters<nLayers>); // static parameters loaded once
  return total;
}

template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::computeRofPerChunk(const TimeFrameGPUParameters& config, const size_t m)
{
  return (m * GB / (float)(config.nTimeFrameChunks) - GpuTimeFrameChunk<nLayers>::computeFixedSizeBytes(config)) / (float)GpuTimeFrameChunk<nLayers>::computeScalingSizeBytes(1, config);
}

/// Interface
template <int nLayers>
Cluster* GpuTimeFrameChunk<nLayers>::getDeviceClusters(const int layer)
{
  return mClustersDevice[layer];
}

template <int nLayers>
unsigned char* GpuTimeFrameChunk<nLayers>::getDeviceUsedClusters(const int layer)
{
  return mUsedClustersDevice[layer];
}

template <int nLayers>
TrackingFrameInfo* GpuTimeFrameChunk<nLayers>::getDeviceTrackingFrameInfo(const int layer)
{
  return mTrackingFrameInfoDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceClusterExternalIndices(const int layer)
{
  return mClusterExternalIndicesDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceIndexTables(const int layer)
{
  return mIndexTablesDevice[layer];
}

template <int nLayers>
Tracklet* GpuTimeFrameChunk<nLayers>::getDeviceTracklets(const int layer)
{
  return mTrackletsDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceTrackletsLookupTables(const int layer)
{
  return mTrackletsLookupTablesDevice[layer];
}

template <int nLayers>
Cell* GpuTimeFrameChunk<nLayers>::getDeviceCells(const int layer)
{
  return mCellsDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceCellsLookupTables(const int layer)
{
  return mCellsLookupTablesDevice[layer];
}

// Load data
template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::loadDataOnDevice(const size_t startRof, const size_t maxRof, const int maxLayers, Stream& stream)
{
  RANGE("load_clusters_data", 5);
  auto nRofs = std::min(maxRof - startRof, mNRof);
  mNPopulatedRof = mTimeFramePtr->getNClustersROFrange(startRof, nRofs, 0).size();
  for (int i = 0; i < maxLayers; ++i) {
    mHostClusters[i] = mTimeFramePtr->getClustersPerROFrange(startRof, nRofs, i);
    if (maxLayers < nLayers) { // Vertexer
      mHostIndexTables[0] = mTimeFramePtr->getIndexTablePerROFrange(startRof, nRofs, 0);
      mHostIndexTables[2] = mTimeFramePtr->getIndexTablePerROFrange(startRof, nRofs, 2);
    } else { // Tracker
      mHostIndexTables[i] = mTimeFramePtr->getIndexTablePerROFrange(startRof, nRofs, i);
    }
    if (mHostClusters[i].size() > mTFGPUParams->clustersPerROfCapacity * nRofs) {
      LOGP(warning, "Excess of expected clusters on layer {}, resizing to config value: {}, will lose information!", i, mTFGPUParams->clustersPerROfCapacity * nRofs);
    }
    checkGPUError(hipMemcpyAsync(mClustersDevice[i],
                                  mHostClusters[i].data(),
                                  (int)std::min(mHostClusters[i].size(), mTFGPUParams->clustersPerROfCapacity * nRofs) * sizeof(Cluster),
                                  hipMemcpyHostToDevice, stream.get()));
    if (mHostIndexTables[i].data()) {
      checkGPUError(hipMemcpyAsync(mIndexTablesDevice[i],
                                    mHostIndexTables[i].data(),
                                    mHostIndexTables[i].size() * sizeof(int),
                                    hipMemcpyHostToDevice, stream.get()));
    }
  }
  return mNPopulatedRof; // return the number of ROFs we loaded the data for.
}

/////////////////////////////////////////////////////////////////////////////////////////
// TimeFrameGPU
/////////////////////////////////////////////////////////////////////////////////////////
template <int nLayers>
TimeFrameGPU<nLayers>::TimeFrameGPU()
{
  mIsGPU = true;
  utils::getDeviceProp(0, true);
}

template <int nLayers>
TimeFrameGPU<nLayers>::~TimeFrameGPU() = default;

template <int nLayers>
void TimeFrameGPU<nLayers>::registerHostMemory(const int maxLayers)
{
  if (mHostRegistered) {
    return;
  } else {
    mHostRegistered = true;
  }
  for (auto iLayer{0}; iLayer < maxLayers; ++iLayer) {
    checkGPUError(hipHostRegister(mClusters[iLayer].data(), mClusters[iLayer].size() * sizeof(Cluster), hipHostRegisterPortable));
    checkGPUError(hipHostRegister(mNClustersPerROF[iLayer].data(), mNClustersPerROF[iLayer].size() * sizeof(int), hipHostRegisterPortable));
    checkGPUError(hipHostRegister(mIndexTables[iLayer].data(), (mStaticTrackingParams.ZBins * mStaticTrackingParams.PhiBins + 1) * mNrof * sizeof(int), hipHostRegisterPortable));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::unregisterHostMemory(const int maxLayers)
{
  if (!mHostRegistered) {
    return;
  }
  for (auto iLayer{0}; iLayer < maxLayers; ++iLayer) {
    checkGPUError(hipHostUnregister(mClusters[iLayer].data()));
    checkGPUError(hipHostUnregister(mNClustersPerROF[iLayer].data()));
    checkGPUError(hipHostUnregister(mIndexTables[iLayer].data()));
  }
  mHostRegistered = false;
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initialise(const int iteration,
                                       const TrackingParameters& trkParam,
                                       const int maxLayers,
                                       const IndexTableUtils* utils,
                                       const TimeFrameGPUParameters* gpuParam)
{
  mGpuStreams.resize(mGpuParams.nTimeFrameChunks);
  auto init = [&](int p) -> void {
    this->initDevice(p, utils, trkParam, *gpuParam, maxLayers);
  };
  std::thread t1{init, mGpuParams.nTimeFrameChunks};
  RANGE("tf_cpu_initialisation", 1);
  o2::its::TimeFrame::initialise(iteration, trkParam, maxLayers);
  registerHostMemory(maxLayers);
  t1.join();
}

template <int nLayers>
void TimeFrameGPU<nLayers>::wipe(const int maxLayers)
{
  unregisterHostMemory(maxLayers);
  for (auto iLayer{0}; iLayer < maxLayers; ++iLayer) {
    checkGPUError(hipFree(mROframesClustersDevice[iLayer]));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initDevice(const int chunks,
                                       const IndexTableUtils* utils,
                                       const TrackingParameters& trkParam,
                                       const TimeFrameGPUParameters& gpuParam,
                                       const int maxLayers)
{
  if (mFirstInit) {

    mGpuParams = gpuParam;
    if (mGpuParams.maxGPUMemoryGB < 0) {
      // Adaptive to available memory, hungry mode
      size_t free;
      checkGPUError(hipMemGetInfo(&free, nullptr));
      mAvailMemGB = (double)free / GB;
      LOGP(info, "Hungry memory mode requested, found {} free GB, going to use all of them", mAvailMemGB);
    } else {
      mAvailMemGB = mGpuParams.maxGPUMemoryGB;
      LOGP(info, "Fixed memory mode requested, will try to use {} GB", mAvailMemGB);
    }

    mStaticTrackingParams.ZBins = trkParam.ZBins;
    mStaticTrackingParams.PhiBins = trkParam.PhiBins;
    checkGPUError(hipMalloc(reinterpret_cast<void**>(&mTrackingParamsDevice), sizeof(gpu::StaticTrackingParameters<nLayers>)));
    checkGPUError(hipMemcpy(mTrackingParamsDevice, &mStaticTrackingParams, sizeof(gpu::StaticTrackingParameters<nLayers>), hipMemcpyHostToDevice));
    if (utils) { // If utils is not nullptr, then its gpu vertexing
      checkGPUError(hipMalloc(reinterpret_cast<void**>(&mIndexTableUtilsDevice), sizeof(IndexTableUtils)));
      checkGPUError(hipMemcpy(mIndexTableUtilsDevice, utils, sizeof(IndexTableUtils), hipMemcpyHostToDevice));
    }
    mMemChunks.resize(chunks, GpuTimeFrameChunk<nLayers>{static_cast<TimeFrame*>(this), mGpuParams});
    mVerticesInChunks.resize(chunks);
    mNVerticesInChunks.resize(chunks);
    mLabelsInChunks.resize(chunks);
    LOGP(debug, "Size of fixed part is: {} MB", GpuTimeFrameChunk<nLayers>::computeFixedSizeBytes(mGpuParams) / MB);
    LOGP(debug, "Size of scaling part is: {} MB", GpuTimeFrameChunk<nLayers>::computeScalingSizeBytes(GpuTimeFrameChunk<nLayers>::computeRofPerChunk(mGpuParams, mAvailMemGB), mGpuParams) / MB);
    LOGP(info, "Allocating {} chunks of {} rofs capacity each.", chunks, GpuTimeFrameChunk<nLayers>::computeRofPerChunk(mGpuParams, mAvailMemGB));

    initDeviceChunks(GpuTimeFrameChunk<nLayers>::computeRofPerChunk(mGpuParams, mAvailMemGB), maxLayers);
    mFirstInit = false;
  }
  for (auto iLayer{0}; iLayer < maxLayers; ++iLayer) {
    checkGPUError(hipMalloc(reinterpret_cast<void**>(&mROframesClustersDevice[iLayer]), mROframesClusters[iLayer].size() * sizeof(int)));
    checkGPUError(hipMemcpy(mROframesClustersDevice[iLayer], mROframesClusters[iLayer].data(), mROframesClusters[iLayer].size() * sizeof(int), hipMemcpyHostToDevice));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initDeviceChunks(const int nRof, const int maxLayers)
{
  if (mDeviceInitialised) {
    return;
  } else {
    mDeviceInitialised = true;
  }
  if (!mMemChunks.size()) {
    LOGP(fatal, "gpu-tracking: TimeFrame GPU chunks not created");
  }
  for (int iChunk{0}; iChunk < mMemChunks.size(); ++iChunk) {
    mMemChunks[iChunk].allocate(nRof, mGpuStreams[iChunk]);
  }
}

template class TimeFrameGPU<7>;
template class GpuTimeFrameChunk<7>;
} // namespace gpu
} // namespace its
} // namespace o2