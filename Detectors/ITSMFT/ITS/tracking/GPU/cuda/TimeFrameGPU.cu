#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

#include "ITStracking/Constants.h"

#include "ITStrackingGPU/Utils.h"
#include "ITStrackingGPU/TimeFrameGPU.h"
#include "ITStrackingGPU/TracerGPU.h"

#include <unistd.h>
#include <thread>

#ifndef __HIPCC__
#define THRUST_NAMESPACE thrust::cuda
#else
#define THRUST_NAMESPACE thrust::hip
#endif

namespace o2
{
namespace its
{
using constants::GB;
using constants::MB;

namespace gpu
{
using utils::checkGPUError;

template <int nLayers>
struct StaticTrackingParameters {
  StaticTrackingParameters<nLayers>& operator=(const StaticTrackingParameters<nLayers>& t) = default;
  void set(const TrackingParameters& pars)
  {
    ClusterSharing = pars.ClusterSharing;
    MinTrackLength = pars.MinTrackLength;
    NSigmaCut = pars.NSigmaCut;
    PVres = pars.PVres;
    DeltaROF = pars.DeltaROF;
    ZBins = pars.ZBins;
    PhiBins = pars.PhiBins;
    CellDeltaTanLambdaSigma = pars.CellDeltaTanLambdaSigma;
  }

  /// General parameters
  int ClusterSharing = 0;
  int MinTrackLength = nLayers;
  float NSigmaCut = 5;
  float PVres = 1.e-2f;
  int DeltaROF = 0;
  int ZBins{256};
  int PhiBins{128};

  /// Cell finding cuts
  float CellDeltaTanLambdaSigma = 0.007f;
};

/////////////////////////////////////////////////////////////////////////////////////////
// GpuPartition
template <int nLayers>
GpuTimeFramePartition<nLayers>::~GpuTimeFramePartition()
{
  if (mAllocated) {
    for (int i = 0; i < nLayers; ++i) {
      checkGPUError(hipFree(mROframesClustersDevice[i]));
      checkGPUError(hipFree(mClustersDevice[i]));
      checkGPUError(hipFree(mUsedClustersDevice[i]));
      checkGPUError(hipFree(mTrackingFrameInfoDevice[i]));
      checkGPUError(hipFree(mClusterExternalIndicesDevice[i]));
      checkGPUError(hipFree(mIndexTablesDevice[i]));
      if (i < nLayers - 1) {
        checkGPUError(hipFree(mTrackletsDevice[i]));
        checkGPUError(hipFree(mTrackletsLookupTablesDevice[i]));
        if (i < nLayers - 2) {
          checkGPUError(hipFree(mCellsDevice[i]));
          checkGPUError(hipFree(mCellsLookupTablesDevice[i]));
        }
      }
    }
    checkGPUError(hipFree(mCUBTmpBufferDevice));
    checkGPUError(hipFree(mFoundTrackletsDevice));
    checkGPUError(hipFree(mFoundCellsDevice));
  }
}

template <int nLayers>
void GpuTimeFramePartition<nLayers>::allocate(const size_t nrof, Stream& stream)
{
  RANGE("device_partition_allocation", 2);
  mNRof = nrof;
  for (int i = 0; i < nLayers; ++i) {
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mROframesClustersDevice[i])), sizeof(int) * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mClustersDevice[i])), sizeof(Cluster) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mUsedClustersDevice[i])), sizeof(unsigned char) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mTrackingFrameInfoDevice[i])), sizeof(TrackingFrameInfo) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mClusterExternalIndicesDevice[i])), sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mIndexTablesDevice[i])), sizeof(int) * (256 * 128 + 1) * nrof, stream.get()));
    if (i < nLayers - 1) {
      checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mTrackletsLookupTablesDevice[i])), sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
      checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mTrackletsDevice[i])), sizeof(Tracklet) * mTFGconf->maxTrackletsPerCluster * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
      if (i < nLayers - 2) {
        checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mCellsLookupTablesDevice[i])), sizeof(int) * mTFGconf->validatedTrackletsCapacity * nrof, stream.get()));
        checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mCellsDevice[i])), sizeof(Cell) * mTFGconf->validatedTrackletsCapacity * nrof, stream.get()));
        if (i < 2) {
          checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&(mNTrackletsPerClusterDevice[i])), sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
        }
      }
    }
  }
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mCUBTmpBufferDevice), mTFGconf->tmpCUBBufferSize * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mLinesDevice), sizeof(Line) * mTFGconf->maxTrackletsPerCluster * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mNFoundLinesDevice), sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mNExclusiveFoundLinesDevice), sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));

  /// Invariant allocations
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mFoundTrackletsDevice), (nLayers - 1) * sizeof(int) * nrof, stream.get()));
  checkGPUError(hipMallocAsync(reinterpret_cast<void**>(&mFoundCellsDevice), (nLayers - 2) * sizeof(int) * nrof, stream.get()));

  mAllocated = true;
}

template <int nLayers>
void GpuTimeFramePartition<nLayers>::reset(const size_t nrof, const Task task, Stream& stream)
{
  RANGE("buffer_reset", 0);
  if ((bool)task) { // Vertexer-only initialisation (cannot be constexpr: due to the presence of gpu raw calls can't be put in header)
    std::vector<std::thread> t;
    for (int i = 0; i < 2; i++) {
      auto thrustTrackletsBegin = thrust::device_ptr<Tracklet>(mTrackletsDevice[i]);
      auto thrustTrackletsEnd = thrustTrackletsBegin + mTFGconf->maxTrackletsPerCluster * mTFGconf->clustersPerROfCapacity * nrof;
      thrust::fill(THRUST_NAMESPACE::par.on(stream.get()), thrustTrackletsBegin, thrustTrackletsEnd, Tracklet{});
      checkGPUError(hipMemsetAsync(mTrackletsLookupTablesDevice[i], 0, sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
      checkGPUError(hipMemsetAsync(mNTrackletsPerClusterDevice[i], 0, sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
    }
  } else {
    for (int i = 0; i < nLayers - 1; ++i) {
      checkGPUError(hipMemsetAsync(mTrackletsLookupTablesDevice[i], 0, sizeof(int) * mTFGconf->clustersPerROfCapacity * nrof, stream.get()));
      auto thrustTrackletsBegin = thrust::device_ptr<Tracklet>(mTrackletsDevice[i]);
      auto thrustTrackletsEnd = thrustTrackletsBegin + mTFGconf->maxTrackletsPerCluster * mTFGconf->clustersPerROfCapacity * nrof;
      thrust::fill(THRUST_NAMESPACE::par.on(stream.get()), thrustTrackletsBegin, thrustTrackletsEnd, Tracklet{});
      if (i < nLayers - 2) {
        checkGPUError(hipMemsetAsync(mCellsLookupTablesDevice[i], 0, sizeof(int) * mTFGconf->cellsLUTsize * nrof, stream.get()));
      }
    }
    checkGPUError(hipMemsetAsync(mFoundCellsDevice, 0, (nLayers - 2) * sizeof(int), stream.get()));
  }
}

template <int nLayers>
size_t GpuTimeFramePartition<nLayers>::computeScalingSizeBytes(const int nrof, const TimeFrameGPUConfig& config)
{
  size_t rofsize = nLayers * sizeof(int);                                                                      // number of clusters per ROF
  rofsize += nLayers * sizeof(Cluster) * config.clustersPerROfCapacity;                                        // clusters
  rofsize += nLayers * sizeof(unsigned char) * config.clustersPerROfCapacity;                                  // used clusters flags
  rofsize += nLayers * sizeof(TrackingFrameInfo) * config.clustersPerROfCapacity;                              // tracking frame info
  rofsize += nLayers * sizeof(int) * config.clustersPerROfCapacity;                                            // external cluster indices
  rofsize += nLayers * sizeof(int) * (256 * 128 + 1);                                                          // index tables
  rofsize += (nLayers - 1) * sizeof(int) * config.clustersPerROfCapacity;                                      // tracklets lookup tables
  rofsize += (nLayers - 1) * sizeof(Tracklet) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity; // tracklets
  rofsize += 2 * sizeof(int) * config.clustersPerROfCapacity;                                                  // tracklets found per cluster (vertexer)
  rofsize += (nLayers - 2) * sizeof(int) * config.validatedTrackletsCapacity;                                  // cells lookup tables
  rofsize += (nLayers - 2) * sizeof(Cell) * config.validatedTrackletsCapacity;                                 // cells
  rofsize += sizeof(Line) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity;                     // lines
  rofsize += sizeof(int) * config.clustersPerROfCapacity;                                                      // found lines
  rofsize += sizeof(int) * config.clustersPerROfCapacity;                                                      // found lines exclusive sum

  rofsize += (nLayers - 1) * sizeof(int); // total found tracklets
  rofsize += (nLayers - 2) * sizeof(int); // total found cells

  return rofsize * nrof;
}

template <int nLayers>
size_t GpuTimeFramePartition<nLayers>::computeFixedSizeBytes(const TimeFrameGPUConfig& config)
{
  size_t total = config.tmpCUBBufferSize;                  // CUB tmp buffers
  total += sizeof(gpu::StaticTrackingParameters<nLayers>); // static parameters loaded once
  return total;
}

template <int nLayers>
size_t GpuTimeFramePartition<nLayers>::computeRofPerPartition(const TimeFrameGPUConfig& config, const size_t m)
{
  return (m * GB / (float)(config.nTimeFramePartitions) - GpuTimeFramePartition<nLayers>::computeFixedSizeBytes(config)) / (float)GpuTimeFramePartition<nLayers>::computeScalingSizeBytes(1, config);
}

/// Interface
template <int nLayers>
int* GpuTimeFramePartition<nLayers>::getDeviceROframesClusters(const int layer)
{
  return mROframesClustersDevice[layer];
}

template <int nLayers>
Cluster* GpuTimeFramePartition<nLayers>::getDeviceClusters(const int layer)
{
  return mClustersDevice[layer];
}

template <int nLayers>
unsigned char* GpuTimeFramePartition<nLayers>::getDeviceUsedClusters(const int layer)
{
  return mUsedClustersDevice[layer];
}

template <int nLayers>
TrackingFrameInfo* GpuTimeFramePartition<nLayers>::getDeviceTrackingFrameInfo(const int layer)
{
  return mTrackingFrameInfoDevice[layer];
}

template <int nLayers>
int* GpuTimeFramePartition<nLayers>::getDeviceClusterExternalIndices(const int layer)
{
  return mClusterExternalIndicesDevice[layer];
}

template <int nLayers>
int* GpuTimeFramePartition<nLayers>::getDeviceIndexTables(const int layer)
{
  return mIndexTablesDevice[layer];
}

template <int nLayers>
Tracklet* GpuTimeFramePartition<nLayers>::getDeviceTracklets(const int layer)
{
  return mTrackletsDevice[layer];
}

template <int nLayers>
int* GpuTimeFramePartition<nLayers>::getDeviceTrackletsLookupTables(const int layer)
{
  return mTrackletsLookupTablesDevice[layer];
}

template <int nLayers>
Cell* GpuTimeFramePartition<nLayers>::getDeviceCells(const int layer)
{
  return mCellsDevice[layer];
}

template <int nLayers>
int* GpuTimeFramePartition<nLayers>::getDeviceCellsLookupTables(const int layer)
{
  return mCellsLookupTablesDevice[layer];
}

// Load data
template <int nLayers>
size_t GpuTimeFramePartition<nLayers>::copyDeviceData(const size_t startRof, const int maxLayers, Stream& stream)
{
  RANGE("load_clusters_data", 5);
  for (int i = 0; i < maxLayers; ++i) {
    mHostClusters[i] = mTimeFramePtr->getClustersPerROFrange(startRof, mNRof, i);
    mHostROframesClusters[i] = mTimeFramePtr->getROframesClustersPerROFrange(startRof, mNRof, i);
    if (mHostClusters[i].size() > mTFGconf->clustersPerROfCapacity * mNRof) {
      LOGP(warning, "Excess of expected clusters on layer {}, resizing to config value: {}, will lose information!", i, mTFGconf->clustersPerROfCapacity * mNRof);
    }
    checkGPUError(hipMemcpyAsync(mClustersDevice[i], mHostClusters[i].data(), (int)std::min(mHostClusters[i].size(), mTFGconf->clustersPerROfCapacity * mNRof) * sizeof(Cluster), hipMemcpyHostToDevice, stream.get()));
    checkGPUError(hipMemcpyAsync(mROframesClustersDevice[i], mHostROframesClusters[i].data(), mHostROframesClusters[i].size() * sizeof(int), hipMemcpyHostToDevice, stream.get()));
  }
  return mHostROframesClusters[0].size(); // We want to return for how much ROFs we loaded the data.
}

/////////////////////////////////////////////////////////////////////////////////////////
// TimeFrameGPU
template <int nLayers>
TimeFrameGPU<nLayers>::TimeFrameGPU()
{
  mIsGPU = true;
  utils::getDeviceProp(0, true);
  if (mGpuConfig.maxGPUMemoryGB < 0) {
    // Adaptive to available memory, hungry mode
    size_t free;
    checkGPUError(hipMemGetInfo(&free, nullptr));
    mAvailMemGB = (double)free / GB;
    LOGP(info, "Hungry memory mode requested, found {} free GB, going to use all of them", mAvailMemGB);
  } else {
    mAvailMemGB = mGpuConfig.maxGPUMemoryGB;
    LOGP(info, "Fixed memory mode requested, will try to use {} GB", mAvailMemGB);
  }
}

template <int nLayers>
TimeFrameGPU<nLayers>::~TimeFrameGPU()
{
  // checkGPUError(hipFree(mTrackingParamsDevice));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::registerHostMemory(const int maxLayers)
{
  if (mHostRegistered) {
    return;
  } else {
    mHostRegistered = true;
  }

  for (auto iLayer{0}; iLayer < maxLayers; ++iLayer) {
    checkGPUError(hipHostRegister(mClusters[iLayer].data(), mClusters[iLayer].size() * sizeof(Cluster), hipHostRegisterPortable));
    checkGPUError(hipHostRegister(mROframesClusters[iLayer].data(), mROframesClusters[iLayer].size() * sizeof(int), hipHostRegisterPortable));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::unregisterHostMemory(const int maxLayers)
{
  if (!mHostRegistered) {
    return;
  }
  for (auto iLayer{0}; iLayer < maxLayers; ++iLayer) {
    checkGPUError(hipHostUnregister(mClusters[iLayer].data()));
    checkGPUError(hipHostUnregister(mROframesClusters[iLayer].data()));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initialise(const int iteration,
                                       const TrackingParameters& trkParam,
                                       const int maxLayers,
                                       const IndexTableUtils* utils)
{
  mGpuStreams.resize(mGpuConfig.nTimeFramePartitions);
  auto init = [&](int p) -> void {
    this->initDevice(p, utils, maxLayers);
  };
  std::thread t1{init, mGpuConfig.nTimeFramePartitions};
  RANGE("tf_cpu_initialisation", 1);
  o2::its::TimeFrame::initialise(iteration, trkParam, maxLayers);
  registerHostMemory(maxLayers);
  t1.join();
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initDevice(const int partitions, const IndexTableUtils* utils, const int maxLayers)
{
  StaticTrackingParameters<nLayers> pars;
  checkGPUError(hipMalloc(reinterpret_cast<void**>(&mTrackingParamsDevice), sizeof(gpu::StaticTrackingParameters<nLayers>)));
  checkGPUError(hipMemcpy(mTrackingParamsDevice, &pars, sizeof(gpu::StaticTrackingParameters<nLayers>), hipMemcpyHostToDevice));
  if (utils) {
    checkGPUError(hipMalloc(reinterpret_cast<void**>(&mDeviceIndexTableUtils), sizeof(IndexTableUtils)));
    checkGPUError(hipMemcpy(mDeviceIndexTableUtils, utils, sizeof(IndexTableUtils), hipMemcpyHostToDevice));
  }
  mMemPartitions.resize(partitions, GpuTimeFramePartition<nLayers>{static_cast<TimeFrame*>(this), mGpuConfig});
  LOGP(debug, "Size of fixed part is: {} MB", GpuTimeFramePartition<nLayers>::computeFixedSizeBytes(mGpuConfig) / MB);
  LOGP(debug, "Size of scaling part is: {} MB", GpuTimeFramePartition<nLayers>::computeScalingSizeBytes(GpuTimeFramePartition<nLayers>::computeRofPerPartition(mGpuConfig, mAvailMemGB), mGpuConfig) / MB);
  LOGP(info, "Allocating {} partitions counting {} rofs each.", partitions, GpuTimeFramePartition<nLayers>::computeRofPerPartition(mGpuConfig, mAvailMemGB));

  initDevicePartitions(GpuTimeFramePartition<nLayers>::computeRofPerPartition(mGpuConfig, mAvailMemGB), maxLayers);
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initDevicePartitions(const int nRof, const int maxLayers)
{
  if (mDeviceInitialised) {
    return;
  } else {
    mDeviceInitialised = true;
  }
  if (!mMemPartitions.size()) {
    LOGP(fatal, "gpu-tracking: TimeFrame GPU partitions not created");
  }
  for (int iPartition{0}; iPartition < mMemPartitions.size(); ++iPartition) {
    mMemPartitions[iPartition].allocate(nRof, mGpuStreams[iPartition]);
    mMemPartitions[iPartition].reset(nRof, maxLayers < nLayers ? gpu::Task::Vertexer : gpu::Task::Tracker, mGpuStreams[iPartition]);
  }
}

template class TimeFrameGPU<7>;
template class GpuTimeFramePartition<7>;
} // namespace gpu
} // namespace its
} // namespace o2
