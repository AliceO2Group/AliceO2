#include "hip/hip_runtime.h"
// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Utils.cu
/// \brief
///

#include "ITStrackingCUDA/Utils.h"

#include <sstream>
#include <stdexcept>

#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "ITStrackingCUDA/Context.h"

#include <iostream>

namespace
{

int roundUp(const int numToRound, const int multiple)
{
  if (multiple == 0) {

    return numToRound;
  }

  int remainder{numToRound % multiple};

  if (remainder == 0) {

    return numToRound;
  }

  return numToRound + multiple - remainder;
}

int findNearestDivisor(const int numToRound, const int divisor)
{

  if (numToRound > divisor) {

    return divisor;
  }

  int result = numToRound;

  while (divisor % result != 0) {

    ++result;
  }

  return result;
}

} // namespace

namespace o2
{
namespace its
{
namespace GPU
{

void Utils::Host::checkCUDAError(const hipError_t error, const char* file, const int line)
{
  if (error != hipSuccess) {

    std::ostringstream errorString{};

    errorString << file << ":" << line << " CUDA API returned error [" << hipGetErrorString(error) << "] (code "
                << error << ")" << std::endl;

    throw std::runtime_error{errorString.str()};
  }
}

dim3 Utils::Host::getBlockSize(const int colsNum)
{
  return getBlockSize(colsNum, 1);
}

dim3 Utils::Host::getBlockSize(const int colsNum, const int rowsNum)
{
  const DeviceProperties& deviceProperties = Context::getInstance().getDeviceProperties();
  return getBlockSize(colsNum, rowsNum, deviceProperties.cudaCores / deviceProperties.maxBlocksPerSM);
}

dim3 Utils::Host::getBlockSize(const int colsNum, const int rowsNum, const int maxThreadsPerBlock)
{
  const DeviceProperties& deviceProperties = Context::getInstance().getDeviceProperties();
  int xThreads = max(min(colsNum, deviceProperties.maxThreadsDim.x), 1);
  int yThreads = max(min(rowsNum, deviceProperties.maxThreadsDim.y), 1);
  const int totalThreads = roundUp(min(xThreads * yThreads, maxThreadsPerBlock),
                                   deviceProperties.warpSize);

  if (xThreads > yThreads) {

    xThreads = findNearestDivisor(xThreads, totalThreads);
    yThreads = totalThreads / xThreads;

  } else {

    yThreads = findNearestDivisor(yThreads, totalThreads);
    xThreads = totalThreads / yThreads;
  }

  return dim3{static_cast<unsigned int>(xThreads), static_cast<unsigned int>(yThreads)};
}

dim3 Utils::Host::getBlocksGrid(const dim3& threadsPerBlock, const int rowsNum)
{

  return getBlocksGrid(threadsPerBlock, rowsNum, 1);
}

dim3 Utils::Host::getBlocksGrid(const dim3& threadsPerBlock, const int rowsNum, const int colsNum)
{

  return dim3{1 + (rowsNum - 1) / threadsPerBlock.x, 1 + (colsNum - 1) / threadsPerBlock.y};
}

void Utils::Host::gpuMalloc(void** p, const int size)
{
  checkCUDAError(hipMalloc(p, size), __FILE__, __LINE__);
}

void Utils::Host::gpuFree(void* p)
{
  checkCUDAError(hipFree(p), __FILE__, __LINE__);
}

void Utils::Host::gpuMemset(void* p, int value, int size)
{
  checkCUDAError(hipMemset(p, value, size), __FILE__, __LINE__);
}

void Utils::Host::gpuMemcpyHostToDevice(void* dst, const void* src, int size)
{
  checkCUDAError(hipMemcpy(dst, src, size, hipMemcpyHostToDevice), __FILE__, __LINE__);
}

void Utils::Host::gpuMemcpyHostToDeviceAsync(void* dst, const void* src, int size, Stream& stream)
{
  checkCUDAError(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream.get()), __FILE__, __LINE__);
}

void Utils::Host::gpuMemcpyDeviceToHost(void* dst, const void* src, int size)
{
  checkCUDAError(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost), __FILE__, __LINE__);
}

void Utils::Host::gpuStartProfiler()
{
  checkCUDAError(hipProfilerStart(), __FILE__, __LINE__);
}

void Utils::Host::gpuStopProfiler()
{
  checkCUDAError(hipProfilerStop(), __FILE__, __LINE__);
}

GPUd() int Utils::Device::getLaneIndex()
{
  uint32_t laneIndex;
  asm volatile("mov.u32 %0, %%laneid;"
               : "=r"(laneIndex));
  return static_cast<int>(laneIndex);
}

GPUd() int Utils::Device::shareToWarp(const int value, const int laneIndex)
{
  cooperative_groups::coalesced_group threadGroup = cooperative_groups::coalesced_threads();
  return threadGroup.shfl(value, laneIndex);
}

GPUd() int Utils::Device::gpuAtomicAdd(int* p, const int incrementSize)
{
  return atomicAdd(p, incrementSize);
}

} // namespace GPU
} // namespace its
} // namespace o2
