#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file DeviceStoreNV.cxx
/// \brief
///

#include "ITStrackingCUDA/DeviceStoreNV.h"

#include <sstream>

#include "ITStrackingCUDA/Stream.h"

namespace
{

using namespace o2::its;

__device__ void fillIndexTables(o2::its::gpu::DeviceStoreNV& primaryVertexContext, const int layerIndex)
{

  const int currentClusterIndex{static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x)};
  const int nextLayerClustersNum{static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size())};

  if (currentClusterIndex < nextLayerClustersNum) {

    const int currentBinIndex{
      primaryVertexContext.getClusters()[layerIndex + 1][currentClusterIndex].indexTableBinIndex};
    int previousBinIndex;

    if (currentClusterIndex == 0) {

      primaryVertexContext.getIndexTables()[layerIndex][0] = 0;
      previousBinIndex = 0;

    } else {

      previousBinIndex = primaryVertexContext.getClusters()[layerIndex + 1][currentClusterIndex - 1].indexTableBinIndex;
    }

    if (currentBinIndex > previousBinIndex) {

      for (int iBin{previousBinIndex + 1}; iBin <= currentBinIndex; ++iBin) {

        primaryVertexContext.getIndexTables()[layerIndex][iBin] = currentClusterIndex;
      }

      previousBinIndex = currentBinIndex;
    }

    if (currentClusterIndex == nextLayerClustersNum - 1) {

      for (int iBin{currentBinIndex + 1}; iBin <= o2::its::constants::its2::ZBins * o2::its::constants::its2::PhiBins;
           iBin++) {

        primaryVertexContext.getIndexTables()[layerIndex][iBin] = nextLayerClustersNum;
      }
    }
  }
}

__device__ void fillTrackletsPerClusterTables(o2::its::gpu::DeviceStoreNV& primaryVertexContext, const int layerIndex)
{
  const int currentClusterIndex{static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x)};
  const int clustersSize{static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size())};

  if (currentClusterIndex < clustersSize) {

    primaryVertexContext.getTrackletsPerClusterTable()[layerIndex][currentClusterIndex] = 0;
  }
}

__device__ void fillCellsPerClusterTables(o2::its::gpu::DeviceStoreNV& primaryVertexContext, const int layerIndex)
{
  const int totalThreadNum{static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size())};
  const int trackletsSize{static_cast<int>(primaryVertexContext.getTracklets()[layerIndex + 1].capacity())};
  const int trackletsPerThread{1 + (trackletsSize - 1) / totalThreadNum};
  const int firstTrackletIndex{static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x) * trackletsPerThread};

  if (firstTrackletIndex < trackletsSize) {

    const int trackletsToSet{min(trackletsSize, firstTrackletIndex + trackletsPerThread) - firstTrackletIndex};
    memset(&primaryVertexContext.getCellsPerTrackletTable()[layerIndex][firstTrackletIndex], 0,
           trackletsToSet * sizeof(int));
  }
}

__global__ void fillDeviceStructures(o2::its::gpu::DeviceStoreNV& primaryVertexContext, const int layerIndex)
{
  fillIndexTables(primaryVertexContext, layerIndex);

  if (layerIndex < o2::its::constants::its2::CellsPerRoad) {

    fillTrackletsPerClusterTables(primaryVertexContext, layerIndex);
  }

  if (layerIndex < o2::its::constants::its2::CellsPerRoad - 1) {

    fillCellsPerClusterTables(primaryVertexContext, layerIndex);
  }
}
} // namespace

namespace o2
{
namespace its
{
namespace gpu
{

DeviceStoreNV::DeviceStoreNV() = default;

UniquePointer<DeviceStoreNV> DeviceStoreNV::initialise(const float3& primaryVertex,
                                                       const std::array<std::vector<Cluster>, constants::its2::LayersNumber>& clusters,
                                                       const std::array<std::vector<Tracklet>, constants::its2::TrackletsPerRoad>& tracklets,
                                                       const std::array<std::vector<Cell>, constants::its2::CellsPerRoad>& cells,
                                                       const std::array<std::vector<int>, constants::its2::CellsPerRoad - 1>& cellsLookupTable,
                                                       const std::array<float, constants::its2::LayersNumber>& rmin,
                                                       const std::array<float, constants::its2::LayersNumber>& rmax)
{
  mPrimaryVertex = UniquePointer<float3>{primaryVertex};

  for (int iLayer{0}; iLayer < constants::its2::LayersNumber; ++iLayer) {
    this->mRmin[iLayer] = rmin[iLayer];
    this->mRmax[iLayer] = rmax[iLayer];

    this->mClusters[iLayer] =
      Vector<Cluster>{&clusters[iLayer][0], static_cast<int>(clusters[iLayer].size())};

    if (iLayer < constants::its2::TrackletsPerRoad) {
      this->mTracklets[iLayer].reset(tracklets[iLayer].capacity());
    }

    if (iLayer < constants::its2::CellsPerRoad) {

      this->mTrackletsLookupTable[iLayer].reset(static_cast<int>(clusters[iLayer + 1].size()));
      this->mTrackletsPerClusterTable[iLayer].reset(static_cast<int>(clusters[iLayer + 1].size()));
      this->mCells[iLayer].reset(static_cast<int>(cells[iLayer].capacity()));
    }

    if (iLayer < constants::its2::CellsPerRoad - 1) {

      this->mCellsLookupTable[iLayer].reset(static_cast<int>(cellsLookupTable[iLayer].size()));
      this->mCellsPerTrackletTable[iLayer].reset(static_cast<int>(cellsLookupTable[iLayer].size()));
    }
  }

  UniquePointer<DeviceStoreNV> gpuContextDevicePointer{*this};

  std::array<Stream, constants::its2::LayersNumber> streamArray;

  for (int iLayer{0}; iLayer < constants::its2::TrackletsPerRoad; ++iLayer) {

    const int nextLayerClustersNum = static_cast<int>(clusters[iLayer + 1].size());

    dim3 threadsPerBlock{utils::host::getBlockSize(nextLayerClustersNum)};
    dim3 blocksGrid{utils::host::getBlocksGrid(threadsPerBlock, nextLayerClustersNum)};

    fillDeviceStructures<<<blocksGrid, threadsPerBlock, 0, streamArray[iLayer].get()>>>(*gpuContextDevicePointer, iLayer);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess) {

      std::ostringstream errorString{};
      errorString << __FILE__ << ":" << __LINE__ << " CUDA API returned error [" << hipGetErrorString(error)
                  << "] (code " << error << ")" << std::endl;

      throw std::runtime_error{errorString.str()};
    }
  }

  return gpuContextDevicePointer;
}

} // namespace gpu
} // namespace its
} // namespace o2
