// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file Stream.cu
/// \brief
///

#include "ITSReconstruction/CA/gpu/Stream.h"

#include <hip/hip_runtime.h>

namespace o2
{
namespace ITS
{
namespace CA
{
namespace GPU
{

Stream::Stream()
{
  hipStreamCreateWithFlags(&mStream, hipStreamNonBlocking);
}

Stream::~Stream()
{
  hipStreamDestroy(mStream);
}

const GPUStream& Stream::get() const
{
  return mStream;
}

}
}
}
}
