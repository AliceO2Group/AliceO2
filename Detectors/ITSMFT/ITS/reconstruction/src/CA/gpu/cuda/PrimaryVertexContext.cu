#include "hip/hip_runtime.h"
// Copyright CERN and copyright holders of ALICE O2. This software is
// distributed under the terms of the GNU General Public License v3 (GPL
// Version 3), copied verbatim in the file "COPYING".
//
// See http://alice-o2.web.cern.ch/license for full licensing information.
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
/// \file PrimaryVertexContext.cxx
/// \brief
///

#include "ITSReconstruction/CA/gpu/PrimaryVertexContext.h"

#include <sstream>

#include "ITSReconstruction/CA/gpu/Stream.h"

namespace {

using namespace o2::ITS::CA;

__device__ void fillIndexTables(GPU::PrimaryVertexContext &primaryVertexContext, const int layerIndex)
{

  const int currentClusterIndex { static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x) };
  const int nextLayerClustersNum { static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size()) };

  if (currentClusterIndex < nextLayerClustersNum) {

    const int currentBinIndex {
        primaryVertexContext.getClusters()[layerIndex + 1][currentClusterIndex].indexTableBinIndex };
    int previousBinIndex;

    if (currentClusterIndex == 0) {

      primaryVertexContext.getIndexTables()[layerIndex][0] = 0;
      previousBinIndex = 0;

    } else {

      previousBinIndex = primaryVertexContext.getClusters()[layerIndex + 1][currentClusterIndex - 1].indexTableBinIndex;
    }

    if (currentBinIndex > previousBinIndex) {

      for (int iBin { previousBinIndex + 1 }; iBin <= currentBinIndex; ++iBin) {

        primaryVertexContext.getIndexTables()[layerIndex][iBin] = currentClusterIndex;
      }

      previousBinIndex = currentBinIndex;
    }

    if (currentClusterIndex == nextLayerClustersNum - 1) {

      for (int iBin { currentBinIndex + 1 }; iBin <= Constants::IndexTable::ZBins * Constants::IndexTable::PhiBins;
          iBin++) {

        primaryVertexContext.getIndexTables()[layerIndex][iBin] = nextLayerClustersNum;
      }
    }
  }
}

__device__ void fillTrackletsPerClusterTables(GPU::PrimaryVertexContext &primaryVertexContext, const int layerIndex)
{
  const int currentClusterIndex { static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x) };
  const int clustersSize { static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size()) };

  if (currentClusterIndex < clustersSize) {

    primaryVertexContext.getTrackletsPerClusterTable()[layerIndex][currentClusterIndex] = 0;
  }
}

__device__ void fillCellsPerClusterTables(GPU::PrimaryVertexContext &primaryVertexContext, const int layerIndex)
{
  const int totalThreadNum { static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size()) };
  const int trackletsSize { static_cast<int>(primaryVertexContext.getTracklets()[layerIndex + 1].capacity()) };
  const int trackletsPerThread { 1 + (trackletsSize - 1) / totalThreadNum };
  const int firstTrackletIndex { static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x) * trackletsPerThread };

  if (firstTrackletIndex < trackletsSize) {

    const int trackletsToSet { min(trackletsSize, firstTrackletIndex + trackletsPerThread) - firstTrackletIndex };
    memset(&primaryVertexContext.getCellsPerTrackletTable()[layerIndex][firstTrackletIndex], 0,
        trackletsToSet * sizeof(int));
  }
}

__global__ void fillDeviceStructures(GPU::PrimaryVertexContext &primaryVertexContext, const int layerIndex)
{
  fillIndexTables(primaryVertexContext, layerIndex);

  if (layerIndex < Constants::ITS::CellsPerRoad) {

    fillTrackletsPerClusterTables(primaryVertexContext, layerIndex);
  }

  if (layerIndex < Constants::ITS::CellsPerRoad - 1) {

    fillCellsPerClusterTables(primaryVertexContext, layerIndex);
  }
}
}

namespace o2
{
namespace ITS
{
namespace CA
{
namespace GPU
{

PrimaryVertexContext::PrimaryVertexContext()
{
  // Nothing to do
}

UniquePointer<PrimaryVertexContext> PrimaryVertexContext::initialize(const float3 &primaryVertex,
    const std::array<std::vector<Cluster>, Constants::ITS::LayersNumber> &clusters,
    const std::array<std::vector<Cell>, Constants::ITS::CellsPerRoad> &cells,
    const std::array<std::vector<int>, Constants::ITS::CellsPerRoad - 1> &cellsLookupTable)
{
  mPrimaryVertex = UniquePointer<float3>{ primaryVertex };

  for (int iLayer { 0 }; iLayer < Constants::ITS::LayersNumber; ++iLayer) {

    this->mClusters[iLayer] =
        Vector<Cluster> { &clusters[iLayer][0], static_cast<int>(clusters[iLayer].size()) };

    if (iLayer < Constants::ITS::TrackletsPerRoad) {

      this->mTracklets[iLayer].reset(static_cast<int>(std::ceil(
          (Constants::Memory::TrackletsMemoryCoefficients[iLayer] * clusters[iLayer].size())
              * clusters[iLayer + 1].size())));
    }

    if (iLayer < Constants::ITS::CellsPerRoad) {

      this->mTrackletsLookupTable[iLayer].reset(static_cast<int>(clusters[iLayer + 1].size()));
      this->mTrackletsPerClusterTable[iLayer].reset(static_cast<int>(clusters[iLayer + 1].size()));
      this->mCells[iLayer].reset(static_cast<int>(cells[iLayer].capacity()));
    }

    if (iLayer < Constants::ITS::CellsPerRoad - 1) {

      this->mCellsLookupTable[iLayer].reset(static_cast<int>(cellsLookupTable[iLayer].size()));
      this->mCellsPerTrackletTable[iLayer].reset(static_cast<int>(cellsLookupTable[iLayer].size()));
    }
  }

  UniquePointer<PrimaryVertexContext> gpuContextDevicePointer { *this };

  std::array<Stream, Constants::ITS::LayersNumber> streamArray;

  for (int iLayer { 0 }; iLayer < Constants::ITS::TrackletsPerRoad; ++iLayer) {

    const int nextLayerClustersNum = static_cast<int>(clusters[iLayer + 1].size());

    dim3 threadsPerBlock { Utils::Host::getBlockSize(nextLayerClustersNum) };
    dim3 blocksGrid { Utils::Host::getBlocksGrid(threadsPerBlock, nextLayerClustersNum) };

    fillDeviceStructures<<< blocksGrid, threadsPerBlock, 0, streamArray[iLayer].get() >>>(*gpuContextDevicePointer, iLayer);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess) {

      std::ostringstream errorString { };
      errorString << __FILE__ << ":" << __LINE__ << " CUDA API returned error [" << hipGetErrorString(error)
          << "] (code " << error << ")" << std::endl;

      throw std::runtime_error { errorString.str() };
    }
  }

  return gpuContextDevicePointer;
}

}
}
}
}
